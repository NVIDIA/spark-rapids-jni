#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.cuh>
#include <cudf/strings/detail/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include "zorder.hpp"

namespace spark_rapids_jni {

std::unique_ptr<cudf::column> interleave_bits(
  cudf::table_view const& tbl,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {

  auto num_columns = tbl.num_columns();
  CUDF_EXPECTS(num_columns > 0, "The input table must have at least one column.");


  CUDF_EXPECTS(is_fixed_width(tbl.begin()->type()), "Only fixed width columns can be used");

  auto const type_id = tbl.begin()->type().id();
  auto const base_size = cudf::size_of(tbl.begin()->type());
  CUDF_EXPECTS(
    std::all_of(tbl.begin(),
                tbl.end(),
                [type_id](cudf::column_view const& col) { return col.type().id() == type_id; }),
    "All columns of the input table must be the same type.");

  // Because the input is a table we know that they all have the same length.
  auto num_rows = tbl.num_rows();

  const cudf::size_type max_bytes_allowed = std::numeric_limits<cudf::size_type>::max();

  int64_t tmp_column_data_size = num_rows * base_size;
  if (tmp_column_data_size > max_bytes_allowed) {
    throw std::logic_error("Input is too large to process");
  }

  int64_t total_output_size = tmp_column_data_size * num_columns;
  if (total_output_size > max_bytes_allowed) {
    throw std::logic_error("Input is too large to process");
  }

  cudf::size_type output_size = static_cast<cudf::size_type>(total_output_size);

  auto input_dv = cudf::table_device_view::create(tbl, stream);

  auto output_data_col = cudf::make_numeric_column(
      cudf::data_type{cudf::type_id::UINT8}, output_size, cudf::mask_state::UNALLOCATED, stream, mr);

  auto output_dv_ptr = cudf::mutable_column_device_view::create(*output_data_col, stream);

  thrust::for_each_n(
          rmm::exec_policy(stream),
          thrust::make_counting_iterator<cudf::size_type>(0),
          output_size,
          [col = *output_dv_ptr, 
           num_columns,
           base_size,
           input = *input_dv] __device__ (cudf::size_type ret_idx) {
             // Flip the "endianness" of the output based off of the number of columns
             cudf::size_type flipped_start_byte_index = (ret_idx / num_columns) * num_columns;
             cudf::size_type flipped_ret_idx = flipped_start_byte_index + (num_columns - 1 - (ret_idx - flipped_start_byte_index));

             // Start with the highest bit for output
             cudf::size_type ret_bit = 7;
             uint8_t ret_byte = 0;
             while (ret_bit >= 0) {
               int64_t total_output_bit = flipped_ret_idx * 8L + ret_bit;

               // The order of the columns needs to be [0 to N] for the highest bit, so flip them too
               cudf::size_type column_idx = num_columns - 1 - (total_output_bit % num_columns);
               auto column = input.column(column_idx);

               // Also we need tp convert the endian byte order when we read the bytes.
               int64_t bit_within_column = total_output_bit / num_columns;
               cudf::size_type le_read_byte_index = bit_within_column / 8;
               cudf::size_type bit_offset = bit_within_column % 8;
               cudf::size_type input_row_number = le_read_byte_index / base_size;
               cudf::size_type start_item_byte_index = input_row_number * base_size;
               cudf::size_type read_byte_index = start_item_byte_index + (base_size - 1 - (le_read_byte_index - start_item_byte_index));

               uint32_t byte_data = column.data<uint8_t>()[read_byte_index];
               bool is_valid = (column.null_mask() ? cudf::bit_is_set(column.null_mask(), input_row_number + column.offset()): true);
               if (!is_valid) {
                 byte_data = 0;
               }
               uint32_t tmp = ((byte_data >> bit_offset) & 1) << ret_bit;
               ret_byte = static_cast<uint8_t>(ret_byte | tmp);

               ret_bit -= 1;
             }
             col.data<uint8_t>()[ret_idx] = ret_byte;
           });
  
  auto offset_begin = thrust::make_constant_iterator(base_size * num_columns);
  auto offsets_column = cudf::strings::detail::make_offsets_child_column(
    offset_begin, offset_begin + num_rows, stream, mr);

  return cudf::make_lists_column(num_rows,
          std::move(offsets_column),
          std::move(output_data_col),
          0,
          rmm::device_buffer(),
          stream,
          mr);
}

} // namespace spark_rapids_jni
