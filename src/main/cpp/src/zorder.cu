#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "zorder.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.cuh>
#include <cudf/strings/detail/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

namespace {

// pretends to be an array of uint32_t, but really only stores
// the data in a long with a set number of bits allocated for
// each item
struct long_backed_array {
  long_backed_array() = delete;
  ~long_backed_array() = default;
  long_backed_array(long_backed_array const&) = default;  ///< Copy constructor
  long_backed_array(long_backed_array&&) = default;  ///< Move constructor
  inline __device__ explicit long_backed_array(int32_t num_bits): data(0), 
    num_bits(num_bits),  mask(static_cast<uint64_t>((1L << num_bits) - 1)) {}

  /**
   * @brief Copy assignment operator
   *
   * @return Reference to this object
   */
  long_backed_array& operator=(long_backed_array const&) = default;
  /**
   * @brief Move assignment operator
   *
   * @return Reference to this object (after transferring ownership)
   */
  long_backed_array& operator=(long_backed_array&&) = default;

  inline __device__ uint32_t operator[](int i) const {
    int32_t offset = num_bits * i;
    return (data >> offset) & mask;
  }

  inline __device__ void set(int i, uint32_t value) {
    int32_t offset = i * num_bits;
    uint64_t masked_data = data & ~(static_cast<uint64_t>(mask) << offset);
    data = masked_data | (static_cast<uint64_t>(value & mask) << offset);
  }

private:
  uint64_t data;
  int32_t num_bits;
  uint32_t mask;
};


// Most of the hilbert index code is based off of the work done by David Moten at
// https://github.com/davidmoten/hilbert-curve, which has the following Note in
// the code too
// This algorithm is derived from work done by John Skilling and published
// in "Programming the Hilbert curve". (c) 2004 American Institute of Physics.
// With thanks also to Paul Chernoch who published a C# algorithm for Skilling's
// work on StackOverflow and
// <a href="https://github.com/paulchernoch/HilbertTransformation">GitHub</a>.
__device__ uint64_t to_hilbert_index(const long_backed_array & transposed_index, const int num_bits, const int num_dimensions) {
  uint64_t b = 0;
  int32_t length = num_bits * num_dimensions;
  int32_t b_index = length - 1;
  uint64_t mask = 1L << (num_bits - 1);
  for (int i = 0; i < num_bits; i++) {
    for (int j = 0; j < num_dimensions; j++) {
      if ((transposed_index[j] & mask) != 0) {
        b |= 1L << b_index;
      }
      b_index--;
    }
    mask >>= 1;
  }
  // b is expected to be BigEndian
  return b;
}

__device__ long_backed_array hilbert_transposed_index(const long_backed_array & point, const int num_bits, const int num_dimensions) {
  uint32_t const M = 1L << (num_bits - 1);
  int32_t const n = num_dimensions;
  long_backed_array x = point;

  uint32_t p, q, t;
  uint32_t i;
  // Inverse undo
  for (q = M; q > 1; q >>= 1) {
    p = q - 1;
    for (i = 0; i < n; i++) {
      if ((x[i] & q) != 0) {
        x.set(0, x[0] ^ p); // invert
      } else {
        t = (x[0] ^ x[i]) & p;
        x.set(0, x[0] ^ t);
        x.set(i, x[i] ^ t);
      }
    }
  } // exchange

  // Gray encode
  for (i = 1; i < n; i++) {
    x.set(i, x[i] ^ x[i - 1]);
  }
  t = 0;
  for (q = M; q > 1; q >>= 1) {
    if ((x[n - 1] & q) != 0) {
      t ^= q - 1;
    }
  }

  for (i = 0; i < n; i++) {
    x.set(i, x[i] ^ t);
  }

  return x;
}


} // namespace

namespace spark_rapids_jni {

std::unique_ptr<cudf::column> interleave_bits(
  cudf::table_view const& tbl,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {

  auto num_columns = tbl.num_columns();
  CUDF_EXPECTS(num_columns > 0, "The input table must have at least one column.");
  CUDF_EXPECTS(is_fixed_width(tbl.begin()->type()), "Only fixed width columns can be used");

  auto const type_id = tbl.begin()->type().id();
  auto const data_type_size = cudf::size_of(tbl.begin()->type());
  CUDF_EXPECTS(
    std::all_of(tbl.begin(),
                tbl.end(),
                [type_id](cudf::column_view const& col) { return col.type().id() == type_id; }),
    "All columns of the input table must be the same type.");

  // Because the input is a table we know that they all have the same length.
  auto num_rows = tbl.num_rows();

  const cudf::size_type max_bytes_allowed = std::numeric_limits<cudf::size_type>::max();

  int64_t total_output_size = static_cast<int64_t>(num_rows) * data_type_size * num_columns;
  CUDF_EXPECTS (total_output_size <= max_bytes_allowed, "Input is too large to process");

  cudf::size_type output_size = static_cast<cudf::size_type>(total_output_size);

  auto input_dv = cudf::table_device_view::create(tbl, stream);

  auto output_data_col = cudf::make_numeric_column(
      cudf::data_type{cudf::type_id::UINT8}, output_size, cudf::mask_state::UNALLOCATED, stream, mr);

  auto output_dv_ptr = cudf::mutable_column_device_view::create(*output_data_col, stream);

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    output_size,
    [col = *output_dv_ptr, 
     num_columns,
     data_type_size,
     input = *input_dv] __device__ (cudf::size_type ret_idx) {
       // The most significant byte needs to come from the most significant column, so we switch the order of the output
       // bytes to match that
       cudf::size_type const flipped_start_byte_index = (ret_idx / num_columns) * num_columns;
       cudf::size_type const flipped_ret_idx = flipped_start_byte_index + (num_columns - 1 - (ret_idx - flipped_start_byte_index));

       uint8_t ret_byte = 0;
       for (cudf::size_type output_bit_offset = 7; output_bit_offset >= 0; output_bit_offset--) {
         // The index (in bits) of the output bit we are computing right now
         int64_t const output_bit_index = flipped_ret_idx * 8L + output_bit_offset;

         // The most significant bit should come from the most significant column, but 0 is
         // our most significant column, so switch the order of the columns.
         cudf::size_type const column_idx = num_columns - 1 - (output_bit_index % num_columns);
         auto column = input.column(column_idx);

         // Also we need to convert the endian byte order when we read the bytes.
         int64_t const bit_index_within_column = output_bit_index / num_columns;
         cudf::size_type const little_endian_read_byte_index = bit_index_within_column / 8;
         cudf::size_type const read_bit_offset = bit_index_within_column % 8;
         cudf::size_type const input_row_number = little_endian_read_byte_index / data_type_size;
         cudf::size_type const start_row_byte_index = input_row_number * data_type_size;
         cudf::size_type const read_byte_index = start_row_byte_index + (data_type_size - 1 - (little_endian_read_byte_index - start_row_byte_index));

         uint32_t const byte_data = column.is_valid(input_row_number) ? column.data<uint8_t>()[read_byte_index] : 0;
         uint32_t const tmp = ((byte_data >> read_bit_offset) & 1) << output_bit_offset;
         ret_byte = static_cast<uint8_t>(ret_byte | tmp);
       }
       col.data<uint8_t>()[ret_idx] = ret_byte;
     });
  
  auto offset_begin = thrust::make_constant_iterator(data_type_size * num_columns);
  auto offsets_column = cudf::strings::detail::make_offsets_child_column(
    offset_begin, offset_begin + num_rows, stream, mr);

  return cudf::make_lists_column(num_rows,
    std::move(offsets_column),
    std::move(output_data_col),
    0,
    rmm::device_buffer(),
    stream,
    mr);
}

std::unique_ptr<cudf::column> hilbert_index(
  int32_t const num_bits,
  cudf::table_view const& tbl,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {
 
  auto num_rows = tbl.num_rows();
  auto num_columns = tbl.num_columns();

  CUDF_EXPECTS(num_bits > 0 && num_bits <= 32, "the number of bits must be >0 and <= 32.");
  CUDF_EXPECTS(num_bits * num_columns <= 64, "we only support up to 64 bits of output right now.");
  CUDF_EXPECTS(num_columns > 0, "at least one column is required.");

  CUDF_EXPECTS(
    std::all_of(tbl.begin(),
                tbl.end(),
                [](cudf::column_view const& col) { return col.type().id() == cudf::type_id::INT32; }),
    "All columns of the input table must be INT32.");

  auto input_dv = cudf::table_device_view::create(tbl, stream);

  auto output_data_col = cudf::make_numeric_column(
      cudf::data_type{cudf::type_id::INT64}, num_rows, cudf::mask_state::UNALLOCATED, stream, mr);

  auto output_dv_ptr = cudf::mutable_column_device_view::create(*output_data_col, stream);

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    num_rows,
    [output_col = *output_dv_ptr,
     num_bits,
     num_columns,
     input = *input_dv] __device__ (cudf::size_type row_index) {
       long_backed_array row(num_bits);
       for (cudf::size_type column_index = 0; column_index < num_columns; column_index++) {
         auto const column = input.column(column_index);
         uint32_t const data = column.is_valid(row_index) ? column.data<uint32_t>()[row_index] : 0;
         row.set(column_index, data);
       }

       auto transposed_index = hilbert_transposed_index(row, num_bits, num_columns);
       output_col.data<uint64_t>()[row_index] = to_hilbert_index(transposed_index, num_bits, num_columns);
     });

  return output_data_col;
}

} // namespace spark_rapids_jni
