#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "zorder.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.cuh>
#include <cudf/strings/detail/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

namespace spark_rapids_jni {

std::unique_ptr<cudf::column> interleave_bits(
  cudf::table_view const& tbl,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {

  auto num_columns = tbl.num_columns();
  CUDF_EXPECTS(num_columns > 0, "The input table must have at least one column.");
  CUDF_EXPECTS(is_fixed_width(tbl.begin()->type()), "Only fixed width columns can be used");

  auto const type_id = tbl.begin()->type().id();
  auto const data_type_size = cudf::size_of(tbl.begin()->type());
  CUDF_EXPECTS(
    std::all_of(tbl.begin(),
                tbl.end(),
                [type_id](cudf::column_view const& col) { return col.type().id() == type_id; }),
    "All columns of the input table must be the same type.");

  // Because the input is a table we know that they all have the same length.
  auto num_rows = tbl.num_rows();

  const cudf::size_type max_bytes_allowed = std::numeric_limits<cudf::size_type>::max();

  int64_t total_output_size = static_cast<int64_t>(num_rows) * data_type_size * num_columns;
  CUDF_EXPECTS (total_output_size <= max_bytes_allowed, "Input is too large to process");

  cudf::size_type output_size = static_cast<cudf::size_type>(total_output_size);

  auto input_dv = cudf::table_device_view::create(tbl, stream);

  auto output_data_col = cudf::make_numeric_column(
      cudf::data_type{cudf::type_id::UINT8}, output_size, cudf::mask_state::UNALLOCATED, stream, mr);

  auto output_dv_ptr = cudf::mutable_column_device_view::create(*output_data_col, stream);

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    output_size,
    [col = *output_dv_ptr, 
     num_columns,
     data_type_size,
     input = *input_dv] __device__ (cudf::size_type ret_idx) {
       // The most significant byte needs to come from the most significant column, so we switch the order of the output
       // bytes to match that
       cudf::size_type const flipped_start_byte_index = (ret_idx / num_columns) * num_columns;
       cudf::size_type const flipped_ret_idx = flipped_start_byte_index + (num_columns - 1 - (ret_idx - flipped_start_byte_index));

       uint8_t ret_byte = 0;
       for (cudf::size_type output_bit_offset = 7; output_bit_offset >= 0; output_bit_offset--) {
         // The index (in bits) of the output bit we are computing right now
         int64_t const output_bit_index = flipped_ret_idx * 8L + output_bit_offset;

         // The most significant bit should come from the most significant column, but 0 is
         // our most significant column, so switch the order of the columns.
         cudf::size_type const column_idx = num_columns - 1 - (output_bit_index % num_columns);
         auto column = input.column(column_idx);

         // Also we need to convert the endian byte order when we read the bytes.
         int64_t const bit_index_within_column = output_bit_index / num_columns;
         cudf::size_type const little_endian_read_byte_index = bit_index_within_column / 8;
         cudf::size_type const read_bit_offset = bit_index_within_column % 8;
         cudf::size_type const input_row_number = little_endian_read_byte_index / data_type_size;
         cudf::size_type const start_row_byte_index = input_row_number * data_type_size;
         cudf::size_type const read_byte_index = start_row_byte_index + (data_type_size - 1 - (little_endian_read_byte_index - start_row_byte_index));

         uint32_t const byte_data = column.is_valid(input_row_number) ? column.data<uint8_t>()[read_byte_index] : 0;
         uint32_t const tmp = ((byte_data >> read_bit_offset) & 1) << output_bit_offset;
         ret_byte = static_cast<uint8_t>(ret_byte | tmp);
       }
       col.data<uint8_t>()[ret_idx] = ret_byte;
     });
  
  auto offset_begin = thrust::make_constant_iterator(data_type_size * num_columns);
  auto offsets_column = cudf::strings::detail::make_offsets_child_column(
    offset_begin, offset_begin + num_rows, stream, mr);

  return cudf::make_lists_column(num_rows,
    std::move(offsets_column),
    std::move(output_data_col),
    0,
    rmm::device_buffer(),
    stream,
    mr);
}

} // namespace spark_rapids_jni
