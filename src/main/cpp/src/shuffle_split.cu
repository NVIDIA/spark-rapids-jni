#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "shuffle_split.hpp"
#include "shuffle_split_detail.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <cstddef>
#include <numeric>

namespace spark_rapids_jni {

using namespace cudf;
using namespace spark_rapids_jni::detail;

namespace {

/**
 * @brief Struct which contains information on a source buffer.
 *
 * The definition of "buffer" used throughout this module is a component piece of a
 * cudf column. So for example, a fixed-width column with validity would have 2 associated
 * buffers : the data itself and the validity buffer.  contiguous_split operates by breaking
 * each column up into it's individual components and copying each one as a separate kernel
 * block.
 */
struct src_buf_info {
  src_buf_info(cudf::type_id _type,
               int _offset_stack_pos,
               int _parent_offsets_index,
               uint8_t const* _data,
               buffer_type _btype,
               size_type _column_offset)
    : type(_type),
      offset_stack_pos(_offset_stack_pos),
      parent_offsets_index(_parent_offsets_index),
      data(_data),
      btype(_btype),
      column_offset(_column_offset)
  {
  }

  cudf::type_id type;
  int offset_stack_pos;      // position in the offset stack buffer
  int parent_offsets_index;  // immediate parent that has offsets, or -1 if none
  uint8_t const* data;
  buffer_type btype;
  size_type column_offset;  // offset in the case of a sliced column
};

/**
 * @brief Struct which contains information on a destination buffer.
 *
 * Similar to src_buf_info, dst_buf_info contains information on a destination buffer we
 * are going to copy to.  If we have N input buffers (which come from X columns), and
 * M partitions, then we have N*M destination buffers.
 */
struct dst_buf_info {
  size_t buf_size;  // total size of buffer in bytes
  buffer_type type;

  int src_buf_index;
  size_t src_offset;
  size_t dst_offset;
};

// The block of functions below are all related:
//
// compute_offset_stack_size()
// count_src_bufs()
// setup_source_buf_info()
// build_output_columns()
//
// Critically, they all traverse the hierarchy of source columns and their children
// in a specific order to guarantee they produce various outputs in a consistent
// way.
//
// So please be careful if you change the way in which these functions and
// functors traverse the hierarchy.

/**
 * @brief Returns whether or not the specified type is a column that contains offsets.
 */
bool is_offset_type(type_id id) { return (id == type_id::STRING or id == type_id::LIST); }

/**
 * @brief Compute total device memory stack size needed to process nested
 * offsets per-output buffer.
 *
 * When determining the range of rows to be copied for each output buffer
 * we have to recursively apply the stack of offsets from our parent columns
 * (lists or strings).  We want to do this computation on the gpu because offsets
 * are stored in device memory.  However we don't want to do recursion on the gpu, so
 * each destination buffer gets a "stack" of space to work with equal in size to
 * it's offset nesting depth.  This function computes the total size of all of those
 * stacks.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param offset_depth Current offset nesting depth
 *
 * @returns Total offset stack size needed for this range of columns.
 */
template <typename InputIter>
size_t compute_offset_stack_size(InputIter begin, InputIter end, int offset_depth = 0)
{
  return std::accumulate(begin, end, 0, [offset_depth](auto stack_size, column_view const& col) {
    auto const num_buffers = 1 + (col.nullable() ? 1 : 0);
    return stack_size + (offset_depth * num_buffers) +
           compute_offset_stack_size(
             col.child_begin(), col.child_end(), offset_depth + is_offset_type(col.type().id()));
  });
}

/**
 * @brief A count of the three fundamental types of buffers. validity, offsets and data
 */
struct src_buf_count {
  size_t validity_buf_count;
  size_t offset_buf_count;
  size_t data_buf_count;
};

/**
 * @brief Count the total number of source buffers, broken down by type (validity, offset, data)
 * we will be copying from.
 *
 * This count includes buffers for all input columns. For example a
 * fixed-width column with validity would be 2 buffers (validity, data).
 * A string column with validity would be 3 buffers (validity, offssets, data (chars))
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 *
 * @returns total number of source buffer per type for this range of columns
 */
template <typename InputIter>
src_buf_count count_src_bufs(InputIter begin, InputIter end)
{
  auto src_buf_count_add = [](src_buf_count const& lhs, src_buf_count const& rhs) -> src_buf_count {
    return src_buf_count{lhs.validity_buf_count + rhs.validity_buf_count,
                         lhs.offset_buf_count + rhs.offset_buf_count,
                         lhs.data_buf_count + rhs.data_buf_count};
  };

  auto buf_iter = thrust::make_transform_iterator(begin, [&](column_view const& col) {
    auto const type = col.type().id();
    // for lists and strings, account for their offset child here instead of recursively
    auto const has_offsets_child =
      type == cudf::type_id::LIST || (type == cudf::type_id::STRING && col.num_children() > 0);
    src_buf_count const counts{
      static_cast<size_t>(col.nullable()),
      static_cast<size_t>(has_offsets_child),
      size_t{
        1}};  // this is 1 for all types because even lists and structs have stubs for data buffers

    auto child_counts = [&]() {
      // strings don't need to recurse. we count their offsets and data right here.
      switch (type) {
        case cudf::type_id::STRING: return src_buf_count{0, 0, 0};

        // offset for lists child is accounted for here
        case cudf::type_id::LIST: {
          auto data_child = col.child_begin() + cudf::lists_column_view::child_column_index;
          return count_src_bufs(data_child, std::next(data_child));
        }

        default: break;
      }
      return count_src_bufs(col.child_begin(), col.child_end());
    };
    return src_buf_count_add(counts, child_counts());
  });
  return std::accumulate(
    buf_iter, buf_iter + std::distance(begin, end), src_buf_count{0, 0, 0}, src_buf_count_add);
}

/**
 * @brief Computes source buffer information for the copy process.
 *
 * For each input column to be split we need to know several pieces of information
 * in the copy kernel.  This function traverses the input columns and prepares this
 * information for the gpu.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param head Beginning of source buffer info array
 * @param validity_cur[in, out] Current validity source buffer info to be read from
 * @param offset_cur[in, out] Current offset source buffer info to be read from
 * @param data_cur[in, out] Current data source buffer info to be read from
 * @param flattened_col_has_validity[out] A vector representing whether each input column has a
 * validity vector or not.
 * @param offset_stack_pos Integer representing our current offset nesting depth
 * (how many list or string levels deep we are)
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param parent_offset_index Index into src_buf_info output array indicating our nearest
 * containing list parent. -1 if we have no list parent
 * @param offset_depth Current offset nesting depth (how many list levels deep we are)
 *
 * @returns next src_buf_output after processing this range of input columns
 */
// setup source buf info
template <typename InputIter>
void setup_source_buf_info(InputIter begin,
                           InputIter end,
                           src_buf_info* head,
                           src_buf_info*& validity_cur,
                           src_buf_info*& offset_cur,
                           src_buf_info*& data_cur,
                           std::vector<int8_t>& flattened_col_has_validity,
                           int& offset_stack_pos,
                           rmm::cuda_stream_view stream,
                           int parent_offset_index = -1,
                           int offset_depth        = 0);

/**
 * @brief Functor that builds source buffer information based on input columns.
 *
 * Called by setup_source_buf_info to build information for a single source column.  This function
 * will recursively call setup_source_buf_info in the case of nested types.
 */
struct buf_info_functor {
  src_buf_info* head;

  template <typename T>
  void operator()(column_view const& col,
                  src_buf_info*& validity_cur,
                  src_buf_info*& offset_cur,
                  src_buf_info*& data_cur,
                  std::vector<int8_t>& flattened_col_has_validity,
                  int& offset_stack_pos,
                  int parent_offset_index,
                  int offset_depth,
                  rmm::cuda_stream_view)
  {
    flattened_col_has_validity.push_back(col.nullable());
    if (col.nullable()) {
      add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
    }

    // info for the data buffer
    *data_cur = src_buf_info(col.type().id(),
                             offset_stack_pos,
                             parent_offset_index,
                             col.head<uint8_t>(),
                             buffer_type::DATA,
                             col.offset());
    data_cur++;

    offset_stack_pos += offset_depth;
  }

  template <typename T, typename... Args>
  std::enable_if_t<std::is_same_v<T, cudf::dictionary32>, void> operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type");
  }

 private:
  void add_null_buffer(column_view const& col,
                       src_buf_info* validity_cur,
                       int offset_stack_pos,
                       int parent_offset_index,
                       int offset_depth)
  {
    // info for the validity buffer
    *validity_cur = src_buf_info(type_id::INT32,
                                 offset_stack_pos,
                                 parent_offset_index,
                                 reinterpret_cast<uint8_t const*>(col.null_mask()),
                                 buffer_type::VALIDITY,
                                 col.offset());
    validity_cur++;

    offset_stack_pos += offset_depth;
  }
};

template <>
void buf_info_functor::operator()<cudf::string_view>(
  column_view const& col,
  src_buf_info*& validity_cur,
  src_buf_info*& offset_cur,
  src_buf_info*& data_cur,
  std::vector<int8_t>& flattened_col_has_validity,
  int& offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view)
{
  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // the way strings are arranged, the strings column itself contains char data, but our child
  // offsets column contains the offsets. So our parent_offset_index is actually our child.

  // string columns don't necessarily have children if they are empty
  auto const has_offsets_child = col.num_children() > 0;

  // string columns contain the underlying chars data.
  *data_cur = src_buf_info(type_id::STRING,
                           offset_stack_pos,
                           // if I have offsets, it's index will be the current offset buffer,
                           // otherwise it's whatever my parent's was.
                           has_offsets_child ? (offset_cur - head) : parent_offset_index,
                           col.head<uint8_t>(),
                           buffer_type::DATA,
                           col.offset());
  data_cur++;
  // if I have offsets, I need to include that in the stack size
  offset_stack_pos += has_offsets_child ? offset_depth + 1 : offset_depth;

  if (has_offsets_child) {
    CUDF_EXPECTS(col.num_children() == 1, "Encountered malformed string column");
    strings_column_view scv(col);

    // info for the offsets buffer
    CUDF_EXPECTS(not scv.offsets().nullable(), "Encountered nullable string offsets column");
    *offset_cur = src_buf_info(
      type_id::INT32,
      offset_stack_pos,
      parent_offset_index,
      // note: offsets can be null in the case where the string column
      // has been created with empty_like().
      reinterpret_cast<uint8_t const*>(scv.offsets().begin<cudf::id_to_type<type_id::INT32>>()),
      buffer_type::OFFSETS,
      col.offset());

    offset_cur++;
    offset_stack_pos += offset_depth;
  }
}

template <>
void buf_info_functor::operator()<cudf::list_view>(column_view const& col,
                                                   src_buf_info*& validity_cur,
                                                   src_buf_info*& offset_cur,
                                                   src_buf_info*& data_cur,
                                                   std::vector<int8_t>& flattened_col_has_validity,
                                                   int& offset_stack_pos,
                                                   int parent_offset_index,
                                                   int offset_depth,
                                                   rmm::cuda_stream_view stream)
{
  lists_column_view lcv(col);

  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // list columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *data_cur = src_buf_info(
    type_id::LIST, offset_stack_pos, parent_offset_index, nullptr, buffer_type::DATA, col.offset());
  data_cur++;
  offset_stack_pos += offset_depth;

  CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed list column");

  // info for the offsets buffer
  *offset_cur = src_buf_info(
    type_id::INT32,
    offset_stack_pos,
    parent_offset_index,
    // note: offsets can be null in the case where the lists column
    // has been created with empty_like().
    reinterpret_cast<uint8_t const*>(lcv.offsets().begin<cudf::id_to_type<type_id::INT32>>()),
    buffer_type::OFFSETS,
    col.offset());

  // since we are crossing an offset boundary, calculate our new depth and parent offset index.
  parent_offset_index = offset_cur - head;
  offset_cur++;
  offset_stack_pos += offset_depth;
  offset_depth++;

  auto child_col = col.child_begin() + lists_column_view::child_column_index;
  setup_source_buf_info(child_col,
                        std::next(child_col),
                        head,
                        validity_cur,
                        offset_cur,
                        data_cur,
                        flattened_col_has_validity,
                        offset_stack_pos,
                        stream,
                        parent_offset_index,
                        offset_depth);
}

template <>
void buf_info_functor::operator()<cudf::struct_view>(
  column_view const& col,
  src_buf_info*& validity_cur,
  src_buf_info*& offset_cur,
  src_buf_info*& data_cur,
  std::vector<int8_t>& flattened_col_has_validity,
  int& offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // struct columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *data_cur = src_buf_info(type_id::STRUCT,
                           offset_stack_pos,
                           parent_offset_index,
                           nullptr,
                           buffer_type::DATA,
                           col.offset());
  data_cur++;
  offset_stack_pos += offset_depth;

  // recurse on children
  cudf::structs_column_view scv(col);
  std::vector<column_view> sliced_children;
  sliced_children.reserve(scv.num_children());
  std::transform(
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(scv.num_children()),
    std::back_inserter(sliced_children),
    [&scv, &stream](size_type child_index) { return scv.get_sliced_child(child_index, stream); });
  setup_source_buf_info(sliced_children.begin(),
                        sliced_children.end(),
                        head,
                        validity_cur,
                        offset_cur,
                        data_cur,
                        flattened_col_has_validity,
                        offset_stack_pos,
                        stream,
                        parent_offset_index,
                        offset_depth);
}

template <typename InputIter>
void setup_source_buf_info(InputIter begin,
                           InputIter end,
                           src_buf_info* head,
                           src_buf_info*& validity_cur,
                           src_buf_info*& offset_cur,
                           src_buf_info*& data_cur,
                           std::vector<int8_t>& flattened_col_has_validity,
                           int& offset_stack_pos,
                           rmm::cuda_stream_view stream,
                           int parent_offset_index,
                           int offset_depth)
{
  std::for_each(begin, end, [&](column_view const& col) {
    cudf::type_dispatcher(col.type(),
                          buf_info_functor{head},
                          col,
                          validity_cur,
                          offset_cur,
                          data_cur,
                          flattened_col_has_validity,
                          offset_stack_pos,
                          parent_offset_index,
                          offset_depth,
                          stream);
  });
}

/**
 * @brief Output iterator for writing values to the dst_offset field of the
 * dst_buf_info struct
 */
struct dst_offset_output_iterator {
  dst_buf_info* c;
  using value_type        = size_t;
  using difference_type   = size_t;
  using pointer           = size_t*;
  using reference         = size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_offset_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  void operator++ __host__ __device__() { c++; }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->dst_offset; }
};

/**
 * @brief Count the total number of flattened columns in a list of input columns.
 *
 * @param begin First column in the range
 * @param end Last column in the range
 * @param depth Current depth in the column hierarchy
 *
 * @return A pair containing the total number of columns and the maximum depth of the hierarchy.
 */
template <typename InputIter>
std::pair<size_t, size_t> count_flattened_columns(InputIter begin, InputIter end, int depth = 0)
{
  auto child_count = [&](column_view const& col, int depth) -> std::pair<size_t, size_t> {
    if (col.type().id() == cudf::type_id::STRUCT) {
      return count_flattened_columns(col.child_begin(), col.child_end(), depth + 1);
    } else if (col.type().id() == cudf::type_id::LIST) {
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      return count_flattened_columns(children.begin(), children.end(), depth + 1);
    }
    return {size_t{0}, depth};
  };

  size_t col_count = 0;
  size_t max_depth = 0;
  std::for_each(begin, end, [&](column_view const& col) {
    auto const cc = child_count(col, depth);
    col_count += (1 + cc.first);
    max_depth = std::max(max_depth, cc.second);
  });

  return {col_count, max_depth};
}

/**
 * @brief Sizes of each of the data sections in a partition.
 *
 * Does not include padding between sections.
 */
struct partition_size_info {
  size_t validity_size;
  size_t offset_size;
  size_t data_size;
};

/**
 * @brief Kernel that packs the per-partition metadata for the output.
 *
 * @param out_buffer The output buffer for the entire split operation
 * @param out_buffer_offsets Per-partition offsets into the output buffer
 * @param num_partitions The number of partitions
 * @param columns_per_partition The number of flattened columns per partition
 * @param split_indices Per-partition row split indices
 * @param flattened_col_has_validity Per-column bool on whether each column contains a validity
 * vector
 * @param partition_size_info Per-partition size information for each buffer type
 *
 */
__global__ void pack_per_partition_metadata_kernel(uint8_t* out_buffer,
                                                   size_t const* out_buffer_offsets,
                                                   size_t num_partitions,
                                                   size_t columns_per_partition,
                                                   size_type const* split_indices,
                                                   int8_t const* flattened_col_has_validity,
                                                   partition_size_info const* partition_sizes)
{
  constexpr uint32_t magic        = 'ODUK';
  constexpr uint32_t kudo_version = 1;

  int const tid = threadIdx.x + (blockIdx.x * blockDim.x);
  auto const threads_per_partition =
    cudf::util::round_up_safe(columns_per_partition, static_cast<size_t>(cudf::detail::warp_size));
  auto const partition_index = tid / threads_per_partition;
  if (partition_index >= num_partitions) { return; }
  auto const col_index = tid % threads_per_partition;

  // start of the metadata buffer for this partition
  uint8_t* buf_start        = out_buffer + out_buffer_offsets[partition_index];
  partition_header* pheader = reinterpret_cast<partition_header*>(buf_start);

  // first thread in each partition stores constant stuff
  if (col_index == 0) {
    pheader->magic_number = magic;
    pheader->version      = kudo_version;

    pheader->offset = split_indices[partition_index];

    // it is possible to get in here with no columns -or- no rows.
    size_type partition_num_rows = 0;
    if (col_index < columns_per_partition) {
      partition_num_rows = split_indices[partition_index + 1] - split_indices[partition_index];
    }
    pheader->num_rows = partition_num_rows;

    auto const& psize      = partition_sizes[partition_index];
    pheader->validity_size = psize.validity_size;
    pheader->offset_size   = psize.offset_size;
    pheader->data_size     = psize.data_size;
  }

  bitmask_type* has_validity =
    reinterpret_cast<bitmask_type*>(buf_start + sizeof(partition_header));

  // store has-validity bits. note that the kudo format only aligns to byte boundaries at the end of
  // the validity section, but we are doing this before anything further is written and we are
  // guaranteed that the overall buffer is padded out to >= 4 bytes.
  bitmask_type mask = __ballot_sync(
    0xffffffff, col_index < columns_per_partition ? flattened_col_has_validity[col_index] : 0);
  if ((col_index % cudf::detail::warp_size == 0) && col_index < columns_per_partition) {
    has_validity[col_index / cudf::detail::warp_size] = mask;
  }
}

/**
 * @brief Copy data from source buffers into the output.
 *
 * Uses Cub batched memcpy
 *
 * @param src_bufs Source buffer pointers
 * @param dst_buf Pointer to the output buffer
 * @param num_copies The number of copy operations to perform
 * @param d_dst_buf_info num_copies sized array of buffer information
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 */
void split_copy(src_buf_info const* src_bufs,
                uint8_t* dst_buf,
                size_t num_bufs,
                dst_buf_info const* d_dst_buf_info,
                rmm::cuda_stream_view stream)
{
  auto input_iter = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<void*>([src_bufs, d_dst_buf_info] __device__(size_t i) {
      auto const& cinfo = d_dst_buf_info[i];
      return reinterpret_cast<void*>(
        const_cast<uint8_t*>(src_bufs[cinfo.src_buf_index].data + cinfo.src_offset));
    }));
  auto output_iter = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<void*>([dst_buf, d_dst_buf_info] __device__(size_t i) {
      auto const& cinfo = d_dst_buf_info[i];
      return reinterpret_cast<void*>(dst_buf + cinfo.dst_offset);
    }));
  auto size_iter = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<size_t>([d_dst_buf_info] __device__(size_t i) {
      auto const& cinfo = d_dst_buf_info[i];
      return cinfo.buf_size;
    }));

  size_t temp_storage_bytes;
  hipcub::DeviceMemcpy::Batched(
    nullptr, temp_storage_bytes, input_iter, output_iter, size_iter, num_bufs, stream);
  rmm::device_buffer temp_storage(
    temp_storage_bytes, stream, cudf::get_current_device_resource_ref());
  hipcub::DeviceMemcpy::Batched(
    temp_storage.data(), temp_storage_bytes, input_iter, output_iter, size_iter, num_bufs, stream);
}

/**
 * @brief Fill in column metadata info for a list of input columns.
 *
 * @param meta Output metadata
 * @param begin Start of the range of columns
 * @param end End of the range of columns
 *
 */
template <typename InputIter>
void populate_column_data(shuffle_split_metadata& meta, InputIter begin, InputIter end)
{
  std::for_each(begin, end, [&meta](column_view const& col) {
    switch (col.type().id()) {
      case cudf::type_id::STRUCT:
        meta.col_info.push_back({col.type().id(), col.num_children()});
        populate_column_data(meta, col.child_begin(), col.child_end());
        break;

      case cudf::type_id::LIST: {
        meta.col_info.push_back({col.type().id(), 1});
        cudf::lists_column_view lcv(col);
        std::vector<cudf::column_view> children({lcv.child()});
        populate_column_data(meta, children.begin(), children.end());
      } break;

      case cudf::type_id::DECIMAL32:
      case cudf::type_id::DECIMAL64:
      case cudf::type_id::DECIMAL128:
        meta.col_info.push_back({col.type().id(), col.type().scale()});
        break;

      default: meta.col_info.push_back({col.type().id(), 0}); break;
    }
  });
}

/**
 * @brief Create the shuffle_split_metadata struct for the split operation
 *
 * @param input The input table
 * @param total_flattened_columns The total number of flattened columns in the input
 *
 * @return The final shuffle_split_metadata struct
 */
shuffle_split_metadata compute_metadata(cudf::table_view const& input,
                                        size_t total_flattened_columns)
{
  // compute the metadata
  shuffle_split_metadata ret;
  ret.col_info.reserve(total_flattened_columns);
  populate_column_data(ret, input.begin(), input.end());
  return ret;
}

};  // anonymous namespace

/**
 * @copydoc spark_rapids_jni::shuffle_split
 */
std::pair<shuffle_split_result, shuffle_split_metadata> shuffle_split(
  cudf::table_view const& input,
  std::vector<size_type> const& splits,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // empty inputs
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    rmm::device_uvector<size_t> empty_offsets(1, stream, mr);
    thrust::fill(rmm::exec_policy(stream), empty_offsets.begin(), empty_offsets.end(), 0);
    return {shuffle_split_result{std::make_unique<rmm::device_buffer>(0, stream, mr),
                                 std::move(empty_offsets)},
            shuffle_split_metadata{compute_metadata(input, 0)}};
  }
  if (splits.size() > 0) {
    CUDF_EXPECTS(splits.back() <= input.column(0).size(),
                 "splits can't exceed size of input columns");
  }
  {
    size_type begin = 0;
    for (size_t i = 0; i < splits.size(); i++) {
      size_type end = splits[i];
      CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.");
      CUDF_EXPECTS(end >= begin, "End index cannot be smaller than the starting index.");
      CUDF_EXPECTS(end <= input.column(0).size(), "Slice range out of bounds.");
      begin = end;
    }
  }

  auto temp_mr = cudf::get_current_device_resource_ref();

  size_t const num_partitions   = splits.size() + 1;
  size_t const num_root_columns = input.num_columns();

  // compute # of source buffers (column data, validity, children), # of partitions
  // and total # of buffers
  src_buf_count const num_src_bufs_by_type = count_src_bufs(input.begin(), input.end());
  size_t const num_src_bufs                = num_src_bufs_by_type.validity_buf_count +
                              num_src_bufs_by_type.offset_buf_count +
                              num_src_bufs_by_type.data_buf_count;
  size_t const num_bufs         = num_src_bufs * num_partitions;
  auto const bufs_per_partition = num_src_bufs;

  // packed block of memory 1. split indices and src_buf_info structs
  size_t const indices_size =
    cudf::util::round_up_safe((num_partitions + 1) * sizeof(size_type), split_align);
  size_t const src_buf_info_size =
    cudf::util::round_up_safe(num_src_bufs * sizeof(src_buf_info), split_align);
  // host-side
  std::vector<uint8_t> h_indices_and_source_info(indices_size + src_buf_info_size);
  size_type* h_indices = reinterpret_cast<size_type*>(h_indices_and_source_info.data());
  src_buf_info* h_src_buf_head =
    reinterpret_cast<src_buf_info*>(h_indices_and_source_info.data() + indices_size);
  src_buf_info* h_validity_buf_info = h_src_buf_head;
  src_buf_info* h_offset_buf_info   = h_validity_buf_info + num_src_bufs_by_type.validity_buf_count;
  src_buf_info* h_data_buf_info     = h_offset_buf_info + num_src_bufs_by_type.offset_buf_count;
  // device-side
  // gpu-only : stack space needed for nested list offset calculation
  int const offset_stack_partition_size = compute_offset_stack_size(input.begin(), input.end());
  size_t const offset_stack_size = offset_stack_partition_size * num_partitions * sizeof(size_type);
  rmm::device_buffer d_indices_and_source_info(indices_size + src_buf_info_size + offset_stack_size,
                                               stream,
                                               rmm::mr::get_current_device_resource());
  auto* d_indices              = reinterpret_cast<size_type*>(d_indices_and_source_info.data());
  src_buf_info* d_src_buf_info = reinterpret_cast<src_buf_info*>(
    reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) + indices_size);
  size_type* d_offset_stack =
    reinterpret_cast<size_type*>(reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) +
                                 indices_size + src_buf_info_size);

  // compute splits -> indices.
  h_indices[0]              = 0;
  h_indices[num_partitions] = input.column(0).size();
  std::copy(splits.begin(), splits.end(), std::next(h_indices));

  // setup source buf info
  auto const total_flattened_columns = count_flattened_columns(input.begin(), input.end()).first;
  std::vector<int8_t> flattened_col_has_validity;
  flattened_col_has_validity.reserve(total_flattened_columns);
  int offset_stack_pos = 0;
  setup_source_buf_info(input.begin(),
                        input.end(),
                        h_src_buf_head,
                        h_validity_buf_info,
                        h_offset_buf_info,
                        h_data_buf_info,
                        flattened_col_has_validity,
                        offset_stack_pos,
                        stream);
  auto d_flattened_col_has_validity =
    cudf::detail::make_device_uvector_async(flattened_col_has_validity, stream.value(), temp_mr);

  // HtoD indices and source buf info to device
  CUDF_CUDA_TRY(hipMemcpyAsync(
    d_indices, h_indices, indices_size + src_buf_info_size, hipMemcpyDefault, stream.value()));

  // packed block of memory 2. partition buffer sizes and dst_buf_info structs
  size_t const partition_sizes_size =
    cudf::util::round_up_safe(num_partitions * sizeof(partition_size_info), split_align);
  size_t const dst_buf_info_size =
    cudf::util::round_up_safe(num_bufs * sizeof(dst_buf_info), split_align);
  // host-side
  std::vector<uint8_t> h_buf_sizes_and_dst_info(partition_sizes_size + dst_buf_info_size);
  size_t* h_buf_sizes = reinterpret_cast<size_t*>(h_buf_sizes_and_dst_info.data());
  dst_buf_info* h_dst_buf_info =
    reinterpret_cast<dst_buf_info*>(h_buf_sizes_and_dst_info.data() + partition_sizes_size);
  // device-side
  rmm::device_buffer d_buf_sizes_and_dst_info(
    partition_sizes_size + dst_buf_info_size, stream, temp_mr);
  partition_size_info* d_partition_sizes =
    reinterpret_cast<partition_size_info*>(d_buf_sizes_and_dst_info.data());
  dst_buf_info* d_dst_buf_info = reinterpret_cast<dst_buf_info*>(
    static_cast<uint8_t*>(d_buf_sizes_and_dst_info.data()) + partition_sizes_size);

  // this has to be a separate allocation because it gets returned.
  rmm::device_uvector<size_t> d_partition_offsets(num_partitions + 1, stream, mr);

  // compute sizes of each buffer in each partition, including alignment.
  thrust::transform(
    rmm::exec_policy_nosync(stream),
    thrust::make_counting_iterator<size_t>(0),
    thrust::make_counting_iterator<size_t>(num_bufs),
    d_dst_buf_info,
    [bufs_per_partition,
     d_indices,
     d_src_buf_info,
     d_offset_stack,
     offset_stack_partition_size] __device__(size_t t) {
      int const partition_index = t / bufs_per_partition;
      int const src_buf_index   = t % bufs_per_partition;
      auto const& src_info      = d_src_buf_info[src_buf_index];

      // apply nested offsets (for lists and string columns).
      //
      // We can't just use the incoming row indices to figure out where to read from in a
      // nested list situation.  We have to apply offsets every time we cross a boundary
      // (list or string).  This loop applies those offsets so that our incoming row_index_start
      // and row_index_end get transformed to our final values.
      //
      int const stack_pos =
        src_info.offset_stack_pos + (partition_index * offset_stack_partition_size);
      size_type* offset_stack  = &d_offset_stack[stack_pos];
      int parent_offsets_index = src_info.parent_offsets_index;
      int stack_size           = 0;
      int root_column_offset   = src_info.column_offset;
      while (parent_offsets_index >= 0) {
        offset_stack[stack_size++] = parent_offsets_index;
        root_column_offset         = d_src_buf_info[parent_offsets_index].column_offset;
        parent_offsets_index       = d_src_buf_info[parent_offsets_index].parent_offsets_index;
      }
      // make sure to include the -column- offset on the root column in our calculation.
      int row_start = d_indices[partition_index] + root_column_offset;
      int row_end   = d_indices[partition_index + 1] + root_column_offset;
      while (stack_size > 0) {
        stack_size--;
        auto const offsets =
          reinterpret_cast<size_type const*>(d_src_buf_info[offset_stack[stack_size]].data);
        // this case can happen when you have empty string or list columns constructed with
        // empty_like()
        if (offsets != nullptr) {
          row_start = offsets[row_start];
          row_end   = offsets[row_end];
        }
      }

      // final element indices and row count
      size_t const src_element_index =
        src_info.btype == buffer_type::VALIDITY ? row_start / 8 : row_start;
      int const num_rows = row_end - row_start;
      // # of rows isn't necessarily the same as # of elements to be copied.
      auto const num_elements = [&]() {
        if ((src_info.btype == buffer_type::OFFSETS) && (src_info.data != nullptr) &&
            (num_rows > 0)) {
          return num_rows + 1;
        } else if (src_info.btype == buffer_type::VALIDITY) {
          return (num_rows + 7) / 8;
        }
        return num_rows;
      }();
      int const element_size =
        src_info.btype == buffer_type::VALIDITY
          ? 1
          : cudf::type_dispatcher(data_type{src_info.type}, size_of_helper{});
      size_t const bytes = static_cast<size_t>(num_elements) * static_cast<size_t>(element_size);

      return dst_buf_info{
        bytes, src_info.btype, src_buf_index, src_element_index * element_size, 0};
    });

  // compute per-partition metadata size
  auto const per_partition_metadata_size =
    compute_per_partition_metadata_size(total_flattened_columns);

  auto partition_keys = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<size_t>([bufs_per_partition] __device__(size_t buf_index) {
      return buf_index / bufs_per_partition;
    }));

  // - compute: size of all validity buffers, size of all offset buffers, size of all data buffers
  auto buf_sizes_by_type = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<partition_size_info>([d_dst_buf_info] __device__(int index) {
      switch (d_dst_buf_info[index].type) {
        case buffer_type::VALIDITY:
          return partition_size_info{d_dst_buf_info[index].buf_size, 0, 0};
        case buffer_type::OFFSETS: return partition_size_info{0, d_dst_buf_info[index].buf_size, 0};
        case buffer_type::DATA: return partition_size_info{0, 0, d_dst_buf_info[index].buf_size};
        default: break;
      }
      return partition_size_info{0, 0, 0};
    }));
  auto buf_size_reduce = cuda::proclaim_return_type<partition_size_info>(
    [] __device__(partition_size_info const& lhs, partition_size_info const& rhs) {
      auto const validity_size = lhs.validity_size + rhs.validity_size;
      auto const offset_size   = lhs.offset_size + rhs.offset_size;
      auto const data_size     = lhs.data_size + rhs.data_size;
      return partition_size_info{validity_size, offset_size, data_size};
    });
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        partition_keys,
                        partition_keys + num_bufs,
                        buf_sizes_by_type,
                        thrust::make_discard_iterator(),
                        d_partition_sizes,
                        thrust::equal_to{},  // key equality check
                        buf_size_reduce);

  // - compute partition start offsets and total output buffer size overall
  auto partition_size_iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<size_t>(
      [num_partitions, d_partition_sizes, per_partition_metadata_size] __device__(size_t i) {
        return i >= num_partitions
                 ? 0
                 : cudf::util::round_up_safe(
                     cudf::util::round_up_safe(
                       per_partition_metadata_size + d_partition_sizes[i].validity_size,
                       validity_pad) +
                       cudf::util::round_up_safe(d_partition_sizes[i].offset_size, offset_pad) +
                       d_partition_sizes[i].data_size,
                     data_pad);
      }));
  thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr),
                         partition_size_iter,
                         partition_size_iter + num_partitions + 1,
                         d_partition_offsets.begin());

  size_t dst_buf_total_size;
  hipMemcpyAsync(&dst_buf_total_size,
                  d_partition_offsets.begin() + num_partitions,
                  sizeof(size_t),
                  hipMemcpyDeviceToHost,
                  stream);

  // generate destination offsets for each of the source copies
  auto buf_sizes = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<size_t>([d_dst_buf_info] __device__(size_t i) {
      return d_dst_buf_info[i].buf_size;
    }));
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                                partition_keys,
                                partition_keys + num_bufs,
                                buf_sizes,
                                dst_offset_output_iterator{d_dst_buf_info});
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(
    rmm::exec_policy_nosync(stream, temp_mr),
    iter,
    iter + num_bufs,
    [per_partition_metadata_size,
     bufs_per_partition,
     d_dst_buf_info,
     d_partition_sizes,
     d_partition_offsets = d_partition_offsets.begin()] __device__(size_type i) {
      auto const partition_index = i / bufs_per_partition;
      auto const& ps             = d_partition_sizes[partition_index];
      // number of bytes from the start of the (validity|offsets|data) section
      auto const buffer_offset  = cuda::proclaim_return_type<size_t>([&] __device__() {
        switch (d_dst_buf_info[i].type) {
          case buffer_type::OFFSETS: return ps.validity_size;
          case buffer_type::DATA: return ps.validity_size + ps.offset_size;
          default: return size_t{0};
        }
      })();
      auto const section_offset = cuda::proclaim_return_type<size_t>([&] __device__() {
        switch (d_dst_buf_info[i].type) {
          case buffer_type::OFFSETS:
            return cudf::util::round_up_safe(per_partition_metadata_size + ps.validity_size,
                                             validity_pad);
          case buffer_type::DATA:
            return cudf::util::round_up_safe(
              cudf::util::round_up_safe(per_partition_metadata_size + ps.validity_size,
                                        validity_pad) +
                ps.offset_size,
              offset_pad);
          default: return per_partition_metadata_size;
        }
      })();
      d_dst_buf_info[i].dst_offset =
        d_partition_offsets[partition_index] +  // offset to the entire partition
        section_offset +                        // partition-relative offset to our section start
        (d_dst_buf_info[i].dst_offset -
         buffer_offset);  // section-relative offset to the start of our buffer
    });

  // allocate output buffer
  stream.synchronize();  // for dst_buf_total_size from above
  rmm::device_buffer dst_buf(dst_buf_total_size, stream, mr);

  // pack per-partition data. one thread per (flattened) column.
  size_type const thread_count_per_partition = cudf::util::round_up_safe(
    total_flattened_columns, static_cast<size_t>(cudf::detail::warp_size));
  cudf::detail::grid_1d const grid{
    thread_count_per_partition * static_cast<size_type>(num_partitions), 128};
  pack_per_partition_metadata_kernel<<<grid.num_blocks,
                                       grid.num_threads_per_block,
                                       0,
                                       stream.value()>>>(reinterpret_cast<uint8_t*>(dst_buf.data()),
                                                         d_partition_offsets.data(),
                                                         num_partitions,
                                                         total_flattened_columns,
                                                         d_indices,
                                                         d_flattened_col_has_validity.data(),
                                                         d_partition_sizes);

  // perform the copy.
  split_copy(
    d_src_buf_info, reinterpret_cast<uint8_t*>(dst_buf.data()), num_bufs, d_dst_buf_info, stream);

  // do this before the synchronize to take advantage of any gpu time we can overlap with (this
  // function only uses the cpu).
  auto metadata = compute_metadata(input, total_flattened_columns);

  stream.synchronize();
  return {shuffle_split_result{std::make_unique<rmm::device_buffer>(std::move(dst_buf)),
                               std::move(d_partition_offsets)},
          std::move(metadata)};
}

}  // namespace spark_rapids_jni