#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/contiguous_split.hpp>
#include <cudf/detail/contiguous_split.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <cstddef>
#include <numeric>

#include "shuffle_split.hpp"
//#include "db_test_workspace.cuh"
//#include "db_test.cuh"

namespace spark_rapids_jni {

using namespace cudf;

namespace {

// align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr std::size_t split_align = 64;

/**
 * @brief Struct which contains information on a source buffer.
 *
 * The definition of "buffer" used throughout this module is a component piece of a
 * cudf column. So for example, a fixed-width column with validity would have 2 associated
 * buffers : the data itself and the validity buffer.  contiguous_split operates by breaking
 * each column up into it's individual components and copying each one as a separate kernel
 * block.
 */
struct src_buf_info {
  src_buf_info(cudf::type_id _type,
               const int* _offsets,
               int _offset_stack_pos,
               int _parent_offsets_index,
               bool _is_validity,
               size_type _column_offset)
    : type(_type),
      offsets(_offsets),
      offset_stack_pos(_offset_stack_pos),
      parent_offsets_index(_parent_offsets_index),
      is_validity(_is_validity),
      column_offset(_column_offset)
  {
  }

  cudf::type_id type;
  const int* offsets;        // a pointer to device memory offsets if I am an offset buffer
  int offset_stack_pos;      // position in the offset stack buffer
  int parent_offsets_index;  // immediate parent that has offsets, or -1 if none
  bool is_validity;          // if I am a validity buffer
  size_type column_offset;   // offset in the case of a sliced column
};

enum class buffer_type {
  VALIDITY,
  OFFSETS,
  DATA
};

/**
 * @brief Struct which contains information on a destination buffer.
 *
 * Similar to src_buf_info, dst_buf_info contains information on a destination buffer we
 * are going to copy to.  If we have N input buffers (which come from X columns), and
 * M partitions, then we have N*M destination buffers.
 */
struct dst_buf_info {
  /*
  // constant across all copy commands for this buffer
  std::size_t buf_size;  // total size of buffer in bytes
  int num_elements;      // # of elements to be copied
  int element_size;      // size of each element in bytes
  int num_rows;  // # of rows to be copied(which may be different from num_elements in the case of
                 // validity or offset buffers)

  int src_element_index;   // element index to start reading from from my associated source buffer
  // std::size_t dst_offset;  // my offset into the per-partition allocation
  //int value_shift;         // amount to shift values down by (for offset buffers)
  //int bit_shift;           // # of bits to shift right by (for validity buffers)
  //size_type valid_count;   // validity count for this block of work
  buffer_type type;

  int src_buf_index;       // source buffer index
  int dst_buf_index;       // destination buffer index
  */
  
  std::size_t buf_size;    // total size of buffer
  buffer_type type;

  int src_buf_index;
  std::size_t src_offset;
  std::size_t dst_offset;
};

/**
 * @brief Copy a single buffer of column data, shifting values (for offset columns),
 * and validity (for validity buffers) as necessary.
 *
 * Copies a single partition of a source column buffer to a destination buffer. Shifts
 * element values by value_shift in the case of a buffer of offsets (value_shift will
 * only ever be > 0 in that case).  Shifts elements bitwise by bit_shift in the case of
 * a validity buffer (bif_shift will only ever be > 0 in that case).  This function assumes
 * value_shift and bit_shift will never be > 0 at the same time.
 *
 * This function expects:
 * - src may be a misaligned address
 * - dst must be an aligned address
 *
 * This function always does the ALU work related to value_shift and bit_shift because it is
 * entirely memory-bandwidth bound.
 *
 * @param dst Destination buffer
 * @param src Source buffer
 * @param t Thread index
 * @param num_elements Number of elements to copy
 * @param element_size Size of each element in bytes
 * @param src_element_index Element index to start copying at
 * @param stride Size of the kernel block
 * @param value_shift Shift incoming 4-byte offset values down by this amount
 * @param bit_shift Shift incoming data right by this many bits
 * @param num_rows Number of rows being copied
 * @param valid_count Optional pointer to a value to store count of set bits
 */
template <int block_size>
__device__ void copy_buffer(uint8_t* __restrict__ dst,
                            uint8_t const* __restrict__ src,
                            int t,
                            std::size_t num_elements,
                            std::size_t element_size,
                            std::size_t src_element_index,
                            uint32_t stride,
                            int value_shift,
                            int bit_shift,
                            std::size_t num_rows,
                            size_type* valid_count)
{
  src += (src_element_index * element_size);

  size_type thread_valid_count = 0;

  // handle misalignment. read 16 bytes in 4 byte reads. write in a single 16 byte store.
  std::size_t const num_bytes = num_elements * element_size;
  // how many bytes we're misaligned from 4-byte alignment
  uint32_t const ofs = reinterpret_cast<uintptr_t>(src) % 4;
  std::size_t pos    = t * 16;
  stride *= 16;
  while (pos + 20 <= num_bytes) {
    // read from the nearest aligned address.
    const uint32_t* in32 = reinterpret_cast<const uint32_t*>((src + pos) - ofs);
    uint4 v              = uint4{in32[0], in32[1], in32[2], in32[3]};
    if (ofs || bit_shift) {
      v.x = __funnelshift_r(v.x, v.y, ofs * 8 + bit_shift);
      v.y = __funnelshift_r(v.y, v.z, ofs * 8 + bit_shift);
      v.z = __funnelshift_r(v.z, v.w, ofs * 8 + bit_shift);
      v.w = __funnelshift_r(v.w, in32[4], ofs * 8 + bit_shift);
    }
    v.x -= value_shift;
    v.y -= value_shift;
    v.z -= value_shift;
    v.w -= value_shift;
    reinterpret_cast<uint4*>(dst)[pos / 16] = v;
    if (valid_count) {
      thread_valid_count += (__popc(v.x) + __popc(v.y) + __popc(v.z) + __popc(v.w));
    }
    pos += stride;
  }

  // copy trailing bytes
  if (t == 0) {
    std::size_t remainder;
    if (num_bytes < 16) {
      remainder = num_bytes;
    } else {
      std::size_t const last_bracket = (num_bytes / 16) * 16;
      remainder                      = num_bytes - last_bracket;
      if (remainder < 4) {
        // we had less than 20 bytes for the last possible 16 byte copy, so copy 16 + the extra
        remainder += 16;
      }
    }

    // if we're performing a value shift (offsets), or a bit shift (validity) the # of bytes and
    // alignment must be a multiple of 4. value shifting and bit shifting are mutually exclusive
    // and will never both be true at the same time.
    if (value_shift || bit_shift) {
      std::size_t idx = (num_bytes - remainder) / 4;
      uint32_t v = remainder > 0 ? (reinterpret_cast<uint32_t const*>(src)[idx] - value_shift) : 0;

      constexpr size_type rows_per_element = 32;
      auto const have_trailing_bits = ((num_elements * rows_per_element) - num_rows) < bit_shift;
      while (remainder) {
        // if we're at the very last word of a validity copy, we do not always need to read the next
        // word to get the final trailing bits.
        auto const read_trailing_bits = bit_shift > 0 && remainder == 4 && have_trailing_bits;
        uint32_t const next           = (read_trailing_bits || remainder > 4)
                                          ? (reinterpret_cast<uint32_t const*>(src)[idx + 1] - value_shift)
                                          : 0;

        uint32_t const val = (v >> bit_shift) | (next << (32 - bit_shift));
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint32_t*>(dst)[idx] = val;
        v                                     = next;
        idx++;
        remainder -= 4;
      }
    } else {
      while (remainder) {
        std::size_t const idx = num_bytes - remainder--;
        uint32_t const val    = reinterpret_cast<uint8_t const*>(src)[idx];
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint8_t*>(dst)[idx] = val;
      }
    }
  }

  if (valid_count) {
    if (num_bytes == 0) {
      if (!t) { *valid_count = 0; }
    } else {
      using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      size_type block_valid_count{BlockReduce(temp_storage).Sum(thread_valid_count)};
      if (!t) {
        // we may have copied more bits than there are actual rows in the output.
        // so we need to subtract off the count of any bits that shouldn't have been
        // considered during the copy step.
        std::size_t const max_row    = (num_bytes * 8);
        std::size_t const slack_bits = max_row > num_rows ? max_row - num_rows : 0;
        auto const slack_mask        = set_most_significant_bits(slack_bits);
        if (slack_mask > 0) {
          uint32_t const last_word = reinterpret_cast<uint32_t*>(dst + (num_bytes - 4))[0];
          block_valid_count -= __popc(last_word & slack_mask);
        }
        *valid_count = block_valid_count;
      }
    }
  }
}

// The block of functions below are all related:
//
// compute_offset_stack_size()
// setup_src_buf_data()
// count_src_bufs()
// setup_source_buf_info()
// build_output_columns()
//
// Critically, they all traverse the hierarchy of source columns and their children
// in a specific order to guarantee they produce various outputs in a consistent
// way.  For example, setup_src_buf_info() produces a series of information
// structs that must appear in the same order that setup_src_buf_data() produces
// buffers.
//
// So please be careful if you change the way in which these functions and
// functors traverse the hierarchy.

/**
 * @brief Returns whether or not the specified type is a column that contains offsets.
 */
bool is_offset_type(type_id id) { return (id == type_id::STRING or id == type_id::LIST); }

/**
 * @brief Compute total device memory stack size needed to process nested
 * offsets per-output buffer.
 *
 * When determining the range of rows to be copied for each output buffer
 * we have to recursively apply the stack of offsets from our parent columns
 * (lists or strings).  We want to do this computation on the gpu because offsets
 * are stored in device memory.  However we don't want to do recursion on the gpu, so
 * each destination buffer gets a "stack" of space to work with equal in size to
 * it's offset nesting depth.  This function computes the total size of all of those
 * stacks.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param offset_depth Current offset nesting depth
 *
 * @returns Total offset stack size needed for this range of columns.
 */
template <typename InputIter>
std::size_t compute_offset_stack_size(InputIter begin, InputIter end, int offset_depth = 0)
{
  return std::accumulate(begin, end, 0, [offset_depth](auto stack_size, column_view const& col) {
    auto const num_buffers = 1 + (col.nullable() ? 1 : 0);
    return stack_size + (offset_depth * num_buffers) +
           compute_offset_stack_size(
             col.child_begin(), col.child_end(), offset_depth + is_offset_type(col.type().id()));
  });
}

/**
 * @brief Retrieve all buffers for a range of source columns.
 *
 * Retrieve the individual buffers that make up a range of input columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param out_buf Iterator into output buffer infos
 *
 * @returns next output buffer iterator
 */
template <typename InputIter, typename OutputIter>
OutputIter setup_src_buf_data(InputIter begin, InputIter end, OutputIter out_buf)
{
  std::for_each(begin, end, [&out_buf](column_view const& col) {
    if (col.nullable()) {
      *out_buf = reinterpret_cast<uint8_t const*>(col.null_mask());
      out_buf++;
    }
    // NOTE: we're always returning the base pointer here.  column-level offset is accounted
    // for later. Also, for some column types (string, list, struct) this pointer will be null
    // because there is no associated data with the root column.
    *out_buf = col.head<uint8_t>();
    out_buf++;

    out_buf = setup_src_buf_data(col.child_begin(), col.child_end(), out_buf);
  });
  return out_buf;
}

/**
 * @brief Count the total number of source buffers, broken down by type (validity, offset, data) 
 * we will be copying from.
 *
 * This count includes buffers for all input columns. For example a
 * fixed-width column with validity would be 2 buffers (data, validity).
 * A string column with validity would be 3 buffers (chars, offsets, validity).
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 *
 * @returns total number of source buffer per type for this range of columns
 */
struct src_buf_count {
  size_t validity_buf_count;
  size_t offset_buf_count;
  size_t data_buf_count;
};
struct src_buf_count_add {
  src_buf_count operator()(src_buf_count const& lhs, src_buf_count const& rhs)
  {
    return {lhs.validity_buf_count + rhs.validity_buf_count,
            lhs.offset_buf_count + rhs.offset_buf_count,
            lhs.data_buf_count + rhs.data_buf_count};
  }
};
template <typename InputIter>
src_buf_count count_src_bufs(InputIter begin, InputIter end)
{
  auto buf_iter = thrust::make_transform_iterator(begin, [](column_view const& col) {
    auto const has_offsets_child = col.type().id() == cudf::type_id::LIST ||
                                   (col.type().id() == cudf::type_id::STRING && col.num_children() > 0);
    src_buf_count const counts{static_cast<size_t>(col.nullable() ? 1 : 0),
                               static_cast<size_t>(has_offsets_child ? 1 : 0),
                               size_t{1}};
    src_buf_count const child_counts = count_src_bufs(col.child_begin(), col.child_end());
    return src_buf_count_add{}(counts, child_counts);
  });
  return std::accumulate(buf_iter, buf_iter + std::distance(begin, end), src_buf_count{0, 0, 0}, src_buf_count_add{});
}

/**
 * @brief Computes source buffer information for the copy kernel.
 *
 * For each input column to be split we need to know several pieces of information
 * in the copy kernel.  This function traverses the input columns and prepares this
 * information for the gpu.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param head Beginning of source buffer info array
 * @param current Current source buffer info to be written to
 * @param offset_stack_pos Integer representing our current offset nesting depth
 * (how many list or string levels deep we are)
 * @param parent_offset_index Index into src_buf_info output array indicating our nearest
 * containing list parent. -1 if we have no list parent
 * @param offset_depth Current offset nesting depth (how many list levels deep we are)
 *
 * @returns next src_buf_output after processing this range of input columns
 */
// setup source buf info
template <typename InputIter>
void setup_source_buf_info(InputIter begin,
                           InputIter end,
                           src_buf_info* head,
                           src_buf_info *& validity_cur,
                           src_buf_info *& offset_cur,
                           src_buf_info *& data_cur,
                           std::vector<int8_t>& flattened_col_has_validity,
                           int& offset_stack_pos,
                           rmm::cuda_stream_view stream,
                           int parent_offset_index = -1,
                           int offset_depth        = 0);

/**
 * @brief Functor that builds source buffer information based on input columns.
 *
 * Called by setup_source_buf_info to build information for a single source column.  This function
 * will recursively call setup_source_buf_info in the case of nested types.
 */
struct buf_info_functor {
  src_buf_info* head;

  template <typename T>
  void operator()(column_view const& col,
                  src_buf_info *& validity_cur,
                  src_buf_info *& offset_cur,
                  src_buf_info *& data_cur,
                  std::vector<int8_t>& flattened_col_has_validity,
                  int& offset_stack_pos,
                  int parent_offset_index,
                  int offset_depth,
                  rmm::cuda_stream_view)
  {
    flattened_col_has_validity.push_back(col.nullable());
    if (col.nullable()) {
      add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
    }

    // info for the data buffer
    *data_cur = src_buf_info(
      col.type().id(), nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
    data_cur++;
    
    offset_stack_pos += offset_depth;
  }

  template <typename T, typename... Args>
  std::enable_if_t<std::is_same_v<T, cudf::dictionary32>, void>
  operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type");
  }

 private:
  void add_null_buffer(column_view const& col,
                       src_buf_info* validity_cur,
                       int offset_stack_pos,
                       int parent_offset_index,
                       int offset_depth)
  {
    // info for the validity buffer
    *validity_cur = src_buf_info(
      type_id::INT32, nullptr, offset_stack_pos, parent_offset_index, true, col.offset());
    validity_cur++;

    offset_stack_pos += offset_depth;
  }
};

template <>
void buf_info_functor::operator()<cudf::string_view>(
  column_view const& col,
  src_buf_info *& validity_cur,
  src_buf_info *& offset_cur,
  src_buf_info *& data_cur,
  std::vector<int8_t>& flattened_col_has_validity,
  int &offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view)
{
  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // the way strings are arranged, the strings column itself contains char data, but our child
  // offsets column actually contains our offsets. So our parent_offset_index is actually our child.

  // string columns don't necessarily have children if they are empty
  auto const has_offsets_child = col.num_children() > 0;

  // string columns contain the underlying chars data.
  *data_cur = src_buf_info(type_id::STRING,
                          nullptr,
                          offset_stack_pos,
                          // if I have offsets, it's index will be the current offset buffer, otherwise
                          // it's whatever my parent's was.
                          has_offsets_child ? (offset_cur - head) : parent_offset_index,
                          false,
                          col.offset());
  data_cur++;
  // if I have offsets, I need to include that in the stack size
  offset_stack_pos += has_offsets_child ? offset_depth + 1 : offset_depth;

  if (has_offsets_child) {
    CUDF_EXPECTS(col.num_children() == 1, "Encountered malformed string column");
    strings_column_view scv(col);

    // info for the offsets buffer
    CUDF_EXPECTS(not scv.offsets().nullable(), "Encountered nullable string offsets column");
    *offset_cur = src_buf_info(type_id::INT32,
                               // note: offsets can be null in the case where the string column
                               // has been created with empty_like().
                               scv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                               offset_stack_pos,
                               parent_offset_index,
                               false,
                               col.offset());

    offset_cur++;
    offset_stack_pos += offset_depth;
  }
}

template <>
void buf_info_functor::operator()<cudf::list_view>(
  column_view const& col,
  src_buf_info *& validity_cur,
  src_buf_info *& offset_cur,
  src_buf_info *& data_cur,
  std::vector<int8_t>& flattened_col_has_validity,
  int &offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  lists_column_view lcv(col);

  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // list columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *data_cur = src_buf_info(
    type_id::LIST, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  data_cur++;
  offset_stack_pos += offset_depth;

  CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed list column");

  // info for the offsets buffer
  *offset_cur        = src_buf_info(type_id::INT32,
                                    // note: offsets can be null in the case where the lists column
                                    // has been created with empty_like().
                                    lcv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                                    offset_stack_pos,
                                    parent_offset_index,
                                    false,
                                    col.offset());
  // since we are crossing an offset boundary, calculate our new depth and parent offset index.  
  parent_offset_index = offset_cur - head;
  offset_cur++;
  offset_stack_pos += offset_depth;
  offset_depth++;

  setup_source_buf_info(col.child_begin() + 1,
                        col.child_end(),
                        head,
                        validity_cur,
                        offset_cur,
                        data_cur,
                        flattened_col_has_validity,
                        offset_stack_pos,
                        stream,
                        parent_offset_index,
                        offset_depth);
}

template <>
void buf_info_functor::operator()<cudf::struct_view>(
  column_view const& col,
  src_buf_info *& validity_cur,
  src_buf_info *& offset_cur,
  src_buf_info *& data_cur,
  std::vector<int8_t>& flattened_col_has_validity,
  int &offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  flattened_col_has_validity.push_back(col.nullable());
  if (col.nullable()) {
    add_null_buffer(col, validity_cur, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // struct columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *data_cur = src_buf_info(
    type_id::STRUCT, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  data_cur++;
  offset_stack_pos += offset_depth;

  // recurse on children
  cudf::structs_column_view scv(col);
  std::vector<column_view> sliced_children;
  sliced_children.reserve(scv.num_children());
  std::transform(
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(scv.num_children()),
    std::back_inserter(sliced_children),
    [&scv, &stream](size_type child_index) { return scv.get_sliced_child(child_index, stream); });
  setup_source_buf_info(sliced_children.begin(),
                        sliced_children.end(),
                        head,
                        validity_cur,
                        offset_cur,
                        data_cur,
                        flattened_col_has_validity,
                        offset_stack_pos,
                        stream,
                        parent_offset_index,
                        offset_depth);
}

template <typename InputIter>
void setup_source_buf_info(InputIter begin,
                           InputIter end,
                           src_buf_info* head,
                           src_buf_info*& validity_cur,
                           src_buf_info*& offset_cur,
                           src_buf_info*& data_cur,
                           std::vector<int8_t>& flattened_col_has_validity,
                           int &offset_stack_pos,
                           rmm::cuda_stream_view stream,
                           int parent_offset_index,
                           int offset_depth)
{
  std::for_each(begin, end, [&](column_view const& col) {
    cudf::type_dispatcher(col.type(),
                          buf_info_functor{head},
                          col,
                          validity_cur,
                          offset_cur,
                          data_cur,
                          flattened_col_has_validity,
                          offset_stack_pos,
                          parent_offset_index,
                          offset_depth,
                          stream);
  });
}

/**
 * @brief Given a set of input columns and processed split buffers, produce
 * output columns.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param out_begin Output iterator of column views
 * @param base_ptr Pointer to the base address of copied data for the working partition
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
template <typename InputIter, typename BufInfo, typename Output>
BufInfo build_output_columns(InputIter begin,
                             InputIter end,
                             BufInfo info_begin,
                             Output out_begin,
                             uint8_t const* const base_ptr)
{
  auto current_info = info_begin;
  std::transform(begin, end, out_begin, [&current_info, base_ptr](column_view const& src) {
    auto [bitmask_ptr, null_count] = [&]() {
      if (src.nullable()) {
        auto const ptr =
          current_info->num_elements == 0
            ? nullptr
            : reinterpret_cast<bitmask_type const*>(base_ptr + current_info->dst_offset);
        auto const null_count = current_info->num_elements == 0
                                  ? 0
                                  : (current_info->num_rows - current_info->valid_count);
        ++current_info;
        return std::pair(ptr, null_count);
      }
      return std::pair(static_cast<bitmask_type const*>(nullptr), 0);
    }();

    // size/data pointer for the column
    auto const size = current_info->num_elements;
    uint8_t const* data_ptr =
      size == 0 || src.head() == nullptr ? nullptr : base_ptr + current_info->dst_offset;
    ++current_info;

    // children
    auto children = std::vector<column_view>{};
    children.reserve(src.num_children());

    current_info = build_output_columns(
      src.child_begin(), src.child_end(), current_info, std::back_inserter(children), base_ptr);

    return column_view{src.type(), size, data_ptr, bitmask_ptr, null_count, 0, std::move(children)};
  });

  return current_info;
}

struct partition_size_info {
  size_t validity_size;
  size_t offset_size;
  size_t data_size;
};

/**
 * @brief Output iterator for writing values to the dst_offset field of the
 * dst_buf_info struct
 */
struct dst_offset_output_iterator {
  dst_buf_info* c;
  using value_type        = std::size_t;
  using difference_type   = std::size_t;
  using pointer           = std::size_t*;
  using reference         = std::size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_offset_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  void operator++ __host__ __device__() { c++; }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->dst_offset; }
};

/**
 * @brief Output iterator for writing values to the valid_count field of the
 * dst_buf_info struct
 */
/*
struct dst_valid_count_output_iterator {
  dst_buf_info* c;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_valid_count_output_iterator operator+ __host__ __device__(int i)
  {
    return dst_valid_count_output_iterator{c + i};
  }

  void operator++ __host__ __device__() { c++; }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->valid_count; }
};
*/

/**
 * @brief Functor for computing size of data elements for a given cudf type.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct size_of_helper {
  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && !std::is_same_v<T, cudf::string_view>, size_t>
    __device__ operator()() const
  {    
    return 0;
  }

  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && std::is_same_v<T, cudf::string_view>, size_t>
    __device__ operator()() const
  {
    return sizeof(cudf::device_storage_type_t<int8_t>);
  }

  template <typename T>
  constexpr std::enable_if_t<is_fixed_width<T>(), size_t> __device__ operator()() const noexcept
  {
    return sizeof(cudf::device_storage_type_t<T>);
  }
};

template <typename InputIter>
std::pair<size_t, size_type> count_flattened_columns(InputIter begin, InputIter end, int depth = 0)
{
  auto child_count = [&](column_view const& col, int depth) -> std::pair<size_type, size_type> {
    if(col.type().id() == cudf::type_id::STRUCT){
      return count_flattened_columns(col.child_begin(), col.child_end(), depth+1);
    } else if(col.type().id() == cudf::type_id::LIST){
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      return count_flattened_columns(children.begin(), children.end(), depth+1);
    }
    return {0, depth};
  };

  size_type col_count = 0;
  size_type max_depth = 0;
  std::for_each(begin, end, [&](column_view const& col){
    auto const cc = child_count(col, depth);
    col_count += (1 + cc.first);
    max_depth = std::max(max_depth, cc.second);
  });

  return {col_count, max_depth};
}

struct partition_header {
  uint32_t    magic_number;
  uint32_t    version;
  uint64_t    offset;
  uint64_t    num_rows;
  uint64_t    validity_size;
  uint64_t    offset_size;
  uint64_t    data_size;
};
constexpr size_t validity_pad = 4;
constexpr size_t offset_pad = 8;
constexpr size_t data_pad = 8;

size_t compute_per_partition_metadata_size(size_t total_columns)
{
  auto const has_validity_length = (total_columns + 7) / 8; // has-validity bit per column
  return sizeof(partition_header) + has_validity_length;
}

__global__ void pack_per_partition_data_kernel(uint8_t* out_buffer,
                                               size_t num_partitions,
                                               size_t columns_per_partition,
                                               size_t bufs_per_partition,
                                               src_buf_info const* src_buf_info,
                                               size_type const* split_indices,
                                               int8_t const* flattened_col_has_validity,
                                               size_t const* out_buffer_offsets,
                                               partition_size_info const* partition_sizes)
{
  constexpr uint32_t magic = 'ODUK';
  constexpr uint32_t kudo_version = 1;

  int const tid = threadIdx.x + blockIdx.x * blockDim.x;
  auto const threads_per_partition = cudf::util::round_up_safe(columns_per_partition, static_cast<size_t>(cudf::detail::warp_size));
  auto const partition_index = tid / threads_per_partition;
  if(partition_index >= num_partitions){
    return;
  }  
  auto const col_index = tid % threads_per_partition;

  // start of the metadata buffer for this partition
  uint8_t* buf_start = out_buffer + out_buffer_offsets[partition_index];
  partition_header* pheader = reinterpret_cast<partition_header*>(buf_start);

  // first thread in each partition stores constant stuff
  if(col_index == 0){
    pheader->magic_number = magic;
    pheader->version = kudo_version;

    pheader->offset = 0;  // TODO

    // it is possible to get in here with no columns -or- no rows.
    size_type partition_num_rows = 0;
    if(col_index < columns_per_partition){
      partition_num_rows = split_indices[partition_index+1] - split_indices[partition_index];
      // printf("CBI: %d %d %d\n", (int)partition_index, (int)col_index, (int)partition_num_rows);
    }
    pheader->num_rows = partition_num_rows;

    auto const& psize = partition_sizes[partition_index];
    pheader->validity_size = psize.validity_size;
    pheader->offset_size = psize.offset_size;
    pheader->data_size = psize.data_size;
  }

  bitmask_type* has_validity = reinterpret_cast<bitmask_type*>(buf_start + sizeof(partition_header));

  // store has-validity bits. note that the kudo format only aligns to byte boundaries here, but we are guaranteed that the overall buffer is
  // padded out to >= 4 bytes.  
  bitmask_type mask = __ballot_sync(0xffffffff, col_index < columns_per_partition ? flattened_col_has_validity[col_index] : 0);
  if((col_index % cudf::detail::warp_size == 0) && col_index < columns_per_partition){
    // printf("HV: %d : %d, %d, %d\n", (int)(col_index / cudf::detail::warp_size), (int)mask, (int)col_index, (int)tid);
    has_validity[col_index / cudf::detail::warp_size] = mask;
  }
}

// perform the copy.
void copy_data(size_t num_src_bufs, uint8_t const **src_bufs, size_t num_bufs, uint8_t* dst_buf, dst_buf_info* d_dst_buf_info, rmm::cuda_stream_view stream)
{
  auto input_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<void*>([src_bufs, d_dst_buf_info] __device__ (size_t i){
    auto const& cinfo = d_dst_buf_info[i];
    //printf("Split src (%d %lu): %lu\n", cinfo.src_buf_index, i, (uint64_t)(src_bufs[cinfo.src_buf_index] + cinfo.src_offset));
    return reinterpret_cast<void*>(const_cast<uint8_t*>(src_bufs[cinfo.src_buf_index] + cinfo.src_offset));
  }));
  auto output_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<void*>([dst_buf, d_dst_buf_info] __device__ (size_t i){
    auto const& cinfo = d_dst_buf_info[i];
    //printf("Split dst (%d %lu): %lu\n", cinfo.src_buf_index, i,(uint64_t)(dst_buf + cinfo.dst_offset));
    return reinterpret_cast<void*>(dst_buf + cinfo.dst_offset);
  }));
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([d_dst_buf_info] __device__ (size_t i){
    auto const& cinfo = d_dst_buf_info[i];
    //printf("Split size (%d %lu): %lu\n", cinfo.src_buf_index, i, cinfo.buf_size);
    return cinfo.buf_size;
  }));

  size_t temp_storage_bytes;
  hipcub::DeviceMemcpy::Batched(nullptr, temp_storage_bytes, input_iter, output_iter, size_iter, num_bufs, stream);
  rmm::device_buffer temp_storage(temp_storage_bytes, stream, cudf::get_current_device_resource_ref());
  hipcub::DeviceMemcpy::Batched(temp_storage.data(), temp_storage_bytes, input_iter, output_iter, size_iter, num_bufs, stream);

  // debug.
  stream.synchronize();
  int whee = 10;
  whee++;
}

template <typename InputIter>
void populate_column_data(shuffle_split_metadata& meta, InputIter begin, InputIter end)
{
  std::for_each(begin, end, [&meta](column_view const& col){
    switch(col.type().id()){
    case cudf::type_id::STRUCT:
      meta.col_info.push_back({col.type().id(), col.num_children()});
      populate_column_data(meta, col.child_begin(), col.child_end());
      break;
    
    case cudf::type_id::LIST: {
      meta.col_info.push_back({col.type().id(), 1});
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      populate_column_data(meta, children.begin(), children.end());
      } break;

    case cudf::type_id::DECIMAL32:
    case cudf::type_id::DECIMAL64:
    case cudf::type_id::DECIMAL128:
      // TODO: scale.
      meta.col_info.push_back({col.type().id(), 0});
      break;

    default:
      meta.col_info.push_back({col.type().id(), 0});
      break;
    }
  });
}

// returns global metadata describing the table and the size of the
// internal per-partition data
shuffle_split_metadata compute_metadata(cudf::table_view const& input, size_t total_flattened_columns)
{
  // compute the metadata
  shuffle_split_metadata ret;
  ret.col_info.reserve(total_flattened_columns);
  populate_column_data(ret, input.begin(), input.end());
  return ret;
}

};  // anonymous namespace

std::pair<shuffle_split_result, shuffle_split_metadata> shuffle_split(cudf::table_view const& input,
                                                                      std::vector<size_type> const& splits,
                                                                      rmm::cuda_stream_view stream,
                                                                      rmm::device_async_resource_ref mr)
{
  // empty inputs
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    rmm::device_uvector<size_t> empty_offsets(1, stream, mr);
    thrust::fill(rmm::exec_policy(stream), empty_offsets.begin(), empty_offsets.end(), 0);
    return {shuffle_split_result{std::make_unique<rmm::device_buffer>(0, stream, mr), std::move(empty_offsets)},
            shuffle_split_metadata{compute_metadata(input, 0)}};
  }
  if (splits.size() > 0) {
    CUDF_EXPECTS(splits.back() <= input.column(0).size(),
                 "splits can't exceed size of input columns");
  }
  {
    size_type begin = 0;
    for (std::size_t i = 0; i < splits.size(); i++) {
      size_type end = splits[i];
      CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.");
      CUDF_EXPECTS(end >= begin, "End index cannot be smaller than the starting index.");
      CUDF_EXPECTS(end <= input.column(0).size(), "Slice range out of bounds.");
      begin = end;
    }
  }

  auto temp_mr = cudf::get_current_device_resource_ref();

  size_t const num_partitions   = splits.size() + 1;
  size_t const num_root_columns = input.num_columns();

  // if inputs are empty, just return num_partitions empty tables
  /*
  if (input.column(0).size() == 0) {
    // sanitize the inputs (to handle corner cases like sliced tables)
    std::vector<std::unique_ptr<column>> empty_columns;
    empty_columns.reserve(input.num_columns());
    std::transform(
      input.begin(), input.end(), std::back_inserter(empty_columns), [](column_view const& col) {
        return cudf::empty_like(col);
      });
    std::vector<cudf::column_view> empty_column_views;
    empty_column_views.reserve(input.num_columns());
    std::transform(empty_columns.begin(),
                   empty_columns.end(),
                   std::back_inserter(empty_column_views),
                   [](std::unique_ptr<column> const& col) { return col->view(); });
    table_view empty_inputs(empty_column_views);

    // build the empty results
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    auto iter = thrust::make_counting_iterator(0);
    std::transform(iter,
                   iter + num_partitions,
                   std::back_inserter(result),
                   [&empty_inputs](int partition_index) {
                     return packed_table{
                       empty_inputs,
                       packed_columns{std::make_unique<std::vector<uint8_t>>(pack_metadata(
                                        empty_inputs, static_cast<uint8_t const*>(nullptr), 0)),
                                      std::make_unique<rmm::device_buffer>()}};
                   });

    return result;
  }
  */

  // compute # of source buffers (column data, validity, children), # of partitions
  // and total # of buffers
  src_buf_count const num_src_bufs_by_type = count_src_bufs(input.begin(), input.end());
  size_t const num_src_bufs = num_src_bufs_by_type.validity_buf_count +
                              num_src_bufs_by_type.offset_buf_count +
                              num_src_bufs_by_type.data_buf_count;
  size_t const num_bufs = num_src_bufs * num_partitions;
  auto const bufs_per_partition = num_src_bufs;

  // packed block of memory 1. split indices and src_buf_info structs
  std::size_t const indices_size =
    cudf::util::round_up_safe((num_partitions + 1) * sizeof(size_type), split_align);
  std::size_t const src_buf_info_size =
    cudf::util::round_up_safe(num_src_bufs * sizeof(src_buf_info), split_align);
  // host-side
  std::vector<uint8_t> h_indices_and_source_info(indices_size + src_buf_info_size);
  size_type* h_indices = reinterpret_cast<size_type*>(h_indices_and_source_info.data());
  src_buf_info* h_src_buf_head =  
    reinterpret_cast<src_buf_info*>(h_indices_and_source_info.data() + indices_size);
  src_buf_info* h_validity_buf_info = h_src_buf_head;
  src_buf_info* h_offset_buf_info = h_validity_buf_info + num_src_bufs_by_type.validity_buf_count;
  src_buf_info* h_data_buf_info = h_offset_buf_info + num_src_bufs_by_type.offset_buf_count;
  // device-side
  // gpu-only : stack space needed for nested list offset calculation
  int const offset_stack_partition_size = compute_offset_stack_size(input.begin(), input.end());
  std::size_t const offset_stack_size =
    offset_stack_partition_size * num_partitions * sizeof(size_type);
  rmm::device_buffer d_indices_and_source_info(indices_size + src_buf_info_size + offset_stack_size,
                                               stream,
                                               rmm::mr::get_current_device_resource());
  auto* d_indices              = reinterpret_cast<size_type*>(d_indices_and_source_info.data());
  src_buf_info* d_src_buf_info = reinterpret_cast<src_buf_info*>(
    reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) + indices_size);
  size_type* d_offset_stack =
    reinterpret_cast<size_type*>(reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) +
                                 indices_size + src_buf_info_size);

  // compute splits -> indices.
  h_indices[0]              = 0;
  h_indices[num_partitions] = input.column(0).size();
  std::copy(splits.begin(), splits.end(), std::next(h_indices));

  // setup source buf info
  auto const total_flattened_columns = count_flattened_columns(input.begin(), input.end()).first;
  std::vector<int8_t> flattened_col_has_validity;
  flattened_col_has_validity.reserve(total_flattened_columns);
  int offset_stack_pos = 0;
  setup_source_buf_info(input.begin(), input.end(), h_src_buf_head, h_validity_buf_info, h_offset_buf_info, h_data_buf_info, flattened_col_has_validity, offset_stack_pos, stream);
  auto d_flattened_col_has_validity = cudf::detail::make_device_uvector_async(flattened_col_has_validity, stream.value(), temp_mr);

  // HtoD indices and source buf info to device
  CUDF_CUDA_TRY(hipMemcpyAsync(
    d_indices, h_indices, indices_size + src_buf_info_size, hipMemcpyDefault, stream.value()));

  // packed block of memory 2. partition buffer sizes and dst_buf_info structs
  std::size_t const partition_sizes_size =
    cudf::util::round_up_safe(num_partitions * sizeof(partition_size_info), split_align);
  std::size_t const dst_buf_info_size =
    cudf::util::round_up_safe(num_bufs * sizeof(dst_buf_info), split_align);
  // host-side
  std::vector<uint8_t> h_buf_sizes_and_dst_info(partition_sizes_size + dst_buf_info_size);
  std::size_t* h_buf_sizes = reinterpret_cast<std::size_t*>(h_buf_sizes_and_dst_info.data());
  dst_buf_info* h_dst_buf_info =
    reinterpret_cast<dst_buf_info*>(h_buf_sizes_and_dst_info.data() + partition_sizes_size);
  // device-side  
  rmm::device_buffer d_buf_sizes_and_dst_info(
    partition_sizes_size + dst_buf_info_size, stream, temp_mr);
  partition_size_info* d_partition_sizes     = reinterpret_cast<partition_size_info*>(d_buf_sizes_and_dst_info.data());
  dst_buf_info* d_dst_buf_info = reinterpret_cast<dst_buf_info*>(
    static_cast<uint8_t*>(d_buf_sizes_and_dst_info.data()) + partition_sizes_size);

  // this has to be a separate allocation because it gets returned.
  rmm::device_uvector<size_t> d_partition_offsets(num_partitions + 1, stream, mr);

  // compute sizes of each buffer in each partition, including alignment.
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<std::size_t>(0),
    thrust::make_counting_iterator<std::size_t>(num_bufs),
    d_dst_buf_info,
    [bufs_per_partition,
     d_indices,
     d_src_buf_info,
     d_offset_stack,
     offset_stack_partition_size] __device__(std::size_t t) {
      int const partition_index   = t / bufs_per_partition;
      int const src_buf_index = t % bufs_per_partition;
      auto const& src_info    = d_src_buf_info[src_buf_index];

      // apply nested offsets (for lists and string columns).
      //
      // We can't just use the incoming row indices to figure out where to read from in a
      // nested list situation.  We have to apply offsets every time we cross a boundary
      // (list or string).  This loop applies those offsets so that our incoming row_index_start
      // and row_index_end get transformed to our final values.
      //
      int const stack_pos = src_info.offset_stack_pos + (partition_index * offset_stack_partition_size);
      size_type* offset_stack  = &d_offset_stack[stack_pos];
      int parent_offsets_index = src_info.parent_offsets_index;
      int stack_size           = 0;
      int root_column_offset   = src_info.column_offset;
      while (parent_offsets_index >= 0) {
        offset_stack[stack_size++] = parent_offsets_index;
        root_column_offset         = d_src_buf_info[parent_offsets_index].column_offset;
        parent_offsets_index       = d_src_buf_info[parent_offsets_index].parent_offsets_index;
      }
      // make sure to include the -column- offset on the root column in our calculation.
      int row_start = d_indices[partition_index] + root_column_offset;
      int row_end   = d_indices[partition_index + 1] + root_column_offset;
      while (stack_size > 0) {
        stack_size--;
        auto const offsets = d_src_buf_info[offset_stack[stack_size]].offsets;
        // this case can happen when you have empty string or list columns constructed with
        // empty_like()
        if (offsets != nullptr) {
          row_start = offsets[row_start];
          row_end   = offsets[row_end];
        }
      }

      // final element indices and row count
      size_t const src_element_index = src_info.is_validity ? row_start / 8 : row_start;
      int const num_rows          = row_end - row_start;
      // if I am an offsets column, all my values need to be shifted
      //int const value_shift = src_info.offsets == nullptr ? 0 : src_info.offsets[row_start];
      // if I am a validity column, we may need to shift bits
      //int const bit_shift = src_info.is_validity ? row_start % 32 : 0;
      // # of rows isn't necessarily the same as # of elements to be copied.
      auto const num_elements = [&]() {
        if (src_info.offsets != nullptr && num_rows > 0) {
          return num_rows + 1;
        } else if (src_info.is_validity) {
          return (num_rows + 7) / 8;
        }
        return num_rows;
      }();
      buffer_type const btype = [&]() {
        if (src_info.offsets != nullptr) {
          return buffer_type::OFFSETS;
        } else if (src_info.is_validity) {
          return buffer_type::VALIDITY;
        }
        return buffer_type::DATA;
      }();
      int const element_size = cudf::type_dispatcher(data_type{src_info.type}, size_of_helper{});
      std::size_t const bytes =
        static_cast<std::size_t>(num_elements) * static_cast<std::size_t>(element_size);
      
      // printf("P: %d %d %lu\n", partition_index, src_buf_index, bytes);

      return dst_buf_info{bytes,
                          btype,
                          src_buf_index,
                          src_element_index * element_size,
                          0};   // dst_offset is computed later
                          //num_elements,
                          //element_size,
                          //num_rows,
                          //src_element_index,
                          // 0,
                          //btype,
                          //src_info.is_validity ? 1 : 0,
                          //value_shift,
                          //bit_shift,
                          //src_info.is_validity ? 1 : 0,
                          //src_buf_index,
                          //split_index};
    });

  // compute per-partition metadata size
  auto const per_partition_metadata_size = compute_per_partition_metadata_size(total_flattened_columns);

  auto partition_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([bufs_per_partition] __device__ (size_t buf_index){
    return buf_index / bufs_per_partition;
  }));

  // - compute: size of all validity buffers, size of all offset buffers, size of all data buffers
  auto buf_sizes_by_type =
    cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<partition_size_info>([d_dst_buf_info] __device__ (int index){ 
      switch(d_dst_buf_info[index].type){
      case buffer_type::VALIDITY: return partition_size_info{d_dst_buf_info[index].buf_size, 0, 0};
      case buffer_type::OFFSETS: return partition_size_info{0, d_dst_buf_info[index].buf_size, 0};
      case buffer_type::DATA: return partition_size_info{0, 0, d_dst_buf_info[index].buf_size};
      default: break;
      }
      return partition_size_info{0, 0, 0};
    }));
  auto buf_size_reduce = cuda::proclaim_return_type<partition_size_info>([] __device__ (partition_size_info const& lhs, partition_size_info const& rhs){
                           auto const validity_size = lhs.validity_size + rhs.validity_size;
                           auto const offset_size = lhs.offset_size + rhs.offset_size;
                           auto const data_size = lhs.data_size + rhs.data_size;
                           return partition_size_info{validity_size, offset_size, data_size};
                        });
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        partition_keys,
                        partition_keys + num_bufs,
                        buf_sizes_by_type,
                        thrust::make_discard_iterator(),
                        d_partition_sizes,
                        thrust::equal_to{}, // key equality check
                        buf_size_reduce);  

  // - compute partition start offsets and total output buffer size overall
  auto partition_size_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([num_partitions, d_partition_sizes, per_partition_metadata_size] __device__ (size_t i){
    return i >= num_partitions ? 0 :
      cudf::util::round_up_safe(cudf::util::round_up_safe(per_partition_metadata_size + d_partition_sizes[i].validity_size, validity_pad) +
                                cudf::util::round_up_safe(d_partition_sizes[i].offset_size, offset_pad) +
                                d_partition_sizes[i].data_size,
                                data_pad);
  }));
  thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                         partition_size_iter,
                         partition_size_iter + num_partitions + 1,
                         d_partition_offsets.begin());

  size_t dst_buf_total_size;
  hipMemcpyAsync(&dst_buf_total_size, d_partition_offsets.begin() + num_partitions, sizeof(size_t), hipMemcpyDeviceToHost, stream);
  
  /*
  {
    std::vector<partition_size_info> h_partition_sizes(num_partitions);
    hipMemcpy(h_partition_sizes.data(), d_partition_sizes, sizeof(partition_size_info) * num_partitions, hipMemcpyDeviceToHost);
    std::vector<size_t> h_partition_offsets(num_partitions + 1);
    hipMemcpy(h_partition_offsets.data(), d_partition_offsets.data(), sizeof(size_t) * (num_partitions + 1), hipMemcpyDeviceToHost);
    printf("Per partition metadata size : %lu\n", per_partition_metadata_size);
    for(size_t idx=0; idx<num_partitions; idx++){
      size_t const partition_total = h_partition_offsets[idx+1] - h_partition_offsets[idx];
      printf("HBS(%lu): %lu, %lu, %lu, %lu, %lu\n", idx, h_partition_sizes[idx].validity_size, 
                                               h_partition_sizes[idx].offset_size, 
                                               h_partition_sizes[idx].data_size,
                                               partition_total,
                                               h_partition_offsets[idx]);
    }
  }
  */
  
  // generate individual buffer offsets
  auto buf_sizes = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([d_dst_buf_info] __device__ (size_t i){
    return d_dst_buf_info[i].buf_size;
  }));  
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                                partition_keys,
                                partition_keys + num_bufs,
                                buf_sizes,
                                dst_offset_output_iterator{d_dst_buf_info});
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(rmm::exec_policy(stream, temp_mr),
                  iter,
                  iter + num_bufs,
                  [per_partition_metadata_size,
                   bufs_per_partition,
                   d_dst_buf_info,
                   d_partition_sizes,
                   d_partition_offsets = d_partition_offsets.begin()]  __device__ (size_type i){

    auto const partition_index = i / bufs_per_partition;
    auto const section_offset = cuda::proclaim_return_type<size_t>([&] __device__ (){
      auto const& ps = d_partition_sizes[partition_index];
      switch(d_dst_buf_info[i].type){
      case buffer_type::OFFSETS: return cudf::util::round_up_safe(per_partition_metadata_size + ps.validity_size, validity_pad);
      case buffer_type::DATA: return cudf::util::round_up_safe(cudf::util::round_up_safe(per_partition_metadata_size + ps.validity_size, validity_pad) + ps.offset_size,
                                                               offset_pad);
      default: return per_partition_metadata_size;
      }
    })();    
    size_t const pre = d_dst_buf_info[i].dst_offset;
    d_dst_buf_info[i].dst_offset += d_partition_offsets[partition_index] + section_offset;
    auto const& ps = d_partition_sizes[partition_index];
    // printf("dst(p:%d %d (%d)): %lu, %lu, %lu %lu (%lu %lu)\n", (int)i, (int)partition_index, (int)d_dst_buf_info[i].type, d_partition_offsets[partition_index], section_offset, d_dst_buf_info[i].dst_offset, pre, ps.validity_size, ps.offset_size);
  });

  // packed block of memory 3. pointers to source and destination buffers (and stack space on the
  // gpu for offset computation)
  std::size_t const src_bufs_size =
    cudf::util::round_up_safe(num_src_bufs * sizeof(uint8_t*), split_align);
  // host-side
  std::vector<uint8_t> h_src_and_dst_buffers(src_bufs_size/* + dst_bufs_size*/);
  uint8_t const** h_src_bufs = reinterpret_cast<uint8_t const**>(h_src_and_dst_buffers.data());
  rmm::device_buffer d_src_and_dst_buffers(src_bufs_size,
                                           stream,
                                           temp_mr);
  auto const** d_src_bufs = reinterpret_cast<uint8_t const**>(d_src_and_dst_buffers.data());

  // setup src buffers
  setup_src_buf_data(input.begin(), input.end(), h_src_bufs);
  
  // HtoD src buffers
  CUDF_CUDA_TRY(hipMemcpyAsync(
    d_src_bufs, h_src_bufs, src_bufs_size, hipMemcpyDefault, stream.value()));

  // allocate output buffer
  rmm::device_buffer dst_buf(dst_buf_total_size, stream, mr);

  // pack per-partition data. one thread per (flattened) column.
  size_type const thread_count_per_partition = cudf::util::round_up_safe(total_flattened_columns, static_cast<size_t>(cudf::detail::warp_size));
  cudf::detail::grid_1d const grid{thread_count_per_partition * static_cast<size_type>(num_partitions), 128};
  pack_per_partition_data_kernel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    reinterpret_cast<uint8_t*>(dst_buf.data()),
    num_partitions,
    total_flattened_columns,
    num_src_bufs,
    d_src_buf_info,
    d_indices,
    d_flattened_col_has_validity.data(),
    d_partition_offsets.data(),
    d_partition_sizes);

  /*
  {
    CUDF_CUDA_TRY(hipMemcpyAsync(
      h_dst_buf_info, d_dst_buf_info, dst_buf_info_size, hipMemcpyDefault, stream.value()));    
    stream.synchronize();
  }
  */

  stream.synchronize();

  // perform the copy.
  copy_data(num_src_bufs, d_src_bufs, num_bufs, reinterpret_cast<uint8_t*>(dst_buf.data()), d_dst_buf_info, stream);

  stream.synchronize();
 
  // return result;
  return {shuffle_split_result{std::make_unique<rmm::device_buffer>(std::move(dst_buf)), std::move(d_partition_offsets)},
          compute_metadata(input, total_flattened_columns)};
}

namespace {

#define OUTPUT_ITERATOR(__name, __T, __field_name)                                                  \
  template<typename __T>                                                                            \
  struct __name##generic_output_iter {                                                              \
    __T* c;                                                                                         \
    using value_type        = decltype(__T::__field_name);                                          \
    using difference_type   = size_t;                                                               \
    using pointer           = decltype(__T::__field_name)*;                                         \
    using reference         = decltype(__T::__field_name)&;                                         \
    using iterator_category = thrust::output_device_iterator_tag;                                   \
                                                                                                    \
    __name##generic_output_iter operator+ __host__ __device__(int i) { return {c + i}; }            \
                                                                                                    \
    __name##generic_output_iter& operator++ __host__ __device__()                                   \
    {                                                                                               \
      c++;                                                                                          \
      return *this;                                                                                 \
    }                                                                                               \
                                                                                                    \
    reference operator[] __device__(int i) { return dereference(c + i); }                           \
    reference operator* __device__() { return dereference(c); }                                     \
                                                                                                    \
  private:                                                                                          \
    reference __device__ dereference(__T* c) { return c->__field_name; }                            \
  };                                                                                                \
  using __name = __name##generic_output_iter<__T>

// per-flattened-column information
struct assemble_column_info {
  cudf::type_id         type;
  bool                  has_validity;
  size_type             num_rows, num_chars;
  size_type             null_count;
  size_type             num_children;
};
OUTPUT_ITERATOR(assemble_column_info_num_rows_output_iter, assemble_column_info, num_rows);
OUTPUT_ITERATOR(assemble_column_info_has_validity_output_iter, assemble_column_info, has_validity);

constexpr size_t bitmask_allocation_size_bytes(size_type number_of_bits, int pad = 1)
{
  return cudf::util::round_up_safe((number_of_bits + 7) / 8, pad);
}

// a copy batch. 1 per block.
struct assemble_batch {
  __device__ assemble_batch(int8_t const* _src, int8_t* _dst, size_t _size, bool _validity, int _value_shift, int _bit_shift):
    src(_src), dst(_dst), size(_size), validity(_validity), value_shift(_value_shift), bit_shift(_bit_shift){}

  int8_t const* src;
  int8_t* dst;
  size_t              size;     // bytes
  bool                validity; // whether or not this is a validity buffer
  int value_shift;              // amount to shift values down by (for offset buffers)
  int bit_shift;                // # of bits to shift left by (for validity buffers)
  size_type valid_count = 0;    // (output) validity count for this block of work
};

struct assemble_buffer_functor {
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, typename BufIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  void operator()(assemble_column_info const& col, BufIter validity_out, BufIter offsets_out, BufIter data_out)
  {
    // validity
    *validity_out = col.has_validity ? alloc_validity(col.num_rows) : rmm::device_buffer(0, stream, mr);

    // no offsets for fixed width types
    
    // data
    auto const data_size = cudf::util::round_up_safe(cudf::type_dispatcher(data_type{col.type}, size_of_helper{}) * col.num_rows, split_align);
    *data_out = rmm::device_buffer(data_size, stream, mr);
  }

  template <typename T, typename BufIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  void operator()(assemble_column_info const& col, BufIter validity_out, BufIter offsets_out, BufIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? alloc_validity(col.num_rows) : rmm::device_buffer(0, stream, mr);    

    // offsets
    auto const offsets_size = cudf::util::round_up_safe(sizeof(size_type) * (col.num_rows + 1), split_align);
    *offsets_out = rmm::device_buffer(offsets_size, stream, mr);

    // no data for lists
  } 

  template <typename T, typename BufIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  void operator()(assemble_column_info const& col, BufIter validity_out, BufIter offsets_out, BufIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? alloc_validity(col.num_rows) : rmm::device_buffer(0, stream, mr);

    // no offsets or data for structs
  }

  template <typename T, typename BufIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::string_view>)>
  void operator()(assemble_column_info const& col, BufIter validity_out, BufIter offsets_out, BufIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? alloc_validity(col.num_rows) : rmm::device_buffer(0, stream, mr);

    // chars TODO
    // auto const chars_size = cudf::util::round_up_safe(sizeof(int8_t) * (col.num_chars + 1), shuffle_split_partition_data_align);
    // out.push_back(rmm::device_buffer(chars_size, stream, mr));
    *data_out = rmm::device_buffer(0, stream, mr);

    // offsets
    auto const offsets_size = cudf::util::round_up_safe(sizeof(size_type) * (col.num_rows + 1), split_align);
    *offsets_out = rmm::device_buffer(offsets_size, stream, mr);    
  }

  template <typename T, typename BufIter, CUDF_ENABLE_IF(!std::is_same_v<T, cudf::struct_view> && 
                                                         !std::is_same_v<T, cudf::list_view> && 
                                                         !std::is_same_v<T, cudf::string_view> && 
                                                         !cudf::is_fixed_width<T>())>
  void operator()(assemble_column_info const& col, BufIter& validity_out, BufIter& offsets_out, BufIter& data_out)
  { 
    CUDF_FAIL("Unsupported type in assemble_buffer_functor");
  }
 
private:
  rmm::device_buffer alloc_validity(size_type num_rows)
  {
    return rmm::device_buffer(bitmask_allocation_size_bytes(num_rows, split_align), stream, mr);
  }
};

struct assemble_column_functor {
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto const validity = buffer;
    // no offsets
    auto const data = buffer + 2;

    out.push_back(std::make_unique<cudf::column>(cudf::data_type{col->type},
                  col->num_rows,
                  std::move(*data),
                  col->has_validity ? std::move(*validity) : rmm::device_buffer{},
                  col->null_count));
    
    return {col + 1, buffer + 3};
  }

  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto const validity = buffer;
    buffer += 3;

    // build children
    std::vector<std::unique_ptr<cudf::column>> children;
    children.reserve(col->num_children);
    auto next = col + 1;
    for(size_type i=0; i<col->num_children; i++){
      std::tie(next, buffer) = cudf::type_dispatcher(cudf::data_type{next->type},
                                                     assemble_column_functor{stream, mr},
                                                     next,
                                                     buffer,
                                                     children);
    }    

    out.push_back(cudf::make_structs_column(col->num_rows,
                                            std::move(children),
                                            col->null_count,
                                            col->has_validity ? std::move(*validity) : rmm::device_buffer{},
                                            stream,
                                            mr));
    return {next, buffer};
  }
    
    /*
  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  size_t operator()(size_t cur, host_span<assemble_column_info const> assemble_data, host_span<rmm::device_buffer> buffers, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto col = assemble_data[cur];
    auto validity = cur;
    auto offsets = col.has_validity ? cur + 1 : cur;
    cur = offsets + 1;

    // build offsets
    auto offsets_col = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                      col.num_rows + 1,
                                                      std::move(buffers[offsets]),
                                                      rmm::device_buffer{},
                                                      0);

    // build the child
    std::vector<std::unique_ptr<cudf::column>> child_col;
    cur = cudf::type_dispatcher(cudf::data_type{col.type},
                                *this,
                                cur,
                                assemble_data,
                                buffers,
                                child_col);
    
    // build the final column
    out.push_back(cudf::make_lists_column(col.num_rows,
                                          std::move(offsets_col),
                                          std::move(child_col.back()),
                                          col.null_count,
                                          col.has_validity ? std::move(buffers[validity]) : rmm::device_buffer{},
                                          stream,
                                          mr));
    return cur;
  }  
  */

  // template <typename T, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::list_view> and !std::is_same_v<T, cudf::struct_view>)>
  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::struct_view>)>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    CUDF_FAIL("Unsupported type in shuffle_assemble");
  }
};

struct assemble_empty_column_functor {
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, typename ColumnIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  ColumnIter operator()(ColumnIter col, std::vector<std::unique_ptr<cudf::column>>& out)
  {    
    out.push_back(std::make_unique<cudf::column>(cudf::data_type{col->type},
                  0,
                  rmm::device_buffer{},
                  rmm::device_buffer{},
                  0));
    
    return {col + 1};
  }

  template <typename T, typename ColumnIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  ColumnIter operator()(ColumnIter col, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    // build children
    std::vector<std::unique_ptr<cudf::column>> children;
    children.reserve(col->num_children);
    auto next = col + 1;
    for(size_type i=0; i<col->num_children; i++){
      next = cudf::type_dispatcher(cudf::data_type{next->type},
                                   assemble_empty_column_functor{stream, mr},
                                   next,
                                   children);
    }    

    out.push_back(cudf::make_structs_column(0,
                                            std::move(children),
                                            0,
                                            rmm::device_buffer{},
                                            stream,
                                            mr));
    return next;
  }
    
    /*
  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  size_t operator()(size_t cur, host_span<assemble_column_info const> assemble_data, host_span<rmm::device_buffer> buffers, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto col = assemble_data[cur];
    auto validity = cur;
    auto offsets = col.has_validity ? cur + 1 : cur;
    cur = offsets + 1;

    // build offsets
    auto offsets_col = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                      col.num_rows + 1,
                                                      std::move(buffers[offsets]),
                                                      rmm::device_buffer{},
                                                      0);

    // build the child
    std::vector<std::unique_ptr<cudf::column>> child_col;
    cur = cudf::type_dispatcher(cudf::data_type{col.type},
                                *this,
                                cur,
                                assemble_data,
                                buffers,
                                child_col);
    
    // build the final column
    out.push_back(cudf::make_lists_column(col.num_rows,
                                          std::move(offsets_col),
                                          std::move(child_col.back()),
                                          col.null_count,
                                          col.has_validity ? std::move(buffers[validity]) : rmm::device_buffer{},
                                          stream,
                                          mr));
    return cur;
  }
  */

  // template <typename T, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::list_view> and !std::is_same_v<T, cudf::struct_view>)>
  template <typename T, typename ColumnIter, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::struct_view>)>
  ColumnIter operator()(ColumnIter col, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    CUDF_FAIL("Unsupported type in shuffle_assemble");
  }
};

// The size that contiguous split uses internally as the GPU unit of work.
// The number of `desired_batch_size` batches equals the number of CUDA blocks
// that will be used for the main kernel launch (`copy_partitions`).
constexpr std::size_t desired_assemble_batch_size = 1 * 1024 * 1024;


// returns:
// - a vector of assemble_column_info structs representing the destination column data.
//   the vector is of length global_metadata.col_info.size()  that is, the flattened list of columns in the table.
//
// - the same vector as above, but in host memory. 
//
// - a vector of assemble_column_info structs, representing the source column data.
//   the vector is of length global_metadata.col_info.size() * the # of partitions. 
//
std::tuple<rmm::device_uvector<assemble_column_info>,
           std::vector<assemble_column_info>,
           rmm::device_uvector<assemble_column_info>,
           size_t>
assemble_build_column_info(shuffle_split_metadata const& h_global_metadata,
                           cudf::device_span<int8_t const> partitions, 
                           cudf::device_span<size_t const> partition_offsets,
                           rmm::cuda_stream_view stream,
                           rmm::device_async_resource_ref mr)
{
  auto temp_mr = cudf::get_current_device_resource_ref();
  rmm::device_uvector<shuffle_split_col_data> global_metadata = cudf::detail::make_device_uvector_async(h_global_metadata.col_info, stream, temp_mr);

  // "columns" here means the number of flattened columns in the entire source table, not just the
  // number of columns at the top level
  auto const num_columns = global_metadata.size();
  size_type const num_partitions = partition_offsets.size() - 1;
  auto const num_column_instances = num_columns * num_partitions;

  // generate per-column data ------------------------------------------------------
  rmm::device_uvector<assemble_column_info> column_info(num_columns, stream, temp_mr);

  // compute:
  //  - indices into the char count data for string columns
  //  - offset into the partition data where has-validity begins
  /*
  rmm::device_uvector<size_type> char_count_indices(num_columns + 1, stream, temp_mr);
  auto cc_index_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([global_metadata = global_metadata.begin(), num_columns] __device__ (size_type i) {
    return i >= num_columns ? 0 : (global_metadata[i].type == cudf::type_id::STRING ? 1 : 0);
  }));
  thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr), cc_index_iter, cc_index_iter + num_columns + 1, char_count_indices.begin());
  size_type const per_partition_num_char_counts = char_count_indices.back_element(stream);
  // the +1 is for the per-partition overall row count at the very beginning
  auto const has_validity_offset = (per_partition_num_char_counts + 1) * sizeof(size_type);
  
  {
    auto h_char_count_indices = cudf::detail::make_std_vector_sync(char_count_indices, stream);
    printf("per_partition_num_char_counts : %d\n", per_partition_num_char_counts);
    printf("has_validity_offset : %lu\n", has_validity_offset);
    for(size_t idx=0; idx<h_char_count_indices.size(); idx++){
      printf("h_char_count_indices(%lu): %d\n", idx, h_char_count_indices[idx]);
    }
  } 
  */

  // compute per-partition metadata size
  auto const per_partition_metadata_size = compute_per_partition_metadata_size(h_global_metadata.col_info.size());

  // compute has-validity
  // note that we are iterating vertically -> horizontally here, with each column's individual piece per partition first.
  auto column_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([num_partitions] __device__ (size_type i){
    return i / num_partitions;
  }));  
  auto has_validity_values = cudf::detail::make_counting_transform_iterator(0, 
    cuda::proclaim_return_type<bool>([num_partitions,
                                      partitions = partitions.data(),
                                      partition_offsets = partition_offsets.begin()]
                                      __device__ (int i) -> bool {
      auto const partition_index = i % num_partitions;
      bitmask_type const*const has_validity_buf = reinterpret_cast<bitmask_type const*>(partitions + partition_offsets[partition_index] + sizeof(partition_header));
      auto const col_index = i / num_partitions;
      
      // int has_validity = has_validity_buf[col_index / 32] & (1 << (col_index % 32)) ? 1 : 0;
      // printf("HVV: %d, %d, %d, %d, %d\n", (int)partition_index, (int)partition_offsets[partition_index], (int)sizeof(partition_header), (int)col_index, (int)has_validity);
      
      return has_validity_buf[col_index / 32] & (1 << (col_index % 32)) ? 1 : 0;
    })
  );
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        column_keys,
                        column_keys +  num_column_instances,
                        has_validity_values,
                        thrust::make_discard_iterator(),
                        assemble_column_info_has_validity_output_iter{column_info.begin()},
                        thrust::equal_to<size_type>{},
                        thrust::logical_or<bool>{});
  /*
  {
    auto h_column_info = cudf::detail::make_std_vector_sync(column_info, stream);
    for(size_t idx=0; idx<h_column_info.size(); idx++){
      printf("h_column_info(%lu): has_validity = %d\n", idx, (int)(h_column_info[idx].has_validity ? 1 : 0));
    }
  }
  */

  //print_span(cudf::device_span<size_t const>(partition_offsets));

  // compute overall row count
  auto row_count_values = cudf::detail::make_counting_transform_iterator(0,
    cuda::proclaim_return_type<size_t>([num_partitions,
                                                 partitions = partitions.data(),
                                                 partition_offsets = partition_offsets.begin()]
                                                 __device__ (int i){
                                                  partition_header const*const pheader = reinterpret_cast<partition_header const*>(partitions + partition_offsets[i]);
                                                  return pheader->num_rows;
                                                 }));
  size_t const row_count =  thrust::reduce(rmm::exec_policy_nosync(stream, temp_mr),
                                            row_count_values,
                                            row_count_values + num_partitions);
  
  // compute char counts for strings
  // note that we are iterating vertically -> horizontally here, with each column's individual piece per partition first.
  // TODO: use an output iterator and write directly to the outgoing assembly_info structs
  /*
  auto cc_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<cudf::size_type>([num_partitions] __device__ (int i){
    return i / num_partitions;
  }));
  auto char_count_values = cudf::detail::make_counting_transform_iterator(0,
    cuda::proclaim_return_type<cudf::size_type>([num_partitions,
                                                 partitions = partitions.data(),
                                                 partition_offsets = partition_offsets.begin(),
                                                 global_metadata = global_metadata.begin()]
                                                 __device__ (int i){
      auto const partition_index = i % num_partitions;
      auto const col_index = i / num_partitions;

      // non-string columns don't have a char count
      auto const column_type = global_metadata[col_index].type;
      if(column_type != cudf::type_id::STRING){
        return 0;
      }

      // string columns
      size_type const*const char_counts = reinterpret_cast<size_type const*>(partitions + partition_offsets[partition_index] + 4);
      // printf("RCI %d : %d, partition_index = %d\n", (int)col_index, char_counts[col_index], (int)partition_index);
      return char_counts[col_index];
    })
  );
  rmm::device_uvector<size_type> char_counts(num_columns, stream, temp_mr);
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        cc_keys, 
                        cc_keys + num_column_instances,
                        char_count_values,
                        thrust::make_discard_iterator(),
                        char_counts.begin());
  print_span(static_cast<cudf::device_span<size_type const>>(char_counts));
  */
  
  // copy type and summed row counts
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(rmm::exec_policy_nosync(stream, temp_mr), iter, iter + num_columns, [row_count,
                                                                                        column_info = column_info.begin(),
                                                                                        global_metadata = global_metadata.begin()
                                                                                        //char_count_indices = char_count_indices.begin(),
                                                                                        //char_counts = char_counts.begin()]
                                                                                        ]
                                                                                        __device__ (size_type col_index){
    auto const& metadata = global_metadata[col_index];
    auto& cinfo = column_info[col_index];
    
    cinfo.type = metadata.type;
    cinfo.null_count = 0; // TODO
    cinfo.num_children = metadata.num_children;
    
    cinfo.num_rows = row_count;
    
    // string columns store the char count separately
    // cinfo.num_chars = cinfo.type == cudf::type_id::STRING ? char_counts[char_count_indices[col_index]] : 0;
  });
    
  /*
  {
    auto h_column_info = cudf::detail::make_std_vector_sync(column_info, stream);
    for(size_t idx=0; idx<h_column_info.size(); idx++){
      printf("col_info[%lu]: type = %d has_validity = %d num_rows = %d num_chars = %d null_count = %d\n", idx,
        (int)h_column_info[idx].type, h_column_info[idx].has_validity ? 1 : 0, h_column_info[idx].num_rows, h_column_info[idx].num_chars, h_column_info[idx].null_count);
    }
  }
  */

  // generate per-column-instance data ------------------------------------------------------

  // has-validity, type, row count
  rmm::device_uvector<assemble_column_info> column_instance_info(num_column_instances, stream, temp_mr);
  thrust::for_each(rmm::exec_policy_nosync(stream, temp_mr), iter, iter + num_column_instances, [//char_count_indices = char_count_indices.begin(),
                                                                                                 column_instance_info = column_instance_info.begin(),
                                                                                                 global_metadata = global_metadata.begin(),
                                                                                                 partitions = partitions.data(),
                                                                                                 partition_offsets = partition_offsets.begin(),
                                                                                                 num_columns]
                                                                                                 __device__ (size_type i){
    auto const partition_index = i / num_columns;
    auto const col_index = i % num_columns;
    auto const col_instance_index = (partition_index * num_columns) + col_index;

    auto const& metadata = global_metadata[col_index];
    auto& cinstance_info = column_instance_info[col_instance_index];

    uint8_t const*const buf_start = reinterpret_cast<uint8_t const*>(partitions + partition_offsets[partition_index]);
    partition_header const*const pheader = reinterpret_cast<partition_header const*>(buf_start);

    bitmask_type const*const has_validity_buf = reinterpret_cast<bitmask_type const*>(buf_start + sizeof(partition_header));
    cinstance_info.has_validity = has_validity_buf[col_index / 32] & (1 << (col_index % 32)) ? 1 : 0;
    
    cinstance_info.type = metadata.type;
    cinstance_info.null_count = 0; // TODO
    cinstance_info.num_children = metadata.num_children;
    
    cinstance_info.num_rows = pheader->num_rows;
    
    // string columns store the char count separately
    /*
    if(metadata.type == cudf::type_id::STRING){
      size_type const*const char_counts = reinterpret_cast<size_type const*>(pheader + 4);
      cinstance_info.num_chars = char_counts[char_count_indices[col_index]];
    }
    */
  });
    
  /*
  {
    auto h_column_instance_info = cudf::detail::make_std_vector_sync(column_instance_info, stream);
    for(size_t idx=0; idx<h_column_instance_info.size(); idx++){
      size_type const partition_index = idx / num_columns;
      size_type const col_index = idx % num_columns;
      size_type const col_instance_index = (partition_index * num_columns) + col_index;

      printf("col_info[%d, %d, %d]: type = %d has_validity = %d num_rows = %d num_chars = %d null_count = %d\n",
        partition_index, col_index, col_instance_index,
        (int)h_column_instance_info[idx].type, h_column_instance_info[idx].has_validity ? 1 : 0, h_column_instance_info[idx].num_rows, h_column_instance_info[idx].num_chars, h_column_instance_info[idx].null_count);
    }
  } 
  */

  // compute per-partition metadata size
  // size_t const metadata_rc_size = ((per_partition_num_char_counts + 1) * sizeof(size_type));  
  // size_t const per_partition_metadata_size = cudf::util::round_up_safe(metadata_rc_size + metadata_has_validity_size, shuffle_split_partition_data_align);
  // size_t const metadata_has_validity_size = per_partition_metadata_size - header_size;

  return {std::move(column_info), cudf::detail::make_std_vector_sync(column_info, stream), std::move(column_instance_info), per_partition_metadata_size};
}

// Important: this returns the size of the buffer -without- padding. just the size of
// the raw bytes containing the actual data.
struct assemble_buffer_size_functor {
  template <typename T, typename OutputIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter validity_out, OutputIter offsets_out, OutputIter data_out)
  {
    // validity
    *validity_out = col.has_validity ? bitmask_allocation_size_bytes(col.num_rows) : 0;

    // no offsets for fixed width types
    *offsets_out = 0;

    // data
    *data_out = cudf::type_dispatcher(data_type{col.type}, size_of_helper{}) * col.num_rows;
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter validity_out, OutputIter offsets_out, OutputIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? bitmask_allocation_size_bytes(col.num_rows) : 0;

    // offsets
    *offsets_out = sizeof(size_type) * (col.num_rows + 1);

    // no data for lists
    *data_out = 0;
  } 

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter validity_out, OutputIter offsets_out, OutputIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? bitmask_allocation_size_bytes(col.num_rows) : 0;

    // no offsets or data for structs
    *offsets_out = 0;
    *data_out = 0;
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::string_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter validity_out, OutputIter offsets_out, OutputIter data_out)
  { 
    // validity
    *validity_out = col.has_validity ? bitmask_allocation_size_bytes(col.num_rows) : 0;

    // chars
    // TODO:
    // *out++ = sizeof(int8_t) * (col.num_chars + 1);
    *data_out = 0;

    // offsets
    *offsets_out = sizeof(size_type) * (col.num_rows + 1);
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(!std::is_same_v<T, cudf::struct_view> && 
                                       !std::is_same_v<T, cudf::list_view> && 
                                       !std::is_same_v<T, cudf::string_view> && 
                                       !cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter validity_out, OutputIter offsets_out, OutputIter data_out)
  {
  }
};

constexpr size_t size_to_batch_count(size_t bytes)
{
  return std::max(std::size_t{1}, util::round_up_unsafe(bytes, desired_assemble_batch_size) / desired_assemble_batch_size);
}

template<typename SizeIterator, typename GroupFunction>
rmm::device_uvector<std::invoke_result_t<GroupFunction>> transform_expand(SizeIterator first,
                                                                          SizeIterator last,
                                                                          GroupFunction op,
                                                                          rmm::cuda_stream_view stream,
                                                                          rmm::device_async_resource_ref mr)
{ 
  auto temp_mr = cudf::get_current_device_resource_ref();

  auto value_count = std::distance(first, last);
  auto size_wrapper = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([value_count, first] __device__ (size_t i){
    return i >= value_count ? 0 : first[i];
  }));
  rmm::device_uvector<size_t> group_offsets(value_count + 1, stream, temp_mr);
  thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                         size_wrapper,
                         size_wrapper + group_offsets.size(),
                         group_offsets.begin());
  size_t total_size = group_offsets.back_element(stream); // note memcpy and device sync
  
  using OutputType = std::invoke_result_t<GroupFunction>;
  rmm::device_uvector<OutputType> result(total_size, stream, mr);
  auto iter = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + total_size,
                    result.begin(),
                    cuda::proclaim_return_type<OutputType>([op, group_offsets_begin = group_offsets.begin(), group_offsets_end = group_offsets.end()] __device__ (size_t i){
                      auto const group_index = thrust::lower_bound(thrust::seq, group_offsets_begin, group_offsets_end, i) - group_offsets_begin;
                      auto const intra_group_index = i - group_offsets_begin[group_index];
                      return op(group_index, intra_group_index);
                    }));

  return result;
}

// returns destination buffers
std::pair<std::vector<rmm::device_buffer>, rmm::device_uvector<assemble_batch>> assemble_build_buffers(rmm::device_uvector<assemble_column_info> const& column_info,
                                                                                                       rmm::device_uvector<assemble_column_info> const& column_instance_info,
                                                                                                       cudf::device_span<int8_t const> partitions,
                                                                                                       cudf::device_span<size_t const> partition_offsets,
                                                                                                       size_t per_partition_metadata_size,
                                                                                                       rmm::cuda_stream_view stream,
                                                                                                       rmm::device_async_resource_ref mr)
{  
  auto temp_mr = cudf::get_current_device_resource_ref();

  auto h_column_info = cudf::detail::make_std_vector_async(column_info, stream);
  auto const num_columns = column_info.size();
  auto const num_partitions = partition_offsets.size() - 1;

/*
  // # of validity buffers, offset buffers, data buffers, per partition.
  // NOTE: to simplify things, we are going to keep a validity buffer for every column, even if it ends up being empty.
  //       this keeps the source<->dst buffer mapping simpler.
  auto buf_count_by_type =
    thrust::make_transform_iterator(column_info.begin(), cuda::proclaim_return_type<partition_size_info>([] __device__ (assemble_column_info const& cinfo){
      return partition_size_info{1,
                                 cinfo.type == type_id::STRING || cinfo.type == type_id::LIST ? size_t{1} : size_t{0},
                                 cinfo.type != type_id::STRING && cinfo.type != type_id::LIST && cinfo.type != type_id::STRUCT ? size_t{1} : size_t{0}};
    }));
  auto buf_count_reduce = cuda::proclaim_return_type<partition_size_info>([per_partition_metadata_size] __device__ (partition_size_info const& lhs, partition_size_info const& rhs){
                            auto const validity_size = lhs.validity_size + rhs.validity_size;
                            auto const offset_size = lhs.offset_size + rhs.offset_size;
                            auto const data_size = lhs.data_size + rhs.data_size;
                            return partition_size_info{validity_size, offset_size, data_size};
                          });
  partition_size_info buf_type_counts = thrust::reduce(rmm::exec_policy(stream, temp_mr),
                                                       buf_count_by_type,
                                                       buf_count_by_type + num_columns,
                                                       partition_size_info{0, 0, 0},
                                                       buf_count_reduce);
  auto const dst_buf_count = buf_type_counts.validity_size + buf_type_counts.offset_size + buf_type_counts.data_size;
  */
  // to simplify things, we will reserve 3 buffers for each column. validity, data, offsets. not every column will use all of them, so those
  // buffers will remain unallocated/zero size. 
  auto const dst_buf_count = num_columns * 3;

  // allocate output buffers. ordered in the array as (validity, offsets, data) per column.
  // TODO: potentially add an option to do this contiguous-split style where we allocate a single
  // huge buffer and wrap it with column_views.
  // std::vector<rmm::device_buffer> assemble_buffers(buf_type_counts.validity_size + buf_type_counts.offset_size + buf_type_counts.data_size);
  std::vector<rmm::device_buffer> assemble_buffers(dst_buf_count);
  auto dst_validity_iter = assemble_buffers.begin();
  auto dst_offsets_iter = assemble_buffers.begin() + 1;
  auto dst_data_iter = assemble_buffers.begin() + 2;
  
  // for each column, a mapping to it's corresponding validity, offset and data buffer
  stream.synchronize(); // for h_column_info
  
  for(size_t idx=0; idx<h_column_info.size(); idx++){
    cudf::type_dispatcher(cudf::data_type{h_column_info[idx].type},
                          assemble_buffer_functor{stream, mr},
                          h_column_info[idx],
                          dst_validity_iter,
                          dst_offsets_iter,
                          dst_data_iter);
    dst_validity_iter += 3;
    dst_offsets_iter += 3;
    dst_data_iter += 3;
  }
  std::vector<int8_t*> h_dst_buffers(assemble_buffers.size());
  std::transform(assemble_buffers.begin(), assemble_buffers.end(), h_dst_buffers.begin(), [](rmm::device_buffer& buf){
    return reinterpret_cast<int8_t*>(buf.data());
  });
  auto dst_buffers = cudf::detail::make_device_uvector_async(h_dst_buffers, stream, temp_mr);
  //auto column_to_buffer_map = cudf::detail::make_device_uvector_async(h_column_to_buffer_map, stream, temp_mr);

  int whee = 10;
  whee++;

  // compute:
  // - row indices by partition
  // - unpadded sizes of the source buffers
  // - offsets into the partition data where each source buffer starts
  size_t const buffers_per_partition = assemble_buffers.size();
  size_t const num_src_buffers = buffers_per_partition * num_partitions;  
  rmm::device_uvector<size_type> partition_row_indices(num_partitions, stream, temp_mr);
  // ordered by partition, with each column containing (validity, offsets, data)
  rmm::device_uvector<size_t> src_sizes_unpadded(num_src_buffers, stream, mr);
  rmm::device_uvector<size_t> src_offsets(num_src_buffers, stream, mr);
  // arranged in destination buffer order, each column containinf (validity, offsets, data)
  rmm::device_uvector<size_t> dst_offsets(num_src_buffers, stream, mr);  
  {
    // generate partition row indices
    auto row_count_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([column_instance_info = column_instance_info.begin(),
                                                                                                                  columns_per_partition = column_info.size()] __device__ (size_type i){
      auto col = column_instance_info[i * columns_per_partition];
      return col.num_rows;
    }));
    thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                           row_count_iter,
                           row_count_iter + num_partitions,
                           partition_row_indices.begin());
    //print_span(cudf::device_span<size_type const>{partition_row_indices});

    // generate unpadded sizes of the source buffers
    auto const num_column_instances = column_instance_info.size();
    auto iter = thrust::make_counting_iterator(0);
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + num_column_instances,
                    [buffers_per_partition,
                     num_columns,
                     column_instance_info = column_instance_info.begin(),
                     src_sizes_unpadded = src_sizes_unpadded.begin()] __device__ (size_type i){

      auto const partition_index = i / num_columns;
      auto const col_index = i % num_columns;
      auto const col_instance_index = (partition_index * num_columns) + col_index;

      auto const& cinfo_instance = column_instance_info[col_instance_index];
      auto const validity_buf_index = (col_index * 3) + (partition_index * buffers_per_partition);
      auto const offset_buf_index = ((col_index * 3) + 1) + (partition_index * buffers_per_partition);
      auto const data_buf_index = ((col_index * 3) + 2) + (partition_index * buffers_per_partition);
      cudf::type_dispatcher(cudf::data_type{cinfo_instance.type},
                            assemble_buffer_size_functor{},
                            cinfo_instance,
                            &src_sizes_unpadded[validity_buf_index],
                            &src_sizes_unpadded[offset_buf_index],
                            &src_sizes_unpadded[data_buf_index]);
      //printf("SSU: %d %d (%d %d %d) (%d %d %d) (%lu %lu %lu)\n", (int)partition_index, (int)col_index, 
        //                                                (int)column_to_buffer_map[validity_buf_index], (int)column_to_buffer_map[offset_buf_index], (int)column_to_buffer_map[data_buf_index],
          //                                              (int)validity_buf_index, (int)offset_buf_index, (int)data_buf_index,
            //                                            src_sizes_unpadded[validity_buf_index], src_sizes_unpadded[offset_buf_index], src_sizes_unpadded[data_buf_index]);
    });
    //print_span(cudf::device_span<size_t const>{src_sizes_unpadded});
        
    // scan to source offsets, by partition
    auto partition_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([buffers_per_partition] __device__ (size_t i){
      return i / buffers_per_partition;
    }));
    thrust::exclusive_scan_by_key(rmm::exec_policy(stream, temp_mr),
                                  partition_keys,
                                  partition_keys + num_src_buffers,
                                  src_sizes_unpadded.begin(),
                                  src_offsets.begin());
    //print_span(cudf::device_span<size_t const>{src_offsets});
    
    // adjust the source offsets:
    // - add metadata offset
    // - take padding into account 
    // - add partition offset
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + num_column_instances,
                    [num_columns,
                     buffers_per_partition,
                     column_instance_info = column_instance_info.begin(),
                     src_offsets = src_offsets.begin(),
                     partition_offsets = partition_offsets.begin(),
                     partitions = partitions.data(),
                     per_partition_metadata_size] __device__ (size_type i){

      auto const partition_index = i / num_columns;
      auto const partition_offset = partition_offsets[partition_index];
      auto const col_index = i % num_columns;
      auto const col_instance_index = (partition_index * num_columns) + col_index;

      partition_header const*const pheader = reinterpret_cast<partition_header const*>(partitions + partition_offset);

      auto const validity_section_offset = partition_offset + per_partition_metadata_size;
      auto const validity_buf_index = (col_index * 3) + (partition_index * buffers_per_partition);
      src_offsets[validity_buf_index] += validity_section_offset;

      auto const offset_section_offset = cudf::util::round_up_safe(validity_section_offset + pheader->validity_size, validity_pad);
      auto const offset_buf_index = ((col_index * 3) + 1) + (partition_index * buffers_per_partition);
      src_offsets[offset_buf_index] += offset_section_offset;
      
      auto const data_section_offset = cudf::util::round_up_safe(offset_section_offset + pheader->offset_size, offset_pad);
      auto const data_buf_index = ((col_index * 3) + 2) + (partition_index * buffers_per_partition);
      src_offsets[data_buf_index] += data_section_offset;

      //printf("MHO: %d, partition_index = %d, partition_offset = %lu, col_index = %d, col_instance_index = %d, validity_offset = %lu, offset_offset = %lu, data_offset = %lu\n", 
        //    i, (int)partition_index, partition_offset, (int)col_index, (int)col_instance_index, validity_section_offset, offset_section_offset, data_section_offset);
    });
    //print_span(cudf::device_span<size_t const>{src_offsets});

    // compute: generate destination buffer offsets
    // NOTE: dst_offsets is arranged in destination buffer order, not source buffer order.
    // We're wasting a little work here as the validity computation has to be redone later.    
    auto dst_buf_key = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([num_partitions] __device__ (size_t i){
      return i / num_partitions;
    }));
    auto size_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([src_sizes_unpadded = src_sizes_unpadded.begin(), num_partitions, buffers_per_partition] __device__ (size_t i){
      auto const dst_buf_index = i / num_partitions;
      auto const partition_index = i % num_partitions;
      auto const src_buf_index = (partition_index * buffers_per_partition) + dst_buf_index;
      return src_sizes_unpadded[src_buf_index];
    }));
    // dst_offsets is arranged     
    thrust::exclusive_scan_by_key(rmm::exec_policy(stream, temp_mr),
                                  dst_buf_key,
                                  dst_buf_key + num_src_buffers,
                                  size_iter,
                                  dst_offsets.begin());
    //print_span(cudf::device_span<size_t const>{dst_offsets});

    // for validity, we need to do a little more work. our destination positions are defined by bit position,
    // not byte position. so round down into the nearest starting bitmask word.    
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                     iter,
                     iter + num_column_instances,
                     [column_info = column_info.begin(),
                      num_columns,
                      buffers_per_partition,
                      partition_row_indices = partition_row_indices.begin(),
                      dst_offsets = dst_offsets.begin()] __device__ (size_t i){
      auto const col_index = i / num_columns;
      auto const partition_index = i % num_columns;
      auto const& cinfo = column_info[col_index];
      if(cinfo.has_validity){
        auto const validity_buf_index = (col_index * 3) + (partition_index * buffers_per_partition);
        dst_offsets[validity_buf_index] = (partition_row_indices[partition_index] / 32) * sizeof(bitmask_type);
      }
    });
    //print_span(cudf::device_span<size_t const>{dst_offsets});
  }
  
  // generate copy batches ------------------------------------

  // generate batches.
  // - validity and offsets will be copied by custom kernels, so we will subdivide them them into batches of 1 MB
  // - data is copied by cub, so we will do no subdivision of the batches under the assumption that cub will make it's
  //   own smart internal decisions
  auto batch_count_iter = cudf::detail::make_counting_transform_iterator(0, 
                                                                         cuda::proclaim_return_type<size_t>([src_sizes_unpadded = src_sizes_unpadded.begin()] __device__ (size_t i){
                                                                           return i % 3 == 2 ? 1 : size_to_batch_count(src_sizes_unpadded[i]);
                                                                         }));
  auto copy_batches = transform_expand(batch_count_iter, 
                                       batch_count_iter + src_sizes_unpadded.size(),
                                       cuda::proclaim_return_type<assemble_batch>([dst_buffers = dst_buffers.begin(),
                                                                                   dst_offsets = dst_offsets.begin(),
                                                                                   partitions = partitions.data(),
                                                                                   buffers_per_partition,
                                                                                   num_partitions,
                                                                                   src_sizes_unpadded = src_sizes_unpadded.begin(),
                                                                                   src_offsets = src_offsets.begin(),
                                                                                   desired_assemble_batch_size = desired_assemble_batch_size,
                                                                                   partition_row_indices = partition_row_indices.begin()] __device__ (size_t src_buf_index, size_t batch_index){
                                         auto const batch_offset = batch_index * desired_assemble_batch_size;
                                         auto const partition_index = src_buf_index / buffers_per_partition;
                                         
                                         auto const src_offset = src_offsets[src_buf_index];
                                        
                                         auto const dst_buf_index = src_buf_index % buffers_per_partition;
                                         auto const dst_offset_index = (dst_buf_index * num_partitions) + partition_index;
                                         auto const dst_offset = dst_offsets[dst_offset_index];

                                         auto const bytes = (src_buf_index % 3 == 2) ? src_sizes_unpadded[src_buf_index] : std::min(src_sizes_unpadded[src_buf_index] - batch_offset, desired_assemble_batch_size);
                                         
                                         /*
                                         printf("ET: partition_index=%lu, src_buf_index=%lu, dst_buf_index=%lu, batch_index=%lu, src_offset=%lu, dst_offset=%lu bytes=%lu bit_shift = %d\n", 
                                           partition_index,
                                           src_buf_index,
                                           dst_buf_index,
                                           batch_index,
                                           src_offset + batch_offset,
                                           dst_offset + batch_offset,
                                           bytes,
                                           partition_row_indices[partition_index] % 32);
                                           */

                                         return assemble_batch {
                                          partitions + src_offset + batch_offset,
                                          dst_buffers[dst_buf_index] + dst_offset + batch_offset,
                                          bytes,
                                          0,  // TODO: handle offsets
                                          partition_row_indices[partition_index] % 32,  // bit shift for the validity copy step
                                          0};
                                         }),
                                       stream,
                                       mr);

  return {std::move(assemble_buffers), std::move(copy_batches)};
}

void assemble_copy(rmm::device_uvector<assemble_batch> const& batches, rmm::cuda_stream_view stream)
{
  // main data copy. everything except validity and offsets
  {
    auto input_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<void*>([batches = batches.begin()] __device__ (size_t i){
      //printf("SRC: %lu\n", (uint64_t)(batches[(i * 3) + 2].src));
      return reinterpret_cast<void*>(const_cast<int8_t*>(batches[(i * 3) + 2].src));
    }));
    auto output_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<void*>([batches = batches.begin()] __device__ (size_t i){
      //printf("DST: %lu\n", (uint64_t)(batches[(i * 3) + 2].dst));
      return reinterpret_cast<void*>(const_cast<int8_t*>(batches[(i * 3) + 2].dst));
    }));
    auto size_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([batches = batches.begin()] __device__ (size_t i){
      //printf("SIZE: %lu\n", (uint64_t)(batches[(i * 3) + 2].size));
      return batches[(i * 3) + 2].size;
    }));

    size_t temp_storage_bytes;
    hipcub::DeviceMemcpy::Batched(nullptr, temp_storage_bytes, input_iter, output_iter, size_iter, batches.size() / 3, stream);
    rmm::device_buffer temp_storage(temp_storage_bytes, stream, cudf::get_current_device_resource_ref());
    hipcub::DeviceMemcpy::Batched(temp_storage.data(), temp_storage_bytes, input_iter, output_iter, size_iter, batches.size() / 3, stream);
  }

#if 0
  // copy validity
  /*
  constexpr int block_size = 256;
  cudf::detail::grid_1d const grid{static_cast<cudf::thread_index_type>(batches.size()), block_size};
  copy_validity<block_size><<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(batches);
  */  
  #endif

  stream.synchronize();
}

// assemble all the columns and the final table from the intermediate buffers
std::unique_ptr<cudf::table> build_table(std::vector<assemble_column_info> const& assembly_data,
                                         std::vector<rmm::device_buffer>& assembly_buffers,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  std::vector<std::unique_ptr<cudf::column>> columns;
  auto column = assembly_data.begin();
  auto buffer = assembly_buffers.begin();
  while(column != assembly_data.end()){
    std::tie(column, buffer) = cudf::type_dispatcher(cudf::data_type{column->type},
                                                     assemble_column_functor{stream, mr},
                                                     column,
                                                     buffer,
                                                     columns);
  }
  return std::make_unique<cudf::table>(std::move(columns));
}

// assemble all the columns and the final table from the intermediate buffers
std::unique_ptr<cudf::table> build_empty_table(std::vector<shuffle_split_col_data> const& col_info,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  std::vector<std::unique_ptr<cudf::column>> columns;
  auto column = col_info.begin();
  while(column != col_info.end()){
    column = cudf::type_dispatcher(cudf::data_type{column->type},
                                   assemble_empty_column_functor{stream, mr},
                                   column,
                                   columns);
  }
  return std::make_unique<cudf::table>(std::move(columns));
}

}; // anonymous namespace

std::unique_ptr<cudf::table> shuffle_assemble(shuffle_split_metadata const& metadata,
                                              cudf::device_span<int8_t const> partitions,
                                              cudf::device_span<size_t const> partition_offsets,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr)
{
  // if the input is empty, just generate an empty table
  if(partition_offsets.size() == 1){
    return build_empty_table(metadata.col_info, stream, mr);
  }

  // generate the info structs representing the flattened column hierarchy. the total number of assembled rows, null counts, etc
  auto [column_info, h_column_info, column_instance_info, per_partition_metadata_size] = assemble_build_column_info(metadata, partitions, partition_offsets, stream, mr);

  // generate the (empty) output buffers based on the column info. note that is not a 1:1 mapping between column info
  // and buffers, since some columns will have validity and some will not.
  auto [dst_buffers, batches] = assemble_build_buffers(column_info, column_instance_info, partitions, partition_offsets, per_partition_metadata_size, stream, mr);  

  // copy the data. note that this does not sync.
  assemble_copy(batches, stream);

  // build the final table while the gpu is performing the copy
  auto ret = build_table(h_column_info, dst_buffers, stream, mr);
  stream.synchronize();
  return ret;
}

};  // namespace spark_rapids_jni
