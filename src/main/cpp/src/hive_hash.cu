#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hash.cuh"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tabulate.h>

namespace spark_rapids_jni {

namespace {

using hive_hash_value_t = int32_t;

constexpr hive_hash_value_t HIVE_HASH_FACTOR = 31;
constexpr hive_hash_value_t HIVE_INIT_HASH   = 0;

constexpr int MAX_STACK_DEPTH = 8;

hive_hash_value_t __device__ inline compute_int(int32_t key) { return key; }

hive_hash_value_t __device__ inline compute_long(int64_t key)
{
  return (static_cast<uint64_t>(key) >> 32) ^ key;
}

hive_hash_value_t __device__ inline compute_bytes(int8_t const* data, cudf::size_type const len)
{
  hive_hash_value_t ret = HIVE_INIT_HASH;
  for (auto i = 0; i < len; i++) {
    ret = ret * HIVE_HASH_FACTOR + static_cast<int32_t>(data[i]);
  }
  return ret;
}

template <typename Key>
struct hive_hash_function {
  // 'seed' is not used in 'hive_hash_function', but required by 'element_hasher'.
  constexpr hive_hash_function(uint32_t) {}

  [[nodiscard]] hive_hash_value_t __device__ inline operator()(Key const& key) const
  {
    CUDF_UNREACHABLE("Unsupported type for hive hash");
  }
};  // struct hive_hash_function

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::string_view>::operator()(
  cudf::string_view const& key) const
{
  auto const data = reinterpret_cast<int8_t const*>(key.data());
  auto const len  = key.size_bytes();
  return compute_bytes(data, len);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<bool>::operator()(bool const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int8_t>::operator()(int8_t const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int16_t>::operator()(
  int16_t const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int32_t>::operator()(
  int32_t const& key) const
{
  return compute_int(key);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int64_t>::operator()(
  int64_t const& key) const
{
  return compute_long(key);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<float>::operator()(float const& key) const
{
  auto normalized = spark_rapids_jni::normalize_nans(key);
  auto* p_int     = reinterpret_cast<int32_t const*>(&normalized);
  return compute_int(*p_int);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<double>::operator()(double const& key) const
{
  auto normalized = spark_rapids_jni::normalize_nans(key);
  auto* p_long    = reinterpret_cast<int64_t const*>(&normalized);
  return compute_long(*p_long);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::timestamp_D>::operator()(
  cudf::timestamp_D const& key) const
{
  auto* p_int = reinterpret_cast<int32_t const*>(&key);
  return compute_int(*p_int);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::timestamp_us>::operator()(
  cudf::timestamp_us const& key) const
{
  auto time_as_long            = *reinterpret_cast<int64_t const*>(&key);
  constexpr int MICRO_PER_SEC  = 1000000;
  constexpr int NANO_PER_MICRO = 1000;

  int64_t ts  = time_as_long / MICRO_PER_SEC;
  int64_t tns = (time_as_long % MICRO_PER_SEC) * NANO_PER_MICRO;

  int64_t result = ts;
  result <<= 30;
  result |= tns;

  result = (static_cast<uint64_t>(result) >> 32) ^ result;
  return static_cast<hive_hash_value_t>(result);
}

/**
 * @brief Computes the hash value of a row in the given table.
 *
 * This functor produces the same result as "HiveHash" in Spark for supported types.
 *
 * @tparam hash_function Hash functor to use for hashing elements. Must be hive_hash_function.
 * @tparam Nullate A cudf::nullate type describing whether to check for nulls.
 */
template <template <typename> class hash_function, typename Nullate>
class hive_device_row_hasher {
 public:
  CUDF_HOST_DEVICE hive_device_row_hasher(Nullate check_nulls, cudf::table_device_view t) noexcept
    : _check_nulls{check_nulls}, _table{t}
  {
    // Error out if passed an unsupported hash_function
    static_assert(std::is_base_of_v<hive_hash_function<int>, hash_function<int>>,
                  "hive_device_row_hasher only supports the 'hive_hash_function' hash function");
  }

  /**
   * @brief Return the hash value of a row in the given table.
   *
   * @param row_index The row index to compute the hash value of
   * @return The hash value of the row
   */
  __device__ auto operator()(cudf::size_type row_index) const noexcept
  {
    return cudf::detail::accumulate(
      _table.begin(),
      _table.end(),
      HIVE_INIT_HASH,
      cuda::proclaim_return_type<hive_hash_value_t>(
        [row_index, nulls = this->_check_nulls] __device__(auto hash, auto const& column) {
          auto cur_hash =
            cudf::type_dispatcher(column.type(), element_hasher_adapter{nulls}, column, row_index);
          return HIVE_HASH_FACTOR * hash + cur_hash;
        }));
  }

 private:
  /**
   * @brief Computes the hash value of an element in the given column.
   */
  class element_hasher_adapter {
   public:
    using hash_functor_t = cudf::experimental::row::hash::element_hasher<hash_function, Nullate>;

    __device__ element_hasher_adapter(Nullate check_nulls) noexcept
      : hash_functor{check_nulls, HIVE_INIT_HASH, HIVE_INIT_HASH}
    {
    }

    template <typename T, CUDF_ENABLE_IF(not cudf::is_nested<T>())>
    __device__ hive_hash_value_t operator()(cudf::column_device_view const& col,
                                            cudf::size_type row_index) const noexcept
    {
      return this->hash_functor.template operator()<T>(col, row_index);
    }

    /**
     * @brief A structure to keep track of the computation for nested types.
     */
    struct col_stack_frame {
     private:
      cudf::column_device_view _column;  // the column has only one row
      hive_hash_value_t _cur_hash;       // current hash value of the column
      int _idx_to_process;               // the index of child or element to process next

     public:
      __device__ col_stack_frame() =
        delete;  // Because the default constructor of `cudf::column_device_view` is deleted

      __device__ col_stack_frame(cudf::column_device_view col)
        : _column(std::move(col)), _idx_to_process(0), _cur_hash(HIVE_INIT_HASH)
      {
      }

      __device__ void update_cur_hash(hive_hash_value_t hash)
      {
        _cur_hash = _cur_hash * HIVE_HASH_FACTOR + hash;
      }

      __device__ hive_hash_value_t get_hash() { return _cur_hash; }

      __device__ int get_and_inc_idx_to_process() { return _idx_to_process++; }

      __device__ int get_idx_to_process() { return _idx_to_process; }

      __device__ cudf::column_device_view get_column() { return _column; }
    };

    /**
     * @brief Functor to compute the hive hash value for a nested column.
     *
     * This functor produces the same result as "HiveHash" in Spark for structs and lists.
     * The pseudocode of Spark's HiveHash function for structs is as follows:
     *
     * hive_hash_value_t hive_hash(NestedType element) {
     *    hive_hash_value_t hash = HIVE_INIT_HASH;
     *    for (int i = 0; i < element.num_child(); i++) {
     *        hash = hash * HIVE_HASH_FACTOR + hive_hash(element.get_child(i));
     *    }
     *    return hash;
     * }
     *
     * In the cases of lists, the hash value is computed by a similar way but we iterate through the
     * list elements instead of through the child columns' elements.
     *
     * This functor uses a stack to simulate the recursive process of the above pseudocode.
     * When an element is popped from the stack, it means that the hash value of it has been
     * computed. Therefore, we should update the parent's `cur_hash` upon popping the element.
     *
     * The algorithm is as follows:
     *
     * 1. Initialize the stack and push the root column into the stack.
     * 2. While the stack is not empty:
     *    a. Get the top element of the stack. Don't pop it until it is processed.
     *    b. If the column is a structs column:
     *        i.  If all child columns are processed, pop the element and update `cur_hash` of its
     *            parent column.
     *        ii. Otherwise, process the next child column.
     *    c. If the column is a lists column, process it by a similar way as structs column but
     *       iterating through the list elements instead of child columns' elements.
     * 3. Return the hash value of the root column.
     *
     * For example, consider the following nested column: `Struct<Struct<int, float>, decimal>`
     *
     *      S1
     *     / \
     *    S2  d
     *   / \
     *  i   f
     *
     * - First, S1 is pushed into the stack. Then, S2 is pushed into the stack.
     * - S2's hash value can be computed directly because its children are of primitive types.
     *   When S2 is popped, S1's `cur_hash` is updated to S2's hash value.
     * - Now the top of the stack is S1. The next child to process is d. S1's `cur_hash` is updated
     *   to `hash(S2) * HIVE_HASH_FACTOR + hash(d)`, which is the hash value of S1.
     * - When S1 is popped, the hash value of the root column is returned.
     *
     * As lists columns have a different interface from structs columns, we need to handle them
     * separately.
     *
     * For example, consider that the input column is of type `List<List<int>>`.
     * Assume that the element at `row_index` is: [[1, 0], null, [2, null]]
     * Since the stack_frame should contain a column that consists of only one row, the input column
     * should be sliced. The sliced column is noted as L1 here.
     *
     *     L1
     *     |
     *     L2
     *     |
     *     i
     *
     * List level L1:
     * |Index|      List<list<int>>    |
     * |-----|-------------------------|
     * |0    |[[1, 0], null, [2, null]]|
     * length: 1
     * Offsets: 0, 3
     *
     * List level L2:
     * |Index|List<int>|
     * |-----|---------|
     * |0    |[1, 0]   |
     * |1    |null     |
     * |2    |[2, null]|
     * length: 3
     * Offsets: 0, 2, 2, 4
     * null_mask: 101
     *
     * Int level i:
     * |Index|int |
     * |-----|----|
     * |0    |1   |
     * |1    |0   |
     * |2    |2   |
     * |3    |null|
     * length: 4
     * null_mask: 0111
     *
     * Since the underlying data loses the null information of the top-level list column, computing
     * hash values using the underlying data merely can yield different results compared to Spark.
     * For example, [[1, 0], [2, null]] has the same underlying data as [[1, 0], null, [2, null]].
     * However, they have different hive hash values, as null values affect the hash value.
     *
     * And sublist structure factors in the hash value. For example, [[1, 0], [2]] and [[1], [0, 2]]
     * have different hive hash values.
     *
     * The computation process for lists columns in this solution is as follows:
     *            L1              List<list<int>>
     *            |
     *            L2              List<int>
     *      /     |     \
     *    L2[0] L2[1] L2[2]       int
     *     |            |
     *    i1           i2         int
     *
     * Note: L2、i1、i2 are all temporary columns, which would not be pushed into the stack.
     * If the child column is of primitive type, the hash value of the list column can be directly
     * computed.
     *
     * @tparam T The type of the column.
     * @param col The column device view.
     * @param row_index The index of the row to compute the hash for.
     * @return The computed hive hash value.
     *
     * @note This function is only enabled for nested column types.
     */
    template <typename T, CUDF_ENABLE_IF(cudf::is_nested<T>())>
    __device__ hive_hash_value_t operator()(cudf::column_device_view const& col,
                                            cudf::size_type row_index) const noexcept
    {
      cudf::column_device_view curr_col = col.slice(row_index, 1);
      // The default constructor of `col_stack_frame` is deleted, so it can not allocate an array
      // of `col_stack_frame` directly.
      // Instead leverage the byte array to create the col_stack_frame array.
      alignas(col_stack_frame) char stack_wrapper[sizeof(col_stack_frame) * MAX_STACK_DEPTH];
      auto col_stack = reinterpret_cast<col_stack_frame*>(stack_wrapper);
      int stack_size = 0;

      col_stack[stack_size++] = col_stack_frame(curr_col);

      while (stack_size > 0) {
        col_stack_frame& top = col_stack[stack_size - 1];
        curr_col             = top.get_column();
        // Do not pop it until it is processed. The definition of `processed` is:
        // - For structs, it is when all child columns are processed.
        // - For lists, it is when all elements in the list are processed.
        if (curr_col.type().id() == cudf::type_id::STRUCT) {
          if (top.get_idx_to_process() == curr_col.num_child_columns()) {
            if (--stack_size > 0) { col_stack[stack_size - 1].update_cur_hash(top.get_hash()); }
          } else {
            auto const structcv = cudf::detail::structs_column_device_view(curr_col);
            while (top.get_idx_to_process() < curr_col.num_child_columns()) {
              auto idx             = top.get_and_inc_idx_to_process();
              auto const child_col = structcv.get_sliced_child(idx);
              // If the child is of primitive type, accumulate child hash into struct hash
              if (child_col.type().id() != cudf::type_id::LIST &&
                  child_col.type().id() != cudf::type_id::STRUCT) {
                auto child_hash =
                  cudf::type_dispatcher<cudf::experimental::dispatch_void_if_nested>(
                    child_col.type(), this->hash_functor, child_col, 0);
                top.update_cur_hash(child_hash);
              } else {
                col_stack[stack_size++] = col_stack_frame(child_col);
                break;
              }
            }
          }
        } else if (curr_col.type().id() == cudf::type_id::LIST) {
          // Get the child column of the list column
          cudf::column_device_view child_col =
            cudf::detail::lists_column_device_view(curr_col).get_sliced_child();
          // If the child column is of primitive type, directly compute the hash value of the list
          if (child_col.type().id() != cudf::type_id::LIST &&
              child_col.type().id() != cudf::type_id::STRUCT) {
            auto single_level_list_hash = cudf::detail::accumulate(
              thrust::counting_iterator(0),
              thrust::counting_iterator(child_col.size()),
              HIVE_INIT_HASH,
              [child_col, hasher = this->hash_functor] __device__(auto hash, auto element_index) {
                auto cur_hash = cudf::type_dispatcher<cudf::experimental::dispatch_void_if_nested>(
                  child_col.type(), hasher, child_col, element_index);
                return HIVE_HASH_FACTOR * hash + cur_hash;
              });
            top.update_cur_hash(single_level_list_hash);
            if (--stack_size > 0) { col_stack[stack_size - 1].update_cur_hash(top.get_hash()); }
          } else {
            if (top.get_idx_to_process() == child_col.size()) {
              if (--stack_size > 0) { col_stack[stack_size - 1].update_cur_hash(top.get_hash()); }
            } else {
              // Push the next element into the stack
              col_stack[stack_size++] =
                col_stack_frame(child_col.slice(top.get_and_inc_idx_to_process(), 1));
            }
          }
        }
      }
      return col_stack[0].get_hash();
    }

   private:
    hash_functor_t const hash_functor;
  };

  Nullate const _check_nulls;
  cudf::table_device_view const _table;
};

void check_nested_depth(cudf::table_view const& input)
{
  using column_checker_fn_t = std::function<int(cudf::column_view const&)>;

  column_checker_fn_t get_nested_depth = [&](cudf::column_view const& col) {
    if (col.type().id() == cudf::type_id::LIST) {
      auto const child_col = cudf::lists_column_view(col).child();
      return 1 + get_nested_depth(child_col);
    } else if (col.type().id() == cudf::type_id::STRUCT) {
      int max_child_depth = 0;
      for (auto child = col.child_begin(); child != col.child_end(); ++child) {
        max_child_depth = std::max(max_child_depth, get_nested_depth(*child));
      }
      return 1 + max_child_depth;
    } else {  // Primitive type
      return 0;
    }
  };

  for (auto i = 0; i < input.num_columns(); i++) {
    cudf::column_view const& col = input.column(i);
    CUDF_EXPECTS(get_nested_depth(col) <= MAX_STACK_DEPTH,
                 "The " + std::to_string(i) +
                   "-th column exceeds the maximum allowed nested depth. " +
                   "Current depth: " + std::to_string(get_nested_depth(col)) + ", " +
                   "Maximum allowed depth: " + std::to_string(MAX_STACK_DEPTH));
  }
}

}  // namespace

std::unique_ptr<cudf::column> hive_hash(cudf::table_view const& input,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  auto output = cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<hive_hash_value_t>()),
                                          input.num_rows(),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) { return output; }

  check_nested_depth(input);

  bool const nullable   = has_nested_nulls(input);
  auto const input_view = cudf::table_device_view::create(input, stream);
  auto output_view      = output->mutable_view();

  // Compute the hash value for each row
  thrust::tabulate(rmm::exec_policy(stream),
                   output_view.begin<hive_hash_value_t>(),
                   output_view.end<hive_hash_value_t>(),
                   hive_device_row_hasher<hive_hash_function, bool>(nullable, *input_view));

  return output;
}

}  // namespace spark_rapids_jni
