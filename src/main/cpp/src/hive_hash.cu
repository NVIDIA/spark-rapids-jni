#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hash.cuh"
#include "hash.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tabulate.h>

namespace spark_rapids_jni {

namespace {

using hive_hash_value_t = int32_t;

constexpr hive_hash_value_t HIVE_HASH_FACTOR = 31;
constexpr hive_hash_value_t HIVE_INIT_HASH   = 0;

struct col_info {
  cudf::type_id type_id;
  cudf::size_type nested_num_children_or_basic_col_idx;  // Number of children for nested types, or
                                                         // col_idx in `basic_cdvs` for basic types
};

hive_hash_value_t __device__ inline compute_int(int32_t key) { return key; }

hive_hash_value_t __device__ inline compute_long(int64_t key)
{
  return (static_cast<uint64_t>(key) >> 32) ^ key;
}

hive_hash_value_t __device__ inline compute_bytes(int8_t const* data, cudf::size_type const len)
{
  hive_hash_value_t ret = HIVE_INIT_HASH;
  for (auto i = 0; i < len; i++) {
    ret = ret * HIVE_HASH_FACTOR + static_cast<int32_t>(data[i]);
  }
  return ret;
}

template <typename Key>
struct hive_hash_function {
  // 'seed' is not used in 'hive_hash_function', but required by 'element_hasher'.
  constexpr hive_hash_function(uint32_t) {}

  [[nodiscard]] hive_hash_value_t __device__ inline operator()(Key const& key) const
  {
    CUDF_UNREACHABLE("Unsupported type for hive hash");
  }
};  // struct hive_hash_function

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::string_view>::operator()(
  cudf::string_view const& key) const
{
  auto const data = reinterpret_cast<int8_t const*>(key.data());
  auto const len  = key.size_bytes();
  return compute_bytes(data, len);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<bool>::operator()(bool const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int8_t>::operator()(int8_t const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int16_t>::operator()(
  int16_t const& key) const
{
  return compute_int(static_cast<int32_t>(key));
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int32_t>::operator()(
  int32_t const& key) const
{
  return compute_int(key);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<int64_t>::operator()(
  int64_t const& key) const
{
  return compute_long(key);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<float>::operator()(float const& key) const
{
  auto normalized = spark_rapids_jni::normalize_nans(key);
  auto* p_int     = reinterpret_cast<int32_t const*>(&normalized);
  return compute_int(*p_int);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<double>::operator()(double const& key) const
{
  auto normalized = spark_rapids_jni::normalize_nans(key);
  auto* p_long    = reinterpret_cast<int64_t const*>(&normalized);
  return compute_long(*p_long);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::timestamp_D>::operator()(
  cudf::timestamp_D const& key) const
{
  auto* p_int = reinterpret_cast<int32_t const*>(&key);
  return compute_int(*p_int);
}

template <>
hive_hash_value_t __device__ inline hive_hash_function<cudf::timestamp_us>::operator()(
  cudf::timestamp_us const& key) const
{
  auto time_as_long            = *reinterpret_cast<int64_t const*>(&key);
  constexpr int MICRO_PER_SEC  = 1000000;
  constexpr int NANO_PER_MICRO = 1000;

  int64_t ts  = time_as_long / MICRO_PER_SEC;
  int64_t tns = (time_as_long % MICRO_PER_SEC) * NANO_PER_MICRO;

  int64_t result = ts;
  result <<= 30;
  result |= tns;

  result = (static_cast<uint64_t>(result) >> 32) ^ result;
  return static_cast<hive_hash_value_t>(result);
}

/**
 * @brief Computes the hash value of a row in the given table.
 *
 * This functor produces the same result as "HiveHash" in Spark for supported types.
 *
 * @tparam hash_function Hash functor to use for hashing elements. Must be hive_hash_function.
 * @tparam Nullate A cudf::nullate type describing whether to check for nulls.
 */
template <template <typename> class hash_function, typename Nullate>
class hive_device_row_hasher {
 public:
  CUDF_HOST_DEVICE hive_device_row_hasher(Nullate check_nulls,
                                          cudf::table_device_view t,
                                          cudf::column_device_view* basic_cdvs,
                                          cudf::size_type* column_map,
                                          col_info* col_infos) noexcept
    : _check_nulls{check_nulls},
      _table{t},
      _basic_cdvs{basic_cdvs},
      _column_map{column_map},
      _col_infos{col_infos}
  {
    // Error out if passed an unsupported hash_function
    static_assert(std::is_base_of_v<hive_hash_function<int>, hash_function<int>>,
                  "hive_device_row_hasher only supports the 'hive_hash_function' hash function");
  }

  /**
   * @brief Return the hash value of a row in the given table.
   *
   * @param row_index The row index to compute the hash value of
   * @return The hash value of the row
   */
  __device__ auto operator()(cudf::size_type row_index) const noexcept
  {
    return cudf::detail::accumulate(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(_table.num_columns()),
      HIVE_INIT_HASH,
      cuda::proclaim_return_type<hive_hash_value_t>([=] __device__(auto hash, auto const col_idx) {
        auto const& column = _table.column(col_idx);
        auto cur_hash      = cudf::type_dispatcher(
          column.type(), element_hasher_adapter{_check_nulls, col_idx, *this}, column, row_index);
        return HIVE_HASH_FACTOR * hash + cur_hash;
      }));
  }

 private:
  /**
   * @brief Computes the hash value of an element in the given column.
   */
  class element_hasher_adapter {
   public:
    using hash_functor_t = cudf::experimental::row::hash::element_hasher<hash_function, Nullate>;

    __device__ element_hasher_adapter(Nullate check_nulls,
                                      cudf::size_type col_idx,
                                      hive_device_row_hasher const& parent) noexcept
      : hash_functor{check_nulls, HIVE_INIT_HASH, HIVE_INIT_HASH},
        _col_idx{col_idx},
        _parent{parent}
    {
    }

    template <typename T, CUDF_ENABLE_IF(not cudf::is_nested<T>())>
    __device__ hive_hash_value_t operator()(cudf::column_device_view const& col,
                                            cudf::size_type row_index) const noexcept
    {
      return this->hash_functor.template operator()<T>(col, row_index);
    }

    /**
     * @brief A structure to keep track of the computation for nested types.
     */
    struct col_stack_frame {
     private:
      cudf::size_type _col_idx;     // the column index in the flattened array
      cudf::size_type _row_idx;     // the index of the row in the column
      int _idx_to_process;          // the index of child or element to process next
      hive_hash_value_t _cur_hash;  // current hash value of the column

     public:
      __device__ col_stack_frame() = default;

      __device__ void init(cudf::size_type col_idx, cudf::size_type row_idx)
      {
        _col_idx        = col_idx;
        _row_idx        = row_idx;
        _idx_to_process = 0;
        _cur_hash       = HIVE_INIT_HASH;
      }

      __device__ void update_cur_hash(hive_hash_value_t hash)
      {
        _cur_hash = _cur_hash * HIVE_HASH_FACTOR + hash;
      }

      __device__ hive_hash_value_t get_hash() const { return _cur_hash; }

      __device__ int get_and_inc_idx_to_process() { return _idx_to_process++; }

      __device__ int get_idx_to_process() const { return _idx_to_process; }

      __device__ cudf::size_type get_col_idx() const { return _col_idx; }

      __device__ cudf::size_type get_row_idx() const { return _row_idx; }
    };

    /**
     * @brief Functor to compute the hive hash value for a nested column.
     *
     * This functor produces the same result as "HiveHash" in Spark for structs and lists.
     * The pseudocode of Spark's HiveHash function for structs is as follows:
     *
     * hive_hash_value_t hive_hash(NestedType element) {
     *    hive_hash_value_t hash = HIVE_INIT_HASH;
     *    for (int i = 0; i < element.num_child(); i++) {
     *        hash = hash * HIVE_HASH_FACTOR + hive_hash(element.get_child(i));
     *    }
     *    return hash;
     * }
     *
     * In the cases of lists, the hash value is computed by a similar way but we iterate through the
     * list elements instead of through the child columns' elements.
     *
     * This functor uses a stack to simulate the recursive process of the above pseudocode.
     * When an element is popped from the stack, it means that the hash value of it has been
     * computed. Therefore, we should update the parent's `cur_hash` upon popping the element.
     *
     * The algorithm is as follows:
     *
     * 1. Initialize the stack and push the root column into the stack.
     * 2. While the stack is not empty:
     *    a. Get the top element of the stack. Don't pop it until it is processed.
     *    b. If the column is a structs column:
     *        i.  If all child columns are processed, pop the element and update `cur_hash` of its
     *            parent column.
     *        ii. Otherwise, process the next child column.
     *    c. If the column is a lists column, process it by a similar way as structs column but
     *       iterating through the list elements instead of child columns' elements.
     * 3. Return the hash value of the root column.
     *
     * For example, consider the following nested column: `Struct<Struct<int, float>, decimal>`
     *
     *      S1
     *     / \
     *    S2  d
     *   / \
     *  i   f
     *
     * - First, S1 is pushed into the stack. Then, S2 is pushed into the stack.
     * - S2's hash value can be computed directly because its children are of primitive types.
     *   When S2 is popped, S1's `cur_hash` is updated to S2's hash value.
     * - Now the top of the stack is S1. The next child to process is d. S1's `cur_hash` is updated
     *   to `hash(S2) * HIVE_HASH_FACTOR + hash(d)`, which is the hash value of S1.
     * - When S1 is popped, the hash value of the root column is returned.
     *
     * As lists columns have a different interface from structs columns, we need to handle them
     * separately.
     *
     * For example, consider that the input column is of type `List<List<int>>`.
     * Assume that the element at `row_index` is: [[1, 0], null, [2, null]]
     * Since the stack_frame should contain a column that consists of only one row, the input column
     * should be sliced. The sliced column is noted as L1 here.
     *
     *     L1
     *     |
     *     L2
     *     |
     *     i
     *
     * List level L1:
     * |Index|      List<list<int>>    |
     * |-----|-------------------------|
     * |0    |[[1, 0], null, [2, null]]|
     * length: 1
     * Offsets: 0, 3
     *
     * List level L2:
     * |Index|List<int>|
     * |-----|---------|
     * |0    |[1, 0]   |
     * |1    |null     |
     * |2    |[2, null]|
     * length: 3
     * Offsets: 0, 2, 2, 4
     * null_mask: 101
     *
     * Int level i:
     * |Index|int |
     * |-----|----|
     * |0    |1   |
     * |1    |0   |
     * |2    |2   |
     * |3    |null|
     * length: 4
     * null_mask: 0111
     *
     * Since the underlying data loses the null information of the top-level list column, computing
     * hash values using the underlying data merely can yield different results compared to Spark.
     * For example, [[1, 0], [2, null]] has the same underlying data as [[1, 0], null, [2, null]].
     * However, they have different hive hash values, as null values affect the hash value.
     *
     * And sublist structure factors in the hash value. For example, [[1, 0], [2]] and [[1], [0, 2]]
     * have different hive hash values.
     *
     * The computation process for lists columns in this solution is as follows:
     *            L1              List<list<int>>
     *            |
     *            L2              List<int>
     *      /     |     \
     *    L2[0] L2[1] L2[2]       int
     *     |            |
     *    i1           i2         int
     *
     * Note: L2、i1、i2 are all temporary columns, which would not be pushed into the stack.
     * If the child column is of primitive type, the hash value of the list column can be directly
     * computed.
     *
     * @tparam T The type of the column.
     * @param col The column device view.
     * @param row_index The index of the row to compute the hash for.
     * @return The computed hive hash value.
     *
     * @note This function is only enabled for nested column types.
     */
    template <typename T, CUDF_ENABLE_IF(cudf::is_nested<T>())>
    __device__ hive_hash_value_t operator()(cudf::column_device_view const&,
                                            cudf::size_type row_index) const noexcept
    {
      auto curr_col_idx = _parent._column_map[_col_idx];
      auto next_col_idx = curr_col_idx;
      auto curr_row_idx = row_index;

      col_stack_frame col_stack[MAX_STACK_DEPTH];
      int stack_size = 0;
      col_stack[stack_size++].init(curr_col_idx, curr_row_idx);

      while (stack_size > 0) {
        col_stack_frame& top      = col_stack[stack_size - 1];
        curr_col_idx              = top.get_col_idx();
        curr_row_idx              = top.get_row_idx();
        auto const& curr_col_info = _parent._col_infos[curr_col_idx];
        // Do not pop it until it is processed. The definition of `processed` is:
        // - For structs, it is when all child columns are processed.
        // - For lists, it is when all elements in the list are processed.
        if (curr_col_info.type_id == cudf::type_id::STRUCT) {
          if (top.get_idx_to_process() == curr_col_info.nested_num_children_or_basic_col_idx) {
            if (--stack_size > 0) { col_stack[stack_size - 1].update_cur_hash(top.get_hash()); }
          } else {
            if (top.get_idx_to_process() == 0) { next_col_idx = curr_col_idx + 1; }
            while (top.get_idx_to_process() < curr_col_info.nested_num_children_or_basic_col_idx) {
              top.get_and_inc_idx_to_process();
              auto child_col_idx     = next_col_idx++;
              auto const& child_info = _parent._col_infos[child_col_idx];
              // If the child is of primitive type, accumulate child hash into struct hash
              if (child_info.type_id != cudf::type_id::LIST &&
                  child_info.type_id != cudf::type_id::STRUCT) {
                auto const& child_col =
                  _parent._basic_cdvs[child_info.nested_num_children_or_basic_col_idx];
                auto child_hash =
                  cudf::type_dispatcher<cudf::experimental::dispatch_void_if_nested>(
                    child_col.type(), this->hash_functor, child_col, curr_row_idx);
                top.update_cur_hash(child_hash);
              } else {
                col_stack[stack_size++].init(child_col_idx, curr_row_idx);
                break;
              }
            }
          }
        } else if (curr_col_info.type_id == cudf::type_id::LIST) {
          // Get the child column of the list column
          auto offsets_col_idx       = curr_col_idx + 1;
          auto child_col_idx         = curr_col_idx + 2;
          auto const& offsets_col    = _parent._basic_cdvs[_parent._col_infos[offsets_col_idx]
                                                          .nested_num_children_or_basic_col_idx];
          auto const& child_col_info = _parent._col_infos[child_col_idx];
          auto child_row_idx_begin   = offsets_col.element<cudf::size_type>(curr_row_idx);
          auto child_row_idx_end     = offsets_col.element<cudf::size_type>(curr_row_idx + 1);

          // If the child column is of primitive type, directly compute the hash value of the list
          if (child_col_info.type_id != cudf::type_id::LIST &&
              child_col_info.type_id != cudf::type_id::STRUCT) {
            auto const& child_col =
              _parent._basic_cdvs[child_col_info.nested_num_children_or_basic_col_idx];
            auto single_level_list_hash = cudf::detail::accumulate(
              thrust::counting_iterator(child_row_idx_begin),
              thrust::counting_iterator(child_row_idx_end),
              HIVE_INIT_HASH,
              [child_col, hasher = this->hash_functor] __device__(auto hash, auto element_index) {
                auto cur_hash = cudf::type_dispatcher<cudf::experimental::dispatch_void_if_nested>(
                  child_col.type(), hasher, child_col, element_index);
                return HIVE_HASH_FACTOR * hash + cur_hash;
              });
            top.update_cur_hash(single_level_list_hash);
            if (--stack_size > 0) {
              col_stack[stack_size - 1].update_cur_hash(top.get_hash());
              next_col_idx = curr_col_idx + 3;
            }
          } else {
            if (top.get_idx_to_process() == child_row_idx_end - child_row_idx_begin) {
              if (--stack_size > 0) { col_stack[stack_size - 1].update_cur_hash(top.get_hash()); }
            } else {
              // Push the next element into the stack
              col_stack[stack_size++].init(child_col_idx,
                                           child_row_idx_begin + top.get_and_inc_idx_to_process());
            }
          }
        }
      }
      return col_stack[0].get_hash();
    }

   private:
    hash_functor_t const hash_functor;
    cudf::size_type const _col_idx;
    hive_device_row_hasher const& _parent;
  };

  Nullate const _check_nulls;
  cudf::table_device_view const _table;
  cudf::column_device_view* _basic_cdvs;
  cudf::size_type* _column_map;
  col_info* _col_infos;
};

void check_nested_depth(cudf::table_view const& input)
{
  using column_checker_fn_t = std::function<int(cudf::column_view const&)>;

  column_checker_fn_t get_nested_depth = [&](cudf::column_view const& col) {
    if (col.type().id() == cudf::type_id::LIST) {
      auto const child_col = cudf::lists_column_view(col).child();
      return 1 + get_nested_depth(child_col);
    } else if (col.type().id() == cudf::type_id::STRUCT) {
      int max_child_depth = 0;
      for (auto child = col.child_begin(); child != col.child_end(); ++child) {
        max_child_depth = std::max(max_child_depth, get_nested_depth(*child));
      }
      return 1 + max_child_depth;
    } else {  // Primitive type
      return 0;
    }
  };

  for (auto i = 0; i < input.num_columns(); i++) {
    cudf::column_view const& col = input.column(i);
    CUDF_EXPECTS(get_nested_depth(col) <= MAX_STACK_DEPTH,
                 "The " + std::to_string(i) +
                   "-th column exceeds the maximum allowed nested depth. " +
                   "Current depth: " + std::to_string(get_nested_depth(col)) + ", " +
                   "Maximum allowed depth: " + std::to_string(MAX_STACK_DEPTH));
  }
}

void flatten_table(std::vector<col_info>& col_infos,
                   std::vector<cudf::column_view>& basic_cvs,
                   cudf::table_view const& input,
                   std::vector<cudf::size_type>& column_map,
                   rmm::cuda_stream_view const& stream)
{
  using column_processer_fn_t = std::function<void(std::vector<col_info>&,
                                                   std::vector<cudf::column_view>&,
                                                   cudf::column_view const&,
                                                   rmm::cuda_stream_view const&)>;
  // Pre-order traversal
  column_processer_fn_t flatten_column = [&](std::vector<col_info>& col_infos,
                                             std::vector<cudf::column_view>& basic_cvs,
                                             cudf::column_view const& col,
                                             rmm::cuda_stream_view const& stream) {
    auto type_id = col.type().id();
    if (type_id == cudf::type_id::LIST) {
      col_infos.push_back(col_info{type_id, col.num_children()});
      auto const list_col = cudf::lists_column_view(col);
      flatten_column(col_infos, basic_cvs, list_col.offsets(), stream);
      flatten_column(col_infos, basic_cvs, list_col.get_sliced_child(stream), stream);
    } else if (type_id == cudf::type_id::STRUCT) {
      col_infos.push_back(col_info{type_id, col.num_children()});
      auto const struct_col = cudf::structs_column_view(col);
      for (auto child_idx = 0; child_idx < col.num_children(); child_idx++) {
        flatten_column(
          col_infos, basic_cvs, struct_col.get_sliced_child(child_idx, stream), stream);
      }
    } else {
      col_infos.push_back(col_info{type_id, static_cast<cudf::size_type>(basic_cvs.size())});
      basic_cvs.push_back(col);
    }
  };

  for (auto const& root_col : input) {
    column_map.push_back(static_cast<cudf::size_type>(col_infos.size()));
    flatten_column(col_infos, basic_cvs, root_col, stream);
  }
}
}  // namespace

std::unique_ptr<cudf::column> hive_hash(cudf::table_view const& input,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  auto output = cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<hive_hash_value_t>()),
                                          input.num_rows(),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) { return output; }

  check_nested_depth(input);

  // `basic_cvs` contains column_views of all basic columns in `input` and basic columns that result
  // from flattening nested columns
  std::vector<cudf::column_view> basic_cvs;
  // `column_map` maps the column index in `input` to the index in `col_infos`
  std::vector<cudf::size_type> column_map;
  // `col_infos` contains information of all columns in `input` and columns that result from
  // flattening nested columns
  std::vector<col_info> col_infos;

  flatten_table(col_infos, basic_cvs, input, column_map, stream);

  [[maybe_unused]] auto [device_view_owners, basic_cdvs] =
    cudf::contiguous_copy_column_device_views<cudf::column_device_view>(basic_cvs, stream);
  auto col_infos_view = cudf::detail::make_device_uvector_async(
    col_infos, stream, cudf::get_current_device_resource_ref());
  auto column_map_view = cudf::detail::make_device_uvector_async(
    column_map, stream, cudf::get_current_device_resource_ref());

  bool const nullable   = has_nested_nulls(input);
  auto const input_view = cudf::table_device_view::create(input, stream);
  auto output_view      = output->mutable_view();

  // Compute the hash value for each row
  thrust::tabulate(
    rmm::exec_policy_nosync(stream),
    output_view.begin<hive_hash_value_t>(),
    output_view.end<hive_hash_value_t>(),
    hive_device_row_hasher<hive_hash_function, bool>(
      nullable, *input_view, basic_cdvs, column_map_view.data(), col_infos_view.data()));

  // Push data from host vectors `first_child_index` and `column_map` to device
  // before they are destroyed.
  stream.synchronize();
  return output;
}

}  // namespace spark_rapids_jni
