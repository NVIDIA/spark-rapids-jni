#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "murmur_hash.cuh"

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

namespace spark_rapids_jni {

namespace {

using bloom_hash_type = spark_rapids_jni::murmur_hash_value_type;

__device__ inline std::pair<cudf::size_type, cudf::bitmask_type> gpu_get_hash_mask(
  bloom_hash_type h, cudf::size_type bloom_filter_bits)
{
  // https://github.com/apache/spark/blob/7bfbeb62cb1dc58d81243d22888faa688bad8064/common/sketch/src/main/java/org/apache/spark/util/sketch/BloomFilterImpl.java#L94
  auto const index      = (h < 0 ? ~h : h) % static_cast<bloom_hash_type>(bloom_filter_bits);
  auto const word_index = cudf::word_index(index);
  auto const bit_index  = cudf::intra_word_index(index);
  return {word_index, (1 << bit_index)};
}

__global__ void gpu_bloom_filter_build(cudf::bitmask_type* const bloom_filter,
                                       cudf::size_type bloom_filter_bits,
                                       cudf::device_span<int64_t const> input,
                                       cudf::size_type num_hashes)
{
  int const tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= input.size()) { return; }

  // https://github.com/apache/spark/blob/7bfbeb62cb1dc58d81243d22888faa688bad8064/common/sketch/src/main/java/org/apache/spark/util/sketch/BloomFilterImpl.java#L87
  bloom_hash_type const h1 = MurmurHash3_32<int64_t>(0)(input[tid]);
  bloom_hash_type const h2 = MurmurHash3_32<int64_t>(h1)(input[tid]);

  // set a bit in the bloom filter for each hashed value
  for (auto idx = 1; idx <= num_hashes; idx++) {
    bloom_hash_type combined_hash = h1 + (idx * h2);

    auto const [word_index, mask] = gpu_get_hash_mask(combined_hash, bloom_filter_bits);
    atomicOr(bloom_filter + word_index, mask);
  }
}

struct bloom_probe_functor {
  cudf::bitmask_type const* const bloom_filter;
  cudf::size_type const bloom_filter_bits;
  cudf::size_type const num_hashes;

  __device__ bool operator()(int64_t input)
  {
    // https://github.com/apache/spark/blob/7bfbeb62cb1dc58d81243d22888faa688bad8064/common/sketch/src/main/java/org/apache/spark/util/sketch/BloomFilterImpl.java#L110
    // this code could be combined with the very similar code in gpu_bloom_filter_build. i've
    // left it this way since the expectation is that we will early out fairly often, whereas
    // in the build case we never early out so doing the additional if() return check is pointless.
    bloom_hash_type const h1 = MurmurHash3_32<int64_t>(0)(input);
    bloom_hash_type const h2 = MurmurHash3_32<int64_t>(h1)(input);

    // set a bit in the bloom filter for each hashed value
    for (auto idx = 1; idx <= num_hashes; idx++) {
      bloom_hash_type combined_hash = h1 + (idx * h2);
      auto const [word_index, mask] = gpu_get_hash_mask(combined_hash, bloom_filter_bits);
      if (!(bloom_filter[word_index] & mask)) { return false; }
    }
    return true;
  }
};

}  // anonymous namespace

rmm::device_uvector<cudf::bitmask_type> bloom_filter_create(cudf::size_type bloom_filter_bits,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr)
{
  rmm::device_uvector<cudf::bitmask_type> out(
    cudf::num_bitmask_words(bloom_filter_bits), stream, mr);
  hipMemsetAsync(out.data(), 0, out.size() * sizeof(cudf::bitmask_type), stream);
  return out;
}

void bloom_filter_build(rmm::device_uvector<cudf::bitmask_type>& bloom_filter,
                        cudf::size_type bloom_filter_bits,
                        cudf::column_view const& input,
                        cudf::size_type num_hashes,
                        rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(input.type() == cudf::data_type{cudf::type_id::INT64} && !input.nullable(),
               "bloom filter input expects a non-nullable column of int64s");
  CUDF_EXPECTS(bloom_filter_bits > 0, "Invalid empty bloom filter size specified");

  constexpr int block_size = 256;
  auto grid                = cudf::detail::grid_1d{input.size(), block_size, 1};
  gpu_bloom_filter_build<<<grid.num_blocks, block_size, 0, stream.value()>>>(
    bloom_filter.data(), bloom_filter_bits, input, num_hashes);
}

rmm::device_uvector<cudf::bitmask_type> bloom_filter_merge(
  rmm::device_uvector<cudf::bitmask_type> const& a,
  rmm::device_uvector<cudf::bitmask_type> const& b,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(a.size() == b.size(),
               "bloom_filter_merge encountered mismatched input filter sizes");

  rmm::device_uvector<cudf::bitmask_type> out(a.size(), stream, mr);
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(0) + a.size(),
    out.begin(),
    [a = a.begin(), b = b.begin()] __device__(cudf::size_type i) { return a[i] | b[i]; });

  return out;
}

std::unique_ptr<cudf::column> bloom_filter_probe(
  cudf::column_view const& input,
  rmm::device_uvector<cudf::bitmask_type> const& bloom_filter,
  cudf::size_type bloom_filter_bits,
  cudf::size_type num_hashes,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input.type() == cudf::data_type{cudf::type_id::INT64} && !input.nullable(),
               "bloom filter input expects a non-nullable column of int64s");
  CUDF_EXPECTS(bloom_filter_bits > 0, "Invalid empty bloom filter");

  auto out = cudf::make_fixed_width_column(
    cudf::data_type{cudf::type_id::BOOL8}, input.size(), cudf::mask_state::UNALLOCATED, stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    input.begin<int64_t>(),
                    input.end<int64_t>(),
                    out->mutable_view().begin<bool>(),
                    bloom_probe_functor{bloom_filter.data(), bloom_filter_bits, num_hashes});
  return out;
}

}  // namespace spark_rapids_jni
