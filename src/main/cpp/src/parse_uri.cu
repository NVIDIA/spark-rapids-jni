#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parse_uri.hpp"

#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/convert/convert_urls.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/string_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>

#include <memory>
#include <optional>

namespace spark_rapids_jni {

using namespace cudf;

namespace detail {

struct uri_parts {
  string_view scheme;
  string_view host;
  string_view authority;
  string_view path;
  string_view fragment;
  string_view query;
  string_view userinfo;
  string_view port;
  string_view opaque;
  uint32_t valid{0};
};

enum class URI_chunks : int8_t {
  PROTOCOL,
  HOST,
  AUTHORITY,
  PATH,
  FRAGMENT,
  QUERY,
  USERINFO,
  PORT,
  OPAQUE
};

enum class chunk_validity : int8_t { VALID, INVALID, FATAL };

namespace {

// Some parsing errors are fatal and some parsing errors simply mean this
// thing doesn't exist or is invalid. For example, just because 280.0.1.16 is
// not a valid IPv4 address simply means if asking for the host the host is null
// but the authority is still 280.0.1.16 and the uri is not considered invalid.
// By contrast, the URI https://[15:6:g:invalid] will not return https for the
// scheme and is considered completely invalid.

constexpr bool is_alpha(char c) { return (c >= 'a' && c <= 'z') || (c >= 'A' && c <= 'Z'); }

constexpr bool is_numeric(char c) { return c >= '0' && c <= '9'; }

constexpr bool is_alphanum(char c) { return is_alpha(c) || is_numeric(c); }

constexpr bool is_hex(char c)
{
  return is_numeric(c) || (c >= 'a' && c <= 'f') || (c >= 'A' && c <= 'F');
}

__device__ thrust::pair<bool, string_view::const_iterator> skip_and_validate_special(
  string_view::const_iterator iter,
  string_view::const_iterator end,
  bool allow_invalid_escapes = false)
{
  while (iter != end) {
    auto const c         = *iter;
    auto const num_bytes = cudf::strings::detail::bytes_in_char_utf8(*iter);
    if (*iter == '%' && !allow_invalid_escapes) {
      // verify following two characters are hexadecimal
      for (int i = 0; i < 2; ++i) {
        ++iter;
        if (iter == end) { return {false, iter}; }

        if (!is_hex(*iter)) { return {false, iter}; }
      }
    } else if (num_bytes > 1) {
      // UTF8 validation means it isn't whitespace and not a control character
      // the normal validation will handle anything single byte, this checks for multiple byte
      // whitespace
      auto const c = *iter;
      // There are multi-byte looking things like extended ASCII characters that are not valid UTF8.
      // Check that here.
      if ((c & 0xC0) != 0x80) { return {false, iter}; }
      if (num_bytes > 2 && ((c & 0xC000) != 0x8000)) { return {false, iter}; }
      if (num_bytes > 3 && ((c & 0xC00000) != 0x800000)) { return {false, iter}; }

      // Validate it isn't a whitespace or control unicode character.
      if ((c >= 0xc280 && c <= 0xc2a0) || c == 0xe19a80 || (c >= 0xe28080 && c <= 0xe2808a) ||
          c == 0xe280af || c == 0xe280a8 || c == 0xe2819f || c == 0xe38080) {
        return {false, iter};
      }
    } else {
      break;
    }
    ++iter;
  }

  return {true, iter};
}

template <typename Predicate>
__device__ bool validate_chunk(string_view s, Predicate fn, bool allow_invalid_escapes = false)
{
  auto iter = s.begin();
  {
    auto [valid, iter_] = skip_and_validate_special(iter, s.end(), allow_invalid_escapes);
    iter                = std::move(iter_);
    if (!valid) { return false; }
  }
  while (iter != s.end()) {
    if (!fn(iter)) { return false; }

    iter++;
    auto [valid, iter_] = skip_and_validate_special(iter, s.end(), allow_invalid_escapes);
    iter                = std::move(iter_);
    if (!valid) { return false; }
  }
  return true;
}

bool __device__ validate_scheme(string_view scheme)
{
  // A scheme simply needs to be an alpha character followed by alphanumeric
  auto iter = scheme.begin();
  if (!is_alpha(*iter)) { return false; }
  while (++iter != scheme.end()) {
    auto const c = *iter;
    if (!is_alphanum(c) && c != '+' && c != '-' && c != '.') { return false; }
  }
  return true;
}

bool __device__ validate_ipv6(string_view s)
{
  constexpr auto max_colons{8};

  if (s.size_bytes() < 2) { return false; }

  bool found_double_colon{false};
  int open_bracket_count{0};
  int close_bracket_count{0};
  int period_count{0};
  int colon_count{0};
  int percent_count{0};
  char previous_char{0};
  int address{0};
  int address_char_count{0};
  bool address_has_hex{false};

  auto const leading_double_colon = [&]() {
    auto iter = s.begin();
    if (*iter == '[') iter++;
    return *iter++ == ':' && *iter == ':';
  }();

  for (auto iter = s.begin(); iter < s.end(); ++iter) {
    auto const c = *iter;

    switch (c) {
      case '[':
        open_bracket_count++;
        if (open_bracket_count > 1) { return false; }
        break;
      case ']':
        close_bracket_count++;
        if (close_bracket_count > 1) { return false; }
        if ((period_count > 0) && (address_has_hex || address > 255)) { return false; }
        break;
      case ':':
        colon_count++;
        if (previous_char == ':') {
          if (found_double_colon) { return false; }
          found_double_colon = true;
        }
        address            = 0;
        address_has_hex    = false;
        address_char_count = 0;
        if (colon_count > max_colons || (colon_count == max_colons && !found_double_colon)) {
          return false;
        }
        // Periods before a colon don't work, periods can be an IPv4 address after this IPv6 address
        // like [1:2:3:4:5:6:d.d.d.d]
        if (period_count > 0 || percent_count > 0) { return false; }
        break;
      case '.':
        period_count++;
        if (percent_count > 0) { return false; }
        if (period_count > 3) { return false; }
        if (address_has_hex) { return false; }
        if (address > 255) { return false; }
        if (colon_count != 6 && !found_double_colon) { return false; }
        // Special case of ::1:2:3:4:5:d.d.d.d has 7 colons - but spark says this is invalid
        // if (colon_count == max_colons && !leading_double_colon) { return false; }
        if (colon_count >= max_colons) { return false; }
        address            = 0;
        address_has_hex    = false;
        address_char_count = 0;
        break;
      case '%':
        // IPv6 can define a device to use for the routing. This is expressed as '%eth0' at the end
        // of the address.
        percent_count++;
        if (percent_count > 1) { return false; }
        if ((period_count > 0) && (address_has_hex || address > 255)) { return false; }
        address            = 0;
        address_has_hex    = false;
        address_char_count = 0;
        break;
      default:
        // after % all bets are off as the device name can be nearly anything
        if (percent_count == 0) {
          if (address_char_count > 3) { return false; }
          address_char_count++;
          address *= 10;
          if (c >= 'a' && c <= 'f') {
            address += 10;
            address += c - 'a';
            address_has_hex = true;
          } else if (c >= 'A' && c <= 'Z') {
            address += 10;
            address += c - 'A';
            address_has_hex = true;
          } else if (c >= '0' && c <= '9') {
            address += c - '0';
          } else {
            return false;
          }
        }
        break;
    }
    previous_char = c;
  }

  return true;
}

bool __device__ validate_ipv4(string_view s)
{
  // dotted quad (0-255).(0-255).(0-255).(0-255)
  int address            = 0;
  int address_char_count = 0;
  int dot_count          = 0;
  for (auto iter = s.begin(); iter < s.end(); ++iter) {
    auto const c = *iter;

    // can't lead with a .
    if ((c < '0' || c > '9') && (iter == s.begin() || c != '.')) { return false; }

    if (c == '.') {
      // verify we saw at least one character and reset values
      if (address_char_count == 0) { return false; }
      address            = 0;
      address_char_count = 0;
      dot_count++;
      continue;
    }

    address_char_count++;
    address *= 10;
    address += c - '0';

    if (address > 255) { return false; }
  }

  // can't end with a .
  if (address_char_count == 0) { return false; }

  // must be 4 portions seperated by 3 dots.
  if (dot_count != 3) { return false; }

  return true;
}

bool __device__ validate_domain_name(string_view name)
{
  // domain name can be alphanum or -.
  // slash can not be the first of last character of the domain name or around a .
  bool last_was_slash          = false;
  bool last_was_period         = false;
  bool numeric_start           = false;
  int characters_before_period = 0;
  for (auto iter = name.begin(); iter < name.end(); ++iter) {
    auto const c = *iter;
    if (!is_alphanum(c) && c != '-' && c != '.') { return false; }

    // the final section can't start with a digit
    if (last_was_period && c >= '0' && c <= '9') {
      numeric_start = true;
    } else {
      numeric_start = false;
    }

    if (c == '-') {
      if (last_was_period || iter == name.begin() || iter == --name.end()) { return false; }
      last_was_slash  = true;
      last_was_period = false;
    } else if (c == '.') {
      if (last_was_slash || last_was_period || characters_before_period == 0) { return false; }
      last_was_period          = true;
      last_was_slash           = false;
      characters_before_period = 0;
    } else {
      last_was_period = false;
      last_was_slash  = false;
      characters_before_period++;
    }
  }

  // numeric start to last part of domain isn't allowed.
  if (numeric_start) { return false; }

  return true;
}

chunk_validity __device__ validate_host(string_view host)
{
  // This can be IPv4, IPv6, or a domain name.
  if (*host.begin() == '[') {
    // If last character is a ], this is IPv6 or invalid.
    if (*(host.end() - 1) != ']') {
      // invalid
      return chunk_validity::FATAL;
    }
    if (!validate_ipv6(host)) { return chunk_validity::FATAL; }

    return chunk_validity::VALID;
  }

  // If there are more [ or ] characters this is invalid.
  // Also need to find the last .
  int last_open_bracket  = -1;
  int last_close_bracket = -1;
  int last_period        = -1;

  // The original plan on this loop was to get fancy and use a reverse iterator and exit when
  // everything was found, but the expectation is there are no brackets in this string, so we have
  // to traverse the entire thing anyway to verify that. The math is easier with a forward iterator,
  // so we're back here.
  for (auto iter = host.begin(); iter < host.end(); ++iter) {
    auto const c = *iter;
    if (c == '[') {
      last_open_bracket = iter.position();
    } else if (c == ']') {
      last_close_bracket = iter.position();
    } else if (c == '.') {
      last_period = iter.position();
    }
  }

  if (last_open_bracket >= 0 || last_close_bracket >= 0) { return chunk_validity::FATAL; }

  // If we didn't find a period or if the last character is a period or the character after the last
  // period is non numeric
  if (last_period < 0 || last_period == host.length() - 1 || host[last_period + 1] < '0' ||
      host[last_period + 1] > '9') {
    // must be domain name or it is invalid
    if (validate_domain_name(host)) { return chunk_validity::VALID; }

    // the only other option is that this is a IPv4 address
  } else if (validate_ipv4(host)) {
    return chunk_validity::VALID;
  }

  return chunk_validity::INVALID;
}

bool __device__ validate_query(string_view query)
{
  // query can be alphanum and _-!.~'()*,;:$&+=?/[]@"
  return validate_chunk(
    query, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c != '!' && c != '"' && c != '$' && !(c >= '&' && c <= ';') && c != '=' &&
          !(c >= '?' && c <= ']' && c != '\\') && !(c >= 'a' && c <= 'z') && c != '_' && c != '~') {
        return false;
      }
      return true;
    }));
}

bool __device__ validate_authority(string_view authority, bool allow_invalid_escapes)
{
  // authority needs to be alphanum and @[]_-!.'()*,;:$&+=
  return validate_chunk(authority,
                        cuda::proclaim_return_type<bool>(
                          [allow_invalid_escapes] __device__(string_view::const_iterator iter) {
                            auto const c = *iter;
                            if (c != '!' && c != '$' && !(c >= '&' && c <= ';' && c != '/') &&
                                c != '=' && !(c >= '@' && c <= '_' && c != '^' && c != '\\') &&
                                !(c >= 'a' && c <= 'z') && c != '~' &&
                                (!allow_invalid_escapes || c != '%')) {
                              return false;
                            }
                            return true;
                          }),
                        allow_invalid_escapes);
}

bool __device__ validate_userinfo(string_view userinfo)
{
  // can't be ] or [ in here
  return validate_chunk(
    userinfo, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c == '[' || c == ']') { return false; }
      return true;
    }));
}

bool __device__ validate_port(string_view port)
{
  // port is positive numeric >=0 according to spark...shrug
  return validate_chunk(
    port, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c < '0' && c > '9') { return false; }
      return true;
    }));
}

bool __device__ validate_path(string_view path)
{
  // path can be alphanum and @[]_-!.~'()*?/&,;:$+=
  return validate_chunk(
    path, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c != '!' && c != '$' && !(c >= '&' && c <= ';') && c != '=' && !(c >= '@' && c <= 'Z') &&
          c != '_' && !(c >= 'a' && c <= 'z') && c != '~') {
        return false;
      }
      return true;
    }));
}

bool __device__ validate_opaque(string_view opaque)
{
  // opaque can be alphanum and @[]_-!.~'()*?/,;:$@+=
  return validate_chunk(
    opaque, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c != '!' && c != '$' && !(c >= '&' && c <= ';') && c != '=' &&
          !(c >= '?' && c <= ']' && c != '\\') && c != '_' && c != '~' && !(c >= 'a' && c <= 'z')) {
        return false;
      }
      return true;
    }));
}

bool __device__ validate_fragment(string_view fragment)
{
  // fragment can be alphanum and @[]_-!.~'()*?/,;:$&+=
  return validate_chunk(
    fragment, cuda::proclaim_return_type<bool>([] __device__(string_view::const_iterator iter) {
      auto const c = *iter;
      if (c != '!' && c != '$' && !(c >= '&' && c <= ';') && c != '=' &&
          !(c >= '?' && c <= ']' && c != '\\') && c != '_' && c != '~' && !(c >= 'a' && c <= 'z')) {
        return false;
      }
      return true;
    }));
}

__device__ std::pair<string_view, bool> find_query_part(string_view haystack, string_view needle)
{
  auto const n_bytes = needle.size_bytes();
  auto h             = haystack.data();
  auto const h_end   = h + haystack.size_bytes();
  auto n             = needle.data();

  // stop matching early after it can no longer contain the string we are searching for
  while (h + n_bytes < h_end) {
    bool match_needle = true;
    for (size_type jdx = 0; jdx < n_bytes; ++jdx) {
      match_needle = (h[jdx] == n[jdx]);
      if (!match_needle) { break; }
    }

    if (match_needle && h[n_bytes] == '=') {
      // we don't care about the matched part, we want the string data after '='
      h += n_bytes + 1;

      // rest of string until end or until '&' is query match
      int match_len = 0;
      auto start    = h;
      while (h < h_end && *h != '&') {
        match_len++;
        h++;
      }

      return {{start, match_len}, true};
    }

    // not match, skip to the next param if possible, which is after a &.
    while (h + n_bytes < h_end && *h != '&') {
      h++;
    }
    h++;  // skip over the & if has, or point to h_end +1
  }

  return {{}, false};
}

uri_parts __device__ validate_uri(const char* str,
                                  int len,
                                  thrust::optional<column_device_view const> query_match,
                                  size_type row_idx)
{
  uri_parts ret;

  auto const original_str = str;
  // look for :/# characters.
  int col      = -1;
  int slash    = -1;
  int hash     = -1;
  int question = -1;
  for (const char* c = str;
       c - str < len && (col == -1 || slash == -1 || hash == -1 || question == -1);
       ++c) {
    switch (*c) {
      case ':':
        if (col == -1) col = c - str;
        break;
      case '/':
        if (slash == -1) slash = c - str;
        break;
      case '#':
        if (hash == -1) hash = c - str;
        break;
      case '?':
        if (question == -1) question = c - str;
        break;
      default: break;
    }
  }

  // anything after the hash is part of the fragment and ignored for this part
  if (hash >= 0) {
    ret.fragment = {str + hash + 1, len - hash - 1};
    if (!validate_fragment(ret.fragment)) {
      ret.valid = 0;
      return ret;
    }
    ret.valid |= (1 << static_cast<int>(URI_chunks::FRAGMENT));

    len = hash;

    if (col > hash) col = -1;
    if (slash > hash) slash = -1;
    if (question > hash) question = -1;
  }

  // if the first ':' is after the other tokens, this doesn't have a scheme or it is invalid
  bool const has_scheme =
    (col != -1) && ((slash == -1) || (col < slash)) && ((hash == -1) || (col < hash));
  if (has_scheme) {
    // we have a scheme up to the :
    ret.scheme = {str, col};
    if (!validate_scheme(ret.scheme)) {
      ret.valid = 0;
      return ret;
    }
    ret.valid |= (1 << static_cast<int>(URI_chunks::PROTOCOL));

    // skip over scheme
    auto const skip = col + 1;
    str += skip;
    len -= skip;
    question -= skip;
    hash -= skip;
    slash -= skip;
  }

  // no more string to parse is generally an error, unless we had no scheme
  if (len <= 0) {
    // If we had a scheme then this is entirely invalid.
    // If no scheme then URI is entirely empty or we only had a fragment
    // This is equivalent to having a path that is present but empty, so mark it ok
    ret.valid = (static_cast<int>(!has_scheme) << static_cast<int>(URI_chunks::PATH));
    return ret;
  }

  // If we have a '/' as the next character or this is still the start of the string, we have a
  // heirarchical uri. If not it is opaque.
  bool const heirarchical = str[0] == '/' || str == original_str;
  if (heirarchical) {
    // a '?' will break this into query and path/authority
    if (question >= 0) {
      ret.query = {str + question + 1, len - question - 1};
      if (!validate_query(ret.query)) {
        ret.valid = 0;
        return ret;
      }

      // Maybe limit the query data if a literal or a column is passed as a filter. This alters the
      // return from the entire query to just a specific parameter. For example, query for the URI
      // http://www.nvidia.com/page?param0=5&param1=2 is param0=5&param1=2, but if the literal is
      // passed as param0, the return would simply be 5.
      if (query_match && query_match->size() > 0) {
        auto const match_idx = row_idx % query_match->size();
        if (query_match->is_null(match_idx)) {
          ret.valid = 0;
          return ret;
        }
        auto in_match = query_match->element<string_view>(match_idx);

        auto const [query, valid] = find_query_part(ret.query, in_match);
        if (!valid) {
          ret.valid = 0;
          return ret;
        }
        ret.query = query;
      }

      ret.valid |= (1 << static_cast<int>(URI_chunks::QUERY));
    }
    auto const path_len = question >= 0 ? question : len;

    if (str[0] == '/' && str[1] == '/') {
      // If we have a '/', we have //authority/path, otherwise we have //authority with no path.
      int next_slash = -1;
      for (int i = 2; i < path_len; ++i) {
        if (str[i] == '/') {
          next_slash = i;
          break;
        }
      }
      ret.authority = {&str[2],
                       next_slash == -1 ? question < 0 ? len - 2 : question - 2 : next_slash - 2};
      if (next_slash > 0) { ret.path = {str + next_slash, path_len - next_slash}; }

      if (ret.authority.size_bytes() > 0) {
        auto ipv6_address = ret.authority.size_bytes() > 2 && *ret.authority.begin() == '[';
        if (!validate_authority(ret.authority, ipv6_address)) {
          ret.valid = 0;
          return ret;
        }
        ret.valid |= (1 << static_cast<int>(URI_chunks::AUTHORITY));

        // Inspect the authority for userinfo, host, and port
        const char* auth   = ret.authority.data();
        auto auth_size     = ret.authority.size_bytes();
        int amp            = -1;
        int closingbracket = -1;
        int last_colon     = -1;
        for (int i = 0; i < auth_size; ++i) {
          switch (auth[i]) {
            case '@':
              if (amp == -1) {
                amp = i;
                if (last_colon > 0) { last_colon = -1; }
                if (closingbracket > 0) { closingbracket = -1; }
              }
              break;
            case ':': last_colon = amp > 0 ? i - amp - 1 : i; break;
            case ']':
              if (closingbracket == -1) closingbracket = amp > 0 ? i - amp : i;
              break;
          }
        }

        if (amp > 0) {
          ret.userinfo = {auth, amp};
          if (!validate_userinfo(ret.userinfo)) {
            ret.valid = 0;
            return ret;
          }
          ret.valid |= (1 << static_cast<int>(URI_chunks::USERINFO));

          // skip over the @
          amp++;

          auth += amp;
          auth_size -= amp;
        }
        if (last_colon > 0 && last_colon > closingbracket) {
          // Found a port, attempt to parse it
          ret.port = {auth + last_colon + 1, auth_size - last_colon - 1};
          if (!validate_port(ret.port)) {
            ret.valid = 0;
            return ret;
          }
          ret.valid |= (1 << static_cast<int>(URI_chunks::PORT));
          ret.host = {auth, last_colon};
        } else {
          ret.host = {auth, auth_size};
        }
        auto host_ret = validate_host(ret.host);
        switch (host_ret) {
          case chunk_validity::FATAL: ret.valid = 0; return ret;
          case chunk_validity::INVALID: ret.host = {}; break;
          case chunk_validity::VALID: ret.valid |= (1 << static_cast<int>(URI_chunks::HOST)); break;
        }
      }
    } else {
      // path with no authority
      ret.path = {str, path_len};
    }

    if (!validate_path(ret.path)) {
      ret.valid = 0;
      return ret;
    }
    ret.valid |= (1 << static_cast<int>(URI_chunks::PATH));
  } else {
    ret.opaque = {str, len};
    if (!validate_opaque(ret.opaque)) {
      ret.valid = 0;
      return ret;
    }
    ret.valid |= (1 << static_cast<int>(URI_chunks::OPAQUE));
  }

  return ret;
}

// A URI is broken into parts or chunks. There are optional chunks and required chunks. A simple URI
// such as `https://www.nvidia.com` is easy to reason about, but it could also be written as
// `www.nvidia.com`, which is still valid. On top of that, there are characters which are allowed in
// certain chunks that are not allowed in others. There have been a multitude of methods attempted
// to get this correct, but at the end of the day, we have to validate the URI completely. This
// means even the simplest task of pulling off every character before the : still requires
// understanding how to validate an ipv6 address. This kernel was originally conceived as a two-pass
// kernel that ran the same code and either filled in offsets or filled in actual data. The problem
// is that to know what characters you need to copy, you need to have parsed the entire string as a
// 2 meg string could have `:/a` at the very end and everything up to that point is protocol or it
// could end in `.com` and now it is a hostname. To prevent the code from parsing it completely for
// length and then parsing it completely to copy the data, we will store off the offset of the
// string of question. The length is already stored in the offset column, so we then have a pointer
// and a number of bytes to copy and the second pass boils down to a series of memcpy calls.

/**
 * @brief Count the number of characters of each string after parsing the protocol.
 *
 * @param in_strings Input string column
 * @param chunk Chunk of URI to return
 * @param base_ptr Pointer to the start of the character data in the strings column
 * @param out_lengths Number of characters in each decode URL
 * @param out_offsets Offsets to the start of the chunks
 * @param out_validity Bitmask of validity data, updated in function
 */
__global__ void parse_uri_char_counter(column_device_view const in_strings,
                                       URI_chunks chunk,
                                       char const* const base_ptr,
                                       size_type* const out_lengths,
                                       size_type* const out_offsets,
                                       bitmask_type* out_validity,
                                       thrust::optional<column_device_view const> query_match)
{
  // thread per row
  auto const tid = cudf::detail::grid_1d::global_thread_id();

  for (thread_index_type tidx = tid; tidx < in_strings.size();
       tidx += cudf::detail::grid_1d::grid_stride()) {
    auto const row_idx = static_cast<size_type>(tidx);
    if (in_strings.is_null(row_idx)) {
      out_lengths[row_idx] = 0;
      continue;
    }

    auto const in_string     = in_strings.element<string_view>(row_idx);
    auto const in_chars      = in_string.data();
    auto const string_length = in_string.size_bytes();

    auto const uri = validate_uri(in_chars, string_length, query_match, row_idx);
    if ((uri.valid & (1 << static_cast<int>(chunk))) == 0) {
      out_lengths[row_idx] = 0;
      clear_bit(out_validity, row_idx);
    } else {
      // stash output offsets and lengths for next kernel to do the copy
      switch (chunk) {
        case URI_chunks::PROTOCOL:
          out_lengths[row_idx] = uri.scheme.size_bytes();
          out_offsets[row_idx] = uri.scheme.data() - base_ptr;
          break;
        case URI_chunks::HOST:
          out_lengths[row_idx] = uri.host.size_bytes();
          out_offsets[row_idx] = uri.host.data() - base_ptr;
          break;
        case URI_chunks::AUTHORITY:
          out_lengths[row_idx] = uri.authority.size_bytes();
          out_offsets[row_idx] = uri.authority.data() - base_ptr;
          break;
        case URI_chunks::PATH:
          out_lengths[row_idx] = uri.path.size_bytes();
          out_offsets[row_idx] = uri.path.data() - base_ptr;
          break;
        case URI_chunks::QUERY:
          out_lengths[row_idx] = uri.query.size_bytes();
          out_offsets[row_idx] = uri.query.data() - base_ptr;
          break;
        case URI_chunks::USERINFO:
          out_lengths[row_idx] = uri.userinfo.size_bytes();
          out_offsets[row_idx] = uri.userinfo.data() - base_ptr;
          break;
        case URI_chunks::PORT:
          out_lengths[row_idx] = uri.port.size_bytes();
          out_offsets[row_idx] = uri.port.data() - base_ptr;
          break;
        case URI_chunks::FRAGMENT:
          out_lengths[row_idx] = uri.fragment.size_bytes();
          out_offsets[row_idx] = uri.fragment.data() - base_ptr;
          break;
        case URI_chunks::OPAQUE:
          out_lengths[row_idx] = uri.opaque.size_bytes();
          out_offsets[row_idx] = uri.opaque.data() - base_ptr;
          break;
      }
    }
  }
}

/**
 * @brief Parse protocol and copy from the input string column to the output char buffer.
 *
 * @param in_strings Input string column
 * @param base_ptr Pointer to the start of the character data in the strings column
 * @param src_offsets Offset value of source strings in in_strings
 * @param offsets Offset value of each string associated with `out_chars`
 * @param out_chars Character buffer for the output string column
 */
__global__ void parse_uri(column_device_view const in_strings,
                          char const* const base_ptr,
                          size_type const* const src_offsets,
                          size_type const* const offsets,
                          char* const out_chars)
{
  auto const tid = cudf::detail::grid_1d::global_thread_id();

  for (thread_index_type tidx = tid; tidx < in_strings.size();
       tidx += cudf::detail::grid_1d::grid_stride()) {
    auto const row_idx = static_cast<size_type>(tidx);
    auto const len     = offsets[row_idx + 1] - offsets[row_idx];

    if (len > 0) {
      for (int i = 0; i < len; i++) {
        out_chars[offsets[row_idx] + i] = base_ptr[src_offsets[row_idx] + i];
      }
    }
  }
}

}  // namespace

std::unique_ptr<column> parse_uri(strings_column_view const& input,
                                  URI_chunks chunk,
                                  std::optional<strings_column_view const> query_match,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = input.size();
  if (strings_count == 0) { return make_empty_column(type_id::STRING); }

  constexpr size_type num_warps_per_threadblock = 4;
  constexpr size_type threadblock_size = num_warps_per_threadblock * cudf::detail::warp_size;
  auto const num_threadblocks =
    std::min(65536, cudf::util::div_rounding_up_unsafe(strings_count, num_warps_per_threadblock));

  auto offset_count    = strings_count + 1;
  auto const d_strings = column_device_view::create(input.parent(), stream);
  auto const d_matches =
    query_match ? column_device_view::create(query_match->parent(), stream)
                : std::unique_ptr<column_device_view, std::function<void(column_device_view*)>>{};

  // build offsets column
  auto offsets_column = make_numeric_column(
    data_type{type_to_id<size_type>()}, offset_count, mask_state::UNALLOCATED, stream, mr);

  // build src offsets buffer
  auto src_offsets = rmm::device_uvector<size_type>(strings_count, stream);

  // copy null mask
  rmm::device_buffer null_mask =
    input.parent().nullable()
      ? cudf::detail::copy_bitmask(input.parent(), stream, mr)
      : cudf::detail::create_null_mask(input.size(), mask_state::ALL_VALID, stream, mr);

  // count number of bytes in each string after parsing and store it in offsets_column
  auto offsets_view         = offsets_column->view();
  auto offsets_mutable_view = offsets_column->mutable_view();
  parse_uri_char_counter<<<num_threadblocks, threadblock_size, 0, stream.value()>>>(
    *d_strings,
    chunk,
    input.chars_begin(stream),
    offsets_mutable_view.begin<size_type>(),
    reinterpret_cast<size_type*>(src_offsets.data()),
    reinterpret_cast<bitmask_type*>(null_mask.data()),
    d_matches ? thrust::optional<column_device_view const>{*d_matches} : thrust::nullopt);

  // use scan to transform number of bytes into offsets
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         offsets_view.begin<size_type>(),
                         offsets_view.end<size_type>(),
                         offsets_mutable_view.begin<size_type>());

  // copy the total number of characters of all strings combined (last element of the offset column)
  // to the host memory
  auto out_chars_bytes = cudf::detail::get_value<size_type>(offsets_view, offset_count - 1, stream);

  // create the chars buffer
  auto d_out_chars = rmm::device_buffer(out_chars_bytes, stream, mr);

  // copy the characters from the input column to the output column
  parse_uri<<<num_threadblocks, threadblock_size, 0, stream.value()>>>(
    *d_strings,
    input.chars_begin(stream),
    reinterpret_cast<size_type*>(src_offsets.data()),
    offsets_column->view().begin<size_type>(),
    static_cast<char*>(d_out_chars.data()));

  auto null_count =
    cudf::null_count(reinterpret_cast<bitmask_type*>(null_mask.data()), 0, strings_count);

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(d_out_chars),
                             null_count,
                             std::move(null_mask));
}

}  // namespace detail

// external API

std::unique_ptr<column> parse_uri_to_protocol(strings_column_view const& input,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::parse_uri(input, detail::URI_chunks::PROTOCOL, std::nullopt, stream, mr);
}

std::unique_ptr<column> parse_uri_to_host(strings_column_view const& input,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::parse_uri(input, detail::URI_chunks::HOST, std::nullopt, stream, mr);
}

std::unique_ptr<column> parse_uri_to_query(strings_column_view const& input,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::parse_uri(input, detail::URI_chunks::QUERY, std::nullopt, stream, mr);
}

std::unique_ptr<cudf::column> parse_uri_to_query(cudf::strings_column_view const& input,
                                                 std::string const& query_match,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  // build string_column_view from incoming query_match string
  auto d_scalar = make_string_scalar(query_match, stream);
  auto col      = make_column_from_scalar(*d_scalar, 1);

  return detail::parse_uri(input, detail::URI_chunks::QUERY, strings_column_view(*col), stream, mr);
}

std::unique_ptr<cudf::column> parse_uri_to_query(cudf::strings_column_view const& input,
                                                 cudf::strings_column_view const& query_match,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(input.size() == query_match.size(), "Query column must be the same size as input!");

  return detail::parse_uri(input, detail::URI_chunks::QUERY, query_match, stream, mr);
}

std::unique_ptr<column> parse_uri_to_path(strings_column_view const& input,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::parse_uri(input, detail::URI_chunks::PATH, std::nullopt, stream, mr);
}
}  // namespace spark_rapids_jni
