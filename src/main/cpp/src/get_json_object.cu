#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"
#include "json_parser.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>

#include <thrust/pair.h>
#include <thrust/tuple.h>

namespace spark_rapids_jni {

namespace detail {

/**
 * write JSON style
 */
enum class write_style { RAW, QUOTED, FLATTEN };

/**
 * path instruction
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  path_instruction_type type;

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};
};

/**
 * JSON generator is used to write out JSON content.
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(char* _output) : output(_output), output_len(0) {}
  __device__ json_generator() : output(nullptr), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator()
  {
    if (nullptr == output) {
      return json_generator();
    } else {
      return json_generator(output + output_len);
    }
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array()
  {
    try_write_comma();

    if (output) { *(output + output_len) = '['; }

    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array()
  {
    if (output) { *(output + output_len) = ']'; }
    output_len++;

    // point to parent array
    array_depth--;

    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma()
  {
    if (need_comma()) {
      // in array context and writes first item
      if (output) { *(output + output_len) = ','; }
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser)
  {
    // first try add comma
    try_write_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      auto copy_to       = output + output_len;
      auto [b, copy_len] = parser.copy_current_structure(copy_to);
      output_len += copy_len;
      return b;
    } else {
      char* copy_to      = nullptr;
      auto [b, copy_len] = parser.copy_current_structure(copy_to);
      output_len += copy_len;
      return b;
    }
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      auto copied = parser.write_unescaped_text(output + output_len);
      output_len += copied;
    } else {
      auto len = parser.compute_unescaped_len();
      output_len += len;
    }
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        size_t child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      if (write_outer_array_tokens) {
        if (insert_comma) {
          *(child_block_begin + child_block_len + 2) = ']';
          move_forward(child_block_begin, child_block_len, 2);
          *(child_block_begin + 1) = '[';
          *(child_block_begin)     = ',';
        } else {
          *(child_block_begin + child_block_len + 1) = ']';
          move_forward(child_block_begin, child_block_len, 1);
          *(child_block_begin) = '[';
        }
      } else {
        if (insert_comma) {
          move_forward(child_block_begin, child_block_len, 1);
          *(child_block_begin) = ',';
        } else {
          // do not need comma && do not need write outer array tokens
          // do nothing, because child generator buff is directly after the
          // parent generator
        }
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ void reset() { output_len = 0; }

  __device__ inline size_t get_output_len() const { return output_len; }
  __device__ inline char* get_output_start_position() const { return output; }
  __device__ inline char* get_current_output_position() const { return output + output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  char* output;
  size_t output_len;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
  int array_depth = 0;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

/**
 *
 * This function is rewritten from above commented recursive function.
 * this function is equivalent to the above commented recursive function.
 */
__device__ bool evaluate_path(json_parser& p,
                              json_generator& root_g,
                              write_style root_style,
                              cudf::device_span<path_instruction const> root_path)
{
  // manually maintained context stack in lieu of calling evaluate_path recursively.
  struct context {
    // current token
    json_token token;

    // which case path that this task is from
    int case_path;

    // used to save current generator
    json_generator g;

    write_style style;

    cudf::device_span<path_instruction const> path;

    // is this context task is done
    bool task_is_done;

    // whether written output
    // if dirty > 0, indicates success
    int dirty;

    // for some case paths
    bool is_first_enter;

    // used to save child JSON generator for case path 8
    json_generator child_g;
  };

  // path max depth limitation
  // There is a same constant in JSONUtil.java, keep them consistent when changing
  // Note: Spark-Rapids should guarantee the path depth is less or equal to this limit,
  // or GPU reports hipErrorIllegalAddress
  constexpr int max_path_depth = 8;

  // define stack; plus 1 indicates root context task needs an extra memory
  context stack[max_path_depth + 1];
  int stack_pos = 0;

  // push context function
  auto push_context = [&stack, &stack_pos](json_token _token,
                                           int _case_path,
                                           json_generator _g,
                                           write_style _style,
                                           cudf::device_span<path_instruction const> _path) {
    // no need to check stack is full
    // because Spark-Rapids already checked maximum length of `path_instruction`
    auto& ctx          = stack[stack_pos];
    ctx.token          = _token;
    ctx.case_path      = _case_path;
    ctx.g              = _g;
    ctx.style          = _style;
    ctx.path           = _path;
    ctx.task_is_done   = false;
    ctx.dirty          = 0;
    ctx.is_first_enter = true;

    stack_pos++;
  };

  // put the first context task
  push_context(p.get_current_token(), -1, root_g, root_style, root_path);

  while (stack_pos > 0) {
    auto& ctx = stack[stack_pos - 1];
    if (!ctx.task_is_done) {
      // task is not done.

      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size()) &&
          ctx.style == write_style::RAW) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.write_raw(p);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          // push back task
          // add child task
          push_context(p.get_current_token(), 2, ctx.g, ctx.style, {nullptr, 0});
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // general case: just copy the child tree verbatim
        if (!(ctx.g.copy_current_structure(p))) {
          // JSON validation check
          return false;
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return false;
              }
            }
            ctx.task_is_done = true;
          } else {
            return false;
          }
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return false; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return false; }
              // push sub task; sub task will update the result of path 4
              push_context(p.get_current_token(),
                           4,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1});
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return false;
              }
            }
          }
          if (!found_expected_child) {
            // did not find any expected sub child
            ctx.task_is_done = true;
            ctx.dirty        = false;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: Wildcard :: xs)
      // case path 5
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_elements(
                 ctx.path, path_instruction_type::WILDCARD, path_instruction_type::WILDCARD)) {
        // special handling for the non-structure preserving double wildcard
        // behavior in Hive
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array();
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          push_context(p.get_current_token(),
                       5,
                       ctx.g,
                       write_style::FLATTEN,
                       {ctx.path.data() + 2, ctx.path.size() - 2});
        } else {
          ctx.g.write_end_array();
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
      // case path 6
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD) &&
               ctx.style != write_style::QUOTED) {
        // retain Flatten, otherwise use Quoted... cannot use Raw within an array
        write_style next_style = write_style::RAW;
        switch (ctx.style) {
          case write_style::RAW: next_style = write_style::QUOTED; break;
          case write_style::FLATTEN: next_style = write_style::FLATTEN; break;
          case write_style::QUOTED: next_style = write_style::QUOTED;  // never happen
        }

        // temporarily buffer child matches, the emitted json will need to be
        // modified slightly if there is only a single element written

        json_generator child_g;
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          // create a child generator with hide outer array tokens mode.
          child_g = ctx.g.new_child_generator();
          // write first [ without output, without update len, only update internal state
          child_g.write_first_start_array_without_output();
        } else {
          child_g = ctx.child_g;
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          // track the number of array elements and only emit an outer array if
          // we've written more than one element, this matches Hive's behavior
          push_context(p.get_current_token(),
                       6,
                       child_g,
                       next_style,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          char* child_g_start = child_g.get_output_start_position();
          size_t child_g_len  = child_g.get_output_len();
          if (ctx.dirty > 1) {
            // add outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ true);
            ctx.task_is_done = true;
          } else if (ctx.dirty == 1) {
            // remove outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ false);
            ctx.task_is_done = true;
          }  // else do not write anything
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array();
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(p.get_current_token(),
                       7,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          ctx.g.write_end_array();
          ctx.task_is_done = true;
        }
      }
      /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
      // case path 8
      else if (json_token::START_ARRAY == ctx.token &&
               thrust::get<0>(path_match_index_wildcard(ctx.path))) {
        int idx = thrust::get<1>(path_match_index_wildcard(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }
        ctx.is_first_enter = false;

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return false;
          }

          if (!p.try_skip_children()) { return false; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          --i;
        }

        // i == 0
        push_context(p.get_current_token(),
                     8,
                     ctx.g,
                     write_style::QUOTED,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case (START_ARRAY, Index(idx) :: xs)
      // case path 9
      else if (json_token::START_ARRAY == ctx.token && thrust::get<0>(path_match_index(ctx.path))) {
        int idx = thrust::get<1>(path_match_index(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return false;
          }

          if (!p.try_skip_children()) { return false; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          --i;
        }

        // i == 0
        push_context(
          p.get_current_token(), 9, ctx.g, ctx.style, {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case _ =>
      // case path 12
      else {
        if (!p.try_skip_children()) { return false; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    } else {
      // current context is done.

      // pop current top context
      stack_pos--;

      // pop parent task
      // update parent task info according to current task result
      if (stack_pos > 0) {
        // peek parent context task
        auto& p_ctx = stack[stack_pos - 1];

        // case (VALUE_STRING, Nil) if style == RawStyle
        // case path 1
        if (1 == ctx.case_path) {
          // never happen
        }
        // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
        // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
        // path 7: case (START_ARRAY, Wildcard :: xs)
        else if (2 == ctx.case_path || 5 == ctx.case_path || 7 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case (START_OBJECT, Named :: xs)
        // case path 4
        else if (4 == ctx.case_path) {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
        // case path 6
        else if (6 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;
        }
        /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
        // case path 8
        // case (START_ARRAY, Index(idx) :: xs)
        // case path 9
        else if (8 == ctx.case_path || 9 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return false; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return false; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case path 3: case (_, Nil)
        // case path 12: case _ =>
        // others
        else {
          // never happen
        }
      } else {
        // has no parent task, stack is empty, will exit
      }
    }
  }

  // copy output len
  root_g.set_output_len(stack[0].g.get_output_len());
  return stack[0].dirty > 0;
}

rmm::device_uvector<path_instruction> construct_path_commands(
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  cudf::string_scalar const& all_names_scalar,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  int name_pos = 0;

  // construct the path commands
  std::vector<path_instruction> path_commands;
  for (auto const& inst : instructions) {
    auto const& [type, name, index] = inst;
    switch (type) {
      case path_instruction_type::SUBSCRIPT:
      case path_instruction_type::KEY:
        // skip SUBSCRIPT and KEY to save stack size in `evaluate_path`
        break;
      case path_instruction_type::WILDCARD:
        path_commands.emplace_back(path_instruction{path_instruction_type::WILDCARD});
        break;
      case path_instruction_type::INDEX:
        path_commands.emplace_back(path_instruction{path_instruction_type::INDEX});
        path_commands.back().index = index;
        break;
      case path_instruction_type::NAMED:
        path_commands.emplace_back(path_instruction{path_instruction_type::NAMED});
        path_commands.back().name =
          cudf::string_view(all_names_scalar.data() + name_pos, name.size());
        name_pos += name.size();
        break;
      default: CUDF_FAIL("Invalid path instruction type");
    }
  }
  // convert to uvector
  return cudf::detail::make_device_uvector_sync(path_commands, stream, mr);
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 *
 * @param input The incoming json string
 * @param input_len Size of the incoming json string
 * @param path_commands_ptr The command buffer to be applied to the string.
 * @param path_commands_size The command buffer size.
 * @param out_buf Buffer user to store the results of the query
 *                (nullptr in the size computation step)
 * @param out_buf_size Size of the output buffer
 * @returns A pair containing the result code and the output buffer.
 */
__device__ thrust::pair<bool, size_t> get_json_object_single(
  char const* input,
  cudf::size_type input_len,
  cudf::device_span<path_instruction const> path_commands,
  char* out_buf,
  size_t out_buf_size)
{
  json_parser j_parser(input, input_len);
  j_parser.next_token();
  // JSON validation check
  if (json_token::ERROR == j_parser.get_current_token()) { return {false, 0}; }

  // First pass: preprocess sizes.
  // Second pass: writes output.
  // The generator automatically determines which pass based on `out_buf`.
  // If `out_buf_size` is zero, pass in `nullptr` to avoid generator writing trash output.
  json_generator generator((out_buf_size == 0) ? nullptr : out_buf);

  bool const success = evaluate_path(
    j_parser, generator, write_style::RAW, {path_commands.data(), path_commands.size()});

  if (!success) {
    // generator may contain trash output, e.g.: generator writes some output,
    // then JSON format is invalid, the previous output becomes trash.
    // set output as zero to tell second step
    generator.set_output_len_zero();
  }

  return {success, generator.get_output_len()};
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel operates in a 2-pass way. On the first pass it computes the
 * output sizes. On the second pass, it fills in the provided output buffers
 * (chars and validity).
 *
 * @param col Device view of the incoming string
 * @param commands JSONPath command buffer
 * @param output_offsets Buffer used to store the string offsets for the results
 *        of the query
 * @param out_buf Buffer used to store the results of the query
 * @param out_validity Output validity buffer
 * @param out_valid_count Output count of # of valid bits
 * @param options Options controlling behavior
 */
template <int block_size>
__launch_bounds__(block_size) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view col,
                              cudf::device_span<path_instruction const> path_commands,
                              cudf::size_type* d_sizes,
                              cudf::detail::input_offsetalator output_offsets,
                              char* out_buf,
                              cudf::bitmask_type* out_validity,
                              cudf::size_type* out_valid_count)
{
  auto tid          = cudf::detail::grid_1d::global_thread_id();
  auto const stride = cudf::detail::grid_1d::grid_stride();

  cudf::size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffff'ffffu, tid < col.size());
  while (tid < col.size()) {
    bool is_valid               = false;
    cudf::string_view const str = col.element<cudf::string_view>(tid);
    if (str.size_bytes() > 0) {
      char* dst = out_buf != nullptr ? out_buf + output_offsets[tid] : nullptr;
      size_t const dst_size =
        out_buf != nullptr ? output_offsets[tid + 1] - output_offsets[tid] : 0;

      // process one single row
      auto [result, output_size] = get_json_object_single(
        str.data(), str.size_bytes(), {path_commands.data(), path_commands.size()}, dst, dst_size);
      if (result) { is_valid = true; }

      // filled in only during the precompute step. during the compute step, the
      // offsets are fed back in so we do -not- want to write them out
      if (out_buf == nullptr) { d_sizes[tid] = static_cast<cudf::size_type>(output_size); }
    } else {
      // valid JSON length is always greater than 0
      // if `str` size len is zero, output len is 0 and `is_valid` is false
      if (out_buf == nullptr) { d_sizes[tid] = 0; }
    }

    // validity filled in only during the output step
    if (out_validity != nullptr) {
      uint32_t mask = __ballot_sync(active_threads, is_valid);
      // 0th lane of the warp writes the validity
      if (!(tid % cudf::detail::warp_size)) {
        out_validity[cudf::word_index(tid)] = mask;
        warp_valid_count += __popc(mask);
      }
    }

    tid += stride;
    active_threads = __ballot_sync(active_threads, tid < col.size());
  }

  // sum the valid counts across the whole block
  if (out_valid_count != nullptr) {
    cudf::size_type block_valid_count =
      cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count, block_valid_count); }
  }
}

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return cudf::make_empty_column(cudf::type_id::STRING);

  // get a string buffer to store all the names and convert to device
  std::string all_names;
  for (auto const& inst : instructions) {
    all_names += std::get<1>(inst);
  }
  cudf::string_scalar all_names_scalar(all_names, true, stream);
  // parse the json_path into a command buffer
  auto path_commands = construct_path_commands(
    instructions, all_names_scalar, stream, rmm::mr::get_current_device_resource());

  // compute output sizes
  auto sizes = rmm::device_uvector<cudf::size_type>(
    input.size(), stream, rmm::mr::get_current_device_resource());
  auto d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(input.offsets());

  constexpr int block_size = 512;
  cudf::detail::grid_1d const grid{input.size(), block_size};
  auto d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  // preprocess sizes (returned in the offsets buffer)
  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *d_input_ptr, path_commands, sizes.data(), d_offsets, nullptr, nullptr, nullptr);

  // convert sizes to offsets
  auto [offsets, output_size] =
    cudf::strings::detail::make_offsets_child_column(sizes.begin(), sizes.end(), stream, mr);
  d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  // allocate output string column
  rmm::device_uvector<char> chars(output_size, stream, mr);

  // potential optimization : if we know that all outputs are valid, we could
  // skip creating the validity mask altogether
  rmm::device_buffer validity =
    cudf::detail::create_null_mask(input.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  // compute results
  rmm::device_scalar<cudf::size_type> d_valid_count{0, stream};

  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *d_input_ptr,
      path_commands,
      sizes.data(),
      d_offsets,
      chars.data(),
      static_cast<cudf::bitmask_type*>(validity.data()),
      d_valid_count.data());

  return make_strings_column(input.size(),
                             std::move(offsets),
                             chars.release(),
                             input.size() - d_valid_count.value(stream),
                             std::move(validity));
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return detail::get_json_object(input, instructions, stream, mr);
}

}  // namespace spark_rapids_jni
