#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"
#include "json_parser.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

namespace spark_rapids_jni {

namespace detail {

// path max depth limitation
// There is a same constant in JSONUtil.java, keep them consistent when changing
// Note: Spark-Rapids should guarantee the path depth is less or equal to this limit,
// or GPU reports hipErrorIllegalAddress
constexpr int max_path_depth = 16;

/**
 * @brief JSON style to write.
 */
enum class write_style : int8_t { RAW, QUOTED, FLATTEN };

/**
 * @brief Instruction along a JSON path.
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};

  path_instruction_type type;
};

/**
 * @brief JSON generator used to write out JSON content.
 *
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(int _offset = 0) : offset(_offset), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator() const
  {
    return json_generator(offset + output_len);
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array(char* out_begin)
  {
    try_write_comma(out_begin);

    out_begin[offset + output_len] = '[';
    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array(char* out_begin)
  {
    out_begin[offset + output_len] = ']';
    output_len++;
    // point to parent array
    array_depth--;
    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() const { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma(char* out_begin)
  {
    if (need_comma()) {
      // in array context and writes first item
      out_begin[offset + output_len] = ',';
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser, char* out_begin)
  {
    // first try add comma
    try_write_comma(out_begin);

    if (array_depth > 0) { is_curr_array_empty = false; }

    auto [b, copy_len] = parser.copy_current_structure(out_begin + offset + output_len);
    output_len += copy_len;
    return b;
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser, char* out_begin)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    auto copied = parser.write_unescaped_text(out_begin + offset + output_len);
    output_len += copied;
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        int child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (write_outer_array_tokens) {
      if (insert_comma) {
        *(child_block_begin + child_block_len + 2) = ']';
        move_forward(child_block_begin, child_block_len, 2);
        *(child_block_begin + 1) = '[';
        *(child_block_begin)     = ',';
      } else {
        *(child_block_begin + child_block_len + 1) = ']';
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = '[';
      }
    } else {
      if (insert_comma) {
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = ',';
      } else {
        // do not need comma && do not need write outer array tokens
        // do nothing, because child generator buff is directly after the
        // parent generator
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  static __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ inline int get_offset() const { return offset; }
  __device__ inline int get_output_len() const { return output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  int offset;  // offset from the global output buffer
  int output_len;

  int array_depth = 0;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

/**
 * @brief The cases that mirro Apache Spark case path in `jsonExpressions.scala#evaluatePath()`.
 */
enum class evaluation_case_path : int8_t {
  INVALID                                           = -1,
  START_ARRAY___EMPTY_PATH___FLATTEN_STYLE          = 2,
  START_OBJECT___MATCHED_NAME_PATH                  = 4,
  START_ARRAY___MATCHED_DOUBLE_WILDCARD             = 5,
  START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED = 6,
  START_ARRAY___MATCHED_WILDCARD                    = 7,
  START_ARRAY___MATCHED_INDEX_AND_WILDCARD          = 8,
  START_ARRAY___MATCHED_INDEX                       = 9
};

/**
 * @brief The struct to store states during processing JSON through different nested levels.
 */
struct context {
  // used to save current generator
  json_generator g;

  // used to save child JSON generator for case path 6
  json_generator child_g;

  cudf::device_span<path_instruction const> path;

  // whether written output
  // if dirty > 0, indicates success
  int dirty;

  // which case path that this task is from
  evaluation_case_path case_path;

  // current token
  json_token token;

  write_style style;

  // for some case paths
  bool is_first_enter;

  // is this context task is done
  bool task_is_done;
};

/**
 * @brief Parse a single json string using the provided command buffer.
 *
 * @param input The incoming json string
 * @param path_commands The command buffer to be applied to the string
 * @param out_buf Buffer user to store the string resulted from the query
 * @return A pair containing the result code and the output size
 */
__device__ thrust::pair<bool, cudf::size_type> evaluate_path(
  char_range input, cudf::device_span<path_instruction const> path_commands, char* out_buf)
{
  json_parser p{input};
  p.next_token();
  if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

  // define stack; plus 1 indicates root context task needs an extra memory
  context stack[max_path_depth + 1];
  int stack_size = 0;

  // push context function
  auto push_context = [&p, &stack, &stack_size](evaluation_case_path _case_path,
                                                json_generator _g,
                                                write_style _style,
                                                cudf::device_span<path_instruction const> _path) {
    // no need to check stack is full
    // because Spark-Rapids already checked maximum length of `path_instruction`
    auto& ctx          = stack[stack_size++];
    ctx.g              = std::move(_g);
    ctx.path           = std::move(_path);
    ctx.dirty          = 0;
    ctx.case_path      = _case_path;
    ctx.token          = p.get_current_token();
    ctx.style          = _style;
    ctx.is_first_enter = true;
    ctx.task_is_done   = false;
  };

  // put the first context task
  push_context(evaluation_case_path::INVALID, json_generator{}, write_style::RAW, path_commands);

  while (stack_size > 0) {
    auto& ctx = stack[stack_size - 1];
    if (!ctx.task_is_done) {
      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size()) &&
          ctx.style == write_style::RAW) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.write_raw(p, out_buf);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // push back task
          // add child task
          push_context(evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE,
                       ctx.g,
                       ctx.style,
                       {nullptr, 0});
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // general case: just copy the child tree verbatim
        if (!(ctx.g.copy_current_structure(p, out_buf))) {
          // JSON validation check
          return {false, 0};
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          // Mark task is done regardless whether the expected child was found.
          ctx.task_is_done = true;
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return {false, 0}; }
              // push sub task; sub task will update the result of path 4
              push_context(evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1});
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          if (!found_expected_child) {
            // did not find any expected sub child
            ctx.task_is_done = true;
            ctx.dirty        = false;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: Wildcard :: xs)
      // case path 5
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_elements(
                 ctx.path, path_instruction_type::WILDCARD, path_instruction_type::WILDCARD)) {
        // special handling for the non-structure preserving double wildcard
        // behavior in Hive
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          push_context(evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD,
                       ctx.g,
                       write_style::FLATTEN,
                       {ctx.path.data() + 2, ctx.path.size() - 2});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
      // case path 6
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD) &&
               ctx.style != write_style::QUOTED) {
        // retain Flatten, otherwise use Quoted... cannot use Raw within an array
        write_style next_style = write_style::RAW;
        switch (ctx.style) {
          case write_style::RAW: next_style = write_style::QUOTED; break;
          case write_style::FLATTEN: next_style = write_style::FLATTEN; break;
          case write_style::QUOTED: next_style = write_style::QUOTED;  // never happen
        }

        // temporarily buffer child matches, the emitted json will need to be
        // modified slightly if there is only a single element written

        json_generator child_g;
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          // create a child generator with hide outer array tokens mode.
          child_g = ctx.g.new_child_generator();
          // write first [ without output, without update len, only update internal state
          child_g.write_first_start_array_without_output();
        } else {
          child_g = ctx.child_g;
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // track the number of array elements and only emit an outer array if
          // we've written more than one element, this matches Hive's behavior
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED,
                       child_g,
                       next_style,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          char* child_g_start = out_buf + child_g.get_offset();
          int child_g_len     = child_g.get_output_len();
          if (ctx.dirty > 1) {
            // add outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ true);
          } else if (ctx.dirty == 1) {
            // remove outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ false);
          }  // else do not write anything

          // Done anyway, since we already reached the end array.
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
      // case path 8
      else if (json_token::START_ARRAY == ctx.token &&
               thrust::get<0>(path_match_index_wildcard(ctx.path))) {
        int idx = thrust::get<1>(path_match_index_wildcard(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
        ctx.is_first_enter = false;

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD,
                     ctx.g,
                     write_style::QUOTED,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case (START_ARRAY, Index(idx) :: xs)
      // case path 9
      else if (json_token::START_ARRAY == ctx.token && thrust::get<0>(path_match_index(ctx.path))) {
        int idx = thrust::get<1>(path_match_index(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX,
                     ctx.g,
                     ctx.style,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case _ =>
      // case path 12
      else {
        if (!p.try_skip_children()) { return {false, 0}; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    }       // if (!ctx.task_is_done)
    else {  // current context is done.
      // pop current top context
      stack_size--;

      // has no parent task, stack is empty, will exit
      if (stack_size == 0) { break; }

      // peek parent context task
      // update parent task info according to current task result
      auto& p_ctx = stack[stack_size - 1];

      switch (ctx.case_path) {
          // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
          // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
          // path 7: case (START_ARRAY, Wildcard :: xs)
        case evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE:
        case evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_OBJECT, Named :: xs)
          // case path 4
        case evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH: {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
          // case path 6
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;

          break;
        }

          /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
          // case path 8
          // case (START_ARRAY, Index(idx) :: xs)
          // case path 9
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return {false, 0}; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

        default:;  // Never happens!
      }            // end switch (ctx.case_path)
    }              // ctx.task_is_done
  }                // while (stack_size > 0)

  auto const success = stack[0].dirty > 0;

  // generator may contain trash output, e.g.: generator writes some output,
  // then JSON format is invalid, the previous output becomes trash.
  // We need to return output size as zero.
  return {success, success ? stack[0].g.get_output_len() : 0};
}

/**
 * @brief Struct storing data such as path instructions, output buffer etc, corresponding to a
 * single JSON path.
 */
struct json_path_processing_data {
  cudf::device_span<path_instruction const> path_commands;
  cudf::detail::input_offsetalator offsets;
  thrust::pair<char const*, cudf::size_type>* out_stringviews;
  char* out_buf;
  int8_t* has_out_of_bound;
};

/**
 * @brief Extract JSON object, from one input row, on one path.
 *
 * @param input The entire input strings column
 * @param path_data Array containing all path data
 * @param row_path_idx The index to identify row index and path index
 */
__device__ void process_row_path(cudf::column_device_view input,
                                 cudf::device_span<json_path_processing_data> path_data,
                                 int64_t row_path_idx)
{
  auto const row_idx  = row_path_idx / path_data.size();
  auto const path_idx = row_path_idx % path_data.size();
  auto const& path    = path_data[path_idx];

  char* const dst          = path.out_buf + path.offsets[row_idx];
  bool is_valid            = false;
  cudf::size_type out_size = 0;

  auto const str = input.element<cudf::string_view>(row_idx);
  if (str.size_bytes() > 0) {
    thrust::tie(is_valid, out_size) = evaluate_path(char_range{str}, path.path_commands, dst);

    auto const max_size = path.offsets[row_idx + 1] - path.offsets[row_idx];
    if (out_size > max_size) { *(path.has_out_of_bound) = 1; }
  }

  // Write out `nullptr` in the output string_view to indicate that the output is a null.
  // The situation `out_stringviews == nullptr` should only happen if the kernel is launched a
  // second time due to out-of-bound write in the first launch.
  if (path.out_stringviews) {
    path.out_stringviews[row_idx] = {is_valid ? dst : nullptr, out_size};
  }
}

/**
 * @brief Kernel for running the JSONPath query, using all threads for processing.
 *
 * This kernel writes out the output strings and their lengths at the same time. If any output
 * length exceed buffer size limit, a boolean flag will be turned on to inform to the caller.
 * In such situation, another (larger) output buffer will be generated and the kernel is launched
 * again. Otherwise, launching this kernel only once is sufficient to produce the desired output.
 *
 * @param input The input JSON strings stored in a strings column
 * @param path_data Array containing all path data
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel_thread_parallel(
    cudf::column_device_view input, cudf::device_span<json_path_processing_data> path_data)
{
  auto const max_tid = static_cast<int64_t>(input.size()) * path_data.size();
  auto const stride  = cudf::detail::grid_1d::grid_stride();

  for (auto tid = cudf::detail::grid_1d::global_thread_id(); tid < max_tid; tid += stride) {
    process_row_path(input, path_data, tid);
  }
}

/**
 * @brief Kernel for running the JSONPath query, using one warp to process a row.
 *
 * The behavior of this kernel should be identical to that of the version using all threads for
 * processing.
 *
 * @param input The input JSON strings stored in a strings column
 * @param path_data Array containing all path data
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel_warp_parallel(cudf::column_device_view input,
                                            cudf::device_span<json_path_processing_data> path_data)
{
  auto const max_tid =
    static_cast<int64_t>(input.size()) * path_data.size() * cudf::detail::warp_size;
  auto const stride  = cudf::detail::grid_1d::grid_stride();
  auto const lane_id = threadIdx.x % cudf::detail::warp_size;

  for (auto tid = cudf::detail::grid_1d::global_thread_id(); tid < max_tid; tid += stride) {
    if (lane_id == 0) {
      auto const warp_idx = tid / cudf::detail::warp_size;
      process_row_path(input, path_data, warp_idx);
    }
    __syncwarp();
  }
}

/**
 * @brief Launch the main kernel.
 *
 * Either a thread-parallel or warp-parallel kernel is launched, depending on the value of
 * `exec_thread_parallel`.
 */
void launch_kernel(bool exec_thread_parallel,
                   cudf::column_device_view const& input,
                   cudf::device_span<json_path_processing_data> path_data,
                   rmm::cuda_stream_view stream)
{
  // We explicitly set the minBlocksPerMultiprocessor parameter in the launch bounds to avoid
  // spilling from the kernel itself. By default NVCC uses a heuristic to find a balance between
  // the maximum number of registers used by a kernel and the parallelism of the kernel.
  // If lots of registers are used the parallelism may suffer. But in our case
  // NVCC gets this wrong and we want to avoid spilling all the time or else
  // the performance is really bad. This essentially tells NVCC to prefer using lots
  // of registers over spilling.
  if (exec_thread_parallel) {
    constexpr int block_size       = 256;
    constexpr int min_block_per_sm = 1;
    auto const num_blocks =
      cudf::util::div_rounding_up_safe(static_cast<std::size_t>(input.size()) * path_data.size(),
                                       static_cast<std::size_t>(block_size));

    get_json_object_kernel_thread_parallel<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(input, path_data);
  } else {
    // The optimal values for block_size and min_block_per_sm were found through testing,
    // which are 128-8 or 256-4.
    constexpr int block_size       = 128;
    constexpr int min_block_per_sm = 8;
    auto const num_blocks          = cudf::util::div_rounding_up_safe(
      static_cast<std::size_t>(input.size()) * path_data.size() * cudf::detail::warp_size,
      static_cast<std::size_t>(block_size));

    get_json_object_kernel_warp_parallel<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(input, path_data);
  }
}

/**
 * @brief Construct the device vector containing necessary data for the input JSON paths.
 *
 * All JSON paths are processed at once, without stream synchronization, to minimize overhead.
 *
 * A tuple of values are returned, however, only the first element is needed for further kernel
 * launch. The remaining are unused but need to be kept alive as they contains data for later
 * asynchronous host-device memcpy.
 */
std::tuple<std::vector<rmm::device_uvector<path_instruction>>,
           std::unique_ptr<std::vector<std::vector<path_instruction>>>,
           cudf::string_scalar,
           std::string>
construct_path_commands(
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    json_paths,
  rmm::cuda_stream_view stream)
{
  // Concatenate all names from path instructions.
  auto h_inst_names = [&] {
    std::size_t length{0};
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { length += name.length(); }
      }
    }
    std::string all_names;
    all_names.reserve(length);
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { all_names += name; }
      }
    }
    return all_names;
  }();
  auto d_inst_names = cudf::string_scalar(h_inst_names, true, stream);

  std::size_t name_pos{0};
  auto h_path_commands = std::make_unique<std::vector<std::vector<path_instruction>>>();
  h_path_commands->reserve(json_paths.size());

  for (auto const& instructions : json_paths) {
    h_path_commands->emplace_back();
    auto& path_commands = h_path_commands->back();
    path_commands.reserve(instructions.size());

    for (auto const& [type, name, index] : instructions) {
      path_commands.emplace_back(path_instruction{type});

      if (type == path_instruction_type::INDEX) {
        path_commands.back().index = index;
      } else if (type == path_instruction_type::NAMED) {
        path_commands.back().name = cudf::string_view(d_inst_names.data() + name_pos, name.size());
        name_pos += name.size();
      } else if (type != path_instruction_type::WILDCARD) {
        CUDF_FAIL("Invalid path instruction type");
      }
    }
  }

  auto d_path_commands = std::vector<rmm::device_uvector<path_instruction>>{};
  d_path_commands.reserve(h_path_commands->size());
  for (auto const& path_commands : *h_path_commands) {
    d_path_commands.emplace_back(cudf::detail::make_device_uvector_async(
      path_commands, stream, rmm::mr::get_current_device_resource()));
  }

  return {std::move(d_path_commands),
          std::move(h_path_commands),
          std::move(d_inst_names),
          std::move(h_inst_names)};
}

std::vector<std::unique_ptr<cudf::column>> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    json_paths,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const num_outputs = json_paths.size();
  std::vector<std::unique_ptr<cudf::column>> output;

  // Input is empty or all nulls - just return all null columns.
  if (input.is_empty() || input.size() == input.null_count()) {
    for (std::size_t idx = 0; idx < num_outputs; ++idx) {
      output.emplace_back(std::make_unique<cudf::column>(input.parent(), stream, mr));
    }
    return output;
  }

  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  auto const in_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());
  auto const [d_json_paths, h_json_paths, d_inst_names, h_inst_names] =
    construct_path_commands(json_paths, stream);

  auto const [max_row_size, sum_row_size] =
    thrust::transform_reduce(rmm::exec_policy(stream),
                             thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(input.size()),
                             cuda::proclaim_return_type<thrust::pair<int64_t, int64_t>>(
                               [in_offsets] __device__(auto const idx) {
                                 auto const size = in_offsets[idx + 1] - in_offsets[idx];
                                 return thrust::pair<int64_t, int64_t>{size, size};
                               }),
                             thrust::pair<int64_t, int64_t>{0, 0},
                             cuda::proclaim_return_type<thrust::pair<int64_t, int64_t>>(
                               [] __device__(auto const& lhs, auto const& rhs) {
                                 return thrust::pair<int64_t, int64_t>{
                                   std::max(lhs.first, rhs.first), lhs.second + rhs.second};
                               }));

  // We will use scratch buffers to store the output strings without knowing their sizes.
  // Since we do not know their sizes, we need to allocate the buffer a bit larger than the input
  // size so that we will not write output strings into an out-of-bound position.
  // Checking out-of-bound needs to be performed in the main kernel to make sure we will not have
  // data corruption.
  auto const scratch_size = [&, max_row_size = max_row_size] {
    // Pad the scratch buffer by an additional size that is a multiple of max row size.
    auto constexpr padding_rows = 10;
    return input.chars_size(stream) + max_row_size * padding_rows;
  }();

  rmm::device_uvector<int8_t> d_has_out_of_bound(num_outputs, stream);
  std::vector<rmm::device_uvector<char>> scratch_buffers;
  std::vector<rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>> out_stringviews;
  std::vector<json_path_processing_data> h_path_data;
  scratch_buffers.reserve(json_paths.size());
  out_stringviews.reserve(json_paths.size());
  h_path_data.reserve(json_paths.size());

  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& instructions = json_paths[idx];
    if (instructions.size() > max_path_depth) { CUDF_FAIL("JSONPath query exceeds maximum depth"); }

    scratch_buffers.emplace_back(rmm::device_uvector<char>(scratch_size, stream));
    out_stringviews.emplace_back(rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{
      static_cast<std::size_t>(input.size()), stream});

    h_path_data.emplace_back(json_path_processing_data{d_json_paths[idx],
                                                       in_offsets,
                                                       out_stringviews.back().data(),
                                                       scratch_buffers.back().data(),
                                                       d_has_out_of_bound.data() + idx});
  }
  auto d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_has_out_of_bound.begin(), d_has_out_of_bound.end(), 0);

  // Threshold to decide on using thread parallel or warp parallel algorithms.
  constexpr int64_t AVG_CHAR_BYTES_THRESHOLD = 256;
  auto const exec_thread_parallel =
    (sum_row_size / (input.size() - input.null_count())) < AVG_CHAR_BYTES_THRESHOLD;
  launch_kernel(exec_thread_parallel, *d_input_ptr, d_path_data, stream);

  // Do not use parallel check since we do not have many elements.
  auto h_has_out_of_bound = cudf::detail::make_host_vector_sync(d_has_out_of_bound, stream);
  auto has_no_oob         = std::none_of(
    h_has_out_of_bound.begin(), h_has_out_of_bound.end(), [](auto const val) { return val != 0; });

  // If we didn't see any out-of-bound write, everything is good so far.
  // Just gather the output strings and return.
  if (has_no_oob) {
    for (auto const& out_sview : out_stringviews) {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
    return output;
  }
  // From here, we had out-of-bound write. Although this is very rare, it may still happen.

  std::vector<std::pair<rmm::device_buffer, cudf::size_type>> out_null_masks_and_null_counts;
  std::vector<std::pair<std::unique_ptr<cudf::column>, int64_t>> out_offsets_and_sizes;
  std::vector<rmm::device_uvector<char>> out_char_buffers;
  std::vector<std::size_t> oob_indices;

  // Check validity from the stored char pointers.
  auto const validator = [] __device__(thrust::pair<char const*, cudf::size_type> const item) {
    return item.first != nullptr;
  };

  // Rebuild the data only for paths that had out of bound write.
  h_path_data.clear();
  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& out_sview = out_stringviews[idx];

    if (h_has_out_of_bound[idx]) {
      oob_indices.emplace_back(idx);
      output.emplace_back(nullptr);  // just placeholder.

      out_null_masks_and_null_counts.emplace_back(
        cudf::detail::valid_if(out_sview.begin(), out_sview.end(), validator, stream, mr));

      // The string sizes computed in the previous kernel call will be used to allocate a new char
      // buffer to store the output.
      auto const size_it = cudf::detail::make_counting_transform_iterator(
        0,
        cuda::proclaim_return_type<cudf::size_type>(
          [string_pairs = out_sview.data()] __device__(auto const idx) {
            return string_pairs[idx].second;
          }));
      out_offsets_and_sizes.emplace_back(cudf::strings::detail::make_offsets_child_column(
        size_it, size_it + input.size(), stream, mr));
      out_char_buffers.emplace_back(
        rmm::device_uvector<char>(out_offsets_and_sizes.back().second, stream, mr));

      h_path_data.emplace_back(
        json_path_processing_data{d_json_paths[idx],
                                  cudf::detail::offsetalator_factory::make_input_iterator(
                                    out_offsets_and_sizes.back().first->view()),
                                  nullptr /*out_stringviews*/,
                                  out_char_buffers.back().data(),
                                  d_has_out_of_bound.data() + idx});
    } else {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
  }
  // These buffers are no longer needed.
  scratch_buffers.clear();
  out_stringviews.clear();

  // Push data to the GPU and launch the kernel again.
  d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_has_out_of_bound.begin(), d_has_out_of_bound.end(), 0);
  launch_kernel(exec_thread_parallel, *d_input_ptr, d_path_data, stream);

  // Check out of bound again to make sure everything looks right.
  h_has_out_of_bound = cudf::detail::make_host_vector_sync(d_has_out_of_bound, stream);
  has_no_oob         = std::none_of(
    h_has_out_of_bound.begin(), h_has_out_of_bound.end(), [](auto const val) { return val != 0; });

  // The last kernel call should not encounter any out-of-bound write.
  // If OOB is still detected, there must be something wrong happened.
  CUDF_EXPECTS(has_no_oob, "Unexpected out-of-bound write in get_json_object kernel.");

  for (auto const idx : oob_indices) {
    output[idx] = cudf::make_strings_column(input.size(),
                                            std::move(out_offsets_and_sizes[idx].first),
                                            out_char_buffers[idx].release(),
                                            out_null_masks_and_null_counts[idx].second,
                                            std::move(out_null_masks_and_null_counts[idx].first));
  }
  return output;
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  return std::move(detail::get_json_object(input, {instructions}, stream, mr).front());
}

std::vector<std::unique_ptr<cudf::column>> get_json_object_multiple_paths(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    json_paths,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  return detail::get_json_object(input, json_paths, stream, mr);
}

}  // namespace spark_rapids_jni
