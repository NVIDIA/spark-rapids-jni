#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"
#include "json_parser.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

#include <numeric>

namespace spark_rapids_jni {

namespace detail {

/**
 * @brief JSON style to write.
 */
enum class write_style : int8_t { RAW, QUOTED, FLATTEN };

/**
 * @brief Instruction along a JSON path.
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};

  path_instruction_type type;
};

/**
 * @brief JSON generator used to write out JSON content.
 *
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(int _offset = 0) : offset(_offset), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator() const
  {
    return json_generator(offset + output_len);
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array(char* out_begin)
  {
    try_write_comma(out_begin);

    out_begin[offset + output_len] = '[';
    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array(char* out_begin)
  {
    out_begin[offset + output_len] = ']';
    output_len++;
    // point to parent array
    array_depth--;
    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() const { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma(char* out_begin)
  {
    if (need_comma()) {
      // in array context and writes first item
      out_begin[offset + output_len] = ',';
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser, char* out_begin)
  {
    // first try add comma
    try_write_comma(out_begin);

    if (array_depth > 0) { is_curr_array_empty = false; }

    auto [b, copy_len] = parser.copy_current_structure(out_begin + offset + output_len);
    output_len += copy_len;
    return b;
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser, char* out_begin)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    auto copied = parser.write_unescaped_text(out_begin + offset + output_len);
    output_len += copied;
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        int child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (write_outer_array_tokens) {
      if (insert_comma) {
        *(child_block_begin + child_block_len + 2) = ']';
        move_forward(child_block_begin, child_block_len, 2);
        *(child_block_begin + 1) = '[';
        *(child_block_begin)     = ',';
      } else {
        *(child_block_begin + child_block_len + 1) = ']';
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = '[';
      }
    } else {
      if (insert_comma) {
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = ',';
      } else {
        // do not need comma && do not need write outer array tokens
        // do nothing, because child generator buff is directly after the
        // parent generator
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  static __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ inline int get_offset() const { return offset; }
  __device__ inline int get_output_len() const { return output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  int offset;  // offset from the global output buffer
  int output_len;

  int array_depth = 0;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

/**
 * @brief The cases that mirro Apache Spark case path in `jsonExpressions.scala#evaluatePath()`.
 */
enum class evaluation_case_path : int8_t {
  INVALID                                           = -1,
  START_ARRAY___EMPTY_PATH___FLATTEN_STYLE          = 2,
  START_OBJECT___MATCHED_NAME_PATH                  = 4,
  START_ARRAY___MATCHED_DOUBLE_WILDCARD             = 5,
  START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED = 6,
  START_ARRAY___MATCHED_WILDCARD                    = 7,
  START_ARRAY___MATCHED_INDEX_AND_WILDCARD          = 8,
  START_ARRAY___MATCHED_INDEX                       = 9
};

/**
 * @brief The struct to store states during processing JSON through different nested levels.
 */
struct context {
  // used to save current generator
  json_generator g;

  // used to save child JSON generator for case path 6
  json_generator child_g;

  cudf::device_span<path_instruction const> path;

  // whether written output
  // if dirty > 0, indicates success
  int dirty;

  // which case path that this task is from
  evaluation_case_path case_path;

  // current token
  json_token token;

  write_style style;

  // for some case paths
  bool is_first_enter;

  // is this context task is done
  bool task_is_done;
};

/**
 * @brief Parse a single json string using the provided command buffer.
 *
 * @param p The JSON parser for input string
 * @param path_commands The command buffer to be applied to the string
 * @param out_buf Buffer user to store the string resulted from the query
 * @param max_path_depth_exceeded A marker to record if the maximum path depth has been reached
 *        during parsing the input string
 * @return A pair containing the result code and the output size
 */
__device__ thrust::pair<bool, cudf::size_type> evaluate_path(
  json_parser& p,
  cudf::device_span<path_instruction const> path_commands,
  char* out_buf,
  int8_t* max_path_depth_exceeded)
{
  p.next_token();
  if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

  // Define stack; plus 1 indicates root context task needs an extra memory.
  context stack[MAX_JSON_PATH_DEPTH + 1];
  int stack_size = 0;

  auto const push_context = [&](evaluation_case_path _case_path,
                                json_generator _g,
                                write_style _style,
                                cudf::device_span<path_instruction const> _path) {
    if (stack_size > MAX_JSON_PATH_DEPTH) {
      *max_path_depth_exceeded = 1;
      // Because no more context is pushed, the evaluation output should be wrong.
      // But that is not important, since we will throw exception after the kernel finishes.
      return;
    }
    auto& ctx          = stack[stack_size++];
    ctx.g              = std::move(_g);
    ctx.path           = std::move(_path);
    ctx.dirty          = 0;
    ctx.case_path      = _case_path;
    ctx.token          = p.get_current_token();
    ctx.style          = _style;
    ctx.is_first_enter = true;
    ctx.task_is_done   = false;
  };

  push_context(evaluation_case_path::INVALID, json_generator{}, write_style::RAW, path_commands);

  while (stack_size > 0) {
    auto& ctx = stack[stack_size - 1];
    if (!ctx.task_is_done) {
      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size()) &&
          ctx.style == write_style::RAW) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.write_raw(p, out_buf);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // push back task
          // add child task
          push_context(evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE,
                       ctx.g,
                       ctx.style,
                       {nullptr, 0});
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // general case: just copy the child tree verbatim
        if (!(ctx.g.copy_current_structure(p, out_buf))) {
          // JSON validation check
          return {false, 0};
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          // Mark task is done regardless whether the expected child was found.
          ctx.task_is_done = true;
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return {false, 0}; }
              // push sub task; sub task will update the result of path 4
              push_context(evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1});
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          if (!found_expected_child) {
            // did not find any expected sub child
            ctx.task_is_done = true;
            ctx.dirty        = false;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: Wildcard :: xs)
      // case path 5
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_elements(
                 ctx.path, path_instruction_type::WILDCARD, path_instruction_type::WILDCARD)) {
        // special handling for the non-structure preserving double wildcard
        // behavior in Hive
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          push_context(evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD,
                       ctx.g,
                       write_style::FLATTEN,
                       {ctx.path.data() + 2, ctx.path.size() - 2});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
      // case path 6
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD) &&
               ctx.style != write_style::QUOTED) {
        // retain Flatten, otherwise use Quoted... cannot use Raw within an array
        write_style next_style = write_style::RAW;
        switch (ctx.style) {
          case write_style::RAW: next_style = write_style::QUOTED; break;
          case write_style::FLATTEN: next_style = write_style::FLATTEN; break;
          case write_style::QUOTED: next_style = write_style::QUOTED;  // never happen
        }

        // temporarily buffer child matches, the emitted json will need to be
        // modified slightly if there is only a single element written

        json_generator child_g;
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          // create a child generator with hide outer array tokens mode.
          child_g = ctx.g.new_child_generator();
          // write first [ without output, without update len, only update internal state
          child_g.write_first_start_array_without_output();
        } else {
          child_g = ctx.child_g;
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // track the number of array elements and only emit an outer array if
          // we've written more than one element, this matches Hive's behavior
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED,
                       child_g,
                       next_style,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          char* child_g_start = out_buf + child_g.get_offset();
          int child_g_len     = child_g.get_output_len();
          if (ctx.dirty > 1) {
            // add outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ true);
          } else if (ctx.dirty == 1) {
            // remove outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ false);
          }  // else do not write anything

          // Done anyway, since we already reached the end array.
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
      // case path 8
      else if (json_token::START_ARRAY == ctx.token &&
               thrust::get<0>(path_match_index_wildcard(ctx.path))) {
        int idx = thrust::get<1>(path_match_index_wildcard(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
        ctx.is_first_enter = false;

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD,
                     ctx.g,
                     write_style::QUOTED,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case (START_ARRAY, Index(idx) :: xs)
      // case path 9
      else if (json_token::START_ARRAY == ctx.token && thrust::get<0>(path_match_index(ctx.path))) {
        int idx = thrust::get<1>(path_match_index(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX,
                     ctx.g,
                     ctx.style,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case _ =>
      // case path 12
      else {
        if (!p.try_skip_children()) { return {false, 0}; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    }       // if (!ctx.task_is_done)
    else {  // current context is done.
      // pop current top context
      stack_size--;

      // has no parent task, stack is empty, will exit
      if (stack_size == 0) { break; }

      // peek parent context task
      // update parent task info according to current task result
      auto& p_ctx = stack[stack_size - 1];

      switch (ctx.case_path) {
          // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
          // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
          // path 7: case (START_ARRAY, Wildcard :: xs)
        case evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE:
        case evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_OBJECT, Named :: xs)
          // case path 4
        case evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH: {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
          // case path 6
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;

          break;
        }

          /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
          // case path 8
          // case (START_ARRAY, Index(idx) :: xs)
          // case path 9
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return {false, 0}; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

        default:;  // Never happens!
      }            // end switch (ctx.case_path)
    }              // ctx.task_is_done
  }                // while (stack_size > 0)

  auto const success = stack[0].dirty > 0;

  // generator may contain trash output, e.g.: generator writes some output,
  // then JSON format is invalid, the previous output becomes trash.
  // We need to return output size as zero.
  return {success, success ? stack[0].g.get_output_len() : 0};
}

/**
 * @brief Struct storing data such as path instructions, output buffer etc, corresponding to a
 * single JSON path.
 */
struct json_path_processing_data {
  cudf::device_span<path_instruction const> path_commands;
  cudf::detail::input_offsetalator offsets;
  thrust::pair<char const*, cudf::size_type>* out_stringviews;
  char* out_buf;
  int8_t* has_out_of_bound;
};

/**
 * @brief Kernel for running the JSONPath query, in which one input row is processed by entire
 * warp (or multiple warps) of threads.
 *
 * The number of warps processing each row is computed as `ceil(num_paths / warp_size)`.
 *
 * We explicitly set a value for `min_block_per_sm` parameter in the launch bounds to avoid
 * spilling from the kernel itself. By default NVCC uses a heuristic to find a balance between
 * the maximum number of registers used by a kernel and the parallelism of the kernel.
 * If lots of registers are used the parallelism may suffer. But in our case NVCC gets this wrong
 * and we want to avoid spilling all the time or else the performance is really bad. This
 * essentially tells NVCC to prefer using lots of registers over spilling.
 *
 * @param input The input JSON strings stored in a strings column
 * @param path_data Array containing all path data
 * @param num_threads_per_row Number of threads processing each input row
 * @param max_path_depth_exceeded A marker to record if the maximum path depth has been reached
 *        during parsing the input string
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view input,
                              cudf::device_span<json_path_processing_data> path_data,
                              std::size_t num_threads_per_row,
                              int8_t* max_path_depth_exceeded)
{
  auto const tidx    = cudf::detail::grid_1d::global_thread_id();
  auto const row_idx = tidx / num_threads_per_row;
  if (row_idx >= input.size()) { return; }

  auto const path_idx = tidx % num_threads_per_row;
  if (path_idx >= path_data.size()) { return; }

  auto const& path         = path_data[path_idx];
  char* const dst          = path.out_buf + path.offsets[row_idx];
  bool is_valid            = false;
  cudf::size_type out_size = 0;

  auto const str = input.element<cudf::string_view>(row_idx);
  if (str.size_bytes() > 0) {
    json_parser p{char_range{str}};
    thrust::tie(is_valid, out_size) =
      evaluate_path(p, path.path_commands, dst, max_path_depth_exceeded);

    // We did not terminate the `evaluate_path` function early to reduce complexity of the code.
    // Instead, if max depth was encountered, we've just continued the evaluation until here
    // then discard the output entirely.
    if (p.max_nesting_depth_exceeded()) {
      *max_path_depth_exceeded = 1;
      return;
    }

    auto const max_size = path.offsets[row_idx + 1] - path.offsets[row_idx];
    if (out_size > max_size) { *(path.has_out_of_bound) = 1; }
  }

  // Write out `nullptr` in the output string_view to indicate that the output is a null.
  // The situation `out_stringviews == nullptr` should only happen if the kernel is launched a
  // second time due to out-of-bound write in the first launch.
  if (path.out_stringviews) {
    path.out_stringviews[row_idx] = {is_valid ? dst : nullptr, out_size};
  }
}

/**
 * @brief A utility class to launch the main kernel.
 */
struct kernel_launcher {
  static void exec(cudf::column_device_view const& input,
                   cudf::device_span<json_path_processing_data> path_data,
                   int8_t* max_path_depth_exceeded,
                   rmm::cuda_stream_view stream)
  {
    // The optimal values for block_size and min_block_per_sm were found through testing,
    // which are either 128-8 or 256-4. The pair 128-8 seems a bit better.
    static constexpr int block_size       = 128;
    static constexpr int min_block_per_sm = 8;

    // The number of threads for processing one input row is at least one warp.
    auto const num_threads_per_row =
      cudf::util::div_rounding_up_safe(path_data.size(),
                                       static_cast<std::size_t>(cudf::detail::warp_size)) *
      cudf::detail::warp_size;
    auto const num_blocks = cudf::util::div_rounding_up_safe(num_threads_per_row * input.size(),
                                                             static_cast<std::size_t>(block_size));
    get_json_object_kernel<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(
        input, path_data, num_threads_per_row, max_path_depth_exceeded);
  }
};

/**
 * @brief Construct the device vector containing necessary data for the input JSON paths.
 *
 * All JSON paths are processed at once, without stream synchronization, to minimize overhead.
 *
 * A tuple of values are returned, however, only the first element is needed for further kernel
 * launch. The remaining are unused but need to be kept alive as they contains data for later
 * asynchronous host-device memcpy.
 */
std::tuple<std::vector<rmm::device_uvector<path_instruction>>,
           std::unique_ptr<std::vector<std::vector<path_instruction>>>,
           cudf::string_scalar,
           std::string>
construct_path_commands(
  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> const&
    json_paths,
  rmm::cuda_stream_view stream)
{
  // Concatenate all names from path instructions.
  auto h_inst_names = [&] {
    std::size_t length{0};
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { length += name.length(); }
      }
    }
    std::string all_names;
    all_names.reserve(length);
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { all_names += name; }
      }
    }
    return all_names;
  }();
  auto d_inst_names = cudf::string_scalar(h_inst_names, true, stream);

  std::size_t name_pos{0};
  auto h_path_commands = std::make_unique<std::vector<std::vector<path_instruction>>>();
  h_path_commands->reserve(json_paths.size());

  for (auto const& instructions : json_paths) {
    h_path_commands->emplace_back();
    auto& path_commands = h_path_commands->back();
    path_commands.reserve(instructions.size());

    for (auto const& [type, name, index] : instructions) {
      path_commands.emplace_back(path_instruction{type});

      if (type == path_instruction_type::INDEX) {
        path_commands.back().index = index;
      } else if (type == path_instruction_type::NAMED) {
        path_commands.back().name = cudf::string_view(d_inst_names.data() + name_pos, name.size());
        name_pos += name.size();
      } else if (type != path_instruction_type::WILDCARD) {
        CUDF_FAIL("Invalid path instruction type");
      }
    }
  }

  auto d_path_commands = std::vector<rmm::device_uvector<path_instruction>>{};
  d_path_commands.reserve(h_path_commands->size());
  for (auto const& path_commands : *h_path_commands) {
    d_path_commands.emplace_back(cudf::detail::make_device_uvector_async(
      path_commands, stream, rmm::mr::get_current_device_resource()));
  }

  return {std::move(d_path_commands),
          std::move(h_path_commands),
          std::move(d_inst_names),
          std::move(h_inst_names)};
}

int64_t calc_scratch_size(cudf::strings_column_view const& input,
                          cudf::detail::input_offsetalator const& in_offsets,
                          rmm::cuda_stream_view stream)
{
  auto const max_row_size = thrust::transform_reduce(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(input.size()),
    cuda::proclaim_return_type<int64_t>(
      [in_offsets] __device__(auto const idx) { return in_offsets[idx + 1] - in_offsets[idx]; }),
    int64_t{0},
    thrust::maximum{});

  // We will use scratch buffers to store the output strings without knowing their sizes.
  // Since we do not know their sizes, we need to allocate the buffer a bit larger than the input
  // size so that we will not write output strings into an out-of-bound position.
  // Checking out-of-bound needs to be performed in the main kernel to make sure we will not have
  // data corruption.
  auto const scratch_size = [&, max_row_size = max_row_size] {
    // Pad the scratch buffer by an additional size that is a multiple of max row size.
    auto constexpr padding_rows = 10;
    return input.chars_size(stream) + max_row_size * padding_rows;
  }();
  return scratch_size;
}

/**
 * @brief Error handling using error markers gathered after kernel launch.
 *
 * If the input JSON has nesting depth exceeds the maximum allowed value, an exception will be
 * thrown as it is unacceptable. Otherwise, out of bound write is checked and returned.
 *
 * @param error_check The array of markers to check for error
 * @return A boolean value indicating if there is any out of bound write
 */
bool check_error(cudf::detail::host_vector<int8_t> const& error_check)
{
  // The last value is to mark if nesting depth has exceeded.
  CUDF_EXPECTS(error_check.back() == 0,
               "The processed input has nesting depth exceeds depth limit.");

  // Do not use parallel check since we do not have many elements.
  // The last element is not related, but its value is already `0` thus just check until
  // the end of the array for simplicity.
  return std::none_of(
    error_check.cbegin(), error_check.cend(), [](auto const val) { return val != 0; });
}

std::vector<std::unique_ptr<cudf::column>> get_json_object_batch(
  cudf::column_device_view const& input,
  cudf::detail::input_offsetalator const& in_offsets,
  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> const&
    json_paths,
  int64_t scratch_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const [d_json_paths, h_json_paths, d_inst_names, h_inst_names] =
    construct_path_commands(json_paths, stream);

  auto const num_outputs = json_paths.size();
  std::vector<std::unique_ptr<cudf::column>> output;

  // The error check array contains markers denoting if there is any out-of-bound write occurs
  // (first `num_outputs` elements), or if the nesting depth exceeded its limits (the last element).
  rmm::device_uvector<int8_t> d_error_check(num_outputs + 1, stream);
  auto const d_max_path_depth_exceeded = d_error_check.data() + num_outputs;

  std::vector<rmm::device_uvector<char>> scratch_buffers;
  std::vector<rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>> out_stringviews;
  std::vector<json_path_processing_data> h_path_data;
  scratch_buffers.reserve(json_paths.size());
  out_stringviews.reserve(json_paths.size());
  h_path_data.reserve(json_paths.size());

  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& path = json_paths[idx];
    if (path.size() > MAX_JSON_PATH_DEPTH) {
      CUDF_FAIL("JSON Path has depth exceeds the maximum allowed value.");
    }

    scratch_buffers.emplace_back(rmm::device_uvector<char>(scratch_size, stream));
    out_stringviews.emplace_back(rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{
      static_cast<std::size_t>(input.size()), stream});

    h_path_data.emplace_back(json_path_processing_data{d_json_paths[idx],
                                                       in_offsets,
                                                       out_stringviews.back().data(),
                                                       scratch_buffers.back().data(),
                                                       d_error_check.data() + idx});
  }
  auto d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_error_check.begin(), d_error_check.end(), 0);

  kernel_launcher::exec(input, d_path_data, d_max_path_depth_exceeded, stream);
  auto h_error_check = cudf::detail::make_host_vector_sync(d_error_check, stream);
  auto has_no_oob    = check_error(h_error_check);

  // If we didn't see any out-of-bound write, everything is good so far.
  // Just gather the output strings and return.
  if (has_no_oob) {
    for (auto const& out_sview : out_stringviews) {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
    return output;
  }
  // From here, we had out-of-bound write. Although this is very rare, it may still happen.

  std::vector<std::pair<rmm::device_buffer, cudf::size_type>> out_null_masks_and_null_counts;
  std::vector<std::pair<std::unique_ptr<cudf::column>, int64_t>> out_offsets_and_sizes;
  std::vector<rmm::device_uvector<char>> out_char_buffers;
  std::vector<std::size_t> oob_indices;

  // Check validity from the stored char pointers.
  auto const validator = [] __device__(thrust::pair<char const*, cudf::size_type> const item) {
    return item.first != nullptr;
  };

  // Rebuild the data only for paths that had out of bound write.
  h_path_data.clear();
  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& out_sview = out_stringviews[idx];

    if (h_error_check[idx]) {
      oob_indices.emplace_back(idx);
      output.emplace_back(nullptr);  // just placeholder.

      out_null_masks_and_null_counts.emplace_back(
        cudf::detail::valid_if(out_sview.begin(), out_sview.end(), validator, stream, mr));

      // The string sizes computed in the previous kernel call will be used to allocate a new char
      // buffer to store the output.
      auto const size_it = cudf::detail::make_counting_transform_iterator(
        0,
        cuda::proclaim_return_type<cudf::size_type>(
          [string_pairs = out_sview.data()] __device__(auto const idx) {
            return string_pairs[idx].second;
          }));
      out_offsets_and_sizes.emplace_back(cudf::strings::detail::make_offsets_child_column(
        size_it, size_it + input.size(), stream, mr));
      out_char_buffers.emplace_back(
        rmm::device_uvector<char>(out_offsets_and_sizes.back().second, stream, mr));

      h_path_data.emplace_back(
        json_path_processing_data{d_json_paths[idx],
                                  cudf::detail::offsetalator_factory::make_input_iterator(
                                    out_offsets_and_sizes.back().first->view()),
                                  nullptr /*out_stringviews*/,
                                  out_char_buffers.back().data(),
                                  d_error_check.data() + idx});
    } else {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
  }
  // These buffers are no longer needed.
  scratch_buffers.clear();
  out_stringviews.clear();

  // Push data to the GPU and launch the kernel again.
  d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_error_check.begin(), d_error_check.end(), 0);
  kernel_launcher::exec(input, d_path_data, d_max_path_depth_exceeded, stream);
  h_error_check = cudf::detail::make_host_vector_sync(d_error_check, stream);
  has_no_oob    = check_error(h_error_check);

  // The last kernel call should not encounter any out-of-bound write.
  // If OOB is still detected, there must be something wrong happened.
  CUDF_EXPECTS(has_no_oob, "Unexpected out-of-bound write in get_json_object kernel.");

  for (std::size_t idx = 0; idx < oob_indices.size(); ++idx) {
    auto const out_idx = oob_indices[idx];
    output[out_idx] =
      cudf::make_strings_column(input.size(),
                                std::move(out_offsets_and_sizes[idx].first),
                                out_char_buffers[idx].release(),
                                out_null_masks_and_null_counts[idx].second,
                                std::move(out_null_masks_and_null_counts[idx].first));
  }
  return output;
}

std::vector<std::unique_ptr<cudf::column>> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>> const&
    json_paths,
  int64_t memory_budget_bytes,
  int32_t parallel_override,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const num_outputs = json_paths.size();

  // Input is empty or all nulls - just return all null columns.
  if (input.is_empty() || input.size() == input.null_count()) {
    std::vector<std::unique_ptr<cudf::column>> output;
    for (std::size_t idx = 0; idx < num_outputs; ++idx) {
      output.emplace_back(std::make_unique<cudf::column>(input.parent(), stream, mr));
    }
    return output;
  }

  std::vector<std::size_t> sorted_indices(json_paths.size());
  std::iota(sorted_indices.begin(), sorted_indices.end(), 0);  // Fill with 0, 1, 2, ...

  // Sort indices based on the corresponding paths.
  std::sort(sorted_indices.begin(), sorted_indices.end(), [&json_paths](size_t i, size_t j) {
    return json_paths[i] < json_paths[j];
  });

  auto const in_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());
  auto const scratch_size = calc_scratch_size(input, in_offsets, stream);
  if (memory_budget_bytes <= 0 && parallel_override <= 0) {
    parallel_override = static_cast<int>(sorted_indices.size());
  }
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  std::vector<std::unique_ptr<cudf::column>> output(num_outputs);

  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> batch;
  std::vector<std::size_t> output_ids;

  std::size_t starting_path = 0;
  while (starting_path < num_outputs) {
    std::size_t at = starting_path;
    batch.resize(0);
    output_ids.resize(0);
    if (parallel_override > 0) {
      int count = 0;
      while (at < num_outputs && count < parallel_override) {
        auto output_location = sorted_indices[at];
        batch.emplace_back(json_paths[output_location]);
        output_ids.push_back(output_location);
        at++;
        count++;
      }
    } else {
      long budget = 0;
      while (at < num_outputs && budget < memory_budget_bytes) {
        auto output_location = sorted_indices[at];
        batch.emplace_back(json_paths[output_location]);
        output_ids.push_back(output_location);
        at++;
        budget += scratch_size;
      }
    }
    auto tmp = get_json_object_batch(*d_input_ptr, in_offsets, batch, scratch_size, stream, mr);
    for (std::size_t i = 0; i < tmp.size(); i++) {
      std::size_t out_i = output_ids[i];
      output[out_i]     = std::move(tmp[i]);
    }
    starting_path = at;
  }
  return output;
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int32_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return std::move(detail::get_json_object(input, {instructions}, -1, -1, stream, mr).front());
}

std::vector<std::unique_ptr<cudf::column>> get_json_object_multiple_paths(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>> const&
    json_paths,
  int64_t memory_budget_bytes,
  int32_t parallel_override,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::get_json_object(
    input, json_paths, memory_budget_bytes, parallel_override, stream, mr);
}

}  // namespace spark_rapids_jni
