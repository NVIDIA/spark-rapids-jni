#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/json/json.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/optional.h>
#include <thrust/pair.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>

namespace spark_rapids_jni {

namespace detail {
// namespace {

/**
 * write JSON style
 */
enum class write_style { raw_style, quoted_style, flatten_style };

thrust::optional<rmm::device_uvector<path_instruction>> parse_path(
  cudf::string_scalar const& json_path, rmm::cuda_stream_view stream)
{
  std::string h_json_path = json_path.to_string();
  JsonPathParser parser;
  auto instructions = parser.parse(h_json_path);
  if (!instructions) { return thrust::nullopt; }
  return thrust::make_optional(cudf::detail::make_device_uvector_sync(
    *instructions, stream, rmm::mr::get_current_device_resource()));
}

/**
 * TODO: JSON generator
 *
 */
template <int max_json_nesting_depth = curr_max_json_nesting_depth>
class json_generator {
 public:
  CUDF_HOST_DEVICE json_generator(char* _output, size_t _output_len)
    : output(_output), output_len(_output_len)
  {
  }
  CUDF_HOST_DEVICE json_generator() : output(nullptr), output_len(0) {}

  // create a nested child generator based on this parent generator
  // child generator is a view
  CUDF_HOST_DEVICE json_generator new_child_generator()
  {
    if (nullptr == output) {
      return json_generator();
    } else {
      return json_generator(output + output_len, 0);
    }
  }

  CUDF_HOST_DEVICE json_generator finish_child_generator(json_generator const& child_generator)
  {
    // logically delete child generator
    output_len += child_generator.get_output_len();
  }

  CUDF_HOST_DEVICE void write_start_array()
  {
    // TODO
  }

  CUDF_HOST_DEVICE void write_end_array()
  {
    // TODO
  }

  CUDF_HOST_DEVICE void copy_current_structure(json_parser<max_json_nesting_depth>& parser)
  {
    // TODO
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  CUDF_HOST_DEVICE void write_raw(json_parser<max_json_nesting_depth>& parser)
  {
    if (output) {
      auto copied = parser.try_copy_raw_text(output + output_len);
      output_len += copied;
    }
  }

  CUDF_HOST_DEVICE inline size_t get_output_len() const { return output_len; }

 private:
  char const* const output;
  size_t output_len;
};

/**
 * @brief Result of calling a parse function.
 *
 * The primary use of this is to distinguish between "success" and
 * "success but no data" return cases.  For example, if you are reading the
 * values of an array you might call a parse function in a while loop. You
 * would want to continue doing this until you either encounter an error
 * (parse_result::ERROR) or you get nothing back (parse_result::EMPTY)
 */
enum class parse_result {
  ERROR,          // failure
  SUCCESS,        // success
  MISSING_FIELD,  // success, but the field is missing
  EMPTY,          // success, but no data
};

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * @param j_state The incoming json string and associated parser
 * @param commands The command buffer to be applied to the string. Always ends
 * with a path_operator_type::END
 * @param output Buffer user to store the results of the query
 * @returns A result code indicating success/fail/empty.
 */
template <int max_json_nesting_depth = curr_max_json_nesting_depth>
__device__ parse_result parse_json_path(json_parser<max_json_nesting_depth>& j_parser,
                                        path_instruction const* path_commands_ptr,
                                        int path_commands_size,
                                        json_generator<max_json_nesting_depth>& output)
{
  // TODO
  return parse_result::SUCCESS;
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * This function exists primarily as a shim for debugging purposes.
 *
 * @param input The incoming json string
 * @param input_len Size of the incoming json string
 * @param commands The command buffer to be applied to the string. Always ends
 * with a path_operator_type::END
 * @param out_buf Buffer user to store the results of the query (nullptr in the
 * size computation step)
 * @param out_buf_size Size of the output buffer
 * @param options Options controlling behavior
 * @returns A pair containing the result code the output buffer.
 */
template <int max_json_nesting_depth = curr_max_json_nesting_depth>
__device__ thrust::pair<parse_result, json_generator<max_json_nesting_depth>>
get_json_object_single(char const* input,
                       cudf::size_type input_len,
                       path_instruction const* path_commands_ptr,
                       int path_commands_size,
                       char* out_buf,
                       size_t out_buf_size,
                       json_parser_options options)
{
  json_parser j_parser(options, input, input_len);
  json_generator generator(out_buf, out_buf_size);
  auto const result = parse_json_path(j_parser, path_commands_ptr, path_commands_size, generator);
  return {result, generator};
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel operates in a 2-pass way.  On the first pass, it computes
 * output sizes.  On the second pass it fills in the provided output buffers
 * (chars and validity)
 *
 * @param col Device view of the incoming string
 * @param commands JSONPath command buffer
 * @param output_offsets Buffer used to store the string offsets for the results
 * of the query
 * @param out_buf Buffer used to store the results of the query
 * @param out_validity Output validity buffer
 * @param out_valid_count Output count of # of valid bits
 * @param options Options controlling behavior
 */
template <int block_size>
__launch_bounds__(block_size) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view col,
                              path_instruction const* path_commands_ptr,
                              int path_commands_size,
                              cudf::size_type* d_sizes,
                              cudf::detail::input_offsetalator output_offsets,
                              thrust::optional<char*> out_buf,
                              thrust::optional<cudf::bitmask_type*> out_validity,
                              thrust::optional<cudf::size_type*> out_valid_count,
                              json_parser_options options)
{
  auto tid          = cudf::detail::grid_1d::global_thread_id();
  auto const stride = cudf::thread_index_type{blockDim.x} * cudf::thread_index_type{gridDim.x};

  cudf::size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffff'ffffu, tid < col.size());
  while (tid < col.size()) {
    bool is_valid               = false;
    cudf::string_view const str = col.element<cudf::string_view>(tid);
    cudf::size_type output_size = 0;
    if (str.size_bytes() > 0) {
      char* dst = out_buf.has_value() ? out_buf.value() + output_offsets[tid] : nullptr;
      size_t const dst_size =
        out_buf.has_value() ? output_offsets[tid + 1] - output_offsets[tid] : 0;

      // process one single row
      auto [result, out] = get_json_object_single(str.data(),
                                                  str.size_bytes(),
                                                  path_commands_ptr,
                                                  path_commands_size,
                                                  dst,
                                                  dst_size,
                                                  options);
      output_size        = out.get_output_len();
      if (result == parse_result::SUCCESS) { is_valid = true; }
    }

    // filled in only during the precompute step. during the compute step, the
    // offsets are fed back in so we do -not- want to write them out
    if (!out_buf.has_value()) { d_sizes[tid] = output_size; }

    // validity filled in only during the output step
    if (out_validity.has_value()) {
      uint32_t mask = __ballot_sync(active_threads, is_valid);
      // 0th lane of the warp writes the validity
      if (!(tid % cudf::detail::warp_size)) {
        out_validity.value()[cudf::word_index(tid)] = mask;
        warp_valid_count += __popc(mask);
      }
    }

    tid += stride;
    active_threads = __ballot_sync(active_threads, tid < col.size());
  }

  // sum the valid counts across the whole block
  if (out_valid_count) {
    cudf::size_type block_valid_count =
      cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count.value(), block_valid_count); }
  }
}

std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              spark_rapids_jni::json_parser_options options,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  if (col.is_empty()) return cudf::make_empty_column(cudf::type_id::STRING);

  // parse the json_path into a command buffer
  auto path_commands_optional = parse_path(json_path, stream);

  // if the json path is empty, return a string column containing all nulls
  if (!path_commands_optional.has_value()) {
    return std::make_unique<cudf::column>(
      cudf::data_type{cudf::type_id::STRING},
      col.size(),
      rmm::device_buffer{0, stream, mr},  // no data
      cudf::detail::create_null_mask(col.size(), cudf::mask_state::ALL_NULL, stream, mr),
      col.size());                        // null count
  }

  // compute output sizes
  auto sizes = rmm::device_uvector<cudf::size_type>(
    col.size(), stream, rmm::mr::get_current_device_resource());
  auto d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(col.offsets());

  constexpr int block_size = 512;
  cudf::detail::grid_1d const grid{col.size(), block_size};
  auto cdv = cudf::column_device_view::create(col.parent(), stream);
  // preprocess sizes (returned in the offsets buffer)
  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      path_commands_optional.value().data(),
      path_commands_optional.value().size(),
      sizes.data(),
      d_offsets,
      thrust::nullopt,
      thrust::nullopt,
      thrust::nullopt,
      options);

  // convert sizes to offsets
  auto [offsets, output_size] =
    cudf::strings::detail::make_offsets_child_column(sizes.begin(), sizes.end(), stream, mr);
  d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  // allocate output string column
  rmm::device_uvector<char> chars(output_size, stream, mr);

  // potential optimization : if we know that all outputs are valid, we could
  // skip creating the validity mask altogether
  rmm::device_buffer validity =
    cudf::detail::create_null_mask(col.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  // compute results
  rmm::device_scalar<cudf::size_type> d_valid_count{0, stream};

  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      path_commands_optional.value().data(),
      path_commands_optional.value().size(),
      sizes.data(),
      d_offsets,
      chars.data(),
      static_cast<cudf::bitmask_type*>(validity.data()),
      d_valid_count.data(),
      options);

  auto result = make_strings_column(col.size(),
                                    std::move(offsets),
                                    chars.release(),
                                    col.size() - d_valid_count.value(stream),
                                    std::move(validity));
  // unmatched array query may result in unsanitized '[' value in the result
  if (cudf::detail::has_nonempty_nulls(result->view(), stream)) {
    result = cudf::detail::purge_nonempty_nulls(result->view(), stream, mr);
  }
  return result;
}

// }  // namespace

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              spark_rapids_jni::json_parser_options options,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  // TODO: main logic
  // return cudf::make_empty_column(cudf::type_to_id<cudf::size_type>());
  return detail::get_json_object(col, json_path, options, stream, mr);
}

}  // namespace spark_rapids_jni
