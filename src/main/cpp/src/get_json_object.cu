#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"
#include "json_parser.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

namespace spark_rapids_jni {

namespace detail {

// path max depth limitation
// There is a same constant in JSONUtil.java, keep them consistent when changing
// Note: Spark-Rapids should guarantee the path depth is less or equal to this limit,
// or GPU reports hipErrorIllegalAddress
constexpr int max_path_depth = 16;

/**
 * write JSON style
 */
enum class write_style { RAW, QUOTED, FLATTEN };

/**
 * path instruction
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  path_instruction_type type;

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};
};

/**
 * JSON generator is used to write out JSON content.
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(char* _output) : output(_output), output_len(0) {}
  __device__ json_generator() : output(nullptr), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator()
  {
    if (nullptr == output) {
      return json_generator();
    } else {
      return json_generator(output + output_len);
    }
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array()
  {
    try_write_comma();

    if (output) { *(output + output_len) = '['; }

    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array()
  {
    if (output) { *(output + output_len) = ']'; }
    output_len++;

    // point to parent array
    array_depth--;

    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma()
  {
    if (need_comma()) {
      // in array context and writes first item
      if (output) { *(output + output_len) = ','; }
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser)
  {
    // first try add comma
    try_write_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      auto copy_to       = output + output_len;
      auto [b, copy_len] = parser.copy_current_structure(copy_to);
      output_len += copy_len;
      return b;
    } else {
      char* copy_to      = nullptr;
      auto [b, copy_len] = parser.copy_current_structure(copy_to);
      output_len += copy_len;
      return b;
    }
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      auto copied = parser.write_unescaped_text(output + output_len);
      output_len += copied;
    } else {
      auto len = parser.compute_unescaped_len();
      output_len += len;
    }
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        size_t child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (nullptr != output) {
      if (write_outer_array_tokens) {
        if (insert_comma) {
          *(child_block_begin + child_block_len + 2) = ']';
          move_forward(child_block_begin, child_block_len, 2);
          *(child_block_begin + 1) = '[';
          *(child_block_begin)     = ',';
        } else {
          *(child_block_begin + child_block_len + 1) = ']';
          move_forward(child_block_begin, child_block_len, 1);
          *(child_block_begin) = '[';
        }
      } else {
        if (insert_comma) {
          move_forward(child_block_begin, child_block_len, 1);
          *(child_block_begin) = ',';
        } else {
          // do not need comma && do not need write outer array tokens
          // do nothing, because child generator buff is directly after the
          // parent generator
        }
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ inline size_t get_output_len() const { return output_len; }
  __device__ inline char* get_output_start_position() const { return output; }
  __device__ inline char* get_current_output_position() const { return output + output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  char* output;
  size_t output_len;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
  int array_depth = 0;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

/**
 *
 * This function is rewritten from above commented recursive function.
 * this function is equivalent to the above commented recursive function.
 */
__device__ bool evaluate_path(json_parser& p,
                              json_generator& root_g,
                              write_style root_style,
                              cudf::device_span<path_instruction const> root_path)
{
  // manually maintained context stack in lieu of calling evaluate_path recursively.
  struct context {
    // current token
    json_token token;

    // which case path that this task is from
    int case_path;

    // used to save current generator
    json_generator g;

    write_style style;

    cudf::device_span<path_instruction const> path;
    // is this context task is done
    bool task_is_done;

    // whether written output
    // if dirty > 0, indicates success
    int dirty;

    // for some case paths
    bool is_first_enter;

    // used to save child JSON generator for case path 8
    json_generator child_g;
  };

  // define stack; plus 1 indicates root context task needs an extra memory
  context stack[max_path_depth + 1];
  int stack_pos = 0;

  // push context function
  auto push_context = [&stack, &stack_pos](json_token _token,
                                           int _case_path,
                                           json_generator _g,
                                           write_style _style,
                                           cudf::device_span<path_instruction const> _path) {
    // no need to check stack is full
    // because Spark-Rapids already checked maximum length of `path_instruction`
    auto& ctx          = stack[stack_pos];
    ctx.token          = _token;
    ctx.case_path      = _case_path;
    ctx.g              = _g;
    ctx.style          = _style;
    ctx.path           = _path;
    ctx.task_is_done   = false;
    ctx.dirty          = 0;
    ctx.is_first_enter = true;

    stack_pos++;
  };

  // put the first context task
  push_context(p.get_current_token(), -1, root_g, root_style, root_path);

  while (stack_pos > 0) {
    auto& ctx = stack[stack_pos - 1];
    if (!ctx.task_is_done) {
      // task is not done.

      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size()) &&
          ctx.style == write_style::RAW) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.write_raw(p);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          // push back task
          // add child task
          push_context(p.get_current_token(), 2, ctx.g, ctx.style, {nullptr, 0});
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // general case: just copy the child tree verbatim
        if (!(ctx.g.copy_current_structure(p))) {
          // JSON validation check
          return false;
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return false;
              }
            }
            ctx.task_is_done = true;
          } else {
            return false;
          }
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return false; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return false; }
              // push sub task; sub task will update the result of path 4
              push_context(p.get_current_token(),
                           4,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1});
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return false; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return false;
              }
            }
          }
          if (!found_expected_child) {
            // did not find any expected sub child
            ctx.task_is_done = true;
            ctx.dirty        = false;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: Wildcard :: xs)
      // case path 5
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_elements(
                 ctx.path, path_instruction_type::WILDCARD, path_instruction_type::WILDCARD)) {
        // special handling for the non-structure preserving double wildcard
        // behavior in Hive
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array();
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          push_context(p.get_current_token(),
                       5,
                       ctx.g,
                       write_style::FLATTEN,
                       {ctx.path.data() + 2, ctx.path.size() - 2});
        } else {
          ctx.g.write_end_array();
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
      // case path 6
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD) &&
               ctx.style != write_style::QUOTED) {
        // retain Flatten, otherwise use Quoted... cannot use Raw within an array
        write_style next_style = write_style::RAW;
        switch (ctx.style) {
          case write_style::RAW: next_style = write_style::QUOTED; break;
          case write_style::FLATTEN: next_style = write_style::FLATTEN; break;
          case write_style::QUOTED: next_style = write_style::QUOTED;  // never happen
        }

        // temporarily buffer child matches, the emitted json will need to be
        // modified slightly if there is only a single element written

        json_generator child_g;
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          // create a child generator with hide outer array tokens mode.
          child_g = ctx.g.new_child_generator();
          // write first [ without output, without update len, only update internal state
          child_g.write_first_start_array_without_output();
        } else {
          child_g = ctx.child_g;
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }
          // track the number of array elements and only emit an outer array if
          // we've written more than one element, this matches Hive's behavior
          push_context(p.get_current_token(),
                       6,
                       child_g,
                       next_style,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          char* child_g_start = child_g.get_output_start_position();
          size_t child_g_len  = child_g.get_output_len();
          if (ctx.dirty > 1) {
            // add outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ true);
            ctx.task_is_done = true;
          } else if (ctx.dirty == 1) {
            // remove outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ false);
            ctx.task_is_done = true;
          }  // else do not write anything
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array();
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(p.get_current_token(),
                       7,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          ctx.g.write_end_array();
          ctx.task_is_done = true;
        }
      }
      /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
      // case path 8
      else if (json_token::START_ARRAY == ctx.token &&
               thrust::get<0>(path_match_index_wildcard(ctx.path))) {
        int idx = thrust::get<1>(path_match_index_wildcard(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }
        ctx.is_first_enter = false;

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return false;
          }

          if (!p.try_skip_children()) { return false; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          --i;
        }

        // i == 0
        push_context(p.get_current_token(),
                     8,
                     ctx.g,
                     write_style::QUOTED,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case (START_ARRAY, Index(idx) :: xs)
      // case path 9
      else if (json_token::START_ARRAY == ctx.token && thrust::get<0>(path_match_index(ctx.path))) {
        int idx = thrust::get<1>(path_match_index(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return false;
          }

          if (!p.try_skip_children()) { return false; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          --i;
        }

        // i == 0
        push_context(
          p.get_current_token(), 9, ctx.g, ctx.style, {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case _ =>
      // case path 12
      else {
        if (!p.try_skip_children()) { return false; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    } else {
      // current context is done.

      // pop current top context
      stack_pos--;

      // pop parent task
      // update parent task info according to current task result
      if (stack_pos > 0) {
        // peek parent context task
        auto& p_ctx = stack[stack_pos - 1];

        // case (VALUE_STRING, Nil) if style == RawStyle
        // case path 1
        if (1 == ctx.case_path) {
          // never happen
        }
        // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
        // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
        // path 7: case (START_ARRAY, Wildcard :: xs)
        else if (2 == ctx.case_path || 5 == ctx.case_path || 7 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case (START_OBJECT, Named :: xs)
        // case path 4
        else if (4 == ctx.case_path) {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
        // case path 6
        else if (6 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;
        }
        /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
        // case path 8
        // case (START_ARRAY, Index(idx) :: xs)
        // case path 9
        else if (8 == ctx.case_path || 9 == ctx.case_path) {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return false; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return false; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;
        }
        // case path 3: case (_, Nil)
        // case path 12: case _ =>
        // others
        else {
          // never happen
        }
      } else {
        // has no parent task, stack is empty, will exit
      }
    }
  }

  // copy output len
  root_g.set_output_len(stack[0].g.get_output_len());
  return stack[0].dirty > 0;
}

rmm::device_uvector<path_instruction> construct_path_commands(
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  cudf::string_scalar const& all_names_scalar,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  int name_pos = 0;

  // construct the path commands
  std::vector<path_instruction> path_commands;
  for (auto const& inst : instructions) {
    auto const& [type, name, index] = inst;
    switch (type) {
      case path_instruction_type::WILDCARD:
        path_commands.emplace_back(path_instruction{path_instruction_type::WILDCARD});
        break;
      case path_instruction_type::INDEX:
        path_commands.emplace_back(path_instruction{path_instruction_type::INDEX});
        path_commands.back().index = index;
        break;
      case path_instruction_type::NAMED:
        path_commands.emplace_back(path_instruction{path_instruction_type::NAMED});
        path_commands.back().name =
          cudf::string_view(all_names_scalar.data() + name_pos, name.size());
        name_pos += name.size();
        break;
      default: CUDF_FAIL("Invalid path instruction type");
    }
  }
  // convert to uvector
  return cudf::detail::make_device_uvector_sync(path_commands, stream, mr);
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 *
 * @param input The incoming json string
 * @param path_commands The command buffer to be applied to the string
 * @param out_buf Buffer user to store the string resulted from the query
 * @returns A pair containing the result code and the output buffer
 */
__device__ thrust::pair<bool, cudf::size_type> get_json_object_single(
  char_range input, cudf::device_span<path_instruction const> path_commands, char* out_buf)
{
  json_parser j_parser(input);
  j_parser.next_token();
  // JSON validation check
  if (json_token::ERROR == j_parser.get_current_token()) { return {false, 0}; }

  json_generator generator(out_buf);

  bool const success = evaluate_path(j_parser, generator, write_style::RAW, path_commands);

  if (!success) {
    // generator may contain trash output, e.g.: generator writes some output,
    // then JSON format is invalid, the previous output becomes trash.
    // set output as zero to tell second step
    generator.set_output_len_zero();
  }

  return {success, static_cast<cudf::size_type>(generator.get_output_len())};
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel writes out the output strings and their lengths at the same time. If any output
 * length exceed buffer size limit, a boolean flag will be turned on to inform to the caller.
 * In such situation, another (larger) output buffer will be generated and the kernel is launched
 * again. Otherwise, launching this kernel only once is sufficient to produce the desired output.
 *
 * @param input The input JSON strings stored in a strings column
 * @param offsets Offsets to the output locations in the output buffer
 * @param path_commands JSONPath command buffer
 * @param out_stringviews The output array to store pointers to the output strings and their sizes
 * @param out_buf Buffer used to store the strings resulted from the query
 * @param has_out_of_bound Flag to indicate if any output string has length exceeds its buffer size
 */
template <int block_size>
// We have 1 for the minBlocksPerMultiprocessor in the launch bounds to avoid spilling from
// the kernel itself. By default NVCC uses a heuristic to find a balance between the
// maximum number of registers used by a kernel and the parallelism of the kernel.
// If lots of registers are used the parallelism may suffer. But in our case
// NVCC gets this wrong and we want to avoid spilling all the time or else
// the performance is really bad. This essentially tells NVCC to prefer using lots
// of registers over spilling.
__launch_bounds__(block_size, 1) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view input,
                              cudf::detail::input_offsetalator offsets,
                              cudf::device_span<path_instruction const> path_commands,
                              thrust::pair<char const*, cudf::size_type>* out_stringviews,
                              char* out_buf,
                              bool* has_out_of_bound)
{
  auto const stride = cudf::detail::grid_1d::grid_stride();
  for (auto tid = cudf::detail::grid_1d::global_thread_id(); tid < input.size(); tid += stride) {
    char* const dst          = out_buf + offsets[tid];
    bool is_valid            = false;
    cudf::size_type out_size = 0;

    auto const str = input.element<cudf::string_view>(tid);
    if (str.size_bytes() > 0) {
      auto const max_size = offsets[tid + 1] - offsets[tid];

      // If `max_size == 0`, do not pass in the dst pointer to prevent writing garbage data.
      thrust::tie(is_valid, out_size) =
        get_json_object_single(str, path_commands, max_size != 0 ? dst : nullptr);
      if (out_size > max_size) { *has_out_of_bound = true; }
    }

    // Write out `nullptr` in the output string_view to indicate that the output is a null.
    // The situation `out_stringviews == nullptr` should only happen if the kernel is launched a
    // second time due to out-of-bound write in the first launch.
    if (out_stringviews) { out_stringviews[tid] = {is_valid ? dst : nullptr, out_size}; }
  }
}

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  if (instructions.size() > max_path_depth) { CUDF_FAIL("JSONPath query exceeds maximum depth"); }
  if (input.is_empty()) { return cudf::make_empty_column(cudf::type_id::STRING); }

  std::string all_names;
  for (auto const& inst : instructions) {
    all_names += std::get<1>(inst);
  }
  auto const all_names_scalar = cudf::string_scalar(all_names, true, stream);
  auto const path_commands    = construct_path_commands(
    instructions, all_names_scalar, stream, rmm::mr::get_current_device_resource());
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  auto const in_offsets  = cudf::detail::offsetalator_factory::make_input_iterator(input.offsets());

  // A buffer to store the output strings without knowing their sizes.
  // Since we do not know their sizes, we need to allocate the buffer a bit larger than the input
  // size so that we will not write output strings into an out-of-bound position.
  // Checking out-of-bound needs to be performed in the main kernel to make sure we will not have
  // data corruption.
  auto const scratch_size = [&] {
    auto const max_row_size = thrust::transform_reduce(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(input.size()),
      cuda::proclaim_return_type<int64_t>(
        [in_offsets] __device__(auto const idx) { return in_offsets[idx + 1] - in_offsets[idx]; }),
      int64_t{0},
      thrust::maximum{});

    // Pad the scratch buffer by an additional size that is a multiple of max row size.
    auto constexpr padding_rows = 10;
    return input.chars_size(stream) + max_row_size * padding_rows;
  }();
  auto output_scratch  = rmm::device_uvector<char>(scratch_size, stream);
  auto out_stringviews = rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{
    static_cast<std::size_t>(input.size()), stream};
  auto has_out_of_bound = rmm::device_scalar<bool>{false, stream};

  constexpr int blocks_per_SM = 1;
  constexpr int block_size    = 256;
  auto const num_blocks       = [&] {
    int device_id{};
    hipDeviceProp_t props{};
    CUDF_CUDA_TRY(hipGetDevice(&device_id));
    CUDF_CUDA_TRY(hipGetDeviceProperties(&props, device_id));
    return props.multiProcessorCount * blocks_per_SM;
  }();

  get_json_object_kernel<block_size>
    <<<num_blocks, block_size, 0, stream.value()>>>(*d_input_ptr,
                                                    in_offsets,
                                                    path_commands,
                                                    out_stringviews.data(),
                                                    output_scratch.data(),
                                                    has_out_of_bound.data());

  // If we didn't see any out-of-bound write, everything is good so far.
  // Just gather the output strings and return.
  if (!has_out_of_bound.value(stream)) {
    return cudf::make_strings_column(out_stringviews, stream, mr);
  }
  // From here, we had out-of-bound write. Although this is very rare, it may still happen.

  // This scratch buffer is no longer needed.
  output_scratch = rmm::device_uvector<char>{0, stream};

  // The string sizes computed in the previous kernel call will be used to allocate a new char
  // buffer to store the output.
  auto const size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [string_pairs = out_stringviews.data()] __device__(auto const idx) {
        return string_pairs[idx].second;
      }));
  auto [offsets, output_size] =
    cudf::strings::detail::make_offsets_child_column(size_it, size_it + input.size(), stream, mr);

  // Also compute the null mask using the stored char pointers.
  auto const validator = [] __device__(thrust::pair<char const*, cudf::size_type> const item) {
    return item.first != nullptr;
  };
  auto [null_mask, null_count] =
    cudf::detail::valid_if(out_stringviews.begin(), out_stringviews.end(), validator, stream, mr);

  // No longer need it from here. Free up memory for now.
  out_stringviews = rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{0, stream};

  auto chars             = rmm::device_uvector<char>(output_size, stream, mr);
  auto const out_offsets = cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  has_out_of_bound.set_value_to_zero_async(stream);
  get_json_object_kernel<block_size>
    <<<num_blocks, block_size, 0, stream.value()>>>(*d_input_ptr,
                                                    out_offsets,
                                                    path_commands,
                                                    nullptr /*out_stringviews*/,
                                                    chars.data(),
                                                    has_out_of_bound.data());

  // This kernel call should not see out-of-bound write. If it is still detected, there must be
  // something wrong happened.
  CUDF_EXPECTS(!has_out_of_bound.value(stream),
               "Unexpected out-of-bound write in get_json_object kernel.");

  return cudf::make_strings_column(
    input.size(), std::move(offsets), chars.release(), null_count, std::move(null_mask));
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  return detail::get_json_object(input, instructions, stream, mr);
}

}  // namespace spark_rapids_jni
