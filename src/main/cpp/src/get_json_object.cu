#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"
#include "json_parser.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

namespace spark_rapids_jni {

namespace detail {

// path max depth limitation
// There is a same constant in JSONUtil.java, keep them consistent when changing
// Note: Spark-Rapids should guarantee the path depth is less or equal to this limit,
// or GPU reports hipErrorIllegalAddress
constexpr int max_path_depth = 16;

/**
 * write JSON style
 */
enum class write_style : int8_t { RAW, QUOTED, FLATTEN };

/**
 * path instruction
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};

  path_instruction_type type;
};

/**
 * JSON generator is used to write out JSON content.
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(int _offset = 0) : offset(_offset), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator() const
  {
    return json_generator(offset + output_len);
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array(char* out_begin)
  {
    try_write_comma(out_begin);

    out_begin[offset + output_len] = '[';
    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array(char* out_begin)
  {
    out_begin[offset + output_len] = ']';
    output_len++;
    // point to parent array
    array_depth--;
    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() const { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma(char* out_begin)
  {
    if (need_comma()) {
      // in array context and writes first item
      out_begin[offset + output_len] = ',';
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser, char* out_begin)
  {
    // first try add comma
    try_write_comma(out_begin);

    if (array_depth > 0) { is_curr_array_empty = false; }

    auto [b, copy_len] = parser.copy_current_structure(out_begin + offset + output_len);
    output_len += copy_len;
    return b;
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser, char* out_begin)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    auto copied = parser.write_unescaped_text(out_begin + offset + output_len);
    output_len += copied;
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        int child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (write_outer_array_tokens) {
      if (insert_comma) {
        *(child_block_begin + child_block_len + 2) = ']';
        move_forward(child_block_begin, child_block_len, 2);
        *(child_block_begin + 1) = '[';
        *(child_block_begin)     = ',';
      } else {
        *(child_block_begin + child_block_len + 1) = ']';
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = '[';
      }
    } else {
      if (insert_comma) {
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = ',';
      } else {
        // do not need comma && do not need write outer array tokens
        // do nothing, because child generator buff is directly after the
        // parent generator
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  static __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ inline int get_offset() const { return offset; }
  __device__ inline int get_output_len() const { return output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  int offset;  // offset from the global output buffer
  int output_len;

  int array_depth = 0;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

enum class evaluation_case_path : int8_t {
  INVALID                                           = -1,
  START_ARRAY___EMPTY_PATH___FLATTEN_STYLE          = 2,
  START_OBJECT___MATCHED_NAME_PATH                  = 4,
  START_ARRAY___MATCHED_DOUBLE_WILDCARD             = 5,
  START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED = 6,
  START_ARRAY___MATCHED_WILDCARD                    = 7,
  START_ARRAY___MATCHED_INDEX_AND_WILDCARD          = 8,
  START_ARRAY___MATCHED_INDEX                       = 9
};

struct context {
  // used to save current generator
  json_generator g;

  // used to save child JSON generator for case path 6
  json_generator child_g;

  cudf::device_span<path_instruction const> path;

  // whether written output
  // if dirty > 0, indicates success
  int dirty;

  // which case path that this task is from
  evaluation_case_path case_path;

  // current token
  json_token token;

  write_style style;

  // for some case paths
  bool is_first_enter;

  // is this context task is done
  bool task_is_done;
};

/**
 * @brief Parse a single json string using the provided command buffer.
 *
 * @param input The incoming json string
 * @param path_commands The command buffer to be applied to the string
 * @param out_buf Buffer user to store the string resulted from the query
 * @return A pair containing the result code and the output size
 */
__device__ thrust::pair<bool, cudf::size_type> evaluate_path(
  char_range input, cudf::device_span<path_instruction const> path_commands, char* out_buf)
{
  json_parser p{input};
  p.next_token();
  if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

  // define stack; plus 1 indicates root context task needs an extra memory
  context stack[max_path_depth + 1];
  int stack_size = 0;

  // push context function
  auto push_context = [&p, &stack, &stack_size](evaluation_case_path _case_path,
                                                json_generator _g,
                                                write_style _style,
                                                cudf::device_span<path_instruction const> _path) {
    // no need to check stack is full
    // because Spark-Rapids already checked maximum length of `path_instruction`
    auto& ctx          = stack[stack_size++];
    ctx.g              = _g;
    ctx.path           = _path;
    ctx.dirty          = 0;
    ctx.case_path      = _case_path;
    ctx.token          = p.get_current_token();
    ctx.style          = _style;
    ctx.is_first_enter = true;
    ctx.task_is_done   = false;
  };

  // put the first context task
  push_context(evaluation_case_path::INVALID, json_generator{}, write_style::RAW, path_commands);

  while (stack_size > 0) {
    auto& ctx = stack[stack_size - 1];
    if (!ctx.task_is_done) {
      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size()) &&
          ctx.style == write_style::RAW) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.write_raw(p, out_buf);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // push back task
          // add child task
          push_context(evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE,
                       ctx.g,
                       ctx.style,
                       {nullptr, 0});
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // general case: just copy the child tree verbatim
        if (!(ctx.g.copy_current_structure(p, out_buf))) {
          // JSON validation check
          return {false, 0};
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          // Mark task is done regardless whether the expected child was found.
          ctx.task_is_done = true;
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return {false, 0}; }
              // push sub task; sub task will update the result of path 4
              push_context(evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1});
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          if (!found_expected_child) {
            // did not find any expected sub child
            ctx.task_is_done = true;
            ctx.dirty        = false;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: Wildcard :: xs)
      // case path 5
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_elements(
                 ctx.path, path_instruction_type::WILDCARD, path_instruction_type::WILDCARD)) {
        // special handling for the non-structure preserving double wildcard
        // behavior in Hive
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          push_context(evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD,
                       ctx.g,
                       write_style::FLATTEN,
                       {ctx.path.data() + 2, ctx.path.size() - 2});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
      // case path 6
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD) &&
               ctx.style != write_style::QUOTED) {
        // retain Flatten, otherwise use Quoted... cannot use Raw within an array
        write_style next_style = write_style::RAW;
        switch (ctx.style) {
          case write_style::RAW: next_style = write_style::QUOTED; break;
          case write_style::FLATTEN: next_style = write_style::FLATTEN; break;
          case write_style::QUOTED: next_style = write_style::QUOTED;  // never happen
        }

        // temporarily buffer child matches, the emitted json will need to be
        // modified slightly if there is only a single element written

        json_generator child_g;
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          // create a child generator with hide outer array tokens mode.
          child_g = ctx.g.new_child_generator();
          // write first [ without output, without update len, only update internal state
          child_g.write_first_start_array_without_output();
        } else {
          child_g = ctx.child_g;
        }

        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // track the number of array elements and only emit an outer array if
          // we've written more than one element, this matches Hive's behavior
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED,
                       child_g,
                       next_style,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          char* child_g_start = out_buf + child_g.get_offset();
          int child_g_len     = child_g.get_output_len();
          if (ctx.dirty > 1) {
            // add outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ true);
          } else if (ctx.dirty == 1) {
            // remove outer array tokens
            ctx.g.write_child_raw_value(
              child_g_start, child_g_len, /* write_outer_array_tokens */ false);
          }  // else do not write anything

          // Done anyway, since we already reached the end array.
          ctx.task_is_done = true;
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_start_array(out_buf);
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1});
        } else {
          ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
      // case path 8
      else if (json_token::START_ARRAY == ctx.token &&
               thrust::get<0>(path_match_index_wildcard(ctx.path))) {
        int idx = thrust::get<1>(path_match_index_wildcard(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
        ctx.is_first_enter = false;

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD,
                     ctx.g,
                     write_style::QUOTED,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case (START_ARRAY, Index(idx) :: xs)
      // case path 9
      else if (json_token::START_ARRAY == ctx.token && thrust::get<0>(path_match_index(ctx.path))) {
        int idx = thrust::get<1>(path_match_index(ctx.path));

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

        int i = idx;
        while (i > 0) {
          if (p.get_current_token() == json_token::END_ARRAY) {
            // terminate, nothing has been written
            return {false, 0};
          }

          if (!p.try_skip_children()) { return {false, 0}; }

          p.next_token();
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          --i;
        }

        // i == 0
        push_context(evaluation_case_path::START_ARRAY___MATCHED_INDEX,
                     ctx.g,
                     ctx.style,
                     {ctx.path.data() + 1, ctx.path.size() - 1});
      }
      // case _ =>
      // case path 12
      else {
        if (!p.try_skip_children()) { return {false, 0}; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    }       // if (!ctx.task_is_done)
    else {  // current context is done.
      // pop current top context
      stack_size--;

      // has no parent task, stack is empty, will exit
      if (stack_size == 0) { break; }

      // peek parent context task
      // update parent task info according to current task result
      auto& p_ctx = stack[stack_size - 1];

      switch (ctx.case_path) {
          // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
          // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
          // path 7: case (START_ARRAY, Wildcard :: xs)
        case evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE:
        case evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_OBJECT, Named :: xs)
          // case path 4
        case evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH: {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
          // case path 6
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;

          break;
        }

          /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
          // case path 8
          // case (START_ARRAY, Index(idx) :: xs)
          // case path 9
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return {false, 0}; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

        default:;  // Never happens!
      }            // end switch (ctx.case_path)

    }  // ctx.task_is_done
  }    // while (stack_size > 0)

  auto const success = stack[0].dirty > 0;

  // generator may contain trash output, e.g.: generator writes some output,
  // then JSON format is invalid, the previous output becomes trash.
  // We need to return output size as zero.
  return {success, success ? stack[0].g.get_output_len() : 0};
}

/**
 * @brief TODO
 */
struct json_path_query_data {
  json_path_query_data(cudf::device_span<path_instruction const> _path,
                       cudf::detail::input_offsetalator _offsets,
                       thrust::pair<char const*, cudf::size_type>* _out_stringviews,
                       char* _out_buf,
                       int8_t* _has_out_of_bound)
    : path_commands{_path},
      offsets{_offsets},
      out_stringviews{_out_stringviews},
      out_buf{_out_buf},
      has_out_of_bound{_has_out_of_bound}
  {
  }
  cudf::device_span<path_instruction const> path_commands;
  cudf::detail::input_offsetalator offsets;
  thrust::pair<char const*, cudf::size_type>* out_stringviews;
  char* out_buf;
  int8_t* has_out_of_bound;
};

__device__ __forceinline__ void process_row(cudf::column_device_view input,
                                            cudf::device_span<json_path_query_data> query_data,
                                            int64_t idx)
{
  auto const row_idx   = idx / query_data.size();
  auto const query_idx = idx % query_data.size();
  auto const& query    = query_data[query_idx];

  char* const dst          = query.out_buf + query.offsets[row_idx];
  bool is_valid            = false;
  cudf::size_type out_size = 0;

  auto const str = input.element<cudf::string_view>(row_idx);
  if (str.size_bytes() > 0) {
    thrust::tie(is_valid, out_size) = evaluate_path(char_range{str}, query.path_commands, dst);

    auto const max_size = query.offsets[row_idx + 1] - query.offsets[row_idx];
    if (out_size > max_size) { *(query.has_out_of_bound) = 1; }
  }

  // Write out `nullptr` in the output string_view to indicate that the output is a null.
  // The situation `out_stringviews == nullptr` should only happen if the kernel is launched a
  // second time due to out-of-bound write in the first launch.
  if (query.out_stringviews) {
    query.out_stringviews[row_idx] = {is_valid ? dst : nullptr, out_size};
  }
}

/**
 * @brief Kernel for running the JSONPath query, processing one row per thread.
 *
 * This kernel writes out the output strings and their lengths at the same time. If any output
 * length exceed buffer size limit, a boolean flag will be turned on to inform to the caller.
 * In such situation, another (larger) output buffer will be generated and the kernel is launched
 * again. Otherwise, launching this kernel only once is sufficient to produce the desired output.
 *
 * @param input The input JSON strings stored in a strings column
 * @param query_data TODO
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel_row_per_thread(cudf::column_device_view input,
                                             cudf::device_span<json_path_query_data> query_data)
{
  auto const max_tid = static_cast<int64_t>(input.size()) * query_data.size();
  auto const stride  = cudf::detail::grid_1d::grid_stride();

  for (auto tid = cudf::detail::grid_1d::global_thread_id(); tid < max_tid; tid += stride) {
    process_row(input, query_data, tid);
  }
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel writes out the output strings and their lengths at the same time. If any output
 * length exceed buffer size limit, a boolean flag will be turned on to inform to the caller.
 * In such situation, another (larger) output buffer will be generated and the kernel is launched
 * again. Otherwise, launching this kernel only once is sufficient to produce the desired output.
 *
 * @param input The input JSON strings stored in a strings column
 * @param offsets Offsets to the output locations in the output buffer
 * @param path_commands JSONPath command buffer
 * @param out_stringviews The output array to store pointers to the output strings and their sizes
 * @param out_buf Buffer used to store the strings resulted from the query
 * @param has_out_of_bound Flag to indicate if any output string has length exceeds its buffer size
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel_row_per_warp(cudf::column_device_view input,
                                           cudf::device_span<json_path_query_data> query_data)
{
  auto const max_tid =
    static_cast<int64_t>(input.size()) * cudf::detail::warp_size * query_data.size();
  auto const stride  = cudf::detail::grid_1d::grid_stride();
  auto const lane_id = threadIdx.x % cudf::detail::warp_size;

  for (auto tid = cudf::detail::grid_1d::global_thread_id(); tid < max_tid; tid += stride) {
    if (lane_id == 0) {
      auto const warp_idx = tid / cudf::detail::warp_size;
      process_row(input, query_data, warp_idx);
    }  // done lane_id == 0
    __syncwarp();
  }
}

void launch_kernel(bool exec_row_per_thread,
                   cudf::column_device_view const& input,
                   cudf::device_span<json_path_query_data> query_data,
                   rmm::cuda_stream_view stream)
{
  auto const get_SM_count = []() {
    int device_id{};
    hipDeviceProp_t props{};
    CUDF_CUDA_TRY(hipGetDevice(&device_id));
    CUDF_CUDA_TRY(hipGetDeviceProperties(&props, device_id));
    return props.multiProcessorCount;
  };

  // We explicitly set the minBlocksPerMultiprocessor parameter in the launch bounds to avoid
  // spilling from the kernel itself. By default NVCC uses a heuristic to find a balance between
  // the maximum number of registers used by a kernel and the parallelism of the kernel.
  // If lots of registers are used the parallelism may suffer. But in our case
  // NVCC gets this wrong and we want to avoid spilling all the time or else
  // the performance is really bad. This essentially tells NVCC to prefer using lots
  // of registers over spilling.
  if (exec_row_per_thread) {
    constexpr int block_size             = 256;
    constexpr int min_block_per_sm       = 1;
    constexpr int block_count_multiplier = 1;
    static auto const num_blocks         = get_SM_count() * block_count_multiplier;
    get_json_object_kernel_row_per_thread<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(input, query_data);
  } else {
    constexpr int block_size             = 512;
    constexpr int min_block_per_sm       = 2;
    constexpr int block_count_multiplier = 8;
    static auto const num_blocks         = get_SM_count() * block_count_multiplier;
    get_json_object_kernel_row_per_warp<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(input, query_data);
  }
}

std::tuple<std::vector<rmm::device_uvector<path_instruction>>,
           std::unique_ptr<std::vector<std::vector<path_instruction>>>,
           cudf::string_scalar,
           std::string>
construct_path_commands(
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    instruction_array,
  rmm::cuda_stream_view stream)
{
  auto h_inst_names = [&] {
    std::size_t length{0};
    for (auto const& instructions : instruction_array) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { length += name.length(); }
      }
    }

    std::string all_names;
    all_names.reserve(length);
    for (auto const& instructions : instruction_array) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { all_names += name; }
      }
    }
    return all_names;
  }();
  auto d_inst_names = cudf::string_scalar(h_inst_names, true, stream);

  std::size_t name_pos{0};
  auto h_path_commands = std::make_unique<std::vector<std::vector<path_instruction>>>();
  h_path_commands->reserve(instruction_array.size());

  for (auto const& instructions : instruction_array) {
    h_path_commands->emplace_back();
    auto& path_commands = h_path_commands->back();
    path_commands.reserve(instructions.size());

    for (auto const& [type, name, index] : instructions) {
      path_commands.emplace_back(path_instruction{type});

      if (type == path_instruction_type::INDEX) {
        path_commands.back().index = index;
      } else if (type == path_instruction_type::NAMED) {
        path_commands.back().name = cudf::string_view(d_inst_names.data() + name_pos, name.size());
        name_pos += name.size();
      } else if (type != path_instruction_type::WILDCARD) {
        CUDF_FAIL("Invalid path instruction type");
      }
    }
  }

  auto d_path_commands = std::vector<rmm::device_uvector<path_instruction>>{};
  d_path_commands.reserve(h_path_commands->size());
  for (auto const& path_commands : *h_path_commands) {
    d_path_commands.emplace_back(cudf::detail::make_device_uvector_async(
      path_commands, stream, rmm::mr::get_current_device_resource()));
  }

  // h_path_commands needs to be kept alive outside of this function due to async copy.
  return {std::move(d_path_commands),
          std::move(h_path_commands),
          std::move(d_inst_names),
          std::move(h_inst_names)};
}

std::vector<std::unique_ptr<cudf::column>> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    instruction_array,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const num_outputs = instruction_array.size();
  std::vector<std::unique_ptr<cudf::column>> output;
  if (input.is_empty()) {
    for (std::size_t idx = 0; idx < num_outputs; ++idx) {
      output.emplace_back(cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING}));
    }
    return output;
  }

  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  auto const in_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());

  auto const [max_row_size, sum_row_size] =
    thrust::transform_reduce(rmm::exec_policy(stream),
                             thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(input.size()),
                             cuda::proclaim_return_type<thrust::pair<int64_t, int64_t>>(
                               [in_offsets] __device__(auto const idx) {
                                 auto const size = in_offsets[idx + 1] - in_offsets[idx];
                                 return thrust::pair<int64_t, int64_t>{size, size};
                               }),
                             thrust::pair<int64_t, int64_t>{0, 0},
                             cuda::proclaim_return_type<thrust::pair<int64_t, int64_t>>(
                               [] __device__(auto const& lhs, auto const& rhs) {
                                 return thrust::pair<int64_t, int64_t>{
                                   std::max(lhs.first, rhs.first), lhs.second + rhs.second};
                               }));

  // A buffer to store the output strings without knowing their sizes.
  // Since we do not know their sizes, we need to allocate the buffer a bit larger than the input
  // size so that we will not write output strings into an out-of-bound position.
  // Checking out-of-bound needs to be performed in the main kernel to make sure we will not have
  // data corruption.
  auto const scratch_size = [&, max_row_size = max_row_size] {
    // Pad the scratch buffer by an additional size that is a multiple of max row size.
    auto constexpr padding_rows = 10;
    return input.chars_size(stream) + max_row_size * padding_rows;
  }();

  std::vector<rmm::device_uvector<char>> scratch_buffers;
  std::vector<rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>> out_stringviews;
  std::vector<json_path_query_data> h_query_data;
  scratch_buffers.reserve(instruction_array.size());
  out_stringviews.reserve(instruction_array.size());
  h_query_data.reserve(instruction_array.size());

  rmm::device_uvector<int8_t> d_has_out_of_bound(num_outputs, stream);

  auto const [d_json_paths, h_json_paths, d_inst_names, h_inst_names] =
    construct_path_commands(instruction_array, stream);

  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& instructions = instruction_array[idx];
    if (instructions.size() > max_path_depth) { CUDF_FAIL("JSONPath query exceeds maximum depth"); }

    scratch_buffers.emplace_back(rmm::device_uvector<char>(scratch_size, stream));
    out_stringviews.emplace_back(rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{
      static_cast<std::size_t>(input.size()), stream});

    h_query_data.emplace_back(d_json_paths[idx],
                              in_offsets,
                              out_stringviews.back().data(),
                              scratch_buffers.back().data(),
                              d_has_out_of_bound.data() + idx);
  }
  auto d_query_data = cudf::detail::make_device_uvector_async(
    h_query_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_has_out_of_bound.begin(), d_has_out_of_bound.end(), 0);

  // Threshold to decide on using row per thread or row per warp functions.
  constexpr int64_t AVG_CHAR_BYTES_THRESHOLD = 128;
  auto const exec_row_per_thread =
    (sum_row_size / (input.size() - input.null_count())) < AVG_CHAR_BYTES_THRESHOLD;
  launch_kernel(exec_row_per_thread, *d_input_ptr, d_query_data, stream);

  auto h_has_out_of_bound = cudf::detail::make_host_vector_sync(d_has_out_of_bound, stream);
  auto has_no_oob         = std::none_of(
    h_has_out_of_bound.begin(), h_has_out_of_bound.end(), [](auto const val) { return val != 0; });

  // If we didn't see any out-of-bound write, everything is good so far.
  // Just gather the output strings and return.
  if (has_no_oob) {
    for (auto const& out_sview : out_stringviews) {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
    return output;
  }
  // From here, we had out-of-bound write. Although this is very rare, it may still happen.

  std::vector<std::pair<rmm::device_buffer, cudf::size_type>> out_null_masks_and_null_counts;
  std::vector<std::pair<std::unique_ptr<cudf::column>, int64_t>> out_offsets_and_sizes;
  std::vector<rmm::device_uvector<char>> out_char_buffers;
  std::vector<std::size_t> oob_indices;

  // Check validity from the stored char pointers.
  auto const validator = [] __device__(thrust::pair<char const*, cudf::size_type> const item) {
    return item.first != nullptr;
  };

  h_query_data.clear();
  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& out_sview = out_stringviews[idx];

    if (h_has_out_of_bound[idx]) {
      oob_indices.emplace_back(idx);
      output.emplace_back(nullptr);  // just placeholder.

      out_null_masks_and_null_counts.emplace_back(
        cudf::detail::valid_if(out_sview.begin(), out_sview.end(), validator, stream, mr));

      // The string sizes computed in the previous kernel call will be used to allocate a new char
      // buffer to store the output.
      auto const size_it = cudf::detail::make_counting_transform_iterator(
        0,
        cuda::proclaim_return_type<cudf::size_type>(
          [string_pairs = out_sview.data()] __device__(auto const idx) {
            return string_pairs[idx].second;
          }));
      out_offsets_and_sizes.emplace_back(cudf::strings::detail::make_offsets_child_column(
        size_it, size_it + input.size(), stream, mr));
      out_char_buffers.emplace_back(
        rmm::device_uvector<char>(out_offsets_and_sizes.back().second, stream, mr));

      h_query_data.emplace_back(d_json_paths[idx],
                                cudf::detail::offsetalator_factory::make_input_iterator(
                                  out_offsets_and_sizes.back().first->view()),
                                nullptr /*out_stringviews*/,
                                out_char_buffers.back().data(),
                                d_has_out_of_bound.data() + idx);
    } else {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
  }
  // These buffers are no longer needed.
  scratch_buffers.clear();
  out_stringviews.clear();

  d_query_data = cudf::detail::make_device_uvector_async(
    h_query_data, stream, rmm::mr::get_current_device_resource());

  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_has_out_of_bound.begin(), d_has_out_of_bound.end(), 0);

  launch_kernel(exec_row_per_thread, *d_input_ptr, d_query_data, stream);

  // Check out of bound again for sure.
  h_has_out_of_bound = cudf::detail::make_host_vector_sync(d_has_out_of_bound, stream);
  has_no_oob         = std::none_of(
    h_has_out_of_bound.begin(), h_has_out_of_bound.end(), [](auto const val) { return val != 0; });

  // The last kernel call should not encounter any out-of-bound write.
  // If it is still detected, there must be something wrong happened.
  CUDF_EXPECTS(has_no_oob, "Unexpected out-of-bound write in get_json_object kernel.");

  for (auto const idx : oob_indices) {
    output[idx] = cudf::make_strings_column(input.size(),
                                            std::move(out_offsets_and_sizes[idx].first),
                                            out_char_buffers[idx].release(),
                                            out_null_masks_and_null_counts[idx].second,
                                            std::move(out_null_masks_and_null_counts[idx].first));
  }
  return output;
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::tuple<path_instruction_type, std::string, int64_t>> const& instructions,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  return std::move(detail::get_json_object(input, {instructions}, stream, mr).front());
}

std::vector<std::unique_ptr<cudf::column>> get_json_object_multiple_paths(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int64_t>>> const&
    instruction_array,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  return detail::get_json_object(input, instruction_array, stream, mr);
}

}  // namespace spark_rapids_jni
