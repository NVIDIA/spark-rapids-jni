#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cast_string.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/convert/int_to_string.cuh>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/generate.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/optional.h>
#include <thrust/transform.h>

#include <cuda/std/climits>
#include <cuda/std/limits>
#include <cuda/std/type_traits>

using namespace cudf;

namespace spark_rapids_jni {

namespace detail {
namespace {
constexpr int max_zeros = 5;

template <typename DecimalType>
struct decimal_to_non_ansi_string_fn {
  column_device_view d_decimals;
  offset_type* d_offsets{};
  char* d_chars{};

  /**
   * @brief Calculates the size of the string required to convert the element, in base-10 format.
   *
   * @note This code does not properly handle a max negative decimal value and will overflow. This isn't an issue here because Spark will not use the full range of values and will never cause this issue.
   *
   * Output format is [-]integer.fraction
   */
  __device__ int32_t compute_output_size(DecimalType value)
  {
    auto const scale = d_decimals.type().scale();

    if (scale >= 0) return strings::detail::count_digits(value) + scale;

    auto const abs_value = numeric::detail::abs(value);
    auto const exp_ten   = numeric::detail::exp10<DecimalType>(-scale);
    auto const fraction  = strings::detail::count_digits(abs_value % exp_ten);
    auto const num_zeros = std::max(0, (-scale - fraction));

    return value == 0 && num_zeros > max_zeros && scale < 0
             ? static_cast<int32_t>(value < 0) +                       // sign if negative
                 strings::detail::count_digits(abs_value / exp_ten) +  // integer
                 2 +                                                   // E-
                 strings::detail::count_digits(num_zeros + 1)
             :  // number of zeros

             static_cast<int32_t>(value < 0) +                       // sign if negative
               strings::detail::count_digits(abs_value / exp_ten) +  // integer
               1 +                                                   // decimal point
               num_zeros +                                           // zeros padding
               fraction;                                             // size of fraction
  }

  /**
   * @brief Converts a decimal element into a string.
   *
   * @note This code does not properly handle a max negative decimal value and will overflow. This isn't an issue here because Spark will not use the full range of values and will never cause this issue.
   *
   * The value is converted into base-10 digits [0-9]
   * plus the decimal point and a negative sign prefix.
   */
  __device__ void decimal_to_non_ansi_string(size_type idx)
  {
    auto const value = d_decimals.element<DecimalType>(idx);
    auto const scale = d_decimals.type().scale();
    char* d_buffer   = d_chars + d_offsets[idx];

    if (scale >= 0) {
      d_buffer += strings::detail::integer_to_string(value, d_buffer);
      thrust::generate_n(thrust::seq, d_buffer, scale, []() { return '0'; });  // add zeros
      return;
    }

    // scale < 0
    // write format:   [-]integer.fraction
    // where integer  = abs(value) / (10^abs(scale))
    //       fraction = abs(value) % (10^abs(scale))
    if (value < 0) *d_buffer++ = '-';  // add sign
    auto const abs_value = numeric::detail::abs(value);
    auto const exp_ten   = numeric::detail::exp10<DecimalType>(-scale);
    auto const num_zeros =
      std::max(0, (-scale - strings::detail::count_digits(abs_value % exp_ten)));

    if (value == 0 && num_zeros > max_zeros) {
      *d_buffer++ = '0';
      *d_buffer++ = 'E';
      *d_buffer++ = '-';
      d_buffer += strings::detail::integer_to_string(num_zeros + 1, d_buffer);
      return;
    }

    d_buffer +=
      strings::detail::integer_to_string(abs_value / exp_ten, d_buffer);  // add the integer part
    *d_buffer++ = '.';                                                    // add decimal point

    thrust::generate_n(thrust::seq, d_buffer, num_zeros, []() { return '0'; });  // add zeros
    d_buffer += num_zeros;

    strings::detail::integer_to_string(abs_value % exp_ten, d_buffer);  // add the fraction part
  }

  __device__ void operator()(size_type idx)
  {
    if (d_decimals.is_null(idx)) {
      if (d_chars == nullptr) { d_offsets[idx] = 0; }
      return;
    }
    if (d_chars != nullptr) {
      decimal_to_non_ansi_string(idx);
    } else {
      d_offsets[idx] = compute_output_size(d_decimals.element<DecimalType>(idx));
    }
  }
};

/**
 * @brief The dispatcher functor for converting fixed-point values into strings.
 */
struct dispatch_decimal_to_non_ansi_string_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;  // underlying value type

    auto const d_column = column_device_view::create(input, stream);

    auto [offsets, chars] = strings::detail::make_strings_children(
      decimal_to_non_ansi_string_fn<DecimalType>{*d_column}, input.size(), stream, mr);

    return make_strings_column(input.size(),
                               std::move(offsets),
                               std::move(chars),
                               input.null_count(),
                               cudf::detail::copy_bitmask(input, stream, mr));
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Values for decimal_to_non_ansi_string function must be a decimal type.");
  }
};

}  // namespace

std::unique_ptr<column> decimal_to_non_ansi_string(column_view const& input,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return make_empty_column(type_id::STRING);
  return type_dispatcher(input.type(), dispatch_decimal_to_non_ansi_string_fn{}, input, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> decimal_to_non_ansi_string(column_view const& input,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::decimal_to_non_ansi_string(input, stream, mr);
}

}  // namespace spark_rapids_jni
