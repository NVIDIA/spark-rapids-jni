#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "from_json.hpp"
#include "get_json_object.hpp"
#include "json_parser.cuh"

// #include <cudf_test/debug_utilities.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/io/json.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

#include <numeric>
#include <unordered_set>

namespace spark_rapids_jni {

namespace detail {

namespace test {

/**
 * @brief JSON style to write.
 */
enum class write_style : int8_t { RAW, QUOTED, FLATTEN };

/**
 * @brief Instruction along a JSON path.
 */
struct path_instruction {
  __device__ inline path_instruction(path_instruction_type _type) : type(_type) {}

  // used when type is named type
  cudf::string_view name;

  // used when type is index
  int index{-1};

  path_instruction_type type;
};

/**
 * @brief JSON generator used to write out JSON content.
 *
 * Because of get_json_object only outputs JSON object as a whole item,
 * it's no need to store internal state for JSON object when outputing,
 * only need to store internal state for JSON array.
 */
class json_generator {
 public:
  __device__ json_generator(int _offset = 0) : offset(_offset), output_len(0) {}

  // create a nested child generator based on this parent generator,
  // child generator is a view, parent and child share the same byte array
  __device__ json_generator new_child_generator() const
  {
    return json_generator(offset + output_len);
  }

  // write [
  // add an extra comma if needed,
  // e.g.: when JSON content is: [[1,2,3]
  // writing a new [ should result: [[1,2,3],[
  __device__ void write_start_array(char* out_begin, char element_delimiter)
  {
    try_write_comma(out_begin, element_delimiter);

    out_begin[offset + output_len] = '[';
    output_len++;
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // write ]
  __device__ void write_end_array(char* out_begin)
  {
    out_begin[offset + output_len] = ']';
    output_len++;
    // point to parent array
    array_depth--;
    // set parent array as non-empty because already had a closed child item.
    is_curr_array_empty = false;
  }

  // write first start array without output, only update internal state
  __device__ void write_first_start_array_without_output()
  {
    // hide the outer start array token
    // Note: do not inc output_len
    array_depth++;
    // new array is empty
    is_curr_array_empty = true;
  }

  // return true if it's in a array context and it's not writing the first item.
  __device__ inline bool need_comma() const { return (array_depth > 0 && !is_curr_array_empty); }

  /**
   * write comma accroding to current generator state
   */
  __device__ void try_write_comma(char* out_begin, char element_delimiter)
  {
    if (need_comma()) {
      // in array context and writes first item
      out_begin[offset + output_len] = element_delimiter;
      output_len++;
    }
  }

  /**
   * copy current structure when parsing. If current token is start
   * object/array, then copy to corresponding matched end object/array. return
   * false if JSON format is invalid return true if JSON format is valid
   */
  __device__ bool copy_current_structure(json_parser& parser,
                                         char* out_begin,
                                         char element_delimiter)
  {
    // first try add comma
    try_write_comma(out_begin, element_delimiter);

    if (array_depth > 0) { is_curr_array_empty = false; }

    // printf("parser line %d\n", __LINE__);

    auto [b, copy_len] = parser.copy_current_structure(out_begin + offset + output_len);
    output_len += copy_len;
    return b;
  }

  static __device__ cudf::size_type write_quote(char* out, bool keep_quotes)
  {
    if (!keep_quotes) { return 0; }
    *out = '"';
    return 1;
  }

  /**
   * Get current text from JSON parser and then write the text
   * Note: Because JSON strings contains '\' to do escape,
   * JSON parser should do unescape to remove '\' and JSON parser
   * then can not return a pointer and length pair (char *, len),
   * For number token, JSON parser can return a pair (char *, len)
   */
  __device__ void write_raw(json_parser& parser, char* out_begin, bool keep_quotes)
  {
    if (array_depth > 0) { is_curr_array_empty = false; }

    output_len += write_quote(out_begin + offset + output_len, keep_quotes);
    output_len += parser.write_unescaped_text(out_begin + offset + output_len);
    output_len += write_quote(out_begin + offset + output_len, keep_quotes);
  }

  __device__ void write_null_placeholder(char* out_begin, char null)
  {
    out_begin[offset + output_len] = null;
    output_len += 1;
    is_curr_array_empty = false;
  }

  /**
   * write child raw value
   * e.g.:
   *
   * write_outer_array_tokens = false
   * need_comma = true
   * [1,2,3]1,2,3
   *        ^
   *        |
   *    child pointer
   * ==>>
   * [1,2,3],1,2,3
   *
   *
   * write_outer_array_tokens = true
   * need_comma = true
   *   [12,3,4
   *     ^
   *     |
   * child pointer
   * ==>>
   *   [1,[2,3,4]
   *
   * For more information about param write_outer_array_tokens, refer to
   * `write_first_start_array_without_output`
   * @param child_block_begin
   * @param child_block_len
   * @param write_outer_array_tokens whether write outer array tokens for child
   * block
   */
  __device__ void write_child_raw_value(char* child_block_begin,
                                        int child_block_len,
                                        bool write_outer_array_tokens)
  {
    bool insert_comma = need_comma();

    if (array_depth > 0) { is_curr_array_empty = false; }

    if (write_outer_array_tokens) {
      if (insert_comma) {
        *(child_block_begin + child_block_len + 2) = ']';
        move_forward(child_block_begin, child_block_len, 2);
        *(child_block_begin + 1) = '[';
        *(child_block_begin)     = ',';
      } else {
        *(child_block_begin + child_block_len + 1) = ']';
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = '[';
      }
    } else {
      if (insert_comma) {
        move_forward(child_block_begin, child_block_len, 1);
        *(child_block_begin) = ',';
      } else {
        // do not need comma && do not need write outer array tokens
        // do nothing, because child generator buff is directly after the
        // parent generator
      }
    }

    // update length
    if (insert_comma) { output_len++; }
    if (write_outer_array_tokens) { output_len += 2; }
    output_len += child_block_len;
  }

  // move memory block forward by specified bytes
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 2, after moving,
  // memory is: 1 2 1 2.
  // e.g.:  memory is: 1 2 0 0, begin is 1, len is 1, after moving,
  // memory is: 1 1 2 0.
  // Note: should move from end to begin to avoid overwrite buffer
  static __device__ void move_forward(char* begin, size_t len, int forward)
  {
    // TODO copy by 8 bytes
    char* pos = begin + len + forward - 1;
    char* e   = begin + forward - 1;
    while (pos > e) {
      *pos = *(pos - forward);
      pos--;
    }
  }

  __device__ inline int get_offset() const { return offset; }
  __device__ inline int get_output_len() const { return output_len; }

  /**
   * generator may contain trash output, e.g.: generator writes some output,
   * then JSON format is invalid, the previous output becomes trash.
   */
  __device__ inline void set_output_len_zero() { output_len = 0; }

  __device__ inline void set_output_len(size_t len) { output_len = len; }

 private:
  int offset;  // offset from the global output buffer
  int output_len;

  int array_depth = 0;

  // whether already worte a item in current array
  // used to decide whether add a comma before writing out a new item.
  bool is_curr_array_empty;
};

/**
 * path evaluator which can run on both CPU and GPU
 */
__device__ inline bool path_is_empty(size_t path_size) { return path_size == 0; }

__device__ inline bool path_match_element(cudf::device_span<path_instruction const> path,
                                          path_instruction_type path_type0)
{
  if (path.size() < 1) { return false; }
  return path.data()[0].type == path_type0;
}

__device__ inline bool path_match_elements(cudf::device_span<path_instruction const> path,
                                           path_instruction_type path_type0,
                                           path_instruction_type path_type1)
{
  if (path.size() < 2) { return false; }
  return path.data()[0].type == path_type0 && path.data()[1].type == path_type1;
}

__device__ inline thrust::tuple<bool, int> path_match_index(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::INDEX);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

__device__ inline thrust::tuple<bool, cudf::string_view> path_match_named(
  cudf::device_span<path_instruction const> path)
{
  auto match = path_match_element(path, path_instruction_type::NAMED);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

__device__ inline thrust::tuple<bool, int> path_match_index_wildcard(
  cudf::device_span<path_instruction const> path)
{
  auto match =
    path_match_elements(path, path_instruction_type::INDEX, path_instruction_type::WILDCARD);
  if (match) {
    return thrust::make_tuple(true, path.data()[0].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

/**
 * @brief The cases that mirro Apache Spark case path in `jsonExpressions.scala#evaluatePath()`.
 */
enum class evaluation_case_path : int8_t {
  INVALID                                           = -1,
  START_ARRAY___EMPTY_PATH___FLATTEN_STYLE          = 2,
  START_OBJECT___MATCHED_NAME_PATH                  = 4,
  START_ARRAY___MATCHED_DOUBLE_WILDCARD             = 5,
  START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED = 6,
  START_ARRAY___MATCHED_WILDCARD                    = 7,
  START_ARRAY___MATCHED_INDEX_AND_WILDCARD          = 8,
  START_ARRAY___MATCHED_INDEX                       = 9
};

/**
 * @brief The struct to store states during processing JSON through different nested levels.
 */
struct context {
  // used to save current generator
  json_generator g;

  // used to save child JSON generator for case path 6
  json_generator child_g;

  cudf::device_span<path_instruction const> path;

  // whether written output
  // if dirty > 0, indicates success
  int dirty;

  // which case path that this task is from
  evaluation_case_path case_path;

  // current token
  json_token token;

  write_style style;

  bool is_in_array;

  // for some case paths
  bool is_first_enter;

  // is this context task is done
  bool task_is_done;
};

/**
 * @brief Parse a single json string using the provided command buffer.
 *
 * @param p The JSON parser for input string
 * @param path_commands The command buffer to be applied to the string
 * TODO: update
 * @param out_buf Buffer user to store the string resulted from the query
 * @param max_path_depth_exceeded A marker to record if the maximum path depth has been reached
 *        during parsing the input string
 * @return A pair containing the result code and the output size
 */
__device__ thrust::pair<bool, cudf::size_type> evaluate_path(
  json_parser& p,
  cudf::device_span<path_instruction const> path_commands,
  cudf::type_id path_type_id,
  bool keep_quotes,
  char element_delimiter,
  char null_placeholder,
  char* out_buf,
  int8_t* max_path_depth_exceeded)
{
  p.next_token();
  if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

  // Define stack; plus 1 indicates root context task needs an extra memory.
  context stack[MAX_JSON_PATH_DEPTH + 1];
  int stack_size = 0;

  auto const push_context = [&](evaluation_case_path _case_path,
                                json_generator _g,
                                write_style _style,
                                cudf::device_span<path_instruction const> _path,
                                bool is_in_array) {
    if (stack_size > MAX_JSON_PATH_DEPTH) {
      *max_path_depth_exceeded = 1;
      // Because no more context is pushed, the evaluation output should be wrong.
      // But that is not important, since we will throw exception after the kernel finishes.
      return;
    }
    auto& ctx          = stack[stack_size++];
    ctx.g              = std::move(_g);
    ctx.path           = std::move(_path);
    ctx.dirty          = 0;
    ctx.case_path      = _case_path;
    ctx.token          = p.get_current_token();
    ctx.style          = _style;
    ctx.is_in_array    = is_in_array;
    ctx.is_first_enter = true;
    ctx.task_is_done   = false;
  };

  push_context(
    evaluation_case_path::INVALID, json_generator{}, write_style::RAW, path_commands, false);

  while (stack_size > 0) {
    auto& ctx = stack[stack_size - 1];
    if (!ctx.task_is_done) {
      // case (VALUE_STRING, Nil) if style == RawStyle
      // case path 1
      if (json_token::VALUE_STRING == ctx.token && path_is_empty(ctx.path.size())) {
        // there is no array wildcard or slice parent, emit this string without
        // quotes write current string in parser to generator
        ctx.g.try_write_comma(out_buf, element_delimiter);
        ctx.g.write_raw(p, out_buf, keep_quotes);
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_ARRAY, Nil) if style == FlattenStyle
      // case path 2
      else if (json_token::START_ARRAY == ctx.token && path_is_empty(ctx.path.size()) &&
               ctx.style == write_style::FLATTEN) {
        // flatten this array into the parent
        if (json_token::END_ARRAY != p.next_token()) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
          // push back task
          // add child task
          push_context(evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE,
                       ctx.g,
                       ctx.style,
                       {nullptr, 0},
                       true);
        } else {
          // END_ARRAY
          ctx.task_is_done = true;
        }
      }
      // case (_, Nil)
      // case path 3
      else if (path_is_empty(ctx.path.size())) {
        // printf("path is empty, path type = %d, token = %d\n",
        //        (int)path_type_id,
        //        (int)p.get_current_token());

        // If this is a struct column, we only need to check to see if there exists a struct.
        if (path_type_id == cudf::type_id::STRUCT || path_type_id == cudf::type_id::LIST) {
          if (path_type_id == cudf::type_id::STRUCT &&
              p.get_current_token() != json_token::START_OBJECT) {
            return {false, 0};
          }
          if (path_type_id == cudf::type_id::LIST &&
              p.get_current_token() != json_token::START_ARRAY) {
            return {false, 0};
          }

          if (path_type_id == cudf::type_id::STRUCT) {
            // Or copy current structure?
            if (!p.try_skip_children()) { return {false, 0}; }
          } else if (!(ctx.g.copy_current_structure(p, nullptr, ','))) {
            // not copy only if there is struct?
            return {false, 0};
          }

          // Just write anything into the output, to mark the output as a non-null row.
          // Such output will be discarded anyway.
          ctx.g.write_start_array(out_buf, element_delimiter);
        } else if (!(ctx.g.copy_current_structure(p, out_buf, element_delimiter))) {
          return {false, 0};
        }
        ctx.dirty        = 1;
        ctx.task_is_done = true;
      }
      // case (START_OBJECT, Named :: xs)
      // case path 4
      else if (json_token::START_OBJECT == ctx.token &&
               thrust::get<0>(path_match_named(ctx.path))) {
        // printf("start object\n");

        if (!ctx.is_first_enter) {
          // 2st enter
          // skip the following children after the expect
          if (ctx.dirty > 0) {
            while (json_token::END_OBJECT != p.next_token()) {
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // skip value of FIELD_NAME
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          // Mark task is done regardless whether the expected child was found.
          ctx.task_is_done = true;
        } else {
          // below is 1st enter
          ctx.is_first_enter = false;
          // match first mached children with expected name
          bool found_expected_child = false;
          while (json_token::END_OBJECT != p.next_token()) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

            // need to try more children
            auto match_named = path_match_named(ctx.path);
            auto named       = thrust::get<1>(match_named);
            // current token is FIELD_NAME
            if (p.match_current_field_name(named)) {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // meets null token, it's not expected, return false
              if (json_token::VALUE_NULL == p.get_current_token()) { return {false, 0}; }
              // push sub task; sub task will update the result of path 4
              push_context(evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH,
                           ctx.g,
                           ctx.style,
                           {ctx.path.data() + 1, ctx.path.size() - 1},
                           ctx.is_in_array);
              found_expected_child = true;
              break;
            } else {
              // skip FIELD_NAME token
              p.next_token();
              // JSON validation check
              if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

              // current child is not expected, skip current child
              if (!p.try_skip_children()) {
                // JSON validation check
                return {false, 0};
              }
            }
          }
          if (!found_expected_child) {
            if (ctx.is_in_array) {
              ctx.g.try_write_comma(out_buf, element_delimiter);
              ctx.g.write_null_placeholder(out_buf, null_placeholder);
              ctx.dirty = 1;
            } else {
              ctx.dirty = false;
            }
            ctx.task_is_done = true;
          }
        }
      }
      // case (START_ARRAY, Wildcard :: xs)
      // case path 7
      else if (json_token::START_ARRAY == ctx.token &&
               path_match_element(ctx.path, path_instruction_type::WILDCARD)) {
        // printf("array *\n");

        if (ctx.is_first_enter) {
          ctx.is_first_enter = false;
          ctx.g.write_first_start_array_without_output();
        }
        if (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }

          // wildcards can have multiple matches, continually update the dirty
          // count
          push_context(evaluation_case_path::START_ARRAY___MATCHED_WILDCARD,
                       ctx.g,
                       write_style::QUOTED,
                       {ctx.path.data() + 1, ctx.path.size() - 1},
                       true);
        } else {
          // ctx.g.write_end_array(out_buf);
          ctx.task_is_done = true;
        }
      }
      // case _ =>
      // case path 12
      else {
        // printf("get obj line %d\n", __LINE__);

        if (!p.try_skip_children()) { return {false, 0}; }
        // default case path, return false for this task
        ctx.dirty        = 0;
        ctx.task_is_done = true;
      }
    }       // if (!ctx.task_is_done)
    else {  // current context is done.
      // pop current top context
      stack_size--;

      // has no parent task, stack is empty, will exit
      if (stack_size == 0) { break; }

      // peek parent context task
      // update parent task info according to current task result
      auto& p_ctx = stack[stack_size - 1];

      switch (ctx.case_path) {
          // path 2: case (START_ARRAY, Nil) if style == FlattenStyle
          // path 5: case (START_ARRAY, Wildcard :: Wildcard :: xs)
          // path 7: case (START_ARRAY, Wildcard :: xs)
        case evaluation_case_path::START_ARRAY___EMPTY_PATH___FLATTEN_STYLE:
        case evaluation_case_path::START_ARRAY___MATCHED_DOUBLE_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_OBJECT, Named :: xs)
          // case path 4
        case evaluation_case_path::START_OBJECT___MATCHED_NAME_PATH: {
          p_ctx.dirty = ctx.dirty;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

          // case (START_ARRAY, Wildcard :: xs) if style != QuotedStyle
          // case path 6
        case evaluation_case_path::START_ARRAY___MATCHED_WILDCARD___STYLE_NOT_QUOTED: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;
          // update child generator for parent task
          p_ctx.child_g = ctx.g;

          break;
        }

          /* case (START_ARRAY, Index(idx) :: (xs@Wildcard :: _)) */
          // case path 8
          // case (START_ARRAY, Index(idx) :: xs)
          // case path 9
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX_AND_WILDCARD:
        case evaluation_case_path::START_ARRAY___MATCHED_INDEX: {
          // collect result from child task
          p_ctx.dirty += ctx.dirty;

          // post logic:
          while (p.next_token() != json_token::END_ARRAY) {
            // JSON validation check
            if (json_token::ERROR == p.get_current_token()) { return {false, 0}; }
            // advance the token stream to the end of the array
            if (!p.try_skip_children()) { return {false, 0}; }
          }
          // task is done
          p_ctx.task_is_done = true;
          // copy generator states to parent task;
          p_ctx.g = ctx.g;

          break;
        }

        default:;  // Never happens!
      }            // end switch (ctx.case_path)
    }              // ctx.task_is_done
  }                // while (stack_size > 0)

  auto const success = stack[0].dirty > 0;

  // generator may contain trash output, e.g.: generator writes some output,
  // then JSON format is invalid, the previous output becomes trash.
  // We need to return output size as zero.
  return {success, success ? stack[0].g.get_output_len() : 0};
}

/**
 * @brief Struct storing data such as path instructions, output buffer etc, corresponding to a
 * single JSON path.
 */
struct json_path_processing_data {
  cudf::device_span<path_instruction const> path_commands;
  cudf::detail::input_offsetalator offsets;
  thrust::pair<char const*, cudf::size_type>* out_stringviews;
  char* out_buf;
  int8_t* has_out_of_bound;
  bool keep_quotes;
  cudf::type_id type_id;
};

/**
 * @brief Kernel for running the JSONPath query, in which one input row is processed by entire
 * warp (or multiple warps) of threads.
 *
 * The number of warps processing each row is computed as `ceil(num_paths / warp_size)`.
 *
 * We explicitly set a value for `min_block_per_sm` parameter in the launch bounds to avoid
 * spilling from the kernel itself. By default NVCC uses a heuristic to find a balance between
 * the maximum number of registers used by a kernel and the parallelism of the kernel.
 * If lots of registers are used the parallelism may suffer. But in our case NVCC gets this wrong
 * and we want to avoid spilling all the time or else the performance is really bad. This
 * essentially tells NVCC to prefer using lots of registers over spilling.
 *
 * TODO update
 * @param input The input JSON strings stored in a strings column
 * @param path_data Array containing all path data
 * @param num_threads_per_row Number of threads processing each input row
 * @param max_path_depth_exceeded A marker to record if the maximum path depth has been reached
 *        during parsing the input string
 */
template <int block_size, int min_block_per_sm>
__launch_bounds__(block_size, min_block_per_sm) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view input,
                              cudf::device_span<json_path_processing_data> path_data,
                              char element_delimiter,
                              char null_placeholder,
                              bool allow_leading_zero_numbers,
                              bool allow_non_numeric_numbers,
                              bool allow_unquoted_control_chars,
                              std::size_t num_threads_per_row,
                              int8_t* max_path_depth_exceeded)
{
  auto const tidx    = cudf::detail::grid_1d::global_thread_id();
  auto const row_idx = tidx / num_threads_per_row;
  if (row_idx >= input.size()) { return; }

  auto const path_idx = tidx % num_threads_per_row;
  if (path_idx >= path_data.size()) { return; }

  auto const& path         = path_data[path_idx];
  char* const dst          = path.out_buf + path.offsets[row_idx];
  bool is_valid            = false;
  cudf::size_type out_size = 0;

  auto const str = input.element<cudf::string_view>(row_idx);
  if (str.size_bytes() > 0) {
    json_parser p{char_range{str}};
    p.set_allow_leading_zero_numbers(allow_leading_zero_numbers);
    p.set_allow_non_numeric_numbers(allow_non_numeric_numbers);
    p.set_allow_unquoted_control_chars(allow_unquoted_control_chars);
    thrust::tie(is_valid, out_size) = evaluate_path(p,
                                                    path.path_commands,
                                                    path.type_id,
                                                    path.keep_quotes,
                                                    element_delimiter,
                                                    null_placeholder,
                                                    dst,
                                                    max_path_depth_exceeded);

    // We did not terminate the `evaluate_path` function early to reduce complexity of the code.
    // Instead, if max depth was encountered, we've just continued the evaluation until here
    // then discard the output entirely.
    if (p.max_nesting_depth_exceeded()) {
      *max_path_depth_exceeded = 1;
      return;
    }

    auto const max_size = path.offsets[row_idx + 1] - path.offsets[row_idx];
    if (out_size > max_size) { *(path.has_out_of_bound) = 1; }
  }

  // Write out `nullptr` in the output string_view to indicate that the output is a null.
  // The situation `out_stringviews == nullptr` should only happen if the kernel is launched a
  // second time due to out-of-bound write in the first launch.
  if (path.out_stringviews) {
    path.out_stringviews[row_idx] = {is_valid ? dst : nullptr, out_size};
  }
}

/**
 * @brief A utility class to launch the main kernel.
 */
struct kernel_launcher {
  static void exec(cudf::column_device_view const& input,
                   cudf::device_span<json_path_processing_data> path_data,
                   char element_delimiter,
                   char null_placeholder,
                   bool allow_leading_zero_numbers,
                   bool allow_non_numeric_numbers,
                   bool allow_unquoted_control_chars,
                   int8_t* max_path_depth_exceeded,
                   rmm::cuda_stream_view stream)
  {
    // The optimal values for block_size and min_block_per_sm were found through testing,
    // which are either 128-8 or 256-4. The pair 128-8 seems a bit better.
    static constexpr int block_size       = 128;
    static constexpr int min_block_per_sm = 8;

    // The number of threads for processing one input row is at least one warp.
    auto const num_threads_per_row =
      cudf::util::div_rounding_up_safe(path_data.size(),
                                       static_cast<std::size_t>(cudf::detail::warp_size)) *
      cudf::detail::warp_size;
    auto const num_blocks = cudf::util::div_rounding_up_safe(num_threads_per_row * input.size(),
                                                             static_cast<std::size_t>(block_size));
    get_json_object_kernel<block_size, min_block_per_sm>
      <<<num_blocks, block_size, 0, stream.value()>>>(input,
                                                      path_data,
                                                      element_delimiter,
                                                      null_placeholder,
                                                      allow_leading_zero_numbers,
                                                      allow_non_numeric_numbers,
                                                      allow_unquoted_control_chars,
                                                      num_threads_per_row,
                                                      max_path_depth_exceeded);
  }
};

/**
 * @brief Construct the device vector containing necessary data for the input JSON paths.
 *
 * All JSON paths are processed at once, without stream synchronization, to minimize overhead.
 *
 * A tuple of values are returned, however, only the first element is needed for further kernel
 * launch. The remaining are unused but need to be kept alive as they contains data for later
 * asynchronous host-device memcpy.
 */
std::tuple<std::vector<rmm::device_uvector<path_instruction>>,
           std::unique_ptr<std::vector<std::vector<path_instruction>>>,
           cudf::string_scalar,
           std::string>
construct_path_commands(
  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> const&
    json_paths,
  rmm::cuda_stream_view stream)
{
  // Concatenate all names from path instructions.
  auto h_inst_names = [&] {
    std::size_t length{0};
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { length += name.length(); }
      }
    }
    std::string all_names;
    all_names.reserve(length);
    for (auto const& instructions : json_paths) {
      for (auto const& [type, name, index] : instructions) {
        if (type == path_instruction_type::NAMED) { all_names += name; }
      }
    }
    return all_names;
  }();
  auto d_inst_names = cudf::string_scalar(h_inst_names, true, stream);

  std::size_t name_pos{0};
  auto h_path_commands = std::make_unique<std::vector<std::vector<path_instruction>>>();
  h_path_commands->reserve(json_paths.size());

  for (auto const& instructions : json_paths) {
    h_path_commands->emplace_back();
    auto& path_commands = h_path_commands->back();
    path_commands.reserve(instructions.size());

    for (auto const& [type, name, index] : instructions) {
      path_commands.emplace_back(path_instruction{type});

      if (type == path_instruction_type::INDEX) {
        path_commands.back().index = index;
      } else if (type == path_instruction_type::NAMED) {
        path_commands.back().name = cudf::string_view(d_inst_names.data() + name_pos, name.size());
        name_pos += name.size();
      } else if (type != path_instruction_type::WILDCARD) {
        CUDF_FAIL("Invalid path instruction type");
      }
    }
  }

  auto d_path_commands = std::vector<rmm::device_uvector<path_instruction>>{};
  d_path_commands.reserve(h_path_commands->size());
  for (auto const& path_commands : *h_path_commands) {
    d_path_commands.emplace_back(cudf::detail::make_device_uvector_async(
      path_commands, stream, rmm::mr::get_current_device_resource()));
  }

  return {std::move(d_path_commands),
          std::move(h_path_commands),
          std::move(d_inst_names),
          std::move(h_inst_names)};
}

int64_t calc_scratch_size(cudf::strings_column_view const& input,
                          cudf::detail::input_offsetalator const& in_offsets,
                          rmm::cuda_stream_view stream)
{
  auto const max_row_size = thrust::transform_reduce(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(input.size()),
    cuda::proclaim_return_type<int64_t>(
      [in_offsets] __device__(auto const idx) { return in_offsets[idx + 1] - in_offsets[idx]; }),
    int64_t{0},
    thrust::maximum{});

  // We will use scratch buffers to store the output strings without knowing their sizes.
  // Since we do not know their sizes, we need to allocate the buffer a bit larger than the input
  // size so that we will not write output strings into an out-of-bound position.
  // Checking out-of-bound needs to be performed in the main kernel to make sure we will not have
  // data corruption.
  auto const scratch_size = [&, max_row_size = max_row_size] {
    // Pad the scratch buffer by an additional size that is a multiple of max row size.
    auto constexpr padding_rows = 10;
    return input.chars_size(stream) + max_row_size * padding_rows;
  }();
  return scratch_size;
}

/**
 * @brief Error handling using error markers gathered after kernel launch.
 *
 * If the input JSON has nesting depth exceeds the maximum allowed value, an exception will be
 * thrown as it is unacceptable. Otherwise, out of bound write is checked and returned.
 *
 * @param error_check The array of markers to check for error
 * @return A boolean value indicating if there is any out of bound write
 */
bool check_error(cudf::detail::host_vector<int8_t> const& error_check)
{
  // The last value is to mark if nesting depth has exceeded.
  CUDF_EXPECTS(error_check.back() == 0,
               "The processed input has nesting depth exceeds depth limit.");

  // Do not use parallel check since we do not have many elements.
  // The last element is not related, but its value is already `0` thus just check until
  // the end of the array for simplicity.
  return std::none_of(
    error_check.cbegin(), error_check.cend(), [](auto const val) { return val != 0; });
}

std::vector<std::unique_ptr<cudf::column>> get_json_object_batch(
  cudf::column_device_view const& input,
  cudf::detail::input_offsetalator const& in_offsets,
  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> const&
    json_paths,
  std::vector<cudf::type_id> const& type_ids,
  std::vector<std::size_t> const& output_ids,
  std::unordered_set<std::size_t> const& keep_quotes,
  char element_delimiter,
  char null_placeholder,
  int64_t scratch_size,
  bool allow_leading_zero_numbers,
  bool allow_non_numeric_numbers,
  bool allow_unquoted_control_chars,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const [d_json_paths, h_json_paths, d_inst_names, h_inst_names] =
    construct_path_commands(json_paths, stream);

  auto const num_outputs = json_paths.size();
  std::vector<std::unique_ptr<cudf::column>> output;

  // The error check array contains markers denoting if there is any out-of-bound write occurs
  // (first `num_outputs` elements), or if the nesting depth exceeded its limits (the last element).
  rmm::device_uvector<int8_t> d_error_check(num_outputs + 1, stream);
  auto const d_max_path_depth_exceeded = d_error_check.data() + num_outputs;

  std::vector<rmm::device_uvector<char>> scratch_buffers;
  std::vector<rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>> out_stringviews;
  std::vector<json_path_processing_data> h_path_data;
  scratch_buffers.reserve(json_paths.size());
  out_stringviews.reserve(json_paths.size());
  h_path_data.reserve(json_paths.size());

  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& path = json_paths[idx];
    if (path.size() > MAX_JSON_PATH_DEPTH) {
      CUDF_FAIL("JSON Path has depth exceeds the maximum allowed value.");
    }

    scratch_buffers.emplace_back(rmm::device_uvector<char>(scratch_size, stream));
    out_stringviews.emplace_back(rmm::device_uvector<thrust::pair<char const*, cudf::size_type>>{
      static_cast<std::size_t>(input.size()), stream});

    // printf("idx: %d, output_ids[idx]: %d\n", (int)idx, (int)output_ids[idx]);
    // printf("keep_quotes.find(output_ids[idx]) != keep_quotes.end(): %d\n",
    //        (int)(keep_quotes.find(output_ids[idx]) != keep_quotes.end()));
    // fflush(stdout);

    h_path_data.emplace_back(
      json_path_processing_data{d_json_paths[idx],
                                in_offsets,
                                out_stringviews.back().data(),
                                scratch_buffers.back().data(),
                                d_error_check.data() + idx,
                                keep_quotes.find(output_ids[idx]) != keep_quotes.end(),
                                type_ids[idx]});
  }
  auto d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_error_check.begin(), d_error_check.end(), 0);

  kernel_launcher::exec(input,
                        d_path_data,
                        element_delimiter,
                        null_placeholder,
                        allow_leading_zero_numbers,
                        allow_non_numeric_numbers,
                        allow_unquoted_control_chars,
                        d_max_path_depth_exceeded,
                        stream);
  auto h_error_check = cudf::detail::make_host_vector_sync(d_error_check, stream);
  auto has_no_oob    = check_error(h_error_check);

  // If we didn't see any out-of-bound write, everything is good so far.
  // Just gather the output strings and return.
  if (has_no_oob) {
    for (auto const& out_sview : out_stringviews) {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
    return output;
  }
  // From here, we had out-of-bound write. Although this is very rare, it may still happen.

  std::vector<std::pair<rmm::device_buffer, cudf::size_type>> out_null_masks_and_null_counts;
  std::vector<std::pair<std::unique_ptr<cudf::column>, int64_t>> out_offsets_and_sizes;
  std::vector<rmm::device_uvector<char>> out_char_buffers;
  std::vector<std::size_t> oob_indices;

  // Check validity from the stored char pointers.
  auto const validator = [] __device__(thrust::pair<char const*, cudf::size_type> const item) {
    return item.first != nullptr;
  };

  // Rebuild the data only for paths that had out of bound write.
  h_path_data.clear();
  for (std::size_t idx = 0; idx < num_outputs; ++idx) {
    auto const& out_sview = out_stringviews[idx];

    if (h_error_check[idx]) {
      oob_indices.emplace_back(idx);
      output.emplace_back(nullptr);  // just placeholder.

      out_null_masks_and_null_counts.emplace_back(
        cudf::detail::valid_if(out_sview.begin(), out_sview.end(), validator, stream, mr));

      // The string sizes computed in the previous kernel call will be used to allocate a new char
      // buffer to store the output.
      auto const size_it = cudf::detail::make_counting_transform_iterator(
        0,
        cuda::proclaim_return_type<cudf::size_type>(
          [string_pairs = out_sview.data()] __device__(auto const idx) {
            return string_pairs[idx].second;
          }));
      out_offsets_and_sizes.emplace_back(cudf::strings::detail::make_offsets_child_column(
        size_it, size_it + input.size(), stream, mr));
      out_char_buffers.emplace_back(
        rmm::device_uvector<char>(out_offsets_and_sizes.back().second, stream, mr));

      h_path_data.emplace_back(
        json_path_processing_data{d_json_paths[idx],
                                  cudf::detail::offsetalator_factory::make_input_iterator(
                                    out_offsets_and_sizes.back().first->view()),
                                  nullptr /*out_stringviews*/,
                                  out_char_buffers.back().data(),
                                  d_error_check.data() + idx,
                                  keep_quotes.find(output_ids[idx]) != keep_quotes.end(),
                                  type_ids[idx]});
    } else {
      output.emplace_back(cudf::make_strings_column(out_sview, stream, mr));
    }
  }
  // These buffers are no longer needed.
  scratch_buffers.clear();
  out_stringviews.clear();

  // Push data to the GPU and launch the kernel again.
  d_path_data = cudf::detail::make_device_uvector_async(
    h_path_data, stream, rmm::mr::get_current_device_resource());
  thrust::uninitialized_fill(
    rmm::exec_policy(stream), d_error_check.begin(), d_error_check.end(), 0);
  kernel_launcher::exec(input,
                        d_path_data,
                        element_delimiter,
                        null_placeholder,
                        allow_leading_zero_numbers,
                        allow_non_numeric_numbers,
                        allow_unquoted_control_chars,
                        d_max_path_depth_exceeded,
                        stream);
  h_error_check = cudf::detail::make_host_vector_sync(d_error_check, stream);
  has_no_oob    = check_error(h_error_check);

  // The last kernel call should not encounter any out-of-bound write.
  // If OOB is still detected, there must be something wrong happened.
  CUDF_EXPECTS(has_no_oob, "Unexpected out-of-bound write in get_json_object kernel.");

  for (std::size_t idx = 0; idx < oob_indices.size(); ++idx) {
    auto const out_idx = oob_indices[idx];
    output[out_idx] =
      cudf::make_strings_column(input.size(),
                                std::move(out_offsets_and_sizes[idx].first),
                                out_char_buffers[idx].release(),
                                out_null_masks_and_null_counts[idx].second,
                                std::move(out_null_masks_and_null_counts[idx].first));
  }
  return output;
}

// TODO: update docs for keep_quotes
std::vector<std::unique_ptr<cudf::column>> get_json_object(
  cudf::strings_column_view const& input,
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>> const&
    json_paths,
  std::vector<cudf::type_id> const& type_ids,
  std::unordered_set<std::size_t> const& keep_quotes,
  char element_delimiter,
  char null_placeholder,
  int64_t memory_budget_bytes,
  int32_t parallel_override,
  bool allow_leading_zero_numbers,
  bool allow_non_numeric_numbers,
  bool allow_unquoted_control_chars,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const num_outputs = json_paths.size();

  // Input is empty or all nulls - just return all null columns.
  if (input.is_empty() || input.size() == input.null_count()) {
    std::vector<std::unique_ptr<cudf::column>> output;
    for (std::size_t idx = 0; idx < num_outputs; ++idx) {
      output.emplace_back(std::make_unique<cudf::column>(input.parent(), stream, mr));
    }
    return output;
  }

  std::vector<std::size_t> sorted_indices(json_paths.size());
  std::iota(sorted_indices.begin(), sorted_indices.end(), 0);  // Fill with 0, 1, 2, ...

  // Sort indices based on the corresponding paths.
  std::sort(sorted_indices.begin(), sorted_indices.end(), [&json_paths](size_t i, size_t j) {
    return json_paths[i] < json_paths[j];
  });

  auto const in_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());
  auto const scratch_size = calc_scratch_size(input, in_offsets, stream);
  if (memory_budget_bytes <= 0 && parallel_override <= 0) {
    parallel_override = static_cast<int>(sorted_indices.size());
  }
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  std::vector<std::unique_ptr<cudf::column>> output(num_outputs);

  // TODO: reserve
  std::vector<cudf::host_span<std::tuple<path_instruction_type, std::string, int32_t> const>> batch;
  std::vector<cudf::type_id> batch_type_ids;
  std::vector<std::size_t> output_ids;

  std::size_t starting_path = 0;
  while (starting_path < num_outputs) {
    std::size_t at = starting_path;
    batch.resize(0);
    batch_type_ids.resize(0);
    output_ids.resize(0);
    if (parallel_override > 0) {
      int count = 0;
      while (at < num_outputs && count < parallel_override) {
        auto output_location = sorted_indices[at];
        batch.emplace_back(json_paths[output_location]);
        batch_type_ids.push_back(type_ids[output_location]);
        output_ids.push_back(output_location);
        at++;
        count++;
      }
    } else {
      long budget = 0;
      while (at < num_outputs && budget < memory_budget_bytes) {
        auto output_location = sorted_indices[at];
        batch.emplace_back(json_paths[output_location]);
        batch_type_ids.push_back(type_ids[output_location]);
        output_ids.push_back(output_location);
        at++;
        budget += scratch_size;
      }
    }
    auto tmp = get_json_object_batch(*d_input_ptr,
                                     in_offsets,
                                     batch,
                                     batch_type_ids,
                                     output_ids,
                                     keep_quotes,
                                     element_delimiter,
                                     null_placeholder,
                                     scratch_size,
                                     allow_leading_zero_numbers,
                                     allow_non_numeric_numbers,
                                     allow_unquoted_control_chars,
                                     stream,
                                     mr);
    for (std::size_t i = 0; i < tmp.size(); i++) {
      std::size_t out_i = output_ids[i];
      output[out_i]     = std::move(tmp[i]);
    }
    starting_path = at;
  }
  return output;
}

}  // namespace test

void travel_path(
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>>& paths,
  std::vector<std::tuple<path_instruction_type, std::string, int32_t>>& current_path,
  std::vector<cudf::type_id>& type_ids,
  std::unordered_set<std::size_t>& keep_quotes,
  bool& has_list_type,
  bool parent_is_list,
  std::string const& name,
  json_schema_element const& column_schema)
{
  bool popped{false};
  current_path.emplace_back(path_instruction_type::NAMED, name, -1);
  if (column_schema.child_types.size() == 0) {  // leaf of the schema
    if (cudf::is_fixed_width(column_schema.type)) {
      // TODO: comment
      keep_quotes.insert(paths.size());
    }
    // printf("column_schema type: %d\n", static_cast<int>(column_schema.type.id()));
    paths.push_back(current_path);  // this will copy
    type_ids.push_back(column_schema.type.id());
  } else {
    if (column_schema.type.id() == cudf::type_id::STRUCT) {
      type_ids.push_back(column_schema.type.id());

      // STRUCT directly under array does not have name field.
      if (parent_is_list) {
        popped = true;
        current_path.pop_back();  // remove the last NAMED instruction.
      }
      paths.push_back(current_path);  // this will copy
                                      // printf("column_schema type: STRUCT\n");
      for (auto const& [child_name, child_schema] : column_schema.child_types) {
        travel_path(paths,
                    current_path,
                    type_ids,
                    keep_quotes,
                    has_list_type,
                    false /*parent_is_list*/,
                    child_name,
                    child_schema);
      }
    } else if (column_schema.type.id() == cudf::type_id::LIST) {
      // printf("column_schema type: LIST\n");

      CUDF_EXPECTS(column_schema.child_types.size() == 1, "TODO");
      has_list_type = true;

      bool has_struct_child{false};
      for (auto const& [child_name, child_schema] : column_schema.child_types) {
        if (child_schema.type.id() == cudf::type_id::STRUCT) {
          has_struct_child = true;
          break;
        }
      }

      // TODO: is this needed, if there is no struct child?
      if (has_struct_child) {
        paths.push_back(current_path);  // this will copy
        type_ids.push_back(column_schema.type.id());
      }

      current_path.emplace_back(path_instruction_type::WILDCARD, "*", -1);

      // Only add a path name if this column is not under a list type.
      if (has_struct_child) {
        for (auto const& [child_name, child_schema] : column_schema.child_types) {
          travel_path(paths,
                      current_path,
                      type_ids,
                      keep_quotes,
                      has_list_type,
                      true /*parent_is_list*/,
                      child_name,
                      child_schema);
        }
      } else {
        auto const child_type = column_schema.child_types.front().second.type;
        if (cudf::is_fixed_width(child_type)) { keep_quotes.insert(paths.size()); }
        paths.push_back(current_path);  // this will copy
        type_ids.push_back(child_type.id());
      }

      current_path.pop_back();  // remove WILDCARD

    } else {
      // TODO
      CUDF_FAIL("Unsupported type");
    }
  }
  // if (column_schema.type.id() != cudf::type_id::STRUCT || !has_list_type) {
  if (!popped) { current_path.pop_back(); }
}

std::tuple<std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>>,
           std::vector<cudf::type_id>,
           std::unordered_set<std::size_t>,
           bool>
flatten_schema_to_paths(std::vector<std::pair<std::string, json_schema_element>> const& schema)
{
  std::vector<std::vector<std::tuple<path_instruction_type, std::string, int32_t>>> paths;
  std::vector<cudf::type_id> type_ids;
  std::unordered_set<std::size_t> keep_quotes;
  bool has_list_type{false};

  std::vector<std::tuple<path_instruction_type, std::string, int32_t>> current_path;
  std::for_each(schema.begin(), schema.end(), [&](auto const& kv) {
    travel_path(paths,
                current_path,
                type_ids,
                keep_quotes,
                has_list_type,
                false /*parent_is_list*/,
                kv.first,
                kv.second);
  });

  return {std::move(paths), std::move(type_ids), std::move(keep_quotes), has_list_type};
}

std::pair<std::unique_ptr<cudf::column>, std::unique_ptr<cudf::column>> extract_lists(
  std::unique_ptr<cudf::column>& input,
  json_schema_element const& column_schema,
  char element_delimiter,
  char null_placeholder,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  if (column_schema.type.id() == cudf::type_id::STRUCT) {
    std::unique_ptr<cudf::column> offsets{nullptr};
    std::vector<std::unique_ptr<cudf::column>> new_children;
    cudf::size_type num_child_rows{-1};
    auto children = std::move(input->release().children);
    for (std::size_t child_idx = 0; child_idx < children.size(); ++child_idx) {
      auto& child = children[child_idx];
      auto [new_child_offsets, new_child] =
        extract_lists(child,
                      column_schema.child_types[child_idx].second,
                      element_delimiter,
                      null_placeholder,
                      stream,
                      mr);
      if (num_child_rows < 0) { num_child_rows = new_child->size(); }
      if (num_child_rows != new_child->size()) {
        // printf("num_child_rows != new_child->size(): %d != %d\n",
        // (int)num_child_rows,
        // (int)new_child->size());
      }
      CUDF_EXPECTS(num_child_rows == new_child->size(), "num_child_rows != new_child->size()");

      if (!offsets) { offsets = std::move(new_child_offsets); }
      new_children.emplace_back(std::move(new_child));
    }

    // return cudf::make_structs_column(
    //             num_child_rows, std::move(children), null_count, std::move(*null_mask), stream,
    //             mr);
    // TODO: fix null mask
    return {std::move(offsets),
            cudf::make_structs_column(num_child_rows, std::move(new_children), 0, {}, stream, mr)};
  }

  // printf("before split:\n");
  // cudf::test::print(input->view());

  auto tmp           = cudf::strings::split_record(cudf::strings_column_view{input->view()},
                                         cudf::string_scalar{std::string{element_delimiter}},
                                         -1,
                                         stream,
                                         mr);
  auto split_content = tmp->release();

  if (input->size() == input->null_count()) {
    return {std::move(split_content.children[cudf::lists_column_view::offsets_column_index]),
            std::move(split_content.children[cudf::lists_column_view::child_column_index])};
  }

  auto const child_cv = split_content.children[cudf::lists_column_view::child_column_index]->view();
  auto const child_strview = cudf::strings_column_view{child_cv};

  // printf("child_cv:\n");
  // cudf::test::print(child_cv);

  // Convert a row index into an invalid value (-1) if that row contains a null placeholder.
  // Don't care about nulls in the child column, as they will be gathered to the output.
  auto const gather_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [null_placeholder,
       offsets = child_strview.offsets().begin<cudf::size_type>(),
       chars   = child_strview.chars_begin(stream)] __device__(cudf::size_type idx) {
        if (offsets[idx + 1] - offsets[idx] == 1) {
          return chars[offsets[idx]] == null_placeholder ? -1 : idx;
        }
        return idx;
      }));

  // TODO: report issue when the input is strings column has null == size
  auto out_child = std::move(cudf::detail::gather(cudf::table_view{{child_cv}},
                                                  gather_it,
                                                  gather_it + child_cv.size(),
                                                  cudf::out_of_bounds_policy::NULLIFY,
                                                  stream,
                                                  mr)
                               ->release()
                               .front());
  // printf("out_child:\n");
  // cudf::test::print(out_child->view());

  if (out_child->null_count() == 0) { out_child->set_null_mask(rmm::device_buffer{}, 0); }

  // auto split_content =
  //   cudf::strings::split_record(cudf::strings_column_view{input->view()},
  //                               cudf::string_scalar{std::string{element_delimiter}},
  //                               -1,
  //                               stream,
  //                               mr)
  //     ->release();
  // printf("after split:\n");
  // cudf::test::print(tmp->view());

  return {std::move(split_content.children[cudf::lists_column_view::offsets_column_index]),
          std::move(out_child)};
}

void assemble_column(std::size_t& column_order,
                     std::vector<std::unique_ptr<cudf::column>>& output,
                     std::vector<std::unique_ptr<cudf::column>>& read_columns,
                     std::string const& name,
                     json_schema_element const& column_schema,
                     char element_delimiter,
                     char null_placeholder,
                     rmm::cuda_stream_view stream,
                     rmm::device_async_resource_ref mr)
{
  if (column_schema.child_types.size() == 0) {  // leaf of the schema
    output.emplace_back(std::move(read_columns[column_order]));
    ++column_order;
  } else {
    if (column_schema.type.id() == cudf::type_id::STRUCT) {
      auto const null_count = read_columns[column_order]->null_count();
      auto const null_mask  = std::move(read_columns[column_order]->release().null_mask);
      ++column_order;

      std::vector<std::unique_ptr<cudf::column>> children;
      for (auto const& [child_name, child_schema] : column_schema.child_types) {
        assemble_column(column_order,
                        children,
                        read_columns,
                        child_name,
                        child_schema,
                        element_delimiter,
                        null_placeholder,
                        stream,
                        mr);
      }

      // TODO: generate null mask from input.
      auto const num_rows = children.front()->size();
      output.emplace_back(cudf::make_structs_column(
        num_rows, std::move(children), null_count, std::move(*null_mask), stream, mr));
    } else if (column_schema.type.id() == cudf::type_id::LIST) {
      // TODO: split LIST into child column
      // For now, just output as a strings column.

      bool has_struct_child{false};
      for (auto const& [child_name, child_schema] : column_schema.child_types) {
        if (child_schema.type.id() == cudf::type_id::STRUCT) {
          has_struct_child = true;
          break;
        }
      }

      auto const num_rows   = read_columns[column_order]->size();
      auto const null_count = read_columns[column_order]->null_count();
      std::unique_ptr<rmm::device_buffer> null_mask{nullptr};

      // printf("num rows: %d\n", num_rows);
      // If there is struct child, ..... TODO
      if (has_struct_child) {
        null_mask = std::move(read_columns[column_order]->release().null_mask);
        ++column_order;
      }

      std::vector<std::unique_ptr<cudf::column>> children;
      for (auto const& [child_name, child_schema] : column_schema.child_types) {
        assemble_column(column_order,
                        children,
                        read_columns,
                        child_name,
                        child_schema,
                        element_delimiter,
                        null_placeholder,
                        stream,
                        mr);
      }

      // printf("line %d\n", __LINE__);
      // cudf::test::print(children.front()->view());

      auto [offsets, child] = extract_lists(children.front(),
                                            column_schema.child_types.front().second,
                                            element_delimiter,
                                            null_placeholder,
                                            stream,
                                            mr);

      // printf("line %d\n", __LINE__);
      // cudf::test::print(child->view());
      // printf("line %d\n", __LINE__);
      // cudf::test::print(offsets->view());

      // TODO: fix null mask
      if (!has_struct_child) { null_mask = std::move(children.front()->release().null_mask); }

      output.emplace_back(cudf::make_lists_column(num_rows,
                                                  std::move(offsets),
                                                  std::move(child),
                                                  null_count,
                                                  std::move(*null_mask),
                                                  stream,
                                                  mr));

      // printf("line %d\n", __LINE__);
      // cudf::test::print(output.back()->view());
    } else {
      CUDF_FAIL("Unsupported type");
    }
  }
}

std::pair<char, char> find_delimiter(cudf::strings_column_view const& input,
                                     rmm::cuda_stream_view stream)
{
  auto constexpr num_levels  = 256;
  auto constexpr lower_level = std::numeric_limits<char>::min();
  auto constexpr upper_level = std::numeric_limits<char>::max();
  auto const num_chars       = input.chars_size(stream);  // stream sync

  // TODO: return when num_chars==0

  rmm::device_uvector<uint32_t> d_histogram(num_levels, stream);
  thrust::fill(rmm::exec_policy(stream), d_histogram.begin(), d_histogram.end(), 0);

  size_t temp_storage_bytes = 0;
  hipcub::DeviceHistogram::HistogramEven(nullptr,
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      d_histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());
  rmm::device_buffer d_temp(temp_storage_bytes, stream);
  hipcub::DeviceHistogram::HistogramEven(d_temp.data(),
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      d_histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());

  auto const zero_level = d_histogram.begin() - lower_level;
  auto first_zero_count_pos =
    thrust::find(rmm::exec_policy(stream), zero_level, d_histogram.end(), 0);
  if (first_zero_count_pos == d_histogram.end()) {
    // Try again...
    first_zero_count_pos =
      thrust::find(rmm::exec_policy(stream), d_histogram.begin(), d_histogram.end(), 0);
    if (first_zero_count_pos == d_histogram.end()) {
      // TODO: change message
      throw std::logic_error(
        "can't find a character suitable as delimiter for combining json strings to json lines "
        "with "
        "custom delimiter");
    }
  }

  auto second_zero_count_pos =
    thrust::find(rmm::exec_policy(stream), first_zero_count_pos + 1, d_histogram.end(), 0);
  if (second_zero_count_pos == d_histogram.end()) {
    // TODO: change message
    throw std::logic_error(
      "can't find a character suitable as delimiter for combining json strings to json lines "
      "with "
      "custom delimiter");
  }

  return {static_cast<char>(first_zero_count_pos - zero_level),
          static_cast<char>(second_zero_count_pos - zero_level)};
}

std::vector<std::unique_ptr<cudf::column>> assemble_output(
  std::vector<std::pair<std::string, json_schema_element>> const& schema,
  std::vector<std::unique_ptr<cudf::column>>& read_columns,
  char element_delimiter,
  char null_placeholder,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  std::vector<std::unique_ptr<cudf::column>> output;
  output.reserve(read_columns.size());

  std::size_t column_order{0};
  std::for_each(schema.begin(), schema.end(), [&](auto const& kv) {
    assemble_column(column_order,
                    output,
                    read_columns,
                    kv.first,
                    kv.second,
                    element_delimiter,
                    null_placeholder,
                    stream,
                    mr);
  });

  return output;
}

std::vector<std::unique_ptr<cudf::column>> from_json_to_structs(
  cudf::strings_column_view const& input,
  std::vector<std::pair<std::string, json_schema_element>> const& schema,
  bool allow_leading_zero_numbers,
  bool allow_non_numeric_numbers,
  bool allow_unquoted_control_chars,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // printf("line %d\n", __LINE__);
  // fflush(stdout);
  auto const [json_paths, type_ids, keep_quotes, has_list_type] = flatten_schema_to_paths(schema);

  // printf("line %d\n", __LINE__);
  // fflush(stdout);

#if 0
  int count{0};
  for (auto const& path : json_paths) {
    printf("\n\npath (%d/%d): \n", count++, (int)json_paths.size());
    for (auto node : path) {
      printf(".%s", std::get<1>(node).c_str());
    }
    printf("\n");
  }

  printf("keep quotes: \n");
  for (auto const i : keep_quotes) {
    printf("%d, ", (int)i);
  }
  printf("\n\n\n");
  fflush(stdout);

  auto ptr  = input.chars_begin(stream);
  auto size = input.chars_size(stream);
  std::vector<char> h_v(size);
  CUDF_CUDA_TRY(
    hipMemcpyAsync(h_v.data(), ptr, sizeof(char) * size, hipMemcpyDefault, stream.value()));
  stream.synchronize();

  printf("input (size = %d): ", (int)size);
  for (auto c : h_v) {
    printf("%c", c);
  }
  printf("\n");

#endif

  // array<struct<a: struct<b: int>>>
  // [{'a': {'b': 1, 'c' : 2}, 'x': []}, {}]

  // This should only run when there is LIST column.
  char delimiter{','}, null_placeholder{'\0'};
  if (has_list_type) { std::tie(delimiter, null_placeholder) = find_delimiter(input, stream); }
  // printf("delimiter: %c (code: %d)\n", delimiter, (int)delimiter);
  // printf("null_placeholder: %c (code: %d)\n", null_placeholder, (int)null_placeholder);

  auto tmp = test::get_json_object(input,
                                   json_paths,
                                   type_ids,
                                   keep_quotes,
                                   delimiter,
                                   null_placeholder,
                                   1024 * 1024 * 1024 * 4L,
                                   -1,
                                   allow_leading_zero_numbers,
                                   allow_non_numeric_numbers,
                                   allow_unquoted_control_chars,
                                   stream,
                                   mr);
  // printf("line %d\n", __LINE__);
  // fflush(stdout);

  if constexpr (0) {
    for (std::size_t i = 0; i < tmp.size(); ++i) {
      auto out  = cudf::strings_column_view{tmp[i]->view()};
      auto ptr  = out.chars_begin(stream);
      auto size = out.chars_size(stream);
      std::vector<char> h_v(size);
      CUDF_CUDA_TRY(
        hipMemcpyAsync(h_v.data(), ptr, sizeof(char) * size, hipMemcpyDefault, stream.value()));
      stream.synchronize();

      printf("out %d / %d (size = %d): ", (int)i, (int)tmp.size(), (int)size);
      for (auto c : h_v) {
        printf("%c", c);
      }
      printf("\n");

      // cudf::test::print(tmp[i]->view());
    }
  }

  return assemble_output(schema, tmp, delimiter, null_placeholder, stream, mr);
}

}  // namespace detail

std::unique_ptr<cudf::column> is_null_or_empty(cudf::strings_column_view const& input,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  rmm::device_uvector<bool> output(input.size(), stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(input.size()),
                    output.begin(),
                    [input = *d_input_ptr] __device__(cudf::size_type idx) -> bool {
                      if (input.is_null(idx)) { return true; }

                      auto const d_str = input.element<cudf::string_view>(idx);
                      int i            = 0;
                      for (; i < d_str.size_bytes(); ++i) {
                        if (d_str[i] != ' ') { break; }
                      }
                      auto const empty = i == d_str.size_bytes();
                      return empty;
                    });

  return std::make_unique<cudf::column>(std::move(output), rmm::device_buffer{}, 0);
}

std::vector<std::unique_ptr<cudf::column>> from_json_to_structs(
  cudf::strings_column_view const& input,
  std::vector<std::pair<std::string, json_schema_element>> const& schema,
  bool allow_leading_zero_numbers,
  bool allow_non_numeric_numbers,
  bool allow_unquoted_control_chars,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_json_to_structs(input,
                                      schema,
                                      allow_leading_zero_numbers,
                                      allow_non_numeric_numbers,
                                      allow_unquoted_control_chars,
                                      stream,
                                      mr);
}

}  // namespace spark_rapids_jni
