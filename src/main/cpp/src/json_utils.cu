#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cast_string.hpp"
#include "json_utils.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/valid_if.cuh>
#include <cudf/strings/contains.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/regex/regex_program.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cub/device/device_histogram.cuh>
#include <cub/device/device_memcpy.cuh>
#include <cub/device/device_segmented_reduce.cuh>
#include <cuda/functional>
#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/uninitialized_fill.h>

#include <limits>

namespace spark_rapids_jni {

namespace detail {

namespace {

constexpr bool not_whitespace(cudf::char_utf8 ch)
{
  return ch != ' ' && ch != '\r' && ch != '\n' && ch != '\t';
}

constexpr bool can_be_delimiter(char c)
{
  // The character list below is from `json_reader_options.set_delimiter`.
  switch (c) {
    case '{':
    case '[':
    case '}':
    case ']':
    case ',':
    case ':':
    case '"':
    case '\'':
    case '\\':
    case ' ':
    case '\t':
    case '\r': return false;
    default: return true;
  }
}

}  // namespace

std::tuple<std::unique_ptr<cudf::column>, std::unique_ptr<rmm::device_buffer>, char> concat_json(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);
  auto const default_mr  = rmm::mr::get_current_device_resource();

  // Check if the input rows are either null, equal to `null` string literal, or empty.
  // This will be used for masking out the input when doing string concatenation.
  rmm::device_uvector<bool> is_valid_input(input.size(), stream, default_mr);

  // Check if the input rows are either null or empty.
  // This will be returned to the caller.
  rmm::device_uvector<bool> is_null_or_empty(input.size(), stream, mr);

  thrust::for_each(
    rmm::exec_policy_nosync(stream),
    thrust::make_counting_iterator(0L),
    thrust::make_counting_iterator(input.size() * static_cast<int64_t>(cudf::detail::warp_size)),
    [input  = *d_input_ptr,
     output = thrust::make_zip_iterator(thrust::make_tuple(
       is_valid_input.begin(), is_null_or_empty.begin()))] __device__(int64_t tidx) {
      // Execute one warp per row to minimize thread divergence.
      if ((tidx % cudf::detail::warp_size) != 0) { return; }
      auto const idx = tidx / cudf::detail::warp_size;

      if (input.is_null(idx)) {
        output[idx] = thrust::make_tuple(false, true);
        return;
      }

      auto const d_str = input.element<cudf::string_view>(idx);
      auto const size  = d_str.size_bytes();
      int i            = 0;
      char ch;

      // Skip the very first whitespace characters.
      for (; i < size; ++i) {
        ch = d_str[i];
        if (not_whitespace(ch)) { break; }
      }

      if (i + 3 < size &&
          (d_str[i] == 'n' && d_str[i + 1] == 'u' && d_str[i + 2] == 'l' && d_str[i + 3] == 'l')) {
        i += 4;

        // Skip the very last whitespace characters.
        bool is_null_literal{true};
        for (; i < size; ++i) {
          ch = d_str[i];
          if (not_whitespace(ch)) {
            is_null_literal = false;
            break;
          }
        }

        // The current row contains only `null` string literal and not any other non-whitespace
        // characters. Such rows need to be masked out as null when doing concatenation.
        if (is_null_literal) {
          output[idx] = thrust::make_tuple(false, false);
          return;
        }
      }

      auto const not_eol = i < size;

      // If the current row is not null or empty, it should start with `{`. Otherwise, we need to
      // replace it by a null. This is necessary for libcudf's JSON reader to work.
      // Note that if we want to support ARRAY schema, we need to check for `[` instead.
      auto constexpr start_character = '{';
      if (not_eol && ch != start_character) {
        output[idx] = thrust::make_tuple(false, false);
        return;
      }

      output[idx] = thrust::make_tuple(not_eol, !not_eol);
    });

  auto constexpr num_levels  = 256;
  auto constexpr lower_level = std::numeric_limits<char>::min();
  auto constexpr upper_level = std::numeric_limits<char>::max();
  auto const num_chars       = input.chars_size(stream);

  rmm::device_uvector<uint32_t> histogram(num_levels, stream, default_mr);
  thrust::uninitialized_fill(
    rmm::exec_policy_nosync(stream), histogram.begin(), histogram.end(), 0);

  size_t temp_storage_bytes = 0;
  hipcub::DeviceHistogram::HistogramEven(nullptr,
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());
  rmm::device_buffer d_temp(temp_storage_bytes, stream);
  hipcub::DeviceHistogram::HistogramEven(d_temp.data(),
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());

  auto const it             = thrust::make_counting_iterator(0);
  auto const zero_level_idx = -lower_level;  // the bin storing count for character `\0`
  auto const zero_level_it  = it + zero_level_idx;
  auto const end            = it + num_levels;

  auto const first_zero_count_pos =
    thrust::find_if(rmm::exec_policy_nosync(stream),
                    zero_level_it,  // ignore the negative characters
                    end,
                    [zero_level_idx, counts = histogram.begin()] __device__(auto idx) -> bool {
                      auto const count = counts[idx];
                      if (count > 0) { return false; }
                      auto const first_non_existing_char = static_cast<char>(idx - zero_level_idx);
                      return can_be_delimiter(first_non_existing_char);
                    });

  // This should never happen since the input should never cover the entire char range.
  if (first_zero_count_pos == end) {
    throw std::logic_error(
      "Cannot find any character suitable as delimiter during joining json strings.");
  }
  auto const delimiter = static_cast<char>(thrust::distance(zero_level_it, first_zero_count_pos));

  auto [null_mask, null_count] = cudf::detail::valid_if(
    is_valid_input.begin(), is_valid_input.end(), thrust::identity{}, stream, default_mr);
  // If the null count doesn't change, that mean we do not have any rows containing `null` string
  // literal or empty rows. In such cases, just use the input column for concatenation.
  auto const input_applied_null =
    null_count == input.null_count()
      ? cudf::column_view{}
      : cudf::column_view{cudf::data_type{cudf::type_id::STRING},
                          input.size(),
                          input.chars_begin(stream),
                          reinterpret_cast<cudf::bitmask_type const*>(null_mask.data()),
                          null_count,
                          0,
                          std::vector<cudf::column_view>{input.offsets()}};

  auto concat_strings = cudf::strings::detail::join_strings(
    null_count == input.null_count() ? input : cudf::strings_column_view{input_applied_null},
    cudf::string_scalar(std::string(1, delimiter), true, stream, default_mr),
    cudf::string_scalar("{}", true, stream, default_mr),
    stream,
    mr);

  return {std::make_unique<cudf::column>(std::move(is_null_or_empty), rmm::device_buffer{}, 0),
          std::move(concat_strings->release().data),
          delimiter};
}

std::unique_ptr<cudf::column> make_structs(std::vector<cudf::column_view> const& children,
                                           cudf::column_view const& is_null,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  if (children.size() == 0) { return nullptr; }

  auto const row_count = children.front().size();
  for (auto const& col : children) {
    CUDF_EXPECTS(col.size() == row_count, "All columns must have the same number of rows.");
  }

  auto const [null_mask, null_count] = cudf::detail::valid_if(
    is_null.begin<bool>(), is_null.end<bool>(), thrust::logical_not{}, stream, mr);

  auto const structs =
    cudf::column_view(cudf::data_type{cudf::type_id::STRUCT},
                      row_count,
                      nullptr,
                      reinterpret_cast<cudf::bitmask_type const*>(null_mask.data()),
                      null_count,
                      0,
                      children);
  return std::make_unique<cudf::column>(structs, stream, mr);
}

namespace {

using string_index_pair = thrust::pair<char const*, cudf::size_type>;

std::pair<std::unique_ptr<cudf::column>, rmm::device_uvector<bool>> cast_strings_to_booleans(
  cudf::column_view const& input, rmm::cuda_stream_view stream, rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    return {cudf::make_empty_column(cudf::data_type{cudf::type_id::BOOL8}),
            rmm::device_uvector<bool>(0, stream)};
  }

  auto output = cudf::make_fixed_width_column(
    cudf::data_type{cudf::type_id::BOOL8}, string_count, cudf::mask_state::UNALLOCATED, stream, mr);
  auto validity = rmm::device_uvector<bool>(string_count, stream);

  auto const input_sv = cudf::strings_column_view{input};
  auto const offsets_it =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());
  auto const d_input_ptr = cudf::column_device_view::create(input, stream);
  auto const is_valid_it = cudf::detail::make_validity_iterator<true>(*d_input_ptr);
  auto const output_it   = thrust::make_zip_iterator(
    thrust::make_tuple(output->mutable_view().begin<bool>(), validity.begin()));
  thrust::tabulate(
    rmm::exec_policy_nosync(stream),
    output_it,
    output_it + string_count,
    [chars = input_sv.chars_begin(stream), offsets = offsets_it, is_valid = is_valid_it] __device__(
      auto idx) -> thrust::tuple<bool, bool> {
      if (is_valid[idx]) {
        auto const start_offset = offsets[idx];
        auto const end_offset   = offsets[idx + 1];
        auto const size         = end_offset - start_offset;
        auto const str          = chars + start_offset;

        if (size == 4 && str[0] == 't' && str[1] == 'r' && str[2] == 'u' && str[3] == 'e') {
          return {true, true};
        }
        if (size == 5 && str[0] == 'f' && str[1] == 'a' && str[2] == 'l' && str[3] == 's' &&
            str[4] == 'e') {
          return {false, true};
        }
      }

      // Either null input, or the input string is neither `true` nor `false`.
      return {false, false};
    });

  // Reset null count, as it is invalidated after calling to `mutable_view()`.
  output->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);

  return {std::move(output), std::move(validity)};
}

std::pair<std::unique_ptr<cudf::column>, rmm::device_uvector<bool>> cast_strings_to_integers(
  cudf::column_view const& input,
  cudf::data_type output_type,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    return {cudf::make_empty_column(output_type), rmm::device_uvector<bool>(0, stream)};
  }

  auto const input_sv = cudf::strings_column_view{input};
  auto const input_offsets_it =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());
  auto const d_input_ptr = cudf::column_device_view::create(input, stream);
  auto const is_valid_it = cudf::detail::make_validity_iterator<true>(*d_input_ptr);

  auto string_pairs = rmm::device_uvector<string_index_pair>(string_count, stream);
  // Since the strings store integer numbers, they should be very short.
  // As such, using one thread per string should be good.
  thrust::tabulate(rmm::exec_policy_nosync(stream),
                   string_pairs.begin(),
                   string_pairs.end(),
                   [chars    = input_sv.chars_begin(stream),
                    offsets  = input_offsets_it,
                    is_valid = is_valid_it] __device__(cudf::size_type idx) -> string_index_pair {
                     if (!is_valid[idx]) { return {nullptr, 0}; }

                     auto const start_offset = offsets[idx];
                     auto const end_offset   = offsets[idx + 1];

                     auto in_ptr = chars + start_offset;
                     auto in_end = chars + end_offset;
                     while (in_ptr != in_end) {
                       if (*in_ptr == '.' || *in_ptr == 'e' || *in_ptr == 'E') {
                         return {nullptr, 0};
                       }
                       ++in_ptr;
                     }

                     return {chars + start_offset, end_offset - start_offset};
                   });

  auto const size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [string_pairs = string_pairs.begin()] __device__(cudf::size_type idx) -> cudf::size_type {
        return string_pairs[idx].second;
      }));
  auto [offsets_column, bytes] =
    cudf::strings::detail::make_offsets_child_column(size_it, size_it + string_count, stream, mr);
  auto chars_data = cudf::strings::detail::make_chars_buffer(
    offsets_column->view(), bytes, string_pairs.begin(), string_count, stream, mr);

  // Don't care about the null mask, as nulls imply empty strings, and will be nullified.
  auto const sanitized_input =
    cudf::make_strings_column(string_count, std::move(offsets_column), chars_data.release(), 0, {});

  auto output = string_to_integer(
    output_type, cudf::strings_column_view{sanitized_input->view()}, false, false, stream, mr);

  return {std::move(output), rmm::device_uvector<bool>(0, stream)};
}

std::pair<std::unique_ptr<cudf::column>, rmm::device_uvector<bool>> cast_strings_to_dates(
  cudf::column_view const& input,
  std::string const& date_regex,
  std::string const& date_format,
  bool error_if_invalid,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    return {cudf::make_empty_column(cudf::data_type{cudf::type_id::TIMESTAMP_DAYS}),
            rmm::device_uvector<bool>(0, stream)};
  }

  // TODO: mr
  auto const removed_quotes = remove_quotes(input, false, stream, mr);

  auto const input_sv   = cudf::strings_column_view{removed_quotes->view()};
  auto const regex_prog = cudf::strings::regex_program::create(
    date_regex, cudf::strings::regex_flags::DEFAULT, cudf::strings::capture_groups::NON_CAPTURE);
  auto const is_matched     = cudf::strings::matches_re(input_sv, *regex_prog, stream);
  auto const is_timestamp   = cudf::strings::is_timestamp(input_sv, date_format, stream);
  auto const d_is_matched   = is_matched->view().begin<bool>();
  auto const d_is_timestamp = is_timestamp->view().begin<bool>();

  auto const d_input_ptr   = cudf::column_device_view::create(removed_quotes->view(), stream);
  auto const is_valid_it   = cudf::detail::make_validity_iterator<true>(*d_input_ptr);
  auto const invalid_count = thrust::count_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(string_count),
    [is_valid = is_valid_it, is_matched = d_is_matched, is_timestamp = d_is_timestamp] __device__(
      auto idx) { return is_valid[idx] && (!is_matched[idx] || !is_timestamp[idx]); });

  if (invalid_count == 0) {
    auto output = cudf::strings::to_timestamps(
      input_sv, cudf::data_type{cudf::type_id::TIMESTAMP_DAYS}, date_format, stream, mr);
    return {std::move(output), rmm::device_uvector<bool>(0, stream)};
  }

  // From here we have invalid_count > 0.
  if (error_if_invalid) { return {nullptr, rmm::device_uvector<bool>(0, stream)}; }

  auto const input_offsets_it =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());
  auto string_pairs = rmm::device_uvector<string_index_pair>(string_count, stream);

  thrust::tabulate(
    rmm::exec_policy_nosync(stream),
    string_pairs.begin(),
    string_pairs.end(),
    [chars        = input_sv.chars_begin(stream),
     offsets      = input_offsets_it,
     is_valid     = is_valid_it,
     is_matched   = d_is_matched,
     is_timestamp = d_is_timestamp] __device__(cudf::size_type idx) -> string_index_pair {
      if (!is_valid[idx] || !is_matched[idx] || !is_timestamp[idx]) { return {nullptr, 0}; }

      auto const start_offset = offsets[idx];
      auto const end_offset   = offsets[idx + 1];
      return {chars + start_offset, end_offset - start_offset};
    });

  auto const size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [string_pairs = string_pairs.begin()] __device__(cudf::size_type idx) -> cudf::size_type {
        return string_pairs[idx].second;
      }));
  auto [offsets_column, bytes] =
    cudf::strings::detail::make_offsets_child_column(size_it, size_it + string_count, stream, mr);
  auto chars_data = cudf::strings::detail::make_chars_buffer(
    offsets_column->view(), bytes, string_pairs.begin(), string_count, stream, mr);

  // Don't care about the null mask, as nulls imply empty strings, and will be nullified.
  auto const sanitized_input =
    cudf::make_strings_column(string_count, std::move(offsets_column), chars_data.release(), 0, {});

  auto output = cudf::strings::to_timestamps(cudf::strings_column_view{sanitized_input->view()},
                                             cudf::data_type{cudf::type_id::TIMESTAMP_DAYS},
                                             date_format,
                                             stream,
                                             mr);

  auto validity = rmm::device_uvector<bool>(string_count, stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    string_pairs.begin(),
                    string_pairs.end(),
                    validity.begin(),
                    [] __device__(string_index_pair const& pair) { return pair.first != nullptr; });

  // Null mask and null count will be updated later from the validity vector.
  return {std::move(output), std::move(validity)};
}

// TODO there is a bug here around 0 https://github.com/NVIDIA/spark-rapids/issues/10898
std::unique_ptr<cudf::column> cast_strings_to_decimals(cudf::column_view const& input,
                                                       int precision,
                                                       int scale,
                                                       bool is_us_locale,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    auto const dtype = [precision, scale]() {
      if (precision <= std::numeric_limits<int32_t>::digits10) {
        return cudf::data_type(cudf::type_id::DECIMAL32, scale);
      } else if (precision <= std::numeric_limits<int64_t>::digits10) {
        return cudf::data_type(cudf::type_id::DECIMAL64, scale);
      } else if (precision <= std::numeric_limits<__int128_t>::digits10) {
        return cudf::data_type(cudf::type_id::DECIMAL128, scale);
      } else {
        CUDF_FAIL("Unable to support decimal with precision " + std::to_string(precision));
      }
    }();
    return cudf::make_empty_column(dtype);
  }

  CUDF_EXPECTS(is_us_locale, "String to decimal conversion is only supported in US locale.");

  auto const input_sv = cudf::strings_column_view{input};
  auto const in_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());

  // Count the number of characters `"`.
  rmm::device_uvector<int8_t> quote_counts(string_count, stream);
  // Count the number of characters `"` and `,` in each string.
  rmm::device_uvector<int8_t> remove_counts(string_count, stream);

  {
    using count_type    = thrust::tuple<int8_t, int8_t>;
    auto const check_it = cudf::detail::make_counting_transform_iterator(
      0,
      cuda::proclaim_return_type<count_type>(
        [chars = input_sv.chars_begin(stream)] __device__(auto idx) {
          auto const c             = chars[idx];
          auto const is_quote      = c == '"';
          auto const should_remove = is_quote || c == ',';
          return count_type{static_cast<int8_t>(is_quote), static_cast<int8_t>(should_remove)};
        }));
    auto const plus_op =
      cuda::proclaim_return_type<count_type>([] __device__(count_type lhs, count_type rhs) {
        return count_type{thrust::get<0>(lhs) + thrust::get<0>(rhs),
                          thrust::get<1>(lhs) + thrust::get<1>(rhs)};
      });

    auto const out_count_it =
      thrust::make_zip_iterator(quote_counts.begin(), remove_counts.begin());

    std::size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedReduce::Reduce(nullptr,
                                       temp_storage_bytes,
                                       check_it,
                                       out_count_it,
                                       string_count,
                                       in_offsets,
                                       in_offsets + 1,
                                       plus_op,
                                       count_type{0, 0},
                                       stream.value());
    auto d_temp_storage = rmm::device_buffer{temp_storage_bytes, stream};
    hipcub::DeviceSegmentedReduce::Reduce(d_temp_storage.data(),
                                       temp_storage_bytes,
                                       check_it,
                                       out_count_it,
                                       string_count,
                                       in_offsets,
                                       in_offsets + 1,
                                       plus_op,
                                       count_type{0, 0},
                                       stream.value());
  }

  auto const out_size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [offsets       = in_offsets,
       quote_counts  = quote_counts.begin(),
       remove_counts = remove_counts.begin()] __device__(auto idx) {
        auto const input_size = offsets[idx + 1] - offsets[idx];
        // If the current row is a non-quoted string, just return the original string.
        if (quote_counts[idx] == 0) { return static_cast<cudf::size_type>(input_size); }
        // Otherwise, we will modify the string, removing characters '"' and ','.
        return static_cast<cudf::size_type>(input_size - remove_counts[idx]);
      }));
  auto [offsets_column, bytes] = cudf::strings::detail::make_offsets_child_column(
    out_size_it, out_size_it + string_count, stream, mr);

  // If the output strings column does not change in its total bytes, we know that it does not have
  // any '"' or ',' characters.
  if (bytes == input_sv.chars_size(stream)) {
    return string_to_decimal(precision, scale, input_sv, false, false, stream, mr);
  }

  auto const out_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(offsets_column->view());
  auto chars_data = rmm::device_uvector<char>(bytes, stream, mr);

  // Since the strings store decimal numbers, they should be very short.
  // As such, using one thread per string should be good.
  thrust::for_each(rmm::exec_policy_nosync(stream),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(string_count),
                   [in_offsets,
                    out_offsets,
                    input  = input_sv.chars_begin(stream),
                    output = chars_data.begin()] __device__(auto idx) {
                     auto const in_size  = in_offsets[idx + 1] - in_offsets[idx];
                     auto const out_size = out_offsets[idx + 1] - out_offsets[idx];
                     if (in_size == 0) { return; }

                     // If the output size is not changed, we are returning the original unquoted
                     // string. Such string may still contain other alphabet characters, but that
                     // should be handled in the conversion function later on.
                     if (in_size == out_size) {
                       memcpy(output + out_offsets[idx], input + in_offsets[idx], in_size);
                     } else {  // copy byte by byte, ignoring '"' and ',' characters.
                       auto in_ptr  = input + in_offsets[idx];
                       auto in_end  = input + in_offsets[idx + 1];
                       auto out_ptr = output + out_offsets[idx];
                       while (in_ptr != in_end) {
                         if (*in_ptr != '"' && *in_ptr != ',') {
                           *out_ptr = *in_ptr;
                           ++out_ptr;
                         }
                         ++in_ptr;
                       }
                     }
                   });

  auto const unquoted_strings = cudf::make_strings_column(string_count,
                                                          std::move(offsets_column),
                                                          chars_data.release(),
                                                          0,
                                                          rmm::device_buffer{0, stream, mr});
  return string_to_decimal(precision,
                           scale,
                           cudf::strings_column_view{unquoted_strings->view()},
                           false,
                           false,
                           stream,
                           mr);
}

std::pair<std::unique_ptr<cudf::column>, rmm::device_uvector<bool>> remove_quotes(
  cudf::column_view const& input,
  bool nullify_if_not_quoted,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    return {cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING}),
            rmm::device_uvector<bool>(0, stream)};
  }

  auto const input_sv = cudf::strings_column_view{input};
  auto const input_offsets_it =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());
  auto const d_input_ptr = cudf::column_device_view::create(input, stream);
  auto const is_valid_it = cudf::detail::make_validity_iterator<true>(*d_input_ptr);

  auto string_pairs = rmm::device_uvector<string_index_pair>(string_count, stream);
  thrust::tabulate(rmm::exec_policy_nosync(stream),
                   string_pairs.begin(),
                   string_pairs.end(),
                   [nullify_if_not_quoted,
                    chars    = input_sv.chars_begin(stream),
                    offsets  = input_offsets_it,
                    is_valid = is_valid_it] __device__(cudf::size_type idx) -> string_index_pair {
                     if (!is_valid[idx]) { return {nullptr, 0}; }

                     auto const start_offset = offsets[idx];
                     auto const end_offset   = offsets[idx + 1];
                     auto const size         = end_offset - start_offset;
                     auto const str          = chars + start_offset;

                     // Need to check for size, since the input string may contain just a single
                     // character `"`. Such input should not be considered as quoted.
                     auto const is_quoted = size > 1 && str[0] == '"' && str[size - 1] == '"';
                     if (nullify_if_not_quoted && !is_quoted) { return {nullptr, 0}; }

                     auto const output_size = is_quoted ? size - 2 : size;
                     return {chars + start_offset + (is_quoted ? 1 : 0), output_size};
                   });

  auto const size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [string_pairs = string_pairs.begin()] __device__(cudf::size_type idx) -> cudf::size_type {
        return string_pairs[idx].second;
      }));
  auto [offsets_column, bytes] =
    cudf::strings::detail::make_offsets_child_column(size_it, size_it + string_count, stream, mr);
  auto chars_data = cudf::strings::detail::make_chars_buffer(
    offsets_column->view(), bytes, string_pairs.begin(), string_count, stream, mr);

  if (nullify_if_not_quoted) {
    auto validity = rmm::device_uvector<bool>(string_count, stream);
    thrust::transform(
      rmm::exec_policy_nosync(stream),
      string_pairs.begin(),
      string_pairs.end(),
      validity.begin(),
      [] __device__(string_index_pair const& pair) { return pair.first != nullptr; });

    // Null mask and null count will be updated later from the validity vector.
    auto output = cudf::make_strings_column(string_count,
                                            std::move(offsets_column),
                                            chars_data.release(),
                                            0,
                                            rmm::device_buffer{0, stream, mr});

    return {std::move(output), std::move(validity)};
  } else {
    auto output = cudf::make_strings_column(string_count,
                                            std::move(offsets_column),
                                            chars_data.release(),
                                            input.null_count(),
                                            cudf::detail::copy_bitmask(input, stream, mr));

    return {std::move(output), rmm::device_uvector<bool>(0, stream)};
  }
}

// TODO: extract commond code for this and `remove_quotes`.
std::pair<std::unique_ptr<cudf::column>, rmm::device_uvector<bool>> remove_quotes_for_floats(
  cudf::column_view const& input, rmm::cuda_stream_view stream, rmm::device_async_resource_ref mr)
{
  auto const string_count = input.size();
  if (string_count == 0) {
    return {cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING}),
            rmm::device_uvector<bool>(0, stream)};
  }

  auto const input_sv = cudf::strings_column_view{input};
  auto const input_offsets_it =
    cudf::detail::offsetalator_factory::make_input_iterator(input_sv.offsets());
  auto const d_input_ptr = cudf::column_device_view::create(input, stream);
  auto const is_valid_it = cudf::detail::make_validity_iterator<true>(*d_input_ptr);

  auto string_pairs = rmm::device_uvector<string_index_pair>(string_count, stream);
  thrust::tabulate(rmm::exec_policy_nosync(stream),
                   string_pairs.begin(),
                   string_pairs.end(),
                   [chars    = input_sv.chars_begin(stream),
                    offsets  = input_offsets_it,
                    is_valid = is_valid_it] __device__(cudf::size_type idx) -> string_index_pair {
                     if (!is_valid[idx]) { return {nullptr, 0}; }

                     auto const start_offset = offsets[idx];
                     auto const end_offset   = offsets[idx + 1];
                     auto const size         = end_offset - start_offset;
                     auto const str          = chars + start_offset;

                     // Need to check for size, since the input string may contain just a single
                     // character `"`. Such input should not be considered as quoted.
                     auto const is_quoted = size > 1 && str[0] == '"' && str[size - 1] == '"';

                     // We check and remove quotes only for the special cases (non-numeric numbers
                     // wrapped in double quotes) that are accepted in `from_json`.
                     // They are "NaN", "+INF", "-INF", "+Infinity", "Infinity", "-Infinity".
                     if (is_quoted) {
                       // "NaN"
                       auto accepted = size == 5 && str[1] == 'N' && str[2] == 'a' && str[3] == 'N';

                       // "+INF" and "-INF"
                       accepted = accepted || (size == 6 && (str[1] == '+' || str[1] == '-') &&
                                               str[2] == 'I' && str[3] == 'N' && str[4] == 'F');

                       // "Infinity"
                       accepted = accepted || (size == 10 && str[1] == 'I' && str[2] == 'n' &&
                                               str[3] == 'f' && str[4] == 'i' && str[5] == 'n' &&
                                               str[6] == 'i' && str[7] == 't' && str[8] == 'y');

                       // "+Infinity" and "-Infinity"
                       accepted = accepted || (size == 11 && (str[1] == '+' || str[1] == '-') &&
                                               str[2] == 'I' && str[3] == 'n' && str[4] == 'f' &&
                                               str[5] == 'i' && str[6] == 'n' && str[7] == 'i' &&
                                               str[8] == 't' && str[9] == 'y');

                       if (accepted) { return {str + 1, size - 2}; }
                     }

                     return {str, size};
                   });

  auto const size_it = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cudf::size_type>(
      [string_pairs = string_pairs.begin()] __device__(cudf::size_type idx) -> cudf::size_type {
        return string_pairs[idx].second;
      }));
  auto [offsets_column, bytes] =
    cudf::strings::detail::make_offsets_child_column(size_it, size_it + string_count, stream, mr);
  auto chars_data = cudf::strings::detail::make_chars_buffer(
    offsets_column->view(), bytes, string_pairs.begin(), string_count, stream, mr);

  auto output = cudf::make_strings_column(string_count,
                                          std::move(offsets_column),
                                          chars_data.release(),
                                          input.null_count(),
                                          cudf::detail::copy_bitmask(input, stream, mr));

  return {std::move(output), rmm::device_uvector<bool>(0, stream)};
}

std::unique_ptr<cudf::column> convert_column_type(cudf::column_view const& input,
                                                  json_schema_element const& schema,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::device_async_resource_ref mr)
{
  return nullptr;
}

}  // namespace

std::unique_ptr<cudf::column> convert_types(
  cudf::table_view const& input,
  std::vector<std::pair<std::string, json_schema_element>> const& schema,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const num_columns = input.num_columns();
  CUDF_EXPECTS(static_cast<std::size_t>(num_columns) == schema.size(),
               "Numbers of columns in the input table is different from schema size.");

  std::vector<std::unique_ptr<cudf::column>> converted_cols(num_columns);
  for (int i = 0; i < num_columns; ++i) {
    converted_cols[i] = convert_column_type(input.column(i), schema[i].second, stream, mr);
  }

  return nullptr;
}

}  // namespace detail

std::tuple<std::unique_ptr<cudf::column>, std::unique_ptr<rmm::device_buffer>, char> concat_json(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::concat_json(input, stream, mr);
}

std::unique_ptr<cudf::column> make_structs(std::vector<cudf::column_view> const& children,
                                           cudf::column_view const& is_null,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::make_structs(children, is_null, stream, mr);
}

std::unique_ptr<cudf::column> convert_types(
  cudf::table_view const& input,
  std::vector<std::pair<std::string, json_schema_element>> const& schema,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::convert_types(input, schema, stream, mr);
}

std::unique_ptr<cudf::column> cast_strings_to_booleans(cudf::column_view const& input,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto [output, validity] = detail::cast_strings_to_booleans(input, stream, mr);
  auto [null_mask, null_count] =
    cudf::detail::valid_if(validity.begin(), validity.end(), thrust::identity{}, stream, mr);
  if (null_count > 0) { output->set_null_mask(std::move(null_mask), null_count); }
  return std::move(output);
}

std::unique_ptr<cudf::column> cast_strings_to_integers(cudf::column_view const& input,
                                                       cudf::data_type output_type,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto [output, validity] = detail::cast_strings_to_integers(input, output_type, stream, mr);
  return std::move(output);
}

std::unique_ptr<cudf::column> cast_strings_to_dates(cudf::column_view const& input,
                                                    std::string const& date_regex,
                                                    std::string const& date_format,
                                                    bool error_if_invalid,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto [output, validity] =
    detail::cast_strings_to_dates(input, date_regex, date_format, error_if_invalid, stream, mr);

  if (output == nullptr) { return nullptr; }
  auto [null_mask, null_count] =
    cudf::detail::valid_if(validity.begin(), validity.end(), thrust::identity{}, stream, mr);
  if (null_count > 0) { output->set_null_mask(std::move(null_mask), null_count); }
  return std::move(output);
}

std::unique_ptr<cudf::column> cast_strings_to_decimals(cudf::column_view const& input,
                                                       int precision,
                                                       int scale,
                                                       bool is_us_locale,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  return detail::cast_strings_to_decimals(input, precision, scale, is_us_locale, stream, mr);
}

std::unique_ptr<cudf::column> remove_quotes(cudf::column_view const& input,
                                            bool nullify_if_not_quoted,
                                            rmm::cuda_stream_view stream,
                                            rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto [output, validity] = detail::remove_quotes(input, nullify_if_not_quoted, stream, mr);
  if (validity.size() > 0) {
    auto [null_mask, null_count] =
      cudf::detail::valid_if(validity.begin(), validity.end(), thrust::identity{}, stream, mr);
    if (null_count > 0) { output->set_null_mask(std::move(null_mask), null_count); }
  }
  return std::move(output);
}

std::unique_ptr<cudf::column> remove_quotes_for_floats(cudf::column_view const& input,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto [output, validity] = detail::remove_quotes_for_floats(input, stream, mr);
  return std::move(output);
}

}  // namespace spark_rapids_jni
