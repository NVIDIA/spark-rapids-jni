#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/valid_if.cuh>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cub/device/device_histogram.cuh>
#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/uninitialized_fill.h>

namespace spark_rapids_jni {

namespace detail {

std::tuple<std::unique_ptr<cudf::column>, std::unique_ptr<rmm::device_buffer>, char> concat_json(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const d_input_ptr = cudf::column_device_view::create(input.parent(), stream);

  rmm::device_uvector<bool> is_valid_input(input.size(), stream);
  rmm::device_uvector<bool> is_null_or_empty(input.size(), stream, mr);
  rmm::device_scalar<char> first_char(stream);
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(input.size()),
    thrust::make_zip_iterator(thrust::make_tuple(is_valid_input.begin(), is_null_or_empty.begin())),
    [input      = *d_input_ptr,
     first_char = first_char.data()] __device__(cudf::size_type idx) -> thrust::tuple<bool, bool> {
      if (input.is_null(idx)) { return {false, false}; }

      auto const d_str = input.element<cudf::string_view>(idx);
      int i            = 0;
      for (; i < d_str.size_bytes(); ++i) {
        if (d_str[i] != ' ') { break; }
      }

      bool is_null_literal{false};
      if (i + 4 <= d_str.size_bytes() && d_str[i] == 'n' && d_str[i + 1] == 'u' &&
          d_str[i + 2] == 'l' && d_str[i + 3] == 'l') {
        is_null_literal = true;
        i += 4;
      }

      for (; i < d_str.size_bytes(); ++i) {
        if (d_str[i] != ' ') {
          is_null_literal = false;
          break;
        }
      }

      if (is_null_literal) { return {false, true}; }

      auto const not_empty = i + 1 < d_str.size_bytes();
      if (idx == 0) { *first_char = not_empty ? d_str[i] : '\0'; }

      return {not_empty, not_empty};
    });
  auto [null_mask, null_count] = cudf::detail::valid_if(
    is_valid_input.begin(), is_valid_input.end(), thrust::identity{}, stream, mr);

  auto constexpr num_levels  = 256;
  auto constexpr lower_level = std::numeric_limits<char>::min();
  auto constexpr upper_level = std::numeric_limits<char>::max();

  char h_first_char;
  CUDF_CUDA_TRY(hipMemcpyAsync(
    &h_first_char, first_char.data(), sizeof(char), hipMemcpyDefault, stream.value()));
  auto const num_chars = input.chars_size(stream);  // stream sync

  // TODO: return when num_chars==0

  rmm::device_uvector<uint32_t> d_histogram(num_levels, stream);
  thrust::uninitialized_fill(rmm::exec_policy(stream), d_histogram.begin(), d_histogram.end(), 0);

  size_t temp_storage_bytes = 0;
  hipcub::DeviceHistogram::HistogramEven(nullptr,
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      d_histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());
  rmm::device_buffer d_temp(temp_storage_bytes, stream);
  hipcub::DeviceHistogram::HistogramEven(d_temp.data(),
                                      temp_storage_bytes,
                                      input.chars_begin(stream),
                                      d_histogram.begin(),
                                      num_levels,
                                      lower_level,
                                      upper_level,
                                      num_chars,
                                      stream.value());

  auto const zero_level = d_histogram.begin() - lower_level;
  auto const first_zero_count_pos =
    thrust::find(rmm::exec_policy(stream), zero_level + '\n', d_histogram.end(), 0);
  if (first_zero_count_pos == d_histogram.end()) {
    throw std::logic_error(
      "can't find a character suitable as delimiter for combining json strings to json lines with "
      "custom delimiter");
  }

  auto const input_applied_null =
    null_count == 0
      ? nullptr
      : cudf::purge_nonempty_nulls(
          cudf::column_view{cudf::data_type{cudf::type_id::STRING},
                            input.size(),
                            input.chars_begin(stream),
                            reinterpret_cast<cudf::bitmask_type const*>(null_mask.data()),
                            null_count,
                            0,
                            std::vector<cudf::column_view>{input.offsets()}},
          stream);

  auto const first_non_existing_char = first_zero_count_pos - zero_level;
  auto all_done                      = cudf::strings::detail::join_strings(
    null_count == 0 ? input : cudf::strings_column_view{input_applied_null->view()},
    cudf::string_scalar(std::string(1, first_non_existing_char), true, stream, mr),
    cudf::string_scalar(h_first_char == '[' ? "[]" : "{}", true, stream, mr),
    stream,
    mr);

  return {std::make_unique<cudf::column>(std::move(is_null_or_empty), rmm::device_buffer{}, 0),
          std::move(all_done->release().data),
          first_non_existing_char};
}

}  // namespace detail

std::tuple<std::unique_ptr<cudf::column>, std::unique_ptr<rmm::device_buffer>, char> concat_json(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::concat_json(input, stream, mr);
}

}  // namespace spark_rapids_jni
