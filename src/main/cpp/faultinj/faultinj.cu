/*
 * Copyright 2020 NVIDIA Corporation. All rights reserved.
 *
 * Sample CUPTI based injection to attach and detach CUPTI
 * For detaching, it uses CUPTI API cuptiFinalize().
 *
 * It is recommended to invoke API cuptiFinalize() in the
 * exit callsite of any CUDA Driver/Runtime API.
 *
 * API cuptiFinalize() destroys and cleans up all the
 * resources associated with CUPTI in the current process.
 * After CUPTI detaches from the process, the process will
 * keep on running with no CUPTI attached to it.
 *
 * CUPTI can be attached by calling any CUPTI API as CUPTI
 * supports lazy initialization. Any subsequent CUPTI API
 * call will reinitialize the CUPTI.
 *
 * You can attach and detach CUPTI any number of times.
 *
 * After building the sample, set the following environment variable
 * export CUDA_INJECTION64_PATH=<full_path>/libCuptiFinalize.so
 * Add CUPTI library in LD_LIBRARY_PATH and run any CUDA sample
 * with runtime more than 10 sec for demonstration of the
 * CUPTI sample
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <cupti.h>

#define STDCALL

#if defined(__cplusplus)
extern "C" {
#endif

// MAcros

#define CUPTI_CALL(call)                                                       \
do {                                                                           \
    CUptiResult _status = call;                                                \
    if (_status != CUPTI_SUCCESS) {                                            \
        const char *errstr;                                                    \
        cuptiGetResultString(_status, &errstr);                                \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #call, errstr);                            \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define BUF_SIZE (8 * 1024 * 1024)  // 8MB
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
    (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

// Global Structure

typedef struct {
    volatile uint32_t initialized;
    CUpti_SubscriberHandle  subscriber;
    volatile uint32_t detachCupti;
    int frequency;
    int tracingEnabled;
    int terminateThread;
    uint64_t kernelsTraced;
} injGlobalControl;
injGlobalControl globalControl;

// Function Declarations

static CUptiResult cuptiInitialize(void);

static void atExitHandler(void);

void CUPTIAPI callbackHandler(void *userdata, CUpti_CallbackDomain domain, CUpti_CallbackId cbid, void *cbInfo);

extern int STDCALL InitializeInjection(void);

#if defined(__cplusplus)
}
#endif

// Function Definitions

static void
globalControlInit(void) {
    globalControl.initialized = 0;
    globalControl.subscriber = 0;
    globalControl.detachCupti = 0;
    globalControl.frequency = 2; // in seconds
    globalControl.tracingEnabled = 0;
    globalControl.terminateThread = 0;
    globalControl.kernelsTraced = 0;
}

void registerAtExitHandler(void) {
    // Register atExitHandler
    atexit(&atExitHandler);
}


static void
atExitHandler(void) {
    globalControl.terminateThread = 1;
    fprintf(stderr, "GERA_DEBUG CUPTI exit handler");
}


static CUptiResult
cuptiInitialize(void) {
    CUptiResult status = CUPTI_SUCCESS;

    CUPTI_CALL(cuptiSubscribe(&globalControl.subscriber, (CUpti_CallbackFunc)callbackHandler, NULL));

    // Subscribe Driver and Runtime callbacks to call cuptiFinalize in the entry/exit callback of these APIs
    // CUPTI_CALL(cuptiEnableDomain(1, globalControl.subscriber, CUPTI_CB_DOMAIN_RUNTIME_API));
    CUPTI_CALL(cuptiEnableDomain(1, globalControl.subscriber, CUPTI_CB_DOMAIN_DRIVER_API));

    return status;
}

static bool
prefix(const char *pre, const char *str) {
    return strncmp(pre, str, strlen(pre)) == 0;
}

__global__ void
faultInjectorKernelAssert(void) {
    assert(0 && "GERA_DEBUG kernelAssert triggered");
}

static void
deviceAssertAndSync(void) {
    faultInjectorKernelAssert<<<1,1>>>();
    // cudaDeviceSynchronize();
}


__global__ void
faultInjectorKernelTrap(void) {
    asm("trap;");
}

static void
deviceAsmTrapAndSync(void) {
    faultInjectorKernelTrap<<<1,1>>>();
    // cudaDeviceSynchronize();
}

void CUPTIAPI
callbackHandler(void *userdata, CUpti_CallbackDomain domain,
    CUpti_CallbackId cbid, void *cbdata) {
    CUpti_CallbackData *cbInfo = (CUpti_CallbackData *)cbdata;
    // Check last error
    CUPTI_CALL(cuptiGetLastError());
    if (cbInfo->callbackSite == CUPTI_API_EXIT) {
        if (prefix("hipModuleLaunchKernel", cbInfo->functionName) && !strstr(cbInfo->symbolName, "faultInjector")) {
            fprintf(stderr, "GERA_DEBUG callbackHandler: cbid=%d domain=%d function=%s symbol=%s\n", cbid, domain, cbInfo->functionName, cbInfo->symbolName);

            // We can either
            // 1) change the return value. Here is the Driver API example
            // *((CUresult *)cbInfo->functionReturnValue) = CUDA_ERROR_OUT_OF_MEMORY;
            // 2) trigger a device-side assert
            deviceAssertAndSync();
            // 3) execute an invalid trap
            // deviceAsmTrapAndSync();
        }
    }
}


int STDCALL
InitializeInjection(void) {

    if (globalControl.initialized) {
        return 1;
    }
    // Init globalControl
    globalControlInit();
    globalControl.initialized = 1;
    globalControl.tracingEnabled = 1;


    registerAtExitHandler();

    // Initialize CUPTI
    CUPTI_CALL(cuptiInitialize());

    return 1;
}
